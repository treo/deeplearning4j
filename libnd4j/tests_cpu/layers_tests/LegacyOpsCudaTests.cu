/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
//

#include "testlayers.h"
#include <NDArray.h>
#include <ShapeUtils.h>
#include <reduce3.h>
#include <ops/declarable/LegacyTransformOp.h>
#include <ops/declarable/LegacyPairwiseTransformOp.h>
#include <ops/declarable/LegacyScalarOp.h>
#include <ops/declarable/LegacyReduceSameOp.h>
#include <ops/declarable/LegacyReduceFloatOp.h>
#include <ops/declarable/LegacyIndexReduceOp.h>
#include <ops/declarable/LegacyBroadcastOp.h>
#include <helpers/TAD.h>
#include <helpers/ConstantTadHelper.h>

using namespace nd4j;
using namespace nd4j::ops;

class LegacyOpsCudaTests : public testing::Test {

};


TEST_F(LegacyOpsCudaTests, test_sortTad_1) {
    auto x = NDArrayFactory::create<float>('c', {3, 5}, {1.f, 3.f, 0.f, 2.f, 4.f,
                                                         6.f, 5.f, 9.f, 7.f, 8.f,
                                                         10.f, 11.f, 14.f, 12.f, 13.f});

    auto e = NDArrayFactory::create<float>('c', {3, 5}, {0.f, 1.f, 2.f, 3.f, 4.f, 5.f, 6.f, 7.f, 8.f, 9.f, 10.f, 11.f, 12.f, 13.f, 14.f});

    int axis = 1;
    auto packX = ConstantTadHelper::getInstance()->tadForDimensions(x.shapeInfo(), axis);

    Nd4jPointer extras[2] = {nullptr, LaunchContext::defaultContext()->getCudaStream()};

    x.syncToDevice();
    NativeOps nativeOps;
    nativeOps.sortTad(extras, x.buffer(), x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(), &axis, 1, packX.platformShapeInfo(), packX.platformOffsets(), false);
    x.tickWriteDevice();

    ASSERT_EQ(e, x);
}