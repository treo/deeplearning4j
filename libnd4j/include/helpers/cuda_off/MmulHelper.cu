#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma (iuriish@yahoo.com)
//
#include <exceptions/cuda_exception.h>
#include <hipblas.h>
#include "../MmulHelper.h"
#include <specials_cuda.h>
#include <helpers/PointersManager.h>

namespace nd4j {


//////////////////////////////////////////////////////////////////////////////
// MXK x KxN = MxN
// C array must be in f order
template <typename T1, typename T2, typename T3>
static __global__ void usualCudaGemm(const bool transA, const bool transB, const int M, const int N, const int K, const double alpha, const void* vA, const int lda, const void* vB, const int ldb, const double beta, void* vC, const int ldc) {

    T1* A = reinterpret_cast<T1*>(const_cast<void*>(vA));
    T2* B = reinterpret_cast<T2*>(const_cast<void*>(vB));
    T3* C = reinterpret_cast<T3*>(vC);

    __shared__ T3 alphaZ, betaZ;
    __shared__ Nd4jLong strideArow, strideAcol, strideBrow, strideBcol;

    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row == 0 && col == 0) {

        alphaZ = alpha;
        betaZ  = beta;

        if(transA) { strideArow = lda; strideAcol = 1; } else { strideArow = 1; strideAcol = lda; }
        if(transB) { strideBrow = ldb; strideBcol = 1; } else { strideBrow = 1; strideBcol = ldb; }
    }

    __syncthreads();

    T3 val = 0;
    if (row < M && col < N)
        for (int i = 0; i < K; i++)
            val = val + A[row * strideArow + i * strideAcol] * B[i * strideBrow + col * strideBcol];

    C[row + col * ldc] = alphaZ * val + betaZ * C[row + col * ldc];
}

////////////////////////////////////////////////////////////////////////
template <typename T1, typename T2, typename T3>
__host__ static void usualGemm(const dim3 &blocksPerGrid, const dim3 &threadsPerBlock, hipStream_t *stream, const bool transA, const bool transB, const int M, const int N, const int K, const double alpha, const void* vA, const int lda, const void* vB, const int ldb, const double beta, void* vC, const int ldc) {

    usualCudaGemm<T1,T2,T3><<<blocksPerGrid, threadsPerBlock, 1024, *stream>>>(transA, transB, M, N, K, alpha, vA, lda, vB, ldb, beta, vC, ldc);
}

//////////////////////////////////////////////////////////////////////////////
// MXN x N = M
template <typename T1, typename T2, typename T3>
static __global__ void usualCudaGemv(const bool transA, const int M, const int N, const double alpha, const void* vA, const int lda, const void* vX, const int incx, const double beta, void* vY, const int incy) {

    T1* A = reinterpret_cast<T1*>(const_cast<void*>(vA));
    T2* X = reinterpret_cast<T2*>(const_cast<void*>(vX));
    T3* Y = reinterpret_cast<T3*>(vY);

    __shared__ T3 alphaZ, betaZ;
    __shared__ Nd4jLong strideArow, strideAcol;

    const int row = blockIdx.x * blockDim.x + threadIdx.x;

    if(row == 0) {

        alphaZ = alpha;
        betaZ  = beta;

        if(transA) { strideArow = lda; strideAcol = 1; } else { strideArow = 1; strideAcol = lda; }
    }

    __syncthreads();

    T3 val = 0;
    if (row < M)
        for (int i = 0; i < N; i++) {
            val = val + A[row * strideArow + i * strideAcol] * X[i * incx];
        }

    Y[row * incy] = alphaZ * val + betaZ * Y[row * incy];
}

////////////////////////////////////////////////////////////////////////
template <typename T1, typename T2, typename T3>
__host__ static void usualGemv(const dim3 &blocksPerGrid, const dim3 &threadsPerBlock, hipStream_t *stream, const bool transA, const int M, const int N, const double alpha, const void* vA, const int lda, const void* vX, const int incx, const double beta, void* vY, const int incy) {

    usualCudaGemv<T1,T2,T3><<<blocksPerGrid, threadsPerBlock, 1024, *stream>>>(transA, M, N, alpha, vA, lda, vX, incx, beta, vY, incy);
}

//////////////////////////////////////////////////////////////////////////////
template <typename T1, typename T2, typename T3>
static __global__ void usualCudaDot(const Nd4jLong length, const double alpha, const void* vX, const Nd4jLong incx, const void* vY, const Nd4jLong incy, const double beta, void* vZ) {

    T1* X = reinterpret_cast<T1*>(const_cast<void*>(vX));
    T2* Y = reinterpret_cast<T2*>(const_cast<void*>(vY));
    T3* Z = reinterpret_cast<T3*>(vZ);

    extern __shared__ char shmem[];
    auto pairwiseMul = reinterpret_cast<T3*>(shmem);

    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < length)
        pairwiseMul[tid] = X[tid * incx] * Y[tid * incy];

    __syncthreads();

    if(tid == 0) {
        T3 sum = 0;
        for(Nd4jLong i = 0; i < length; ++i)
            sum = sum + pairwiseMul[i];
        *Z = (T3)alpha * sum + (T3)beta * *Z;
    }
}

////////////////////////////////////////////////////////////////////////
template <typename T1, typename T2, typename T3>
__host__ static void usualDot(const dim3 &blocksPerGrid, const dim3 &threadsPerBlock, hipStream_t *stream, const Nd4jLong length, const double alpha, const void* vX, const Nd4jLong incx, const void* vY, const Nd4jLong incy, const double beta, void* vZ) {

    usualCudaDot<T1,T2,T3><<<blocksPerGrid, threadsPerBlock, length*sizeof(T3) + 128, *stream>>>(length, alpha, vX, incx, vY, incy, beta, vZ);
}

//////////////////////////////////////////////////////////////////////////////
// MXK x KxN = MxN
NDArray* MmulHelper::mmulMxM(const NDArray* A, const NDArray* B, NDArray* C, double alpha, double beta, const char outOrder) {

    if(A->rankOf() != 2)
        throw std::runtime_error("MmulHelper::mmulMxM cuda: rank of A array is not equal 2 !");
    if(B->rankOf() != 2)
        throw std::runtime_error("MmulHelper::mmulMxM cuda: rank of B array is not equal 2 !");

    auto M = A->sizeAt(0);
    auto K = A->sizeAt(1);
    auto N = B->sizeAt(1);

    if(C != nullptr && C->rankOf() != 2)
        throw std::runtime_error("MmulHelper::mmulMxM cuda: rank of C array is not equal 2 !");
    if(B->sizeAt(0) != K)
        throw std::runtime_error("MmulHelper::mmulMxM cuda: B array has wrong number of rows !");
    if(C != nullptr && C->sizeAt(0) != M)
        throw std::runtime_error("MmulHelper::mmulMxM cuda: C array has wrong number of rows !");
    if(C != nullptr && C->sizeAt(1) != N)
        throw std::runtime_error("MmulHelper::mmulMxM cuda: C array has wrong number of columns !");

    if(C == nullptr)
        C = new NDArray(outOrder, {M,N}, DataTypeUtils::pickPairwiseResultType(A->dataType(), B->dataType()), A->getContext());

    NDArray *pA(const_cast<NDArray*>(A)), *pB(const_cast<NDArray*>(B)), *pC(const_cast<NDArray*>(C));
    std::vector<NDArray*> toDelete;

    if(A->ews() != 1) {
        pA = pA->dup('f');
        toDelete.push_back(pA);
    }
    if(B->ews() != 1) {
        pB = pB->dup('f');
        toDelete.push_back(pB);
    }
    if(C->ews() != 1) {
        pC = pC->dup('f');
        toDelete.push_back(pC);
    }

    if(pC->ordering() != 'f') {
        auto temp = pA;
        pA = new NDArray(pB  ->permute({1,0}));
        pB = new NDArray(temp->permute({1,0}));
        pC = new NDArray(pC  ->permute({1,0}));
        toDelete.push_back(pA);
        toDelete.push_back(pB);
        toDelete.push_back(pC);
        M = pA->sizeAt(0);
        K = pA->sizeAt(1);
        N = pB->sizeAt(1);
    }

    const auto aOrder = pA->ordering();
    const auto bOrder = pB->ordering();

    const bool transA = aOrder != 'f';
    const bool transB = bOrder != 'f';

    const hipblasOperation_t transAblas = transA ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    const hipblasOperation_t transBblas = transB ? HIPBLAS_OP_T : HIPBLAS_OP_N;

    const int lda = aOrder == 'f' ? M : K;
    const int ldb = bOrder == 'f' ? K : N;
    const int ldc = M; // cOrder == 'f' ? M : N;

    const auto aType = pA->dataType();
    const auto bType = pB->dataType();
    const auto cType = pC->dataType();

    auto handle = reinterpret_cast<hipblasHandle_t *>(A->getContext()->getCublasHandle());
    auto stream = A->getContext()->getCudaStream();

    auto status = hipblasSetStream(*handle, *stream);
    if (status != HIPBLAS_STATUS_SUCCESS) throw cuda_exception::build("MmulHelper::mmulMxM cuda failed !", status);

    const bool AB(aType == bType), AC(aType == cType), ABC(AB && AC);

    NDArray::prepareSpecialUse({pC}, {pA, pB});

    // choose appropriate cuda gemm api depending on data types
    if(ABC && aType == DataType::DOUBLE) {
        status = hipblasDgemm(*handle, transAblas, transBblas, M, N, K, &alpha, (double*)pA->getSpecialBuffer(), lda, (double*)pB->getSpecialBuffer(), ldb, &beta, (double*)pC->getSpecialBuffer(), ldc);
    }
    else if(ABC && aType == DataType::FLOAT32) {
        float alphaF(alpha), betaF(beta);
        status = hipblasSgemm(*handle, transAblas, transBblas, M, N, K, &alphaF, (float*)pA->getSpecialBuffer(), lda, (float*)pB->getSpecialBuffer(), ldb, &betaF, (float*)pC->getSpecialBuffer(), ldc);
    }
    else if(ABC && aType == DataType::HALF) {
        float16 alphaH(alpha), betaH(beta);
        status = hipblasHgemm(*handle, transAblas, transBblas, M, N, K, &alphaH.data, (__half*)pA->getSpecialBuffer(), lda, (__half*)pB->getSpecialBuffer(), ldb, &betaH.data, (__half*)pC->getSpecialBuffer(), ldc);
    }
    else if(AB && aType == DataType::INT8 && cType == DataType::FLOAT32) {
           float alphaF(alpha), betaF(beta);
           status = cublasSgemmEx(*handle, transAblas, transBblas, M, N, K, &alphaF, pA->getSpecialBuffer(), HIP_R_8I, lda, pB->getSpecialBuffer(), HIP_R_8I, ldb, &betaF, pC->getSpecialBuffer(), HIP_R_32F, ldc);
    }
    else if(AB && aType == DataType::HALF && cType == DataType::FLOAT32) {
        float alphaF(alpha), betaF(beta);
        status = cublasSgemmEx(*handle, transAblas, transBblas, M, N, K, &alphaF, pA->getSpecialBuffer(), HIP_R_16F, lda, pB->getSpecialBuffer(), HIP_R_16F, ldb, &betaF, pC->getSpecialBuffer(), HIP_R_32F, ldc);
    }
    else {
        dim3 threadsPerBlock(N, M);
        dim3 blocksPerGrid(1, 1);
        if (M*N > 512){
            threadsPerBlock.x = threadsPerBlock.y = 512;
            blocksPerGrid.x = math::nd4j_ceil<double, int>(static_cast<double>(N) / threadsPerBlock.x);    // cols
            blocksPerGrid.y = math::nd4j_ceil<double, int>(static_cast<double>(M) / threadsPerBlock.y);    // rows
        }

        //BUILD_TRIPLE_SELECTOR(aType, bType, cType, usualGemm, (blocksPerGrid, threadsPerBlock, stream, transA, transB, M, N, K, alpha, pA->getSpecialBuffer(), lda, pB->getSpecialBuffer(), ldb, beta, pC->getSpecialBuffer(), ldc), LIBND4J_TYPES, FLOAT_TYPES, FLOAT_TYPES);
        BUILD_SINGLE_SELECTOR_THRICE(aType, usualGemm, (blocksPerGrid, threadsPerBlock, stream, transA, transB, M, N, K, alpha, pA->getSpecialBuffer(), lda, pB->getSpecialBuffer(), ldb, beta, pC->getSpecialBuffer(), ldc), LIBND4J_TYPES)
    }

    if (status != HIPBLAS_STATUS_SUCCESS) throw cuda_exception::build("MmulHelper::mmulMxM cuda failed !", status);

    auto cudaResult = hipStreamSynchronize(*stream);
    if (cudaResult != 0) throw cuda_exception::build("MmulHelper::mmulMxM cuda failed !", cudaResult);

    NDArray::registerSpecialUse({pC}, {pA, pB});

    if(C->ews() != 1)
        C->assign(pC);

    for(int i = toDelete.size() - 1; i >= 0; --i)
        delete toDelete[i];

    return C;
}

////////////////////////////////////////////////////////////////////////////
// MXN x N = M
NDArray* MmulHelper::mmulMxV(const NDArray* A, const NDArray* X, nd4j::NDArray* Y, const double alpha, const double beta, const char outOrder) {

    int xLenDim, yLenDim(0);

    if(A->rankOf() != 2)
        throw std::runtime_error("MmulHelper::mmulMxV cuda: rank of A array is not equal 2 !");
    if(!shape::isCommonVector(X->getShapeInfo(), xLenDim))
        throw std::runtime_error("MmulHelper::mmulMxV cuda: X array must be vector !");

    const auto M = A->sizeAt(0);
    const auto N = A->sizeAt(1);

    if(Y != nullptr && !shape::isCommonVector(Y->getShapeInfo(), yLenDim))
        throw std::runtime_error("MmulHelper::mmulMxV cuda: Y array must be vector !");
    if(X->lengthOf() != N)
        throw std::runtime_error("MmulHelper::mmulMxV cuda: X vector has wrong length !");
    if(Y != nullptr && Y->lengthOf() != M)
        throw std::runtime_error("MmulHelper::mmulMxV cuda: Y array has wrong length !");

    if(Y == nullptr)
        Y = new NDArray(outOrder, {M}, DataTypeUtils::pickPairwiseResultType(A->dataType(), X->dataType()), A->getContext());

    NDArray *pA(const_cast<NDArray*>(A));

    if(A->ews() != 1)
        pA = pA->dup('f');

    const bool transA = pA->ordering() == 'c';

    const hipblasOperation_t transAblas = transA ? HIPBLAS_OP_T : HIPBLAS_OP_N;

    int lda, lta;
    if(transA) { lda = N; lta = M; }
    else       { lda = M; lta = N; }

    const int incx = X->stridesOf()[xLenDim];
    const int incy = Y->stridesOf()[yLenDim];

    const auto aType = pA->dataType();
    const auto xType = X->dataType();
    const auto yType = Y->dataType();

    auto handle = reinterpret_cast<hipblasHandle_t *>(A->getContext()->getCublasHandle());
    auto stream = A->getContext()->getCudaStream();

    auto status = hipblasSetStream(*handle, *stream);
    if (status != HIPBLAS_STATUS_SUCCESS) throw cuda_exception::build("MmulHelper::mmulMxV cuda failed !", status);

    const bool AX(aType == xType), AY(aType == yType), AXY(AX && AY);

    NDArray::prepareSpecialUse({Y}, {pA, X});

    // choose appropriate cuda gemm api depending on data types
    if(AXY && aType == DataType::DOUBLE) {
        status = hipblasDgemv(*handle, transAblas, lda, lta, &alpha, (double*)pA->getSpecialBuffer(), lda, (double*)X->getSpecialBuffer(), incx, &beta, (double*)Y->getSpecialBuffer(), incy);
    }
    else if(AXY && aType == DataType::FLOAT32) {
        float alphaF(alpha), betaF(beta);
        status = hipblasSgemv(*handle, transAblas, lda, lta, &alphaF, (float*)pA->getSpecialBuffer(), lda, (float*)X->getSpecialBuffer(), incx, &betaF, (float*)Y->getSpecialBuffer(), incy);
    }
    else {
        dim3 threadsPerBlock(M);
        dim3 blocksPerGrid(1);
        if (M > 512){
            threadsPerBlock.x = 512;
            blocksPerGrid.x = math::nd4j_ceil<double, int>(static_cast<double>(M) / threadsPerBlock.x);    // rows
        }
        //BUILD_TRIPLE_SELECTOR(aType, xType, yType, usualGemv, (blocksPerGrid, threadsPerBlock, stream, transA, M, N, alpha, pA->getSpecialBuffer(), lda, X->getSpecialBuffer(), incx, beta, Y->getSpecialBuffer(), incy), LIBND4J_TYPES, FLOAT_TYPES, FLOAT_TYPES);
        BUILD_SINGLE_SELECTOR_THRICE(xType, usualGemv, (blocksPerGrid, threadsPerBlock, stream, transA, M, N, alpha, pA->getSpecialBuffer(), lda, X->getSpecialBuffer(), incx, beta, Y->getSpecialBuffer(), incy), LIBND4J_TYPES)
    }

    if (status != HIPBLAS_STATUS_SUCCESS) throw cuda_exception::build("MmulHelper::mmulMxV cuda failed !", status);

    auto cudaResult = hipStreamSynchronize(*stream);
    if (cudaResult != 0) throw cuda_exception::build("MmulHelper::mmulMxV cuda failed !", cudaResult);

    NDArray::registerSpecialUse({Y}, {pA, X});

    if(pA != A)
        delete pA;

    return Y;
}

////////////////////////////////////////////////////////////////////////////
// (X * Y) = Z[0]
NDArray* MmulHelper::dot(const NDArray* X, const NDArray* Y, nd4j::NDArray* Z, const double alpha, const double beta) {

    int xLenDim(0), yLenDim(0);

    if(!shape::isCommonVector(X->getShapeInfo(), xLenDim))
        throw std::runtime_error("MmulHelper::dot cuda: X array must be vector !");
    if(!shape::isCommonVector(Y->getShapeInfo(), yLenDim))
        throw std::runtime_error("MmulHelper::dot cuda: Y array must be vector !");
    if(Z != nullptr && !Z->isScalar())
        throw std::runtime_error("MmulHelper::dot cuda: Z array must be scalar !");

    const auto length = X->lengthOf();

    if(Y->lengthOf() != length)
        throw std::runtime_error("MmulHelper::dot cuda: lengths of input vectors are different !");

    if(Z == nullptr)
        Z = new NDArray(DataTypeUtils::pickPairwiseResultType(X->dataType(), Y->dataType()), X->getContext());

    const Nd4jLong incx = X->stridesOf()[xLenDim];
    const Nd4jLong incy = Y->stridesOf()[yLenDim];

    const auto xType = X->dataType();
    const auto yType = Y->dataType();
    const auto zType = Z->dataType();

    if(!X->isActualOnDeviceSide())  X->syncToDevice();
    if(!Y->isActualOnDeviceSide())  Y->syncToDevice();
    if(!Z->isActualOnDeviceSide())  Z->syncToDevice();

    hipStream_t* stream = X->getContext()->getCudaStream();

    dim3 threadsPerBlock(512);
    dim3 blocksPerGrid(1);
    if (length > 512)
        threadsPerBlock.x = math::nd4j_ceil<double, int>(static_cast<double>(length) / 512);

    NDArray::prepareSpecialUse({Z}, {X, Y});

    //BUILD_TRIPLE_SELECTOR(xType, yType, zType, usualDot, (blocksPerGrid, threadsPerBlock, stream, length, alpha, X->getSpecialBuffer(), incx, Y->getSpecialBuffer(), incy, beta, Z->getSpecialBuffer()), LIBND4J_TYPES, FLOAT_TYPES, FLOAT_TYPES);
    BUILD_SINGLE_SELECTOR_THRICE(xType, usualDot, (blocksPerGrid, threadsPerBlock, stream, length, alpha, X->getSpecialBuffer(), incx, Y->getSpecialBuffer(), incy, beta, Z->getSpecialBuffer()), LIBND4J_TYPES)

    auto cudaResult = hipStreamSynchronize(*stream);
    if (cudaResult != 0) throw cuda_exception::build("MmulHelper::dot cuda failed !", cudaResult);

    NDArray::registerSpecialUse({Z}, {X, Y});

    return Z;
}

//BUILD_TRIPLE_TEMPLATE(template void usualGemm, (const dim3 &blocksPerGrid, const dim3 &threadsPerBlock, hipStream_t *stream, const bool transA, const bool transB, const int M, const int N, const int K, const double alpha, const void* vA, const int lda, const void* vB, const int ldb, const double beta, void* vC, const int ldc), LIBND4J_TYPES, FLOAT_TYPES, FLOAT_TYPES);
//BUILD_TRIPLE_TEMPLATE(template void usualGemv, (const dim3 &blocksPerGrid, const dim3 &threadsPerBlock, hipStream_t *stream, const bool transA, const int M, const int N, const double alpha, const void* vA, const int lda, const void* vB, const int incx, const double beta, void* vC, const int incy), LIBND4J_TYPES, FLOAT_TYPES, FLOAT_TYPES);
//BUILD_TRIPLE_TEMPLATE(template void usualDot,  (const dim3 &blocksPerGrid, const dim3 &threadsPerBlock, hipStream_t *stream, const Nd4jLong length, const double alpha, const void* vX, const Nd4jLong incx, const void* vY, const Nd4jLong incy, const double beta, void* vZ), LIBND4J_TYPES, FLOAT_TYPES, FLOAT_TYPES);

}