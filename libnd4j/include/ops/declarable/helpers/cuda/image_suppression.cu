#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author sgazeos@gmail.com
//

#include <ops/declarable/helpers/image_suppression.h>
#include <NDArrayFactory.h>

namespace nd4j {
namespace ops {
namespace helpers {

    template <typename T>
    static __global__ void nonMaxSuppressionV2Kernel() {

    }

    template <typename T>
    static __device__ bool needToSuppressWithThreshold(T* boxes, Nd4jLong* boxesShape, int previousIndex, int nextIndex, T threshold) {
        Nd4jLong previous0[] = {previousIndex, 0};
        Nd4jLong previous1[] = {previousIndex, 1};
        Nd4jLong previous2[] = {previousIndex, 2};
        Nd4jLong previous3[] = {previousIndex, 3};
        Nd4jLong next0[] = {nextIndex, 0};
        Nd4jLong next1[] = {nextIndex, 1};
        Nd4jLong next2[] = {nextIndex, 2};
        Nd4jLong next3[] = {nextIndex, 3};

        T minYPrev = nd4j::math::nd4j_min(boxes[shape::getOffset(0, shape::shapeOf(boxesShape), shape::stride(boxesShape), previous0, 2)], boxes[shape::getOffset(0, shape::shapeOf(boxesShape), shape::stride(boxesShape), previous2, 2)]);
        T minXPrev = nd4j::math::nd4j_min(boxes[shape::getOffset(0, shape::shapeOf(boxesShape), shape::stride(boxesShape), previous1, 2)], boxes[shape::getOffset(0, shape::shapeOf(boxesShape), shape::stride(boxesShape), previous3, 2)]);
        T maxYPrev = nd4j::math::nd4j_max(boxes[shape::getOffset(0, shape::shapeOf(boxesShape), shape::stride(boxesShape), previous0, 2)], boxes[shape::getOffset(0, shape::shapeOf(boxesShape), shape::stride(boxesShape), previous2, 2)]);
        T maxXPrev = nd4j::math::nd4j_max(boxes[shape::getOffset(0, shape::shapeOf(boxesShape), shape::stride(boxesShape), previous1, 2)], boxes[shape::getOffset(0, shape::shapeOf(boxesShape), shape::stride(boxesShape), previous3, 2)]);
        T minYNext = nd4j::math::nd4j_min(boxes[shape::getOffset(0, shape::shapeOf(boxesShape), shape::stride(boxesShape), next0, 2)], boxes[shape::getOffset(0, shape::shapeOf(boxesShape), shape::stride(boxesShape), next2, 2)]);
        T minXNext = nd4j::math::nd4j_min(boxes[shape::getOffset(0, shape::shapeOf(boxesShape), shape::stride(boxesShape), next1, 2)], boxes[shape::getOffset(0, shape::shapeOf(boxesShape), shape::stride(boxesShape), next3, 2)]);
        T maxYNext = nd4j::math::nd4j_max(boxes[shape::getOffset(0, shape::shapeOf(boxesShape), shape::stride(boxesShape), next0, 2)], boxes[shape::getOffset(0, shape::shapeOf(boxesShape), shape::stride(boxesShape), next2, 2)]);
        T maxXNext = nd4j::math::nd4j_max(boxes[shape::getOffset(0, shape::shapeOf(boxesShape), shape::stride(boxesShape), next1, 2)], boxes[shape::getOffset(0, shape::shapeOf(boxesShape), shape::stride(boxesShape), next3, 2)]);

        T areaPrev = (maxYPrev - minYPrev) * (maxXPrev - minXPrev);
        T areaNext = (maxYNext - minYNext) * (maxXNext - minXNext);

        if (areaNext <= T(0.f) || areaPrev <= T(0.f)) return false;

        T minIntersectionY = nd4j::math::nd4j_max(minYPrev, minYNext);
        T minIntersectionX = nd4j::math::nd4j_max(minXPrev, minXNext);
        T maxIntersectionY = nd4j::math::nd4j_min(maxYPrev, maxYNext);
        T maxIntersectionX = nd4j::math::nd4j_min(maxXPrev, maxXNext);
        T intersectionArea =
                nd4j::math::nd4j_max(T(maxIntersectionY - minIntersectionY), T(0.0f)) *
                nd4j::math::nd4j_max(T(maxIntersectionX - minIntersectionX), T(0.0f));
        T intersectionValue = intersectionArea / (areaPrev + areaNext - intersectionArea);
        return intersectionValue > threshold;
    };

    template <typename T>
    static __device__ bool needToSelect(T* boxes, Nd4jLong* boxesShape, Nd4jLong* indices, int* selectedIndices, int current, int numSelected, T threshold) {
        bool shouldSelect = true;
        for (int j = numSelected - 1; j >= 0; --j) {
            if (needToSuppressWithThreshold<T>(boxes, boxesShape, indices[current], indices[selectedIndices[j]], threshold)) {
                shouldSelect = false;
                break;
            }
        }
        return shouldSelect;
    }

    template <typename T>
    static __global__ void nonMaxSuppressionKernel(T* boxes, Nd4jLong* boxesShape, Nd4jLong* indices, int* selected, int* selectedIndices, Nd4jLong numBoxes, T* output, Nd4jLong* outputShape, T threshold) {
        __shared__ bool canContinue;
        __shared__ int numSelected;
        __shared__ Nd4jLong outputLen;

        if (threadIdx.x == 0) {
            canContinue = true;
            numSelected = 0;
            outputLen = shape::length(outputShape);
        }
        __syncthreads();

        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = blockDim.x * gridDim.x;

        for (int i = start; i < numBoxes && canContinue; i += step) {
            //if (selected.size() >= output->lengthOf()) break;
            bool shouldSelect = needToSelect<T>(boxes, boxesShape, indices, selectedIndices, i, numSelected, threshold);
            // Overlapping boxes are likely to have similar scores,
            // therefore we iterate through the selected boxes backwards.

            if (shouldSelect) {
                selected[numSelected] = indices[i];
                output[numSelected] = indices[i];
                selectedIndices[numSelected++] = i;
            }

            if (numSelected == outputLen) {
                canContinue = false;
                break;
            }
        }
    }

    template <typename T>
    static void nonMaxSuppressionV2_(nd4j::LaunchContext* context, NDArray* boxes, NDArray* scales, int maxSize, double threshold, NDArray* output) {
        NDArray indices = NDArrayFactory::create<Nd4jLong>({scales->lengthOf()});
        indices.linspace(0);
        // TO DO: sort indices using scales as value row
        //std::sort(indices.begin(), indices.end(), [scales](int i, int j) {return scales->e<T>(i) > scales->e<T>(j);});

        NDArray selected = NDArrayFactory::create<int>({output->lengthOf()});

        NDArray selectedIndices = NDArrayFactory::create<int>({output->lengthOf()});
        int numSelected = 0;
        int numBoxes = boxes->sizeAt(0);
        auto stream = context->getCudaStream();
        T* boxesBuf = reinterpret_cast<T*>(boxes->specialBuffer());
        Nd4jLong* indicesData = reinterpret_cast<Nd4jLong*>(indices.specialBuffer());
        int* selectedData = reinterpret_cast<int*>(selected.specialBuffer());
        int* selectedIndicesData = reinterpret_cast<int*>(selectedIndices.specialBuffer());
        T* outputBuf = reinterpret_cast<T*>(output->specialBuffer());
        nonMaxSuppressionKernel<T><<<1, 512, 1024, *stream>>>(boxesBuf, boxes->specialShapeInfo(), indicesData, selectedData, selectedIndicesData, numBoxes, outputBuf, output->specialShapeInfo(), T(threshold));
//        for (int i = 0; i < boxes->sizeAt(0); ++i) {
//            if (selected.size() >= output->lengthOf()) break;
//            bool shouldSelect = true;
//            // Overlapping boxes are likely to have similar scores,
//            // therefore we iterate through the selected boxes backwards.
//            for (int j = numSelected - 1; j >= 0; --j) {
//                if (needToSuppressWithThreshold(*boxes, indices[i], indices[selectedIndices[j]], T(threshold)) {
//                    shouldSelect = false;
//                    break;
//                }
//            }
//            if (shouldSelect) {
//                selected.push_back(indices[i]);
//                selectedIndices[numSelected++] = i;
//            }
//        }
//        for (size_t e = 0; e < selected.size(); ++e)
//            output->p<int>(e, selected[e]);
//
    }

    void nonMaxSuppressionV2(nd4j::LaunchContext * context, NDArray* boxes, NDArray* scales, int maxSize, double threshold, NDArray* output) {
        BUILD_SINGLE_SELECTOR(output->dataType(), nonMaxSuppressionV2_, (context, boxes, scales, maxSize, threshold, output), NUMERIC_TYPES);
    }
    BUILD_SINGLE_TEMPLATE(template void nonMaxSuppressionV2_, (nd4j::LaunchContext * context, NDArray* boxes, NDArray* scales, int maxSize, double threshold, NDArray* output), NUMERIC_TYPES);

}
}
}