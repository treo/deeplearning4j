#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author raver119@gmail.com
//

#include <ops/declarable/helpers/roll.h>
#include <helpers/ConstantTadHelper.h>
#include <helpers/PointersManager.h>

namespace nd4j {
namespace ops {
namespace helpers {

    template <typename T>
    static void _CUDA_D rollKernelLinearStage1Dev(void *vx, Nd4jLong *xShapeInfo, void *vz, Nd4jLong *zShapeInfo, Nd4jLong fullLength, int actualShift) {
        auto x = reinterpret_cast<T*>(vx);
        auto z = reinterpret_cast<T*>(vz);

        auto xEws = shape::elementWiseStride(xShapeInfo);
        auto zEws = shape::elementWiseStride(zShapeInfo);

        auto xOrder = shape::order(xShapeInfo);
        auto zOrder = shape::order(zShapeInfo);

        auto tid = threadIdx.x + blockIdx.x * blockDim.x;

        if (xEws > 0 && zEws > 0 && xOrder == zOrder) {
            for (int i = tid; i < actualShift; i += blockDim.x * gridDim.x) {
                int sourceIndex = fullLength - actualShift + i;

                auto eA = x[sourceIndex * xEws];
                auto eB = x[i * xEws];

                z[i * zEws] = eA;
                z[sourceIndex * zEws] = eB;
            }
        } else {
            for (int i = tid; i < actualShift; i += blockDim.x * gridDim.x) {
                int sourceIndex = fullLength - actualShift + i;

                auto xOffsetA = shape::getIndexOffset(i, xShapeInfo, fullLength);
                auto xOffsetB = shape::getIndexOffset(sourceIndex, xShapeInfo, fullLength);

                auto zOffsetA = shape::getIndexOffset(i, zShapeInfo, fullLength);
                auto zOffsetB = shape::getIndexOffset(sourceIndex, zShapeInfo, fullLength);

                auto eA = x[xOffsetA];
                auto eB = x[xOffsetB];

                z[zOffsetA] = eB;
                z[zOffsetB] = eA;
            }
        }
    }

    template <typename T>
    static void _CUDA_G rollKernelLinearStage1(void *vx, Nd4jLong *xShapeInfo, void *vz, Nd4jLong *zShapeInfo, Nd4jLong fullLength, int actualShift) {
        rollKernelLinearStage1Dev<T>(vx, xShapeInfo, vz, zShapeInfo, fullLength, actualShift);
    }

    template <typename T>
    static void _CUDA_G rollKernelLinearStage2(void *vx, Nd4jLong *xShapeInfo, void *vz, Nd4jLong *zShapeInfo, Nd4jLong fullLength, int actualShift, int shiftCount) {
        auto x = reinterpret_cast<T*>(vx);
        auto z = reinterpret_cast<T*>(vz);

        auto xEws = shape::elementWiseStride(xShapeInfo);
        auto zEws = shape::elementWiseStride(zShapeInfo);

        auto xOrder = shape::order(xShapeInfo);
        auto zOrder = shape::order(zShapeInfo);

        auto tid = threadIdx.x + blockIdx.x * blockDim.x;

        if (xEws > 0 && zEws > 0 && xOrder == zOrder) {
            for (int count = 1; count < shiftCount; ++count) {
                for (int i = tid; i < actualShift; i += blockDim.x * gridDim.x) {
                    int destinationIndex = fullLength - (count + 1) * actualShift + i;
                    int sourceIndex = fullLength - count * actualShift + i;

                    auto eA = x[sourceIndex * xEws];
                    auto eB = x[destinationIndex * xEws];

                    z[destinationIndex * zEws] = eA;
                    z[sourceIndex * zEws] = eB;
                }

                __syncthreads();
            }
        } else {
            for (int count = 1; count < shiftCount; ++count) {
                for (int i = tid; i < actualShift; i += blockDim.x * gridDim.x) {
                    int destinationIndex = fullLength - (count + 1) * actualShift + i;
                    int sourceIndex = fullLength - count * actualShift + i;

                    auto xOffsetA = shape::getIndexOffset(destinationIndex, xShapeInfo, fullLength);
                    auto xOffsetB = shape::getIndexOffset(sourceIndex, xShapeInfo, fullLength);

                    auto zOffsetA = shape::getIndexOffset(destinationIndex, zShapeInfo, fullLength);
                    auto zOffsetB = shape::getIndexOffset(sourceIndex, zShapeInfo, fullLength);

                    auto eA = x[xOffsetA];
                    auto eB = x[xOffsetB];

                    z[zOffsetA] = eB;
                    z[zOffsetB] = eA;
                }

                __syncthreads();
            }
        }
    }

    template <typename T>
    static void _CUDA_G rollKernelLinearStage3(void *vx, Nd4jLong *xShapeInfo, void *vz, Nd4jLong *zShapeInfo, Nd4jLong fullLength, int actualShift, int remainShift) {
        auto x = reinterpret_cast<T*>(vx);
        auto z = reinterpret_cast<T*>(vz);

        auto xEws = shape::elementWiseStride(xShapeInfo);
        auto zEws = shape::elementWiseStride(zShapeInfo);

        auto xOrder = shape::order(xShapeInfo);
        auto zOrder = shape::order(zShapeInfo);

        auto tid = threadIdx.x + blockIdx.x * blockDim.x;

        if (xEws > 0 && zEws > 0 && xOrder == zOrder) {
            for (int i = tid ; i < actualShift; i += blockDim.x * gridDim.x) {
                int remainIdx = i + actualShift;
                int sourceIndex = remainIdx + remainShift;

                auto eA = x[sourceIndex * xEws];
                auto eB = x[remainIdx * xEws];

                z[remainIdx * zEws] = eA;
                z[sourceIndex * zEws] = eB;
            }
        } else {
            for (int i = tid; i < actualShift; i += blockDim.x * gridDim.x) {
                int remainIdx = i + actualShift;
                int sourceIndex = remainIdx + remainShift;

                auto xOffsetA = shape::getIndexOffset(remainIdx, xShapeInfo, fullLength);
                auto xOffsetB = shape::getIndexOffset(sourceIndex, xShapeInfo, fullLength);

                auto zOffsetA = shape::getIndexOffset(remainIdx, zShapeInfo, fullLength);
                auto zOffsetB = shape::getIndexOffset(sourceIndex, zShapeInfo, fullLength);

                auto eA = x[xOffsetA];
                auto eB = x[xOffsetB];

                z[zOffsetA] = eB;
                z[zOffsetB] = eA;
            }
        }
    }

    template <typename T>
    static void _CUDA_D swapTadsKernel(void *vx, void *vz, Nd4jLong *zShapeInfo, Nd4jLong tadLength) {
        auto x = reinterpret_cast<T*>(vx);
        auto z = reinterpret_cast<T*>(vz);

        auto zEws = shape::elementWiseStride(zShapeInfo);

        auto zOrder = shape::order(zShapeInfo);

        auto tid = threadIdx.x + blockIdx.x * blockDim.x;

        if (zEws > 0) {
            for (int e = threadIdx.x; e < tadLength; e += blockDim.x) {
                auto eA = x[e * zEws];
                auto eB = z[e * zEws];

                x[e * zEws] = eB;
                z[e * zEws] = eA;
            }
        } else {
            for (int e = threadIdx.x; e < tadLength; e += blockDim.x) {
                auto zOffset = shape::getIndexOffset(e, zShapeInfo, tadLength);

                auto eA = x[zOffset];
                auto eB = z[zOffset];

                x[zOffset] = eB;
                z[zOffset] = eA;
            }
        }
    }

    template <typename T>
    static void _CUDA_G rollKernelFullAnyDimensionStage1(void *vx, Nd4jLong *xTadShapeInfo, Nd4jLong *xTadOffsets, void *vz, Nd4jLong *zTadShapeInfo, Nd4jLong *zTadOffsets, int numTads, Nd4jLong tadLength, int dim, Nd4jLong sizeAt, int theShift) {
        auto x = reinterpret_cast<T *>(vx);
        auto z = reinterpret_cast<T *>(vz);

        for (int e = blockIdx.x + theShift; e < sizeAt - theShift; e += gridDim.x) {
            int sourceIndex = dim * sizeAt + e - theShift;
            int targetIndex = dim * sizeAt + e;

            swapTadsKernel<T>(z + xTadOffsets[sourceIndex], z + xTadOffsets[targetIndex], zTadShapeInfo, tadLength);
        }
    }

    template <typename T>
    static void _CUDA_G rollKernelFullAnyDimensionStage2(void *vx, Nd4jLong *xTadShapeInfo, Nd4jLong *xTadOffsets, void *vz, Nd4jLong *zTadShapeInfo, Nd4jLong *zTadOffsets, int numTads, Nd4jLong tadLength, int dim, Nd4jLong sizeAt, int theShift) {
        auto x = reinterpret_cast<T *>(vx);
        auto z = reinterpret_cast<T *>(vz);

        for (int e = blockIdx.x; e < theShift; e += gridDim.x) {
            int sourceIndex = dim * sizeAt + sizeAt - theShift + e;
            int targetIndex = dim * sizeAt + e;

            swapTadsKernel<T>(z + zTadOffsets[sourceIndex], z + zTadOffsets[targetIndex], zTadShapeInfo, tadLength);
        }
    }

    template <typename T>
    static void rollFunctorFull_(NDArray* input, NDArray* output, int shift, std::vector<int> const& axis, bool inplace){
        if (!inplace)
            output->assign(input);

        for (int axe: axis) {
            if (axe == input->rankOf() - 1) { // last dimension
                std::unique_ptr<ResultSet> listOfTensors(output->allTensorsAlongDimension({axe}));
                std::unique_ptr<ResultSet> listOfOutTensors(output->allTensorsAlongDimension({axe}));
                int fullLen = listOfTensors->size();
                int theShift = shift;
                if (theShift > 0) {
                    theShift %= fullLen;
                }
                else {
                    theShift -= fullLen * (theShift / fullLen - 1);
                }
                for (int k = 0; k < fullLen; k++) {
                    rollFunctorLinear(output->getContext(), listOfTensors->at(k), listOfOutTensors->at(k), theShift, true);
                }
            } else {
                std::vector<int> dims(input->rankOf() - axe - 1);
                for (int i = 0; i < dims.size(); ++i)
                    dims[i] = axe + 1 + i;

                auto packZ = ConstantTadHelper::getInstance()->tadForDimensions(output->shapeInfo(), dims);

                int numTads = packZ.numberOfTads();
                int sizeAt = input->sizeAt(axe);
                auto tadLength = shape::length(packZ.primaryShapeInfo());

                int theShift = shift;

                if (theShift > 0)
                    theShift %= sizeAt;
                else
                    theShift -= sizeAt * (theShift / sizeAt - 1);

                if (theShift) {
                    for (int dim = 0; dim < numTads / sizeAt; ++dim) {

                        rollKernelFullAnyDimensionStage1<T><<<1, 256, 1024, *(output->getContext()->getCudaStream())>>>(output->specialBuffer(), packZ.platformShapeInfo(), packZ.platformOffsets(), output->specialBuffer(), packZ.platformShapeInfo(), packZ.platformOffsets(), numTads, tadLength, dim, sizeAt, theShift);

                        rollKernelFullAnyDimensionStage2<T><<<1, 256, 1024, *(output->getContext()->getCudaStream())>>>(output->specialBuffer(), packZ.platformShapeInfo(), packZ.platformOffsets(), output->specialBuffer(), packZ.platformShapeInfo(), packZ.platformOffsets(), numTads, tadLength, dim, sizeAt, theShift);
                    }
                }
            }
        }
    }

    template <typename T>
    static void rollFunctorLinear_(NDArray* input, NDArray* output, int shift, bool inplace){
        if (!inplace)
            output->assign(input);

        auto fullLen = input->lengthOf();
        int actualShift = shift; // % fullLen; // shift already non-negative then
        if (actualShift < 0) {
            actualShift -= fullLen * (actualShift / fullLen - 1);
        }
        else
            actualShift %= fullLen;

        if (actualShift) {
            int shiftCount = fullLen / actualShift - 1;
            int remainShift = fullLen % actualShift;

            // stage 1) swap last actualShift elements with first ones.
            rollKernelLinearStage1<T><<<1, 1, 1024, *(output->getContext()->getCudaStream())>>>(output->specialBuffer(), output->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(), fullLen, actualShift);

            // stage 2) swap swapped actualShift elements with rest remainShiftCount times.
            rollKernelLinearStage2<T><<<1, 1, 1024, *(output->getContext()->getCudaStream())>>>(output->specialBuffer(), output->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(), fullLen, actualShift, shiftCount);

            // FIXME: no parallelism here :(
            // stage 3) swap remainer of items.
            if (remainShift && shiftCount)
                rollKernelLinearStage3<T><<<1, 1, 1024, *(output->getContext()->getCudaStream())>>>(output->specialBuffer(), output->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(), fullLen, actualShift, remainShift);
        }
    }

    void rollFunctorFull(nd4j::LaunchContext * context, NDArray* input, NDArray* output, int shift, std::vector<int> const& axis, bool inplace){
        input->syncToDevice();

        BUILD_SINGLE_SELECTOR(input->dataType(), rollFunctorFull_, (input, output, shift, axis, inplace), LIBND4J_TYPES);

        output->tickWriteDevice();
    }

    void rollFunctorLinear(nd4j::LaunchContext * context, NDArray* input, NDArray* output, int shift, bool inplace){
        input->syncToDevice();

        BUILD_SINGLE_SELECTOR(input->dataType(), rollFunctorLinear_, (input, output, shift, inplace), LIBND4J_TYPES);

        output->tickWriteDevice();
    }

    BUILD_SINGLE_TEMPLATE(template void rollFunctorLinear_, (NDArray* input, NDArray* output, int shift, bool inplace), LIBND4J_TYPES);
    BUILD_SINGLE_TEMPLATE(template void rollFunctorFull_, (NDArray* input, NDArray* output, int shift, std::vector<int> const& axis, bool inplace), LIBND4J_TYPES);
}
}
}