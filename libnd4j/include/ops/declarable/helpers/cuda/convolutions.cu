#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

#include <ops/declarable/helpers/convolutions.h>
#include <ops/declarable/helpers/im2col.h>
#include <ops/declarable/helpers/col2im.h>
#include <exceptions/cuda_exception.h>
#include <NDArrayFactory.h>
#include <MmulHelper.h>
#include <PointersManager.h>

namespace nd4j {
namespace ops  {

//////////////////////////////////////////////////////////////////////////
// vol [bS, iC, iD, iH, iW] is convoluted to col [bS, iC, kD, kH, kW, oD, oH, oW]
template <typename T>
static __global__ void vol2colCuda(const void* volume, const Nd4jLong* volShapeInfo, void* column, const Nd4jLong* colShapeInfo,  const int sD, const int sH, const int sW, const int pD, const int pH, const int pW, const int dD, const int dH, const int dW) {

    const T* vol = reinterpret_cast<const T*>(volume);
          T* col = reinterpret_cast<T*>(column);

    __shared__ int colRank, volRank;
    __shared__ Nd4jLong colLen, iD, iH, iW;
    __shared__ Nd4jLong *sharedMem;

    if (threadIdx.x == 0) {

        extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<Nd4jLong*>(shmem);

        volRank = 5;
        colRank = 8;

        colLen = shape::length(colShapeInfo);

        iD = volShapeInfo[3];
        iH = volShapeInfo[4];
        iW = volShapeInfo[5];
    }

    __syncthreads();

    const auto colInd = threadIdx.x + blockIdx.x * blockDim.x;

    if(colInd >= colLen)
        return;

    auto coords = sharedMem + threadIdx.x * colRank;

    shape::index2coords(colRank, colShapeInfo + 1, colInd, colLen, coords);

    // const auto colW = coords[7];
    // const auto colH = coords[6];
    // const auto colD = coords[5];
    // const auto kCol = coords[4];
    // const auto kRow = coords[3];
    // const auto kDep = coords[2];
    // const auto c    = coords[1];
    // const auto b    = coords[0];

    const auto colOffset = shape::getOffset(0, colShapeInfo + 1, colShapeInfo + colRank + 1, coords, colRank);

    coords[2] = (-pD + coords[2] * dD) + coords[5] * sD;     // const auto volDep = (-pD + kDep * dD) + colD * sD;
    coords[3] = (-pH + coords[3] * dH) + coords[6] * sH;     // const auto volRow = (-pH + kRow * dH) + colH * sH;
    coords[4] = (-pW + coords[4] * dW) + coords[7] * sW;     // const auto volCol = (-pW + kCol * dW) + colW * sW;

    if (static_cast<unsigned>(coords[2]) >= static_cast<unsigned>(iD) || static_cast<unsigned>(coords[3]) >= static_cast<unsigned>(iH) || static_cast<unsigned>(coords[4]) >= static_cast<unsigned>(iW))
        col[colOffset] = static_cast<T>(0.);
    else
        col[colOffset] = vol[shape::getOffset(0, volShapeInfo + 1, volShapeInfo + volRank + 1, coords, volRank)];
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
static void vol2colCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                                const void* volume, const Nd4jLong* volShapeInfo,
                                      void* column, const Nd4jLong* colShapeInfo,
                                const int sD, const int sH, const int sW, const int pD, const int pH, const int pW, const int dD, const int dH, const int dW) {

    vol2colCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(volume, volShapeInfo, column, colShapeInfo,  sD, sH, sW, pD, pH, pW, dD, dH, dW);
}
BUILD_SINGLE_TEMPLATE(template void vol2colCudaLauncher, (const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t* stream, const void *vol, const Nd4jLong *volShapeInfo, void *col, const Nd4jLong *colShapeInfo, const int sD, const int sH, const int sW, const int pD, const int pH, const int pW, const int dD, const int dH, const int dW), FLOAT_TYPES);

//////////////////////////////////////////////////////////////////////////
void ConvolutionUtils::vol2col(nd4j::graph::Context& block, const NDArray& vol, NDArray& col, const int sD, const int sH, const int sW, const int pD, const int pH, const int pW, const int dD, const int dH, const int dW) {

    PointersManager manager(block.launchContext(), "vol2col");

    const int threadsPerBlock = MAX_NUM_THREADS / 4;
    const int blocksPerGrid = (col.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
    const int sharedMem = col.rankOf() * sizeof(Nd4jLong) * threadsPerBlock  + 128;

    NDArray::prepareSpecialUse({&col}, {&vol});
    BUILD_SINGLE_SELECTOR(vol.dataType(), vol2colCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, block.launchContext()->getCudaStream(), vol.getSpecialBuffer(), vol.getSpecialShapeInfo(), col.specialBuffer(), col.specialShapeInfo(), sD, sH, sW, pD, pH, pW, dD, dH, dW), FLOAT_TYPES);
    NDArray::registerSpecialUse({&col}, {&vol});

    manager.synchronize();
}


        void ConvolutionUtils::conv2d(nd4j::graph::Context & block, const NDArray* input, const NDArray* weights, const NDArray* bias, NDArray* output, const int kH, const int kW, const int sH, const int sW, int pH, int pW, const int dH, const int dW, const int isSameMode, const int isNCHW) {

        }

        void ConvolutionUtils::conv2d(nd4j::graph::Context & block, const std::vector<NDArray*>& inArrs, NDArray* output, const std::vector<int>& intArgs) {

        }

        void ConvolutionUtils::conv2dBP(nd4j::graph::Context & block, const std::vector<NDArray*>& inArrs, const std::vector<NDArray*>& outArrs, const std::vector<int>& intArgs) {

        }

        void ConvolutionUtils::conv2dBP(nd4j::graph::Context & block, const NDArray* input, const NDArray* weights, const NDArray* bias, const NDArray* gradO, NDArray* gradI, NDArray* gradW, NDArray* gradB, const int kH, const int kW, const int sH, const int sW, int pH, int pW, const int dH, const int dW, const int isSameMode, const int isNCHW) {

        }

        void ConvolutionUtils::depthwiseConv2d(nd4j::graph::Context & block, const NDArray* input, const NDArray* weights, const NDArray* bias, NDArray* output, const int kH, const int kW, const int sH, const int sW, int pH, int pW, const int dH, const int dW, const int isSameMode, const int isNCHW) {

        }

        void ConvolutionUtils::depthwiseConv2dBP(nd4j::graph::Context & block, const NDArray* input, const NDArray* weights, const NDArray* bias, const NDArray* gradO, NDArray* gradI, NDArray* gradW, NDArray* gradB, const int kH, const int kW, const int sH, const int sW, int pH, int pW, const int dH, const int dW, const int isSameMode, const int isNCHW) {

        }

        void ConvolutionUtils::sconv2d(nd4j::graph::Context & block, const NDArray* input, const NDArray* weightsDepth, const NDArray* weightsPoint, const NDArray* bias,  NDArray* output, const int kH, const int kW, const int sH, const int sW, int pH, int pW, const int dH, const int dW, const int isSameMode, const int isNCHW) {

        }



        void ConvolutionUtils::col2vol(nd4j::graph::Context & block, const NDArray& col, NDArray& vol, const int sD, const int sH, const int sW, const int pD, const int pH, const int pW, const int dD, const int dH, const int dW) {

        }

        void ConvolutionUtils::upsampling2d(nd4j::graph::Context & block, const NDArray& input, NDArray& output, const int factorH, const int factorW, const bool isNCHW) {

        }

        void ConvolutionUtils::upsampling3d(nd4j::graph::Context & block, const NDArray& input, NDArray& output, const int factorD, const int factorH, const int factorW, const bool isNCDHW) {

        }

        void ConvolutionUtils::upsampling2dBP(nd4j::graph::Context & block, const NDArray& gradO, NDArray& gradI, const bool isNCHW) {

        }

        void ConvolutionUtils::upsampling3dBP(nd4j::graph::Context & block, const NDArray& gradO, NDArray& gradI, const bool isNCDHW) {

        }

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Z>
static __global__ void avgPooling2dCuda(const void *vx, const Nd4jLong *xShapeInfo, void *vz, const Nd4jLong *zShapeInfo, const int kH, const int kW, const int sH, const int sW, const int pH, const int pW, const int dH, const int dW, const int extraParam0) {

    // input is  [bS, iC, iH, iW]
    // output is [bS, iC, oH, oW]

    const auto x = reinterpret_cast<const X*>(vx);
          auto z = reinterpret_cast<Z*>(vz);

    __shared__ int bS, iC, oH, oW, iH, iW, strideB, strideC, strideY, strideX, strideOB, strideOC, strideOY, strideOX, length, kHEff, kWEff;

    if (threadIdx.x == 0) {

        bS = shape::sizeAt(xShapeInfo, 0);
        iC = shape::sizeAt(xShapeInfo, 1);
        oH = shape::sizeAt(zShapeInfo, 2);
        oW = shape::sizeAt(zShapeInfo, 3);
        iH = shape::sizeAt(xShapeInfo, 2);
        iW = shape::sizeAt(xShapeInfo, 3);

        strideB = shape::stride(xShapeInfo)[0];
        strideC = shape::stride(xShapeInfo)[1];
        strideY = shape::stride(xShapeInfo)[2];
        strideX = shape::stride(xShapeInfo)[3];

        strideOB = shape::stride(zShapeInfo)[0];
        strideOC = shape::stride(zShapeInfo)[1];
        strideOY = shape::stride(zShapeInfo)[2];
        strideOX = shape::stride(zShapeInfo)[3];

        length = shape::length(zShapeInfo);

        //Replace kernel H/W with *effective* kernel H/W accounting for dilatyon
        kHEff = kH + (kH-1)*(dH-1);
        kWEff = kW + (kW-1)*(dW-1);
    }

    __syncthreads();

    int tid = blockIdx.x * gridDim.x + threadIdx.x;

    for (int index = tid; index < length; index += blockDim.x * gridDim.x) {

        const int pw = index % oW;
        const int ph = (index / oW) % oH;
        const int c = (index / oW / oH) % iC;
        const int n = index / oW / oH / iC;

        int hstart = sH * ph - pH;
        int wstart = sW * pw - pW;
        int hend = hstart + kHEff;
        int wend = wstart + kWEff;

        if(hstart < 0){
            int f = nd4j::math::nd4j_ceil<Z,int>((Z) -hstart / (Z)dH);
            hstart += f * dH;
        }
        if(wstart < 0){
            int f = nd4j::math::nd4j_ceil<Z,int>((Z) -wstart / (Z) dW);
            wstart += f * dW;
        }
        if(hend > iH){
            int f = nd4j::math::nd4j_ceil<Z,int>((Z) (hend-iH) / (Z) dH);
            hend -= f * dH;
        }
        if(wend > iW){
            int f = nd4j::math::nd4j_ceil<Z,int>((Z) (wend-iW) / (Z) dW);
            wend -= f * dW;
        }

        //Accounts for dilation
        int pool_size = nd4j::math::nd4j_ceil<double,int>((double) (hend-hstart) / (double) dH) * nd4j::math::nd4j_ceil<double,int>((double) (wend-wstart) / (double) dW);

        Z sum = 0.0f;

        const X *inSlice = x + (n * strideB + c * strideC);

        for (int h = hstart; h < hend; h += dH)
            for (int w = wstart; w < wend; w += dW)
                sum += static_cast<Z>(inSlice[h * strideY + w * strideX]);

        int divide_factor = pool_size;  //Case 0: exclude padding
        if (extraParam0 == 1)     //Case 1: include padding
            divide_factor = kH * kW;

        z[n * strideOB + c * strideOC + pw * strideOX + ph * strideOY] = sum / static_cast<Z>(divide_factor);
    }
}

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Z>
static void avgPooling2dCudaLauncher(nd4j::LaunchContext & block, void *vx, Nd4jLong *vxShapeInfo, void *vz, Nd4jLong *vzShapeInfo, const int kH, const int kW, const int sH, const int sW, const int pH, const int pW, const int dH, const int dW, const int extraParam0) {
    avgPooling2dCuda<X, Z><<<512, 512, 4192, *block.getCudaStream()>>>(vx, vxShapeInfo, vz, vzShapeInfo, kH, kW, sH, sW, pH, pW, dH, dW, extraParam0);
}

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Z>
static __global__ void pnormPooling2dCuda(const void *vx, const Nd4jLong *xShapeInfo, void *vz, const Nd4jLong *zShapeInfo, const int kH, const int kW, const int sH, const int sW, const int pH, const int pW, const int dH, const int dW, const int extraParam0) {

    // input is  [bS, iC, iH, iW]
    // output is [bS, iC, oH, oW]

    const auto x = reinterpret_cast<const X*>(vx);
          auto z = reinterpret_cast<Z*>(vz);

    __shared__ int bS, iC, oH, oW, iH, iW, strideB, strideC, strideY, strideX, strideOB, strideOC, strideOY, strideOX, length, kHEff, kWEff;
    __shared__ bool fOrder;

    if (threadIdx.x == 0) {

        bS = shape::sizeAt(xShapeInfo, 0);
        iC = shape::sizeAt(xShapeInfo, 1);
        oH = shape::sizeAt(zShapeInfo, 2);
        oW = shape::sizeAt(zShapeInfo, 3);
        iH = shape::sizeAt(xShapeInfo, 2);
        iW = shape::sizeAt(xShapeInfo, 3);

        strideB = shape::stride(xShapeInfo)[0];
        strideC = shape::stride(xShapeInfo)[1];
        strideY = shape::stride(xShapeInfo)[2];
        strideX = shape::stride(xShapeInfo)[3];

        strideOB = shape::stride(zShapeInfo)[0];
        strideOC = shape::stride(zShapeInfo)[1];
        strideOY = shape::stride(zShapeInfo)[2];
        strideOX = shape::stride(zShapeInfo)[3];

        length = shape::length(zShapeInfo);

        //Replace kernel H/W with *effective* kernel H/W accounting for dilatyon
        kHEff = kH + (kH-1)*(dH-1);
        kWEff = kW + (kW-1)*(dW-1);
    }

    __syncthreads();

    int tid = blockIdx.x * gridDim.x + threadIdx.x;

    for (int index = tid; index < length; index += blockDim.x * gridDim.x) {

        const int pw = index % oW;
        const int ph = (index / oW) % oH;
        const int c = (index / oW / oH) % iC;
        const int n = index / oW / oH / iC;

        int hstart = sH * ph - pH;
        int wstart = sW * pw - pW;
        int hend = hstart + kHEff;
        int wend = wstart + kWEff;

        if (hstart < 0) {
            int f = nd4j::math::nd4j_ceil<Z, int>((Z) -hstart / (Z) dH);
            hstart += f * dH;
        }
        if (wstart < 0) {
            int f = nd4j::math::nd4j_ceil<Z, int>((Z) -wstart / (Z) dW);
            wstart += f * dW;
        }
        if (hend > iH) {
            int f = nd4j::math::nd4j_ceil<Z, int>((Z) (hend - iH) / (Z) dH);
            hend -= f * dH;
        }
        if (wend > iW) {
            int f = nd4j::math::nd4j_ceil<Z, int>((Z) (wend - iW) / (Z) dW);
            wend -= f * dW;
        }
        //Accounts for dilation
        int pool_size = nd4j::math::nd4j_ceil<double, int>((double) (hend - hstart) / (double) dH) *
                        nd4j::math::nd4j_ceil<double, int>((double) (wend - wstart) / (double) dW);

        Z sum = 0.f;

        const X *inSlice = x + (n * strideB + c * strideC);

        for (int h = hstart; h < hend; h += dH)
            for (int w = wstart; w < wend; w += dW)
                sum += nd4j::math::nd4j_pow<Z, Z, Z>(static_cast<Z>(nd4j::math::nd4j_abs<X>(inSlice[h * strideY + w * strideX])), extraParam0);

        z[n * strideOB + c * strideOC + pw * strideOX + ph * strideOY] = nd4j::math::nd4j_pow<Z, Z, Z>(sum, (Z) 1.0f / extraParam0);
    }
}

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Z>
static void pnormPooling2dCudaLauncher(nd4j::LaunchContext & block, void *vx, Nd4jLong *vxShapeInfo, void *vz, Nd4jLong *vzShapeInfo, const int kH, const int kW, const int sH, const int sW, const int pH, const int pW, const int dH, const int dW, const int extraParam0) {
    pnormPooling2dCuda<X, Z><<<512, 512, 4192, *block.getCudaStream()>>>(vx, vxShapeInfo, vz, vzShapeInfo, kH, kW, sH, sW, pH, pW, dH, dW, extraParam0);
}

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Z>
static __global__ void maxPooling2dCuda(const void *vx, const Nd4jLong *xShapeInfo, void *vz, const Nd4jLong *zShapeInfo, const int kH, const int kW, const int sH, const int sW, const int pH, const int pW, const int dH, const int dW, const int extraParam0) {

    // input is  [bS, iC, iH, iW]
    // output is [bS, iC, oH, oW]

    const auto x = reinterpret_cast<const X*>(vx);
          auto z = reinterpret_cast<Z*>(vz);

    __shared__ int bS, iC, oH, oW, iH, iW, strideB, strideC, strideY, strideX, strideOB, strideOC, strideOY, strideOX, length, kHEff, kWEff;
    __shared__ bool fOrder;

    if (threadIdx.x == 0) {

        bS = shape::sizeAt(xShapeInfo, 0);
        iC = shape::sizeAt(xShapeInfo, 1);
        oH = shape::sizeAt(zShapeInfo, 2);
        oW = shape::sizeAt(zShapeInfo, 3);
        iH = shape::sizeAt(xShapeInfo, 2);
        iW = shape::sizeAt(xShapeInfo, 3);

        strideB = shape::stride(xShapeInfo)[0];
        strideC = shape::stride(xShapeInfo)[1];
        strideY = shape::stride(xShapeInfo)[2];
        strideX = shape::stride(xShapeInfo)[3];

        strideOB = shape::stride(zShapeInfo)[0];
        strideOC = shape::stride(zShapeInfo)[1];
        strideOY = shape::stride(zShapeInfo)[2];
        strideOX = shape::stride(zShapeInfo)[3];

        length = shape::length(zShapeInfo);

        //Replace kernel H/W with *effective* kernel H/W accounting for dilatyon
        kHEff = kH + (kH-1)*(dH-1);
        kWEff = kW + (kW-1)*(dW-1);
    }

    __syncthreads();

    int tid = blockIdx.x * gridDim.x + threadIdx.x;

    for (int index = tid; index < length; index += blockDim.x * gridDim.x) {

        const int pw = index % oW;
        const int ph = (index / oW) % oH;
        const int c = (index / oW / oH) % iC;
        const int n = index / oW / oH / iC;

        int hstart = sH * ph - pH;
        int wstart = sW * pw - pW;
        int hend = hstart + kHEff;
        int wend = wstart + kWEff;

        if(hstart < 0){
            int f = nd4j::math::nd4j_ceil<Z,int>((Z) -hstart / (Z)dH);
            hstart += f * dH;
        }
        if(wstart < 0){
            int f = nd4j::math::nd4j_ceil<Z,int>((Z) -wstart / (Z) dW);
            wstart += f * dW;
        }
        if(hend > iH){
            int f = nd4j::math::nd4j_ceil<Z,int>((Z) (hend-iH) / (Z) dH);
            hend -= f * dH;
        }
        if(wend > iW){
            int f = nd4j::math::nd4j_ceil<Z,int>((Z) (wend-iW) / (Z) dW);
            wend -= f * dW;
        }
        //Accounts for dilation
        int pool_size = nd4j::math::nd4j_ceil<double,int>((double) (hend-hstart) / (double) dH) * nd4j::math::nd4j_ceil<double,int>((double) (wend-wstart) / (double) dW);

        Z max = -nd4j::DataTypeUtils::max<Z>();

        const X *inSlice = x + (n * strideB + c * strideC);

        for (int h = hstart; h < hend; h += dH) {
            for (int w = wstart; w < wend; w += dW) {
                Z v = static_cast<Z>(inSlice[h * strideY + w * strideX]);
                if (v > max)
                    max = v;
            }
        }

        z[n * strideOB + c * strideOC + pw * strideOX + ph * strideOY] = max;
    }
}

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Z>
static void maxPooling2dCudaLauncher(nd4j::LaunchContext & block, void *vx, Nd4jLong *vxShapeInfo, void *vz, Nd4jLong *vzShapeInfo, const int kH, const int kW, const int sH, const int sW, const int pH, const int pW, const int dH, const int dW, const int extraParam0) {
    maxPooling2dCuda<X,Z><<<512, 512, 4192, *block.getCudaStream()>>>(vx, vxShapeInfo, vz, vzShapeInfo, kH, kW, sH, sW, pH, pW, dH, dW, extraParam0);
}

//////////////////////////////////////////////////////////////////////////
void ConvolutionUtils::pooling2d(nd4j::graph::Context& block, const NDArray& input, NDArray& output, const int kH, const int kW, const int sH, const int sW, const int pH, const int pW, const int dH, const int dW, const PoolingType poolingMode, const int extraParam0) {

    if(!input.isActualOnDeviceSide()) input.syncToDevice();

    switch (poolingMode) {

        case MAX_POOL: {
                BUILD_DOUBLE_SELECTOR(input.dataType(), output.dataType(), maxPooling2dCudaLauncher, (*block.launchContext(), input.getSpecialBuffer(), input.getSpecialShapeInfo(), output.getSpecialBuffer(), output.getSpecialShapeInfo(), kH, kW, sH, sW, pH, pW, dH, dW, extraParam0), LIBND4J_TYPES, FLOAT_TYPES);
            }
            break;
        case AVG_POOL: {
                BUILD_DOUBLE_SELECTOR(input.dataType(), output.dataType(), avgPooling2dCudaLauncher, (*block.launchContext(), input.getSpecialBuffer(), input.getSpecialShapeInfo(), output.getSpecialBuffer(), output.getSpecialShapeInfo(), kH, kW, sH, sW, pH, pW, dH, dW, extraParam0), LIBND4J_TYPES, FLOAT_TYPES);
            }
            break;
        case PNORM_POOL: {
                BUILD_DOUBLE_SELECTOR(input.dataType(), output.dataType(), pnormPooling2dCudaLauncher, (*block.launchContext(), input.getSpecialBuffer(), input.getSpecialShapeInfo(), output.getSpecialBuffer(), output.getSpecialShapeInfo(), kH, kW, sH, sW, pH, pW, dH, dW, extraParam0), LIBND4J_TYPES, FLOAT_TYPES);
            }
            break;
        default:
            throw std::runtime_error("Pooling2D: Unknown PoolingType used");
    }

    output.tickWriteDevice();
    input.tickReadDevice();

    auto result = hipStreamSynchronize(*block.launchContext()->getCudaStream());
    if (result != 0)
        throw cuda_exception::build("Pooling2D failed", result);
}




        void ConvolutionUtils::pooling3d(nd4j::graph::Context & block, const NDArray& input, NDArray& output, const int kD, const int kH, const int kW, const int sD, const int sH, const int sW, const int pD, const int pH, const int pW, const int dD, const int dH, const int dW, const int poolingMode, const int extraParam0) {

        }

        void ConvolutionUtils::pooling2dBP(nd4j::graph::Context & block, const NDArray& input, const NDArray& gradO, NDArray& gradI, const int kH, const int kW, const int sH, const int sW, const int pH, const int pW, const int dH, const int dW, const int poolingMode, const int extraParam0) {

        }

        void ConvolutionUtils::pooling3dBP(nd4j::graph::Context  &block, const NDArray& input, const NDArray& gradO, NDArray& gradI, const int kD, const int kH, const int kW, const int sD, const int sH, const int sW, const int pD, const int pH, const int pW, const int dD, const int dH, const int dW, const int poolingMode, const int extraParam0) {

        }



BUILD_DOUBLE_TEMPLATE(template void maxPooling2dCudaLauncher, (nd4j::LaunchContext & block, void *vx, Nd4jLong *vxShapeInfo, void *vz, Nd4jLong *vzShapeInfo, const int kH, const int kW, const int sH, const int sW, const int pH, const int pW, const int dH, const int dW, const int extraParam0), LIBND4J_TYPES, FLOAT_TYPES);
BUILD_DOUBLE_TEMPLATE(template void pnormPooling2dCudaLauncher, (nd4j::LaunchContext & block, void *vx, Nd4jLong *vxShapeInfo, void *vz, Nd4jLong *vzShapeInfo, const int kH, const int kW, const int sH, const int sW, const int pH, const int pW, const int dH, const int dW, const int extraParam0), LIBND4J_TYPES, FLOAT_TYPES);
BUILD_DOUBLE_TEMPLATE(template void avgPooling2dCudaLauncher, (nd4j::LaunchContext & block, void *vx, Nd4jLong *vxShapeInfo, void *vz, Nd4jLong *vzShapeInfo, const int kH, const int kW, const int sH, const int sW, const int pH, const int pW, const int dH, const int dW, const int extraParam0), LIBND4J_TYPES, FLOAT_TYPES);



}
}