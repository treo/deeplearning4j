#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author raver119@gmail.com
//

#include <ops/declarable/helpers/lrn.h>
#include <Status.h>
#include <ConstantTadHelper.h>

namespace nd4j {
namespace ops {
namespace helpers {

    template <typename T>
    static _CUDA_G void lrnKernel(void *vx, Nd4jLong *xTadShapeInfo, Nd4jLong *xTadOffsets, void *vz, Nd4jLong *zTadShapeInfo, Nd4jLong *zTadOffsets, Nd4jLong numTads, Nd4jLong tadLength, int depth, double bias, double alpha, double beta) {
        extern __shared__ char sharedChar[];
        __shared__ T* shared;
        if (threadIdx.x == 0)
            shared = reinterpret_cast<T*>(sharedChar);
        __syncthreads();


        auto xEws = shape::elementWiseStride(xTadShapeInfo);
        auto zEws = shape::elementWiseStride(zTadShapeInfo);

        auto xOrder = shape::order(xTadShapeInfo);
        auto zOrder = shape::order(zTadShapeInfo);

        const T tbias  = static_cast<T>(bias);
        const T tbeta  = static_cast<T>(beta);
        const T talpha = static_cast<T>(alpha);


        for (uint i = blockIdx.x; i < numTads; i += gridDim.x) {
            auto x = reinterpret_cast<T*>(vx) + xTadOffsets[i];
            auto z = reinterpret_cast<T*>(vz) + zTadOffsets[i];

            // load everything into shared memory
            shared[threadIdx.x] = x[threadIdx.x * xEws];
            __syncthreads();

            const uint begin = nd4j::math::nd4j_max<int>(0, threadIdx.x - depth);
            const uint last  = depth + threadIdx.x + 1;
            const uint end   = nd4j::math::nd4j_min<int>(last, tadLength);

            T prev = 0.;
            for (int s = begin; s < end; s++)
                prev = prev + shared[s] * shared[s];

            z[threadIdx.x * zEws] = shared[threadIdx.x] / nd4j::math::nd4j_pow<T, T, T>(tbias + alpha * prev, tbeta);
        }
    }

    template <typename X, typename Z>
    static _CUDA_G void lrnBPKernel(void *vx, Nd4jLong *xTadShapeInfo, Nd4jLong *xTadOffsets, void *vz, Nd4jLong *zTadShapeInfo, Nd4jLong *zTadOffsets, Nd4jLong numTads, Nd4jLong tadLength, int depth, double bias, double alpha, double beta) {
        extern __shared__ char sharedChar[];
        __shared__ X* sharedX;
        __shared__ Z* sharedY;

        if (threadIdx.x == 0) {
            sharedX = reinterpret_cast<X*>(sharedChar);
            sharedY = reinterpret_cast<Z*>(sharedX + blockDim.x);
        }

        __syncthreads();


        auto xEws = shape::elementWiseStride(xTadShapeInfo);
        auto zEws = shape::elementWiseStride(zTadShapeInfo);

        auto xOrder = shape::order(xTadShapeInfo);
        auto zOrder = shape::order(zTadShapeInfo);

        const Z tbias  = static_cast<Z>(bias);
        const Z tbeta  = static_cast<Z>(beta);
        const Z talpha = static_cast<Z>(alpha);
        const Z coeff  = talpha * tbeta;



        for (uint i = blockIdx.x; i < numTads; i += gridDim.x) {
            auto x = reinterpret_cast<X*>(vx) + xTadOffsets[i];
            auto z = reinterpret_cast<Z*>(vz) + zTadOffsets[i];

            const uint begin = nd4j::math::nd4j_max<int>(0, threadIdx.x - depth);
            const uint last  = depth + threadIdx.x + 1;
            const uint end   = nd4j::math::nd4j_min<int>(last, tadLength);

            // load everything into shared memory
            sharedX[threadIdx.x] = x[threadIdx.x * xEws];
            sharedY[threadIdx.x] = 0.f;
            __syncthreads();


            for (int s = begin; s < end; s++)
                sharedY[threadIdx.x] = sharedY[threadIdx.x] + sharedX[s] * sharedX[s];
            __syncthreads();

            Z factor[1024];
            Z init = tbias + talpha * sharedY[threadIdx.x];

            Z prev = 0.f;
            for (uint s = begin; s < end; ++s) {
                factor[s] = nd4j::math::nd4j_pow<Z, Z, Z>(tbias + talpha * sharedY[s], -tbeta - 1);
                prev = prev + sharedX[s] * factor[s];
            }

            z[threadIdx.x * zEws] = factor[threadIdx.x] * init - 2 * sharedX[threadIdx.x] * coeff * prev;
        }
    }


    template <typename X, typename Z>
    static void lrnBP_(nd4j::graph::Context& block, const NDArray& input, const NDArray& gradO, NDArray& gradI, const int depth, const float bias, const float alpha, const float beta) {
        auto rank = input.rankOf();
        auto packX = ConstantTadHelper::getInstance()->tadForDimensions(input.getShapeInfo(), {rank - 1});
        auto packZ = ConstantTadHelper::getInstance()->tadForDimensions(gradI.getShapeInfo(), {rank - 1});

        const auto tadLength = shape::length(packX.primaryShapeInfo());
        const int numBlocks = nd4j::math::nd4j_min<Nd4jLong>(1024, packX.numberOfTads());
        const int numThreads = tadLength;

        if (tadLength > 1024 || tadLength < 1)
            throw std::runtime_error("LRN: tadLength > 1024 isn't implemented yet");

        lrnBPKernel<X, Z><<<numBlocks, numThreads, numThreads * sizeof(X) + numThreads * sizeof(Z) + numThreads * sizeof(Z) + 1024, *block.launchContext()->getCudaStream()>>>(input.getSpecialBuffer(), packX.platformShapeInfo(), packX.platformOffsets(), gradI.specialBuffer(), packZ.platformShapeInfo(), packZ.platformOffsets(), packX.numberOfTads(),  tadLength, depth, bias, alpha, beta);

        gradI.tickWriteDevice();
        gradI *= gradO;
    }

    void lrnBP(nd4j::graph::Context& block, const NDArray& input, const NDArray& gradO, NDArray& gradI, const int depth, const float bias, const float alpha, const float beta) {
        input.syncToDevice();
        gradO.syncToDevice();

        BUILD_DOUBLE_SELECTOR(input.dataType(), gradO.dataType(), lrnBP_, (block, input, gradO, gradI, depth, bias, alpha, beta), LIBND4J_TYPES, FLOAT_TYPES);

        gradI.tickWriteDevice();
    }

    template <typename T>
    static void lrnFunctor_(nd4j::graph::Context& block, NDArray* input, NDArray* output, int depth, double bias, double alpha, double beta) {
        auto rank = input->rankOf();
        auto packX = ConstantTadHelper::getInstance()->tadForDimensions(input->shapeInfo(), {rank - 1});
        auto packZ = ConstantTadHelper::getInstance()->tadForDimensions(output->shapeInfo(), {rank - 1});

        const auto tadLength = shape::length(packX.primaryShapeInfo());
        const int numBlocks = nd4j::math::nd4j_min<Nd4jLong>(1024, packX.numberOfTads());
        const int numThreads = tadLength;

        if (tadLength > 1024 || tadLength < 1)
            throw std::runtime_error("LRN: tadLength > 1024 isn't implemented yet");

        lrnKernel<T><<<numBlocks, numThreads, numThreads * sizeof(T), *block.launchContext()->getCudaStream()>>>(input->specialBuffer(), packX.platformShapeInfo(), packX.platformOffsets(), output->specialBuffer(), packZ.platformShapeInfo(), packZ.platformOffsets(), packX.numberOfTads(), tadLength, depth, bias, alpha, beta);
    }

    int lrnFunctor(nd4j::graph::Context& block, NDArray* input, NDArray* output, int depth, double bias, double alpha, double beta) {
        input->syncToDevice();

        BUILD_SINGLE_SELECTOR(input->dataType(), lrnFunctor_, (block, input, output, depth, bias, alpha, beta), FLOAT_TYPES);

        output->tickWriteDevice();

        return Status::OK();
    }
}
}
}
