#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author raver119@gmail.com
//  @author Yurii Shyrma (iuriish@yahoo.com)
//

#include <exceptions/cuda_exception.h>
#include <hipblas.h>
#include <specials_cuda.h>
#include <op_boilerplate.h>
#include <types/float16.h>
#include <ops/declarable/helpers/batched_gemm.h>
#include <PointersManager.h>


namespace nd4j {
namespace ops {
namespace helpers {

//////////////////////////////////////////////////////////////////////////////
// bsxMXK x bSxKxN = bSxMxN
void bgemm(const std::vector<NDArray*>& vA, const std::vector<NDArray*>& vB, std::vector<NDArray*>& vC, const NDArray* alphas, const NDArray* betas, int transA, int transB, int M, int N, int K, const int lda, const int ldb, const int ldc) {

    const auto bS = vA.size();      // batch size

    std::vector<NDArray*> pA(bS), pB(bS), pC(bS);

    std::vector<NDArray*> toDelete;

    for(int i = 0; i < bS; ++i) {

        if(vA[i]->ews() != 1) {
            pA[i] = vA[i]->dup('f');
            toDelete.emplace_back(pA[i]);
        }
        else
            pA[i] = vA[i];

        if(vB[i]->ews() != 1) {
            pB[i] = vB[i]->dup('f');
            toDelete.emplace_back(pB[i]);
        }
        else
            pB[i] = vB[i];

        if(vC[i]->ews() != 1) {
            pC[i] = vC[i]->dup('f');
            toDelete.emplace_back(pC[i]);
        }
        else
            pC[i] = vC[i];

        if(pC[i]->ordering() != 'f') {
            auto temp = pA[i];
            pA[i] = new NDArray(pB[i]->permute({1,0}));
            pB[i] = new NDArray(temp ->permute({1,0}));
            pC[i] = new NDArray(pC[i]->permute({1,0}));
            toDelete.push_back(pA[i]);
            toDelete.push_back(pB[i]);
            toDelete.push_back(pC[i]);
            M = pA[i]->sizeAt(0);
            K = pA[i]->sizeAt(1);
            N = pB[i]->sizeAt(1);
        }

        NDArray::prepareSpecialUse ({pC[i]}, {pA[i], pB[i]});
        NDArray::registerSpecialUse({pC[i]}, {pA[i], pB[i]});
    }

    NDArray::prepareSpecialUse ({}, {alphas, betas});
    NDArray::registerSpecialUse({}, {alphas, betas});

    std::vector<void*> pAbuffs(bS), pBbuffs(bS), pCbuffs(bS);
    for(int i = 0; i < bS; ++i) {
        pAbuffs[i] = pA[i]->getSpecialBuffer();
        pBbuffs[i] = pB[i]->getSpecialBuffer();
        pCbuffs[i] = pC[i]->getSpecialBuffer();
    }

    nd4j::LaunchContext* context = vA[0]->getContext();
    PointersManager manager(context, "helpers::bgemm cuda");

    const void** aBuffers = reinterpret_cast<const void**>(manager.replicatePointer(pAbuffs.data(), bS * sizeof(void*)));
    const void** bBuffers = reinterpret_cast<const void**>(manager.replicatePointer(pBbuffs.data(), bS * sizeof(void*)));
          void** cBuffers = reinterpret_cast<void**>(manager.replicatePointer(pCbuffs.data(), bS * sizeof(void*)));

    // const auto aOrder = pA->ordering();
    // const auto bOrder = pB->ordering();

    // const bool transA = aOrder != 'f';
    // const bool transB = bOrder != 'f';

    const hipblasOperation_t transAblas = transA == 112 ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    const hipblasOperation_t transBblas = transB == 112 ? HIPBLAS_OP_T : HIPBLAS_OP_N;

    // const int lda = aOrder == 'f' ? M : K;
    // const int ldb = bOrder == 'f' ? K : N;
    // const int ldc = M; // cOrder == 'f' ? M : N;

    const auto aType = pA[0]->dataType();
    const auto bType = pB[0]->dataType();
    const auto cType = pC[0]->dataType();

    auto handle = reinterpret_cast<hipblasHandle_t*>(context->getCublasHandle());
    auto stream = context->getCudaStream();

    auto status = hipblasSetStream(*handle, *stream);

    if (status != HIPBLAS_STATUS_SUCCESS)
        throw cuda_exception::build("MmulHelper::mmulMxM cuda failed !", status);

    const bool AB(aType == bType), AC(aType == cType), ABC(AB && AC);

    // choose appropriate cuda gemm api depending on data types
    if(ABC && aType == DataType::DOUBLE) {
        double alpha = alphas->e<double>(0);
        double beta  = betas->e<double>(0);
        status = hipblasDgemmBatched(*handle, transAblas, transBblas, M, N, K, &alpha, (const double**)aBuffers, lda, (const double**)bBuffers, ldb, &beta, (double**)cBuffers, ldc, bS);
    }
    else if(ABC && aType == DataType::FLOAT32) {
        float alpha = alphas->e<float>(0);
        float beta  = betas->e<float>(0);
        status = hipblasSgemmBatched(*handle, transAblas, transBblas, M, N, K, &alpha, (const float**)aBuffers, lda, (const float**)bBuffers, ldb, &beta, (float**)cBuffers, ldc, bS);
    }
    else if(ABC && aType == DataType::HALF) {
        __half alpha = alphas->e<float>(0);
        __half beta  = betas->e<float>(0);
        status = hipblasHgemmBatched(*handle, transAblas, transBblas, M, N, K, &alpha, (const __half**)aBuffers, lda, (const __half**)bBuffers, ldb, &beta, (__half**)cBuffers, ldc, bS);
    }
    else if(AB && aType == DataType::INT8 && cType == DataType::FLOAT32) {
        float alpha = alphas->e<float>(0);
        float beta  = betas->e<float>(0);
        status = hipblasGemmBatchedEx(*handle, transAblas, transBblas, M, N, K, &alpha, aBuffers, HIP_R_8I, lda, bBuffers, HIP_R_8I, ldb, &beta, cBuffers, HIP_R_32F, ldc, bS, HIP_R_32F, HIPBLAS_GEMM_DEFAULT);
    }
    else if(AB && aType == DataType::HALF && cType == DataType::FLOAT32) {
        float alpha = alphas->e<float>(0);
        float beta  = betas->e<float>(0);
        status = hipblasGemmBatchedEx(*handle, transAblas, transBblas, M, N, K, &alpha, aBuffers, HIP_R_16F, lda, bBuffers, HIP_R_16F, ldb, &beta, cBuffers, HIP_R_32F, ldc, bS, HIP_R_32F, HIPBLAS_GEMM_DEFAULT);
    }
    else
        throw std::runtime_error("batched gemm cuda: this mode is not implemented yet !");

    if (status != HIPBLAS_STATUS_SUCCESS)
        throw cuda_exception::build("MmulHelper::mmulMxM cuda failed !", status);

    auto cudaResult = hipStreamSynchronize(*stream);
    if (cudaResult != 0)
        throw cuda_exception::build("MmulHelper::mmulMxM cuda failed !", cudaResult);

    for(int i = 0; i < bS; ++i)
    if(vC[i]->ews() != 1)
        vC[i]->assign(pC[i]);

    for(int i = toDelete.size() - 1; i >= 0; --i)
        delete toDelete[i];
}

}
}
}

