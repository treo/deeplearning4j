#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com), created on 19.04.2018
// @author raver119@gmail.com
//

#include <op_boilerplate.h>
#include <ops/declarable/helpers/activations.h>
#include <ShapeUtils.h>
#include <numeric>
#include <PointersManager.h>

namespace nd4j    {
namespace ops     {
namespace helpers {

///////////////////////////////////////////////////////////////////
template<typename X, typename Y>
__global__ void preluCuda(const void *vx, const Nd4jLong *xShapeInfo,
								 const void *vy, const Nd4jLong *yShapeInfo,
									   void *vz) {

	const auto x = reinterpret_cast<const X*>(vx);
	const auto y = reinterpret_cast<const Y*>(vy);
		  auto z = reinterpret_cast<X*>(vz);

	__shared__ Nd4jLong  len;

	if (threadIdx.x == 0)
		len = shape::length(xShapeInfo);

	__syncthreads();

	const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
	const auto totalThreads = gridDim.x * blockDim.x;

	for (int i = tid; i < len; i += totalThreads) {

		const auto xzOffset = shape::getIndexOffset(i, xShapeInfo, len);
		const auto xVal     = x[xzOffset];

		if(xVal < 0)
			z[xzOffset] = xVal * y[shape::subArrayOffset(i, xShapeInfo, yShapeInfo)];
		else
			z[xzOffset] = xVal;
	}
}

///////////////////////////////////////////////////////////////////
template<typename X, typename Y>
linkage void preluCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const hipStream_t *stream, const void *vx, const Nd4jLong *xShapeInfo, const void *vy, const Nd4jLong *yShapeInfo, void *vz) {

	preluCuda<X, Y><<<blocksPerGrid, threadsPerBlock, 1024, *stream>>>(vx, xShapeInfo, vy, yShapeInfo, vz);
}

///////////////////////////////////////////////////////////////////
void prelu(nd4j::LaunchContext * context, const NDArray& input, const NDArray& alpha, NDArray& output) {
	if(!input.isActualOnDeviceSide()) input.syncToDevice();
	if(!alpha.isActualOnDeviceSide()) alpha.syncToDevice();

	const auto xType = input.dataType();
	const auto yType = alpha.dataType();
	int threadsPerBlock = MAX_NUM_THREADS;
	int blocksPerGrid = (input.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;

	BUILD_DOUBLE_SELECTOR(xType, yType, preluCudaLauncher, (blocksPerGrid, threadsPerBlock, context->getCudaStream(), input.getSpecialBuffer(), input.getSpecialShapeInfo(), alpha.getSpecialBuffer(), alpha.getSpecialShapeInfo(), output.getSpecialBuffer()), LIBND4J_TYPES, FLOAT_TYPES);

	input.tickReadHost();
	alpha.tickReadHost();
	output.tickWriteDevice();
}

///////////////////////////////////////////////////////////////////
template<typename T>
__global__ void softMaxForVectorCuda(const void *vx, const Nd4jLong *xzShapeInfo, void *vz) {

	// logic of this kernel is based on assumption gridDim = 1

	const auto x = reinterpret_cast<const T*>(vx);
		  auto z = reinterpret_cast<T*>(vz);

	__shared__ Nd4jLong  len;
	__shared__ int numOfIters;
	__shared__ T* shmem;

	if (threadIdx.x == 0) {
		extern __shared__ char shared[];
		shmem = reinterpret_cast<T*>(shared);
		len = shape::length(xzShapeInfo);
		numOfIters = (len + blockDim.x - 1) / blockDim.x;   // ceil (len / blockDim.x)
	}
	__syncthreads();

	T temp = -DataTypeUtils::max<T>();	// set start value to compare with at first iteration, FIXME: what if T is unsigned ??

	// ************ evaluate max element in input array x ************ //
	for (int i = 0; i < numOfIters; ++i) {

		const Nd4jLong elemIdx = i * blockDim.x + threadIdx.x;
		if(elemIdx < len) {
			const Nd4jLong offset = shape::getIndexOffset(elemIdx, xzShapeInfo, len);
			shmem[threadIdx.x] = (threadIdx.x != 0) ? x[offset] : nd4j::math::nd4j_max<T>(x[offset], temp);	// take into account max element evaluated on previous iteration and stored in temp
		}
		else
			shmem[threadIdx.x] = -DataTypeUtils::max<T>();	// FIXME: what if T is unsigned ??

		__syncthreads();

		for (int s = blockDim.x / 2; s > 0; s /= 2) {
			if(threadIdx.x < s)
				shmem[threadIdx.x] = nd4j::math::nd4j_max<T>(shmem[threadIdx.x], shmem[threadIdx.x + s]);
			__syncthreads();
		}

		temp = shmem[0];	// save max value calculated at current iteration
	}

	const T max = temp;
	temp = 0;

	// ************ evaluate value of exp(x[offset] - max) per each element, store it to shared memory shmem ************ //
	// at the same evaluate sum of exponents, sum will be stored in shmem[0]
	for (int i = 0; i < numOfIters; ++i) {

		const Nd4jLong elemIdx = i * blockDim.x + threadIdx.x;
		if(elemIdx < len) {
			const Nd4jLong offset = shape::getIndexOffset(elemIdx, xzShapeInfo, len);
			z[offset] = nd4j::math::nd4j_exp<T, T>(x[offset] - max);
			shmem[threadIdx.x] = (threadIdx.x != 0) ? z[offset] : (z[offset] + temp); // take into account sum element evaluated on previous iteration and stored in temp
		}
		else
			shmem[threadIdx.x] = 0;

		__syncthreads();

		for (int s = blockDim.x / 2; s > 0; s /= 2) {
			if(threadIdx.x < s)
				shmem[threadIdx.x] += shmem[threadIdx.x + s];
			__syncthreads();
		}

		temp = shmem[0];	// save sum calculated at current iteration
	}

	// ************ evaluate z[offset] / sum  ************ //
	for (int i = 0; i < numOfIters; ++i) {
		const Nd4jLong elemIdx = i * blockDim.x + threadIdx.x;
		if(elemIdx >= len) continue;
		const Nd4jLong offset = shape::getIndexOffset(elemIdx, xzShapeInfo, len);
		z[offset] /= shmem[0];
	}
}

///////////////////////////////////////////////////////////////////
template <typename T>
linkage void softMaxForVectorCudaLauncher(const hipStream_t* stream, const void *vx, const Nd4jLong *xzShapeInfo, void *vz) {

	softMaxForVectorCuda<T><<<1, MAX_NUM_THREADS, MAX_NUM_THREADS * sizeof(T) + 512, *stream>>>(vx, xzShapeInfo, vz);
}

//////////////////////////////////////////////////////////////////////////
void softmax(nd4j::LaunchContext * context, const NDArray& input, NDArray& output, const int dimension) {

	if(!input.isActualOnDeviceSide()) input.syncToDevice();
	const int rank = input.rankOf();

	if(input.isVector()) {

		if(rank == 1 || input.sizeAt(dimension) != 1) {
			BUILD_SINGLE_SELECTOR(input.dataType(), softMaxForVectorCudaLauncher, (context->getCudaStream(), input.getSpecialBuffer(), input.getSpecialShapeInfo(), output.getSpecialBuffer()), FLOAT_TYPES);
			input.tickReadDevice();
		}
		else
			output = 1.;
	}
	else {

		auto maxAlongDim = const_cast<NDArray&>(input).reduceAlongDims(reduce::Max, {dimension}, true);
		(input - maxAlongDim).applyTransform(transform::Exp, &output); // output contains exponents temporarily
		auto sumAlongDim = output.reduceAlongDims(reduce::Sum, {dimension}, true);
		output /= sumAlongDim;
		input.tickReadDevice();
	}

	PointersManager manager(context, "helpers::softmax");
	manager.synchronize();

	output.tickWriteDevice();
}

///////////////////////////////////////////////////////////////////
template<typename T>
__global__  void logSoftMaxForVectorCuda(const void *vx, const Nd4jLong *xzShapeInfo, void *vz) {

	// logic of this kernel is based on assumption gridDim = 1

	const auto x = reinterpret_cast<const T*>(vx);
		  auto z = reinterpret_cast<T*>(vz);

	__shared__ Nd4jLong  len;
	__shared__ int numOfIters;
	__shared__ T* shmem;

	if (threadIdx.x == 0) {
		extern __shared__ char shared[];
		shmem = reinterpret_cast<T*>(shared);
		len = shape::length(xzShapeInfo);
		numOfIters = (len + blockDim.x - 1) / blockDim.x;   // ceil (len / blockDim.x)
	}
	__syncthreads();

	T temp = -DataTypeUtils::max<T>();	// set start value to compare with at first iteration, FIXME: what if T is unsigned ??

	// ************ evaluate max element in input array x ************ //
	for (int i = 0; i < numOfIters; ++i) {

		const Nd4jLong elemIdx = i * blockDim.x + threadIdx.x;
		if(elemIdx < len) {
			const Nd4jLong offset = shape::getIndexOffset(elemIdx, xzShapeInfo, len);
			shmem[threadIdx.x] = (threadIdx.x != 0) ? x[offset] : nd4j::math::nd4j_max<T>(x[offset], temp);	// take into account max element evaluated on previous iteration and stored in temp
		}
		else
			shmem[threadIdx.x] = -DataTypeUtils::max<T>();	// FIXME: what if T is unsigned ??

		__syncthreads();

		for (int s = blockDim.x / 2; s > 0; s /= 2) {
			if(threadIdx.x < s)
				shmem[threadIdx.x] = nd4j::math::nd4j_max<T>(shmem[threadIdx.x], shmem[threadIdx.x + s]);
			__syncthreads();
		}

		temp = shmem[0];	// save max value calculated at current iteration
	}

	const T max = temp;
	temp = 0;

	// ************ evaluate value of exp(x[offset] - max) per each element, store it to shared memory shmem ************ //
	// at the same evaluate sum of exponents, sum will be stored in shmem[0]
	for (int i = 0; i < numOfIters; ++i) {

		const Nd4jLong elemIdx = i * blockDim.x + threadIdx.x;
		if(elemIdx < len) {
			const Nd4jLong offset = shape::getIndexOffset(elemIdx, xzShapeInfo, len);
			z[offset] = nd4j::math::nd4j_exp<T, T>(x[offset] - max);
			shmem[threadIdx.x] = (threadIdx.x != 0) ? z[offset] : (z[offset] + temp); // take into account sum element evaluated on previous iteration and stored in temp
		}
		else
			shmem[threadIdx.x] = 0;

		__syncthreads();

		for (int s = blockDim.x / 2; s > 0; s /= 2) {
			if(threadIdx.x < s)
				shmem[threadIdx.x] += shmem[threadIdx.x + s];
			__syncthreads();
		}

		temp = shmem[0];	// save sum calculated at current iteration
	}

	// ************ evaluate log(z[offset] / sum)  ************ //
	for (int i = 0; i < numOfIters; ++i) {
		const Nd4jLong elemIdx = i * blockDim.x + threadIdx.x;
		if(elemIdx >= len) continue;
		const Nd4jLong offset = shape::getIndexOffset(elemIdx, xzShapeInfo, len);
		z[offset] = nd4j::math::nd4j_log<T,T>(z[offset] / shmem[0]);
	}
}

///////////////////////////////////////////////////////////////////
template <typename T>
linkage void logSoftMaxForVectorCudaLauncher(const hipStream_t* stream, const void *vx, const Nd4jLong *xzShapeInfo, void *vz) {

	logSoftMaxForVectorCuda<T><<<1, MAX_NUM_THREADS, MAX_NUM_THREADS * sizeof(T) + 512, *stream>>>(vx, xzShapeInfo, vz);
}

//////////////////////////////////////////////////////////////////////////
void logSoftmax(nd4j::LaunchContext * context, const NDArray& input, NDArray& output, const int dimension) {

	if(!input.isActualOnDeviceSide()) input.syncToDevice();
	const int rank = input.rankOf();

	if(input.isVector()) {

		if(rank == 1 || input.sizeAt(dimension) != 1) {
			BUILD_SINGLE_SELECTOR(input.dataType(), logSoftMaxForVectorCudaLauncher, (context->getCudaStream(), input.getSpecialBuffer(), input.getSpecialShapeInfo(), output.getSpecialBuffer()), FLOAT_TYPES);
			input.tickReadDevice();
		}
		else
			output = 0.;
	}
	else {

		auto maxAlongDim = const_cast<NDArray&>(input).reduceAlongDims(reduce::Max, {dimension}, true);
		(input - maxAlongDim).applyTransform(transform::Exp, &output); // output contains exponents temporarily
		auto sumAlongDim = output.reduceAlongDims(reduce::Sum, {dimension}, true);
		output /= sumAlongDim;
		output.applyTransform(transform::Log);
		input.tickReadDevice();
	}

	PointersManager manager(context, "helpers::logSoftmax");
	manager.synchronize();

	output.tickWriteDevice();
}

///////////////////////////////////////////////////////////////////
template<typename T>
__global__ linkage void softMaxDerivForVectorCuda(const void *vx, const Nd4jLong *xzShapeInfo, void *vz) {

	// logic of this kernel is based on assumption gridDim = 1

	const auto x = reinterpret_cast<const T*>(vx);
		  auto z = reinterpret_cast<T*>(vz);

	__shared__ Nd4jLong  len;
	__shared__ int numOfIters;
	__shared__ T* shmem;

	if (threadIdx.x == 0) {
		extern __shared__ char shared[];
		shmem = reinterpret_cast<T*>(shared);
		len = shape::length(xzShapeInfo);
		numOfIters = (len + blockDim.x - 1) / blockDim.x;   // ceil (len / blockDim.x)
	}
	__syncthreads();

	T temp = -DataTypeUtils::max<T>();	// set start value to compare with at first iteration, FIXME: what if T is unsigned ??

	// ************ evaluate max element in input array x ************ //
	for (int i = 0; i < numOfIters; ++i) {

		const Nd4jLong elemIdx = i * blockDim.x + threadIdx.x;
		if(elemIdx < len) {
			const Nd4jLong offset = shape::getIndexOffset(elemIdx, xzShapeInfo, len);
			shmem[threadIdx.x] = (threadIdx.x != 0) ? x[offset] : nd4j::math::nd4j_max<T>(x[offset], temp);	// take into account max element evaluated on previous iteration and stored in temp
		}
		else
			shmem[threadIdx.x] = -DataTypeUtils::max<T>();	// FIXME: what if T is unsigned ??

		__syncthreads();

		for (int s = blockDim.x / 2; s > 0; s /= 2) {
			if(threadIdx.x < s)
				shmem[threadIdx.x] = nd4j::math::nd4j_max<T>(shmem[threadIdx.x], shmem[threadIdx.x + s]);
			__syncthreads();
		}

		temp = shmem[0];	// save max value calculated at current iteration
	}

	const T max = temp;
	temp = 0;

	// ************ evaluate value of exp(x[offset] - max) per each element, store it to shared memory shmem ************ //
	// at the same evaluate sum of exponents, sum will be stored in shmem[0]
	for (int i = 0; i < numOfIters; ++i) {

		const Nd4jLong elemIdx = i * blockDim.x + threadIdx.x;
		if(elemIdx < len) {
			const Nd4jLong offset = shape::getIndexOffset(elemIdx, xzShapeInfo, len);
			z[offset] = nd4j::math::nd4j_exp<T, T>(x[offset] - max);
			shmem[threadIdx.x] = (threadIdx.x != 0) ? z[offset] : (z[offset] + temp); // take into account sum element evaluated on previous iteration and stored in temp
		}
		else
			shmem[threadIdx.x] = 0;

		__syncthreads();

		for (int s = blockDim.x / 2; s > 0; s /= 2) {
			if(threadIdx.x < s)
				shmem[threadIdx.x] += shmem[threadIdx.x + s];
			__syncthreads();
		}

		temp = shmem[0];	// save sum calculated at current iteration
	}

	// ************ evaluate (z[offset] / sum) and derivative z[offset] = z[offset] * (1 - z[offset]) ************ //
	for (int i = 0; i < numOfIters; ++i) {
		const Nd4jLong elemIdx = i * blockDim.x + threadIdx.x;
		if(elemIdx >= len) continue;
		const Nd4jLong offset = shape::getIndexOffset(elemIdx, xzShapeInfo, len);
		z[offset] /= shmem[0];
		z[offset] *= (1.f - z[offset]);		// derivative
	}
}

///////////////////////////////////////////////////////////////////
template <typename T>
linkage void softMaxDerivForVectorCudaLauncher(const hipStream_t* stream, const void *vx, const Nd4jLong *xzShapeInfo, void *vz) {

	softMaxDerivForVectorCuda<T><<<1, MAX_NUM_THREADS, MAX_NUM_THREADS * sizeof(T) + 512, *stream>>>(vx, xzShapeInfo, vz);
}

///////////////////////////////////////////////////////////////////
void softmaxDerivative(nd4j::LaunchContext * context, const NDArray& input, NDArray& output, const int dimension) {

	if(!input.isActualOnDeviceSide()) input.syncToDevice();
	const int rank = input.rankOf();
	int temp;

	if(shape::isCommonVector(input.getShapeInfo(), temp)) {

		BUILD_SINGLE_SELECTOR(input.dataType(), softMaxDerivForVectorCudaLauncher, (context->getCudaStream(), input.getSpecialBuffer(), input.getSpecialShapeInfo(), output.getSpecialBuffer()), FLOAT_TYPES);
		input.tickReadDevice();
	}
	else {

		auto maxAlongDim = const_cast<NDArray&>(input).reduceAlongDims(reduce::Max, {dimension}, true);
		(input - maxAlongDim).applyTransform(transform::Exp, &output); // output contains exponents temporarily
		auto sumAlongDim = output.reduceAlongDims(reduce::Sum, {dimension}, true);
		output /= sumAlongDim;
		output *= (1.f - output);	// derivative
		input.tickReadDevice();
	}

	PointersManager manager(context, "helpers::softmaxDerivative");
	manager.synchronize();

	output.tickWriteDevice();
}

///////////////////////////////////////////////////////////////////
template<typename X, typename Y>
__global__ linkage void preluBPCuda(const void *vIn,    const Nd4jLong *inShapeInfo,
								   const void *vAlpha, const Nd4jLong *alphaShapeInfo,
								   const void *vdLdO,  const Nd4jLong *dLdOShapeInfo,
										 void *vdLdI,  const Nd4jLong *dLdIShapeInfo,
										 void *vdLdA,  const Nd4jLong *dLdAShapeInfo) {

	const auto in    = reinterpret_cast<const X*>(vIn);
	const auto alpha = reinterpret_cast<const Y*>(vAlpha);
	const auto dLdO  = reinterpret_cast<const Y*>(vdLdO);
		  auto dLdI  = reinterpret_cast<Y*>(vdLdI);
		  auto dLdA  = reinterpret_cast<Y*>(vdLdA);

	__shared__ Nd4jLong alphaLen;

	if (threadIdx.x == 0)
		alphaLen = shape::length(alphaShapeInfo);

	__syncthreads();

	const auto i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= alphaLen) return;

	Nd4jLong inputIdxs[MAX_RANK*2];
	int numIdxs = shape::outerArrayOffsets(inputIdxs, i, inShapeInfo, alphaShapeInfo);
	Nd4jLong dLdOIdxs[MAX_RANK*2];
	shape::outerArrayOffsets(dLdOIdxs, i, dLdOShapeInfo, alphaShapeInfo);
	Nd4jLong dLdIIdxs[MAX_RANK*2];
	shape::outerArrayOffsets(dLdIIdxs, i, dLdIShapeInfo, alphaShapeInfo);

	const auto alphaOffset = shape::getIndexOffset(i, alphaShapeInfo, alphaLen);
	const auto dLdAOffset  = shape::getIndexOffset(i, dLdAShapeInfo, alphaLen);

	for(Nd4jLong j = 0; j < numIdxs; ++j) {

		const auto inInd   = inputIdxs[j];
		const auto dLdOInd = dLdOIdxs[j];
		const auto dLdIInd = dLdIIdxs[j];

		if(in[inInd] < 0) {
			dLdI[dLdIInd] = dLdO[dLdOInd] * alpha[alphaOffset];
			auto prevVal = dLdA[dLdAOffset];
			prevVal = prevVal + dLdO[dLdOInd] * in[inInd];
			dLdA[dLdAOffset] = prevVal;
		}
		else
			dLdI[dLdIInd] = dLdO[dLdOInd];
	}
}


template<typename X, typename Y>
__host__ linkage void preluBPCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const hipStream_t *stream, const void *vIn, const Nd4jLong *inShapeInfo, const void *vAlpha, const Nd4jLong *alphaShapeInfo, const void *vdLdO,  const Nd4jLong *dLdOShapeInfo, void *vdLdI,  const Nd4jLong *dLdIShapeInfo, void *vdLdA,  const Nd4jLong *dLdAShapeInfo) {

	preluBPCuda<X, Y><<<blocksPerGrid, threadsPerBlock, 1024, *stream>>>(vIn, inShapeInfo, vAlpha, alphaShapeInfo, vdLdO, dLdOShapeInfo, vdLdI, dLdIShapeInfo, vdLdA, dLdAShapeInfo);
}


	//////////////////////////////////////////////////////////////////////////
	void preluBP(nd4j::LaunchContext * context, const NDArray& input, const NDArray& alpha, const NDArray& dLdO, NDArray& dLdI, NDArray& dLdA) {

		if(!input.isActualOnDeviceSide()) input.syncToDevice();
		if(!alpha.isActualOnDeviceSide()) alpha.syncToDevice();
		if(!dLdO.isActualOnDeviceSide())  dLdO.syncToDevice();

		const auto xType = input.dataType();
		const auto zType = dLdO.dataType();

		int threadsPerBlock = MAX_NUM_THREADS;
		int blocksPerGrid = (alpha.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;

		BUILD_DOUBLE_SELECTOR(xType, zType, preluBPCudaLauncher, (blocksPerGrid, threadsPerBlock, context->getCudaStream(), input.getSpecialBuffer(), input.getSpecialShapeInfo(), alpha.getSpecialBuffer(), alpha.getSpecialShapeInfo(), dLdO.getSpecialBuffer(),  dLdO.getSpecialShapeInfo(), dLdI.getSpecialBuffer(), dLdI.getSpecialShapeInfo(), dLdA.getSpecialBuffer(), dLdA.getSpecialShapeInfo()), LIBND4J_TYPES, FLOAT_TYPES);

		input.tickReadHost();
		alpha.tickReadHost();
		dLdO.tickReadHost();
		dLdI.tickWriteDevice();
		dLdA.tickWriteDevice();

	}


	template <typename T>
	linkage void thresholdRelu_(NDArray const& input, double threshold, NDArray& output) {
		auto routine = LAMBDA_T(_x, threshold) {
			return _x > (T)threshold ? _x: (T)0.f;
		};
		const_cast<NDArray&>(input).applyLambda(routine, &output);
	}

	void thresholdRelu(nd4j::LaunchContext * context, NDArray const& input, double threshold, NDArray& output) {
		BUILD_SINGLE_SELECTOR(input.dataType(), thresholdRelu_, (input, threshold, output), FLOAT_TYPES);
	}

	template <typename T>
	linkage void thresholdReluDerivative_(NDArray* input, double theta, NDArray* dLdO, NDArray* output) {

	}

	void thresholdReluDerivative(nd4j::LaunchContext * context, NDArray* input, double threshold, NDArray* dLdO, NDArray* output) {
		BUILD_SINGLE_SELECTOR(input->dataType(), thresholdReluDerivative_, (input, threshold, dLdO, output), FLOAT_TYPES);
	}


BUILD_SINGLE_TEMPLATE(template void thresholdReluDerivative_, (NDArray* input, double threshold, NDArray* dLdO, NDArray* output), FLOAT_TYPES);
BUILD_DOUBLE_TEMPLATE(template void preluCudaLauncher,   (const int blocksPerGrid, const int threadsPerBlock, const hipStream_t *stream, const void *vx, const Nd4jLong *xShapeInfo, const void *vy, const Nd4jLong *yShapeInfo, void *vz), LIBND4J_TYPES, FLOAT_TYPES);
BUILD_DOUBLE_TEMPLATE(template void preluBPCudaLauncher, (const int blocksPerGrid, const int threadsPerBlock, const hipStream_t *stream, const void *vIn, const Nd4jLong *inShapeInfo, const void *vAlpha, const Nd4jLong *alphaShapeInfo, const void *vdLdO,  const Nd4jLong *dLdOShapeInfo, void *vdLdI,  const Nd4jLong *dLdIShapeInfo, void *vdLdA,  const Nd4jLong *dLdAShapeInfo), LIBND4J_TYPES, FLOAT_TYPES);
BUILD_SINGLE_TEMPLATE(template void softMaxForVectorCudaLauncher, (const hipStream_t* stream, const void *vx, const Nd4jLong *xzShapeInfo, void *vz), FLOAT_TYPES);
BUILD_SINGLE_TEMPLATE(template void softMaxDerivForVectorCudaLauncher, (const hipStream_t* stream, const void *vx, const Nd4jLong *xzShapeInfo, void *vz), FLOAT_TYPES);


}
}
}

