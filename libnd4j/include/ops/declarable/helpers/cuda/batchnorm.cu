#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma, created on 25.02.2018
//


#include<ops/declarable/helpers/batchnorm.h>
#include <helpers/ShapeUtils.h>
#include <OmpLaunchHelper.h>
#include <ConstantTadHelper.h>
#include <PointersManager.h>

namespace nd4j 	  {
namespace ops 	  {
namespace helpers {


//////////////////////////////////////////////////////////////////////////
template<typename T>
__global__ static void batchnormCuda(const void* vx, const Nd4jLong* xShapeInfo,
									const void* vMean, const Nd4jLong* meanShapeInfo,
									const void* vVariance, const Nd4jLong* varianceShapeInfo,
									const void* vGamma, const Nd4jLong* gammaShapeInfo,
									const void* vBeta, const Nd4jLong* betaShapeInfo,
										  void* vz, const Nd4jLong* zShapeInfo,
									const Nd4jLong* xTadShapeInfo, const Nd4jLong* xTadOffsets,
									const Nd4jLong* zTadShapeInfo, const Nd4jLong* zTadOffsets,
									const T epsilon) {

	const auto x    	= reinterpret_cast<const T*>(vx);
          auto z        = reinterpret_cast<T*>(vz);
	const auto mean 	= reinterpret_cast<const T*>(vMean);
	const auto variance = reinterpret_cast<const T*>(vVariance);
	const auto gamma    = reinterpret_cast<const T*>(vGamma);
	const auto beta     = reinterpret_cast<const T*>(vBeta);

    // maxRank = xRank = zRank, minRank = meanRank = varianceRank = gammaRank = betaRank
    __shared__ Nd4jLong minLen, tadLen, totalThreads;

    if (threadIdx.x == 0) {

        totalThreads = gridDim.x * blockDim.x;

        minLen = shape::length(meanShapeInfo);
        tadLen = shape::length(xShapeInfo) / minLen;
    }
    __syncthreads();

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (uint i = tid; i < minLen; i += totalThreads) {

		const auto meanOffset     = shape::getIndexOffset(i, meanShapeInfo, minLen);
    	const auto varianceOffset = shape::getIndexOffset(i, varianceShapeInfo, minLen);

    	T sigmaInvGam = 1. / nd4j::math::nd4j_sqrt<T, T>(variance[varianceOffset] + epsilon);

    	if(gamma != nullptr)
    		sigmaInvGam *= gamma[shape::getIndexOffset(i, gammaShapeInfo, minLen)];

		auto betaOffset = 0;
    	if(beta != nullptr)
    		betaOffset = shape::getIndexOffset(i, betaShapeInfo, minLen);

    	const auto xTad = x + xTadOffsets[i];
    		  auto zTad = z + zTadOffsets[i];

    	for (uint j = 0; j < tadLen; ++j) {

    		const auto xTadOffset = shape::getIndexOffset(j, xTadShapeInfo, tadLen);
    		const auto zTadOffset = shape::getIndexOffset(j, zTadShapeInfo, tadLen);

    		zTad[zTadOffset] = (xTad[xTadOffset] - mean[meanOffset]) * sigmaInvGam;

    		if(beta != nullptr)
				zTad[zTadOffset] += beta[betaOffset];
    	}
    }
}

//////////////////////////////////////////////////////////////////////////
template<typename T>
__global__ static void batchnormCuda2(const void* vx, const Nd4jLong* xShapeInfo,
                                    const void* vMean, const Nd4jLong* meanShapeInfo,
                                    const void* vVariance, const Nd4jLong* varianceShapeInfo,
                                    const void* vGamma, const Nd4jLong* gammaShapeInfo,
                                    const void* vBeta, const Nd4jLong* betaShapeInfo,
                                          void* vz, const Nd4jLong* zShapeInfo,
                                    const int numDims, const int* dims,
                                    const T epsilon) {

    const auto x        = reinterpret_cast<const T*>(vx);
          auto z        = reinterpret_cast<T*>(vz);
    const auto mean     = reinterpret_cast<const T*>(vMean);
    const auto variance = reinterpret_cast<const T*>(vVariance);
    const auto gamma    = reinterpret_cast<const T*>(vGamma);
    const auto beta     = reinterpret_cast<const T*>(vBeta);

    __shared__ int xRank, minRank;       // xRank == zRank. minRank = meanRank = varianceRank = gammaRank = betaRank
    __shared__ Nd4jLong xLen, totalThreads, *sharedMem; // xLen = zLen


    if (threadIdx.x == 0) {

        extern __shared__ unsigned char shmem[];
        sharedMem    = reinterpret_cast<Nd4jLong*>(shmem);
        totalThreads = gridDim.x * blockDim.x;

        xLen    = shape::length(xShapeInfo);
        xRank   = shape::rank(xShapeInfo);
        minRank = shape::rank(meanShapeInfo);
    }
    __syncthreads();

    auto coords = sharedMem + threadIdx.x * xRank;
    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (uint i = tid; i < xLen; i += totalThreads) {

        shape::index2coords(xRank, shape::shapeOf(const_cast<Nd4jLong*>(xShapeInfo)), i, xLen, coords);

        const auto xOffset = shape::getOffset(0, shape::shapeOf(const_cast<Nd4jLong*>(xShapeInfo)), shape::stride(const_cast<Nd4jLong*>(xShapeInfo)), coords, xRank);
        const auto zOffset = shape::getOffset(0, shape::shapeOf(const_cast<Nd4jLong*>(zShapeInfo)), shape::stride(const_cast<Nd4jLong*>(zShapeInfo)), coords, xRank);

        if(minRank == xRank) {
            for (uint i = 0, j = 0; i < xRank; ++i) {
                if(j < numDims && i != dims[j])
                    coords[i] = 0;
                else
                    ++j;
            }
        }
        else    // minRank = numDims = 1 in this case
            coords[0] = coords[dims[0]];

        const auto meanOffset     = shape::getOffset(0, shape::shapeOf(const_cast<Nd4jLong*>(meanShapeInfo)), shape::stride(const_cast<Nd4jLong*>(meanShapeInfo)), coords, minRank);
        const auto varianceOffset = shape::getOffset(0, shape::shapeOf(const_cast<Nd4jLong*>(varianceShapeInfo)), shape::stride(const_cast<Nd4jLong*>(varianceShapeInfo)), coords, minRank);

        T sigmaInvGam = 1. / nd4j::math::nd4j_sqrt<T, T>(variance[varianceOffset] + epsilon);

        if(gamma != nullptr) {
            const auto gammaOffset = shape::getOffset(0, shape::shapeOf(const_cast<Nd4jLong*>(gammaShapeInfo)), shape::stride(const_cast<Nd4jLong*>(gammaShapeInfo)), coords, minRank);
            sigmaInvGam *= gamma[gammaOffset];
        }

        z[zOffset] = (x[xOffset] - mean[meanOffset]) * sigmaInvGam;

        if(beta != nullptr) {
            const auto betaOffset = shape::getOffset(0, shape::shapeOf(const_cast<Nd4jLong*>(betaShapeInfo)), shape::stride(const_cast<Nd4jLong*>(betaShapeInfo)), coords, minRank);
            z[zOffset] += beta[betaOffset];
        }
    }
}

///////////////////////////////////////////////////////////////////
template<typename T>
__host__ static void batchnormCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const hipStream_t *stream,
											const void* vx, const Nd4jLong* xShapeInfo,
                                           	const void* vMean, const Nd4jLong* meanShapeInfo,
											const void* vVariance, const Nd4jLong* varianceShapeInfo,
											const void* vGamma, const Nd4jLong* gammaShapeInfo,
											const void* vBeta, const Nd4jLong* betaShapeInfo,
												  void* vz, const Nd4jLong* zShapeInfo,
											const Nd4jLong* xTadShapeInfo, const Nd4jLong* xTadOffsets,
											const Nd4jLong* zTadShapeInfo, const Nd4jLong* zTadOffsets,
											const double epsilon) {

    batchnormCuda<T><<<blocksPerGrid, threadsPerBlock, 1024, *stream>>>(vx, xShapeInfo, vMean, meanShapeInfo, vVariance, varianceShapeInfo, vGamma, gammaShapeInfo, vBeta, betaShapeInfo, vz, zShapeInfo, xTadShapeInfo, xTadOffsets, zTadShapeInfo, zTadOffsets, static_cast<T>(epsilon));
}
BUILD_SINGLE_TEMPLATE(template void batchnormCudaLauncher, (const int blocksPerGrid, const int threadsPerBlock, const hipStream_t *stream, const void* vx, const Nd4jLong* xShapeInfo, const void* vMean, const Nd4jLong* meanShapeInfo, const void* vVariance, const Nd4jLong* varianceShapeInfo, const void* vGamma, const Nd4jLong* gammaShapeInfo, const void* vBeta, const Nd4jLong* betaShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const Nd4jLong* xTadShapeInfo, const Nd4jLong* xTadOffsets, const Nd4jLong* zTadShapeInfo, const Nd4jLong* zTadOffsets, const double epsilon), FLOAT_TYPES);

///////////////////////////////////////////////////////////////////
template<typename T>
__host__ static void batchnormCudaLauncher2(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                                            const void* vx, const Nd4jLong* xShapeInfo,
                                            const void* vMean, const Nd4jLong* meanShapeInfo,
                                            const void* vVariance, const Nd4jLong* varianceShapeInfo,
                                            const void* vGamma, const Nd4jLong* gammaShapeInfo,
                                            const void* vBeta, const Nd4jLong* betaShapeInfo,
                                                  void* vz, const Nd4jLong* zShapeInfo,
                                            const int numDims, const int* dims,
                                            const double epsilon) {

    batchnormCuda2<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vMean, meanShapeInfo, vVariance, varianceShapeInfo, vGamma, gammaShapeInfo, vBeta, betaShapeInfo, vz, zShapeInfo, numDims, dims, static_cast<T>(epsilon));
}
BUILD_SINGLE_TEMPLATE(template void batchnormCudaLauncher2, (const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream, const void* vx, const Nd4jLong* xShapeInfo, const void* vMean, const Nd4jLong* meanShapeInfo, const void* vVariance, const Nd4jLong* varianceShapeInfo, const void* vGamma, const Nd4jLong* gammaShapeInfo, const void* vBeta, const Nd4jLong* betaShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const int numDims, const int* dims, const double epsilon), FLOAT_TYPES);

//////////////////////////////////////////////////////////////////////////
void batchnorm(const NDArray* input, const NDArray* mean, const NDArray* variance, const NDArray* gamma, const NDArray* beta, NDArray* output, const std::vector<int>& axes, const double epsilon) {

	std::vector<int> dimsToExclude = ShapeUtils::evalDimsToExclude(input->rankOf(), axes);

	auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimsToExclude);
    auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->shapeInfo(), dimsToExclude);

    const int threadsPerBlock = MAX_NUM_THREADS / 2;
    const int blocksPerGrid = (mean->lengthOf() + threadsPerBlock - 1) / threadsPerBlock;

    PointersManager manager(input->getContext(), "batchnorm");

    NDArray::prepareSpecialUse({output}, {input, mean, variance, gamma, beta});
    BUILD_SINGLE_SELECTOR(input->dataType(), batchnormCudaLauncher, (blocksPerGrid, threadsPerBlock, input->getContext()->getCudaStream(), input->getSpecialBuffer(), input->getSpecialShapeInfo(), mean->getSpecialBuffer(), mean->getSpecialShapeInfo(), variance->getSpecialBuffer(), variance->getSpecialShapeInfo(), gamma ? gamma->getSpecialBuffer() : nullptr, gamma ? gamma->getSpecialShapeInfo() : nullptr, beta ? beta->getSpecialBuffer() : nullptr, beta ? beta->getSpecialShapeInfo() : nullptr, output->specialBuffer(), output->specialShapeInfo(), packX.platformShapeInfo(), packX.platformOffsets(), packZ.platformShapeInfo(), packZ.platformOffsets(), epsilon), FLOAT_TYPES);
    NDArray::registerSpecialUse({output}, {input, mean, variance, gamma, beta});

    manager.synchronize();


    // const int threadsPerBlock = MAX_NUM_THREADS / 4;
    // const int blocksPerGrid = (input->lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
    // const int sharedMem = sizeof(Nd4jLong) * threadsPerBlock * input->rankOf() + 128;

    // PointersManager manager(input->getContext(), "batchnorm");

    // const int* dims = reinterpret_cast<int*>(manager.replicatePointer(axes.data(), axes.size() * sizeof(int)));

    // NDArray::prepareSpecialUse({output}, {input, mean, variance, gamma, beta});
    // BUILD_SINGLE_SELECTOR(input->dataType(), batchnormCudaLauncher2, (blocksPerGrid, threadsPerBlock, sharedMem, input->getContext()->getCudaStream(), input->getSpecialBuffer(), input->getSpecialShapeInfo(), mean->getSpecialBuffer(), mean->getSpecialShapeInfo(), variance->getSpecialBuffer(), variance->getSpecialShapeInfo(), gamma ? gamma->getSpecialBuffer() : nullptr, gamma ? gamma->getSpecialShapeInfo() : nullptr, beta ? beta->getSpecialBuffer() : nullptr, beta ? beta->getSpecialShapeInfo() : nullptr, output->specialBuffer(), output->specialShapeInfo(), axes.size(), dims, epsilon), FLOAT_TYPES);
    // NDArray::registerSpecialUse({output}, {input, mean, variance, gamma, beta});

    // manager.synchronize();
}


}
}
}

