#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com), created on 07.03.2019
//


#include <ops/declarable/helpers/gather.h>
#include <numeric>
#include <PointersManager.h>
#include <ShapeUtils.h>

namespace nd4j    {
namespace ops     {
namespace helpers {

    template<typename X, typename Y>
    __global__ static void gatherCudaLinearKernel(const void* vx, const Nd4jLong* xShapeInfo, const void* vy, const Nd4jLong* yShapeInfo,
    void* vz, const Nd4jLong* zShapeInfo) {


    __shared__ const X* x;
    __shared__ const Y* y;
    __shared__ X* z;
    __shared__ Nd4jLong xLen, yLen, zLen;

    if (threadIdx.x == 0) {
        x = reinterpret_cast<const X*>(vx);
        z = reinterpret_cast<X*>(vz);
        y = reinterpret_cast<const Y *>(vy);
        xLen = shape::length(xShapeInfo);
        yLen = shape::length(yShapeInfo);
        zLen = shape::length(zShapeInfo);
    }
    __syncthreads();
    //const Nd4jLong zLen = shape::length(zShapeInfo);
    auto start = blockIdx.x * blockDim.x + threadIdx.x;
    auto step = blockDim.x * gridDim.x;

    for (int j = start; j < zLen; j += step) {
        auto zIndex = shape::getIndexOffset(j, zShapeInfo, zLen);
        auto yIndex = shape::getIndexOffset(j, yShapeInfo, yLen);
        auto xIndex = shape::getIndexOffset(y[yIndex], xShapeInfo, xLen);
        //printf("%lld , %lld\n", zIndex, xIndex);
        z[zIndex] = x[xIndex];
    }
}

//////////////////////////////////////////////////////////////////////
template<typename X, typename Y>
__global__ static void gatherCuda(const int numOfSubArrs,
                                    const void* vx, const Nd4jLong* xShapeInfo, const Nd4jLong* xOffsets,
                                    const void* vy, const Nd4jLong* yShapeInfo,
                                          void* vz, const Nd4jLong* zShapeInfo, const Nd4jLong* zOffsets) {

    const Y* y = reinterpret_cast<const Y*>(vy);
    __shared__ const X* x;
    __shared__ X* z;

    const Nd4jLong len = shape::length(xShapeInfo);
    //const Nd4jLong zLen = shape::length(zShapeInfo);
    for (int i = blockIdx.x; i < numOfSubArrs; i += gridDim.x) {

        if (threadIdx.x == 0) {

            x = reinterpret_cast<const X*>(vx) + xOffsets[y[shape::getIndexOffset(i, yShapeInfo, numOfSubArrs)]];
            z = reinterpret_cast<X*>(vz) + zOffsets[i];
        }
        __syncthreads();

        for (int j = threadIdx.x; j < len; j += blockDim.x) {
            auto zIndex = shape::getIndexOffset(j, zShapeInfo, len);
            auto xIndex = shape::getIndexOffset(j, xShapeInfo, len);
            //printf("%lld , %lld\n", zIndex, xIndex);
            z[zIndex] = x[xIndex];
        }
        __syncthreads();
    }
}

template<typename X, typename Y>
__host__ static void gatherCudaLinear(const hipStream_t *stream, const void* vx, const Nd4jLong* xShapeInfo, const void* vy, const Nd4jLong* yShapeInfo,
                                            void* vz, const Nd4jLong* zShapeInfo) {
    gatherCudaLinearKernel<X,Y><<<128, 256, 1024, *stream>>>(vx, xShapeInfo, vy, yShapeInfo, vz, zShapeInfo);
}

//////////////////////////////////////////////////////////////////////
template<typename X, typename Y>
__host__ static void gatherCudaLauncher(const hipStream_t *stream, const int numOfSubArrs,
                                    const void* vx, const Nd4jLong* xShapeInfo, const Nd4jLong* xOffsets,
                                    const void* vy, const Nd4jLong* yShapeInfo,
                                          void* vz, const Nd4jLong* zShapeInfo, const Nd4jLong* zOffsets) {
    gatherCuda<X,Y><<<numOfSubArrs, MAX_NUM_THREADS, 1024, *stream>>>(numOfSubArrs, vx, xShapeInfo, xOffsets, vy, yShapeInfo, vz, zShapeInfo, zOffsets);
}

//////////////////////////////////////////////////////////////////////
void gather(nd4j::LaunchContext * context, const NDArray* input, const NDArray* indices, NDArray* output, const std::vector<int>& intArgs) {

    const int inputRank = input->rankOf();
    int axis = intArgs.size() > 0 ? intArgs[0] : 0;
    if(axis < 0)
        axis += inputRank;

    const int numOfIntArgs = intArgs.size();

    if (indices == nullptr && numOfIntArgs == 2) { // scalar case
        output->assign((*input)(intArgs[1], {axis}));
    }
    else if (indices != nullptr && indices->isScalar()) {

        if(input->rankOf() <= 1) { //For scalar indices, rank 0 or 1 input: can't do tensor along dimension 0 as this is whole array... instead, we want to get a scalar
            auto idx = indices->e<Nd4jLong>(0);
            auto scalarNDArray = input->e(idx);
            output->assign(scalarNDArray);
        }
        else {
            NDArray inSubArr = (*input)(indices->e<Nd4jLong>(0), {axis});
            output->assign(inSubArr);
        }
    }
    else {

        NDArray* pIndices = const_cast<NDArray*>(indices);
        if(indices == nullptr)
            pIndices = new NDArray(input->ordering(), {numOfIntArgs-1}, std::vector<double>(intArgs.begin() + 1, intArgs.end()), DataType::INT64, input->getContext());

        std::vector<int> dimsOut(pIndices->rankOf());
        std::iota(dimsOut.begin(), dimsOut.end(), axis);   // fill with axis, axis+1, ... axis+pIndices->rankOf()-1

        const Nd4jLong numOfSubArrs = pIndices->lengthOf();

        Nd4jLong *outSubArrShapeInfo(nullptr), *inSubArrShapeInfo(nullptr), *outSubArrOffsets(nullptr), *inSubArrOffsets(nullptr);
        input-> getSubArrShapeAndOffsets({axis},  inSubArrShapeInfo,  inSubArrOffsets);
        output->getSubArrShapeAndOffsets(dimsOut, outSubArrShapeInfo, outSubArrOffsets);
        if (output->rankOf() > 1) {
            PointersManager manager(context, "gather");
            auto xShapeInfo = reinterpret_cast<Nd4jLong *>(manager.replicatePointer(inSubArrShapeInfo,
                                                                                    shape::shapeInfoByteLength(
                                                                                            inSubArrShapeInfo)));
            auto zShapeInfo = reinterpret_cast<Nd4jLong *>(manager.replicatePointer(outSubArrShapeInfo,
                                                                                    shape::shapeInfoByteLength(
                                                                                            outSubArrShapeInfo)));
            auto xOffsets = reinterpret_cast<Nd4jLong *>(manager.replicatePointer(inSubArrOffsets, (input->lengthOf() /
                                                                                                    shape::length(
                                                                                                            inSubArrShapeInfo)) *
                                                                                                   sizeof(Nd4jLong)));
            auto zOffsets = reinterpret_cast<Nd4jLong *>(manager.replicatePointer(outSubArrOffsets,
                                                                                  (output->lengthOf() /
                                                                                   shape::length(outSubArrShapeInfo)) *
                                                                                  sizeof(Nd4jLong)));

            NDArray::prepareSpecialUse({output}, {input, pIndices});
            BUILD_DOUBLE_SELECTOR(input->dataType(), pIndices->dataType(), gatherCudaLauncher, (context->getCudaStream(), numOfSubArrs, input->getSpecialBuffer(), xShapeInfo, xOffsets, pIndices->getSpecialBuffer(), pIndices->getSpecialShapeInfo(), output->getSpecialBuffer(), zShapeInfo, zOffsets), NUMERIC_TYPES, INTEGER_TYPES);
            NDArray::registerSpecialUse({output}, {input, pIndices});
            manager.synchronize();
        }
        else {
            NDArray::prepareSpecialUse({output}, {input, pIndices});
            BUILD_DOUBLE_SELECTOR(input->dataType(), pIndices->dataType(), gatherCudaLinear, (context->getCudaStream(), input->getSpecialBuffer(), input->getSpecialShapeInfo(), pIndices->getSpecialBuffer(), pIndices->getSpecialShapeInfo(), output->specialBuffer(), output->specialShapeInfo()), NUMERIC_TYPES, INTEGER_TYPES);
            NDArray::registerSpecialUse({output}, {input, pIndices});

        }

        if(indices == nullptr)
            delete pIndices;

    }
}


BUILD_DOUBLE_TEMPLATE(template void gatherCudaLauncher, (const hipStream_t *stream, const int numOfSubArrs, const void* vx, const Nd4jLong* xShapeInfo, const Nd4jLong* xOffsets, const void* vy, const Nd4jLong* yShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const Nd4jLong* zOffsets), NUMERIC_TYPES, INTEGER_TYPES);
BUILD_DOUBLE_TEMPLATE(template void gatherCudaLinear, (const hipStream_t *stream, const void* vx, const Nd4jLong* xShapeInfo, const void* vy, const Nd4jLong* yShapeInfo, void* vz, const Nd4jLong* zShapeInfo), NUMERIC_TYPES, INTEGER_TYPES);



}
}
}