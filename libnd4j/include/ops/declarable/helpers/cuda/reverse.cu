#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma, created on 16.04.2018
//

#include <ops/declarable/helpers/reverse.h>
#include <helpers/ShapeUtils.h>
#include <array/ResultSet.h>
#include <TAD.h>
#include <PointersManager.h>
#include <ConstantTadHelper.h>


namespace nd4j    {
namespace ops     {
namespace helpers {

    template <typename T>
    inline void __device__ indexSwap(T* arr, Nd4jLong idx1, Nd4jLong idx2) {
        T tmp = arr[idx1];
        arr[idx1] = arr[idx2];
        arr[idx2] = tmp;
    }
//    template <typename T>
//    void reverseArray(nd4j::LaunchContext * context, void* inArr, Nd4jLong *inShapeBuffer, void *result, Nd4jLong *zShapeBuffer, int numOfElemsToReverse = 0);

    /////////////////////////////////////////////////////////////////////////////////////
    template <typename T>
    static __global__ void reverseArrayInplaceKernel(void *input, Nd4jLong *inputShape, Nd4jLong numOfElemsToReverse) {
        const auto tid = blockIdx.x * gridDim.x + threadIdx.x;
        const auto step = gridDim.x * blockDim.x;
        __shared__ Nd4jLong length;
        __shared__ int linearStatus;
        __shared__ T* inputArr;
        if (threadIdx.x == 0) {
            length = shape::length(inputShape);
            linearStatus = shape::elementWiseStride(inputShape);
            inputArr = reinterpret_cast<T*>(input);
        }
        __syncthreads();

        for (Nd4jLong e = tid; e < numOfElemsToReverse / 2; e += step) {
            if (linearStatus == 1) {
                auto idx = numOfElemsToReverse - e - 1;
                indexSwap(inputArr, e, idx);
            }
            else if (linearStatus > 1) {
                auto idx1 = (numOfElemsToReverse - e - 1) * linearStatus;
                Nd4jLong idx2 =  e * linearStatus;
                indexSwap(inputArr, idx1, idx2);
            }
            else {
                auto inOffset  = shape::getIndexOffset(e, inputShape, length);
                auto outOffset = shape::getIndexOffset(numOfElemsToReverse - e - 1, inputShape, length);
                indexSwap(inputArr, inOffset, outOffset);
            }
        }
    }

    template <typename T>
    static __global__ void reverseArrayKernel(void* input, Nd4jLong *inputShape, void* output, Nd4jLong *outputShape, Nd4jLong numOfElemsToReverse) {
        const auto tid = blockIdx.x * gridDim.x + threadIdx.x;
        const auto step = gridDim.x * blockDim.x;
        __shared__ Nd4jLong length;
        __shared__ int linearStatus;
        __shared__ T* inputArr;
        __shared__ T* outputArr;
        __shared__ char inputOrder, outputOrder;

        if (threadIdx.x == 0) {
            length = shape::length(inputShape);
            linearStatus = (shape::elementWiseStride(inputShape) == shape::elementWiseStride(outputShape)) && (inputOrder == outputOrder)? shape::elementWiseStride(inputShape):0;

            char inputOrder = shape::order(inputShape);
            char outputOrder = shape::order(outputShape);
            inputArr = reinterpret_cast<T*>(input);
            outputArr = reinterpret_cast<T*>(output);
        }
        __syncthreads();

        for (Nd4jLong e = tid; e < length; e += step) {
            if (e < numOfElemsToReverse ) {
                if (linearStatus == 1) {
                    auto idx = numOfElemsToReverse - e - 1;
                    outputArr[idx] = inputArr[e];
                } else if (linearStatus > 1) {
                    auto idx1 = (numOfElemsToReverse - e - 1) * linearStatus;
                    Nd4jLong idx2 = e * linearStatus;
                    outputArr[idx1] = inputArr[idx2];
                } else {
                    auto inOffset = shape::getIndexOffset(e, inputShape, length);
                    auto outOffset = shape::getIndexOffset(numOfElemsToReverse - e - 1, outputShape, length);
                    outputArr[outOffset] = inputArr[inOffset];
                }
            }
            else {
                if (linearStatus == 1) {
                    outputArr[e] = inputArr[e];
                } else if (linearStatus > 1) {
                    auto idx1 = e * linearStatus;
                    Nd4jLong idx2 = e * linearStatus;
                    outputArr[idx1] = inputArr[idx2];
                } else {
                    auto inOffset = shape::getIndexOffset(e, inputShape, length);
                    auto outOffset = shape::getIndexOffset(e, outputShape, length);
                    outputArr[outOffset] = inputArr[inOffset];
                }
            }
        }

        //printf("\n");
    }

    template<typename T>
    static void reverseArray(nd4j::LaunchContext * context, NDArray* input, NDArray* output, int numOfElemsToReverse) {
        auto stream = context->getCudaStream();
        Nd4jLong numOfReverse = numOfElemsToReverse;
        if (numOfElemsToReverse == 0)
            numOfReverse = input->lengthOf();
        if (input == output) {
            reverseArrayInplaceKernel<T><<<256, 512, 8192, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), numOfReverse);
        }
        else {
            reverseArrayKernel<T><<<256, 512, 8192, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(), numOfReverse);
        }
    }


    ///////////////////////////////////////////////////////////////////
    template <typename T>
    static void _reverseSequence(nd4j::LaunchContext * context, const NDArray* input, const NDArray* seqLengths, NDArray* output, int seqDim, const int batchDim){
        int posOfNonUnityDim = -1;
        seqLengths->syncToHost();
        auto stream = context->getCudaStream();

        NDArray::prepareSpecialUse({output}, {input, seqLengths});
        if(input->isVector() || shape::isLikeVector(input->getShapeInfo(), posOfNonUnityDim) || seqLengths->lengthOf() == 1) {
            int numOfElemsToReverse = seqLengths->e<int>(0);
//            printf("Length %d\n", numOfElemsToReverse);
//            input->printBuffer("INPUT");
            if((seqDim == 0 && input->sizeAt(0) == 1) || (batchDim == posOfNonUnityDim))
                output->assign(input);
            else
                reverseArrayKernel<T><<<256, 512, 8192, *stream>>>(input->getSpecialBuffer(), input->getSpecialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(), numOfElemsToReverse);//helpers::reverseArray<T>(context, const_cast<NDArray*>(input), output, numOfElemsToReverse);
        }
        else {

            if(seqDim > batchDim)
                --seqDim;

            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {batchDim});

            auto inSubArrsSet  = input->allTensorsAlongDimension(dimensions);
            auto outSubArrsSet = output->allTensorsAlongDimension(dimensions);

// #pragma omp parallel for schedule(guided)  if(inSubArrsSet->size() > Environment::getInstance()->elementwiseThreshold())
            for(int i = 0; i < inSubArrsSet->size(); ++i) {

                int numOfElemsToReverse = seqLengths->e<int>(i);

                if(numOfElemsToReverse == 0 || numOfElemsToReverse == 1) {
                    outSubArrsSet->at(i)->assign(inSubArrsSet->at(i));
                }
                else {
                    auto inInnerSet  = inSubArrsSet->at(i)->allTensorsAlongDimension({seqDim});
                    auto outInnerSet = outSubArrsSet->at(i)->allTensorsAlongDimension({seqDim});
                    for(int j = 0; j < inInnerSet->size(); ++j)
                        reverseArray<T>(context, inInnerSet->at(j), outInnerSet->at(j), numOfElemsToReverse);

                    delete inInnerSet;
                    delete outInnerSet;
                }
            }
            delete inSubArrsSet;
            delete outSubArrsSet;
        }
        NDArray::registerSpecialUse({output}, {input, seqLengths});
    }

    void reverseSequence(nd4j::LaunchContext * context, const NDArray* input, const NDArray* seqLengths, NDArray* output, int seqDim, const int batchDim) {
        BUILD_SINGLE_SELECTOR(input->dataType(), _reverseSequence, (context, input, seqLengths, output, seqDim, batchDim), LIBND4J_TYPES);
    }

    //////////////////////////////////////////////////////////////////////////
    void reverse(nd4j::LaunchContext * context, const NDArray* input, NDArray* output, const std::vector<int>* intArgs, bool isBackProp) {
        // we need to reverse axis only if that's new op
        std::vector<int> dimensions = isBackProp ? ShapeUtils::evalDimsToExclude(input->rankOf(), *intArgs) : *intArgs;
        std::vector<int> axis = ShapeUtils::evalDimsToExclude(input->rankOf(), dimensions);
        auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), axis);
        auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), axis);

        auto listOut = output->allTensorsAlongDimension(dimensions);
        auto listIn  = input->allTensorsAlongDimension(dimensions);

        NDArray *subArrIn, *subArrOut;

        NDArray::prepareSpecialUse({output}, {input});
        for(int i = 0; i < listIn->size(); ++i) {               // listIn->size() = listOut->size()
            subArrIn   = listIn->at(i);
            subArrOut  = listOut->at(i);
            BUILD_SINGLE_SELECTOR(input->dataType(), reverseArray, (context, subArrIn, subArrOut, 0), LIBND4J_TYPES);
        }
        //BUILD_SINGLE_SELECTOR(input->dataType(), reverseArray, (context, const_cast<NDArray*>(input), output, (int)0), LIBND4J_TYPES);
        NDArray::registerSpecialUse({output}, {input});
        delete listOut;
        delete listIn;
    }

BUILD_SINGLE_TEMPLATE(template void reverseArray, (nd4j::LaunchContext * context, NDArray *inArr, NDArray *outArr, int numOfElemsToReverse), LIBND4J_TYPES);

}
}
}

