#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author GS <sgazeos@gmail.com>
//

#include <ops/declarable/helpers/segment.h>
#include <NDArrayFactory.h>
#include <helpers/ShapeUtils.h>
#include <helpers/TAD.h>
#include <exceptions/cuda_exception.h>
#include <PointersManager.h>
#include <ConstantTadHelper.h>

namespace nd4j {
namespace ops {
namespace helpers {
    // -------------------------------------------------------------------------------------------------------------- //
    // Segment ops linear kernels
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static __global__ void segmentMaxLinearKernel(void* input, Nd4jLong* inputShape, int* starts, int* lengths, Nd4jLong numOfClasses, void* output, Nd4jLong* outputShape) {
         __shared__ T* val;
         __shared__ Nd4jLong xLen, zLen, segment, zIndex;
         __shared__ T* x;
         __shared__ T* z;
         __shared__ int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
            threadsPerSegment = (gridDim.x + numOfClasses - 1) / numOfClasses;
            segment = blockIdx.x / threadsPerSegment;
            x = reinterpret_cast<T*>(input);
            z = reinterpret_cast<T*>(output);
            extern __shared__ unsigned char shmem[];
            val = reinterpret_cast<T*>(shmem);
            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);

            if (segment < numOfClasses) {
                zIndex = shape::getIndexOffset(segment, outputShape, zLen);
                start = starts[segment];
                finish = start + lengths[segment];
                z[zIndex] = x[shape::getIndexOffset(start, inputShape, xLen)];
                val[segment] = z[zIndex];
            }

        }
        __syncthreads();

         for (auto e = start + threadIdx.x + 1; e < finish; e += blockDim.x) {
             auto xIndex = shape::getIndexOffset(e, inputShape, xLen);
             nd4j::math::atomics::nd4j_atomicMax(&z[zIndex], x[xIndex]);
         }
    }
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static __global__ void unsortedSegmentMaxLinearKernel(void* input, Nd4jLong* inputShape, void* indices, Nd4jLong* indicesShape, int* starts, int* lengths, Nd4jLong numOfClasses, void* output, Nd4jLong* outputShape) {
        __shared__ T* val;
        __shared__ Nd4jLong xLen, zLen, segment, zIndex;
        __shared__ T* x;
        __shared__ T* z;
        __shared__ I* y; //int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
//            threadsPerSegment = (gridDim.x + numOfClasses - 1) / numOfClasses;
            segment = blockIdx.x;// / threadsPerSegment;
            x = reinterpret_cast<T*>(input);
            z = reinterpret_cast<T*>(output);
            y = reinterpret_cast<I*>(indices);
//            extern __shared__ unsigned char shmem[];
//            val = reinterpret_cast<T*>(shmem);
            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);

//            if (segment < numOfClasses) {
            zIndex = shape::getIndexOffset(segment, outputShape, zLen);
            //start = starts[segment];
            //finish = start + lengths[segment];
            if (lengths[segment] > 0)
                z[zIndex] = x[shape::getIndexOffset(starts[segment], inputShape, xLen)];
            else
                z[zIndex] = -DataTypeUtils::max<T>();
//                val[segment] = z[zIndex];
//            }

        }
        __syncthreads();
        if (lengths[segment] > 0)
        for (auto e = threadIdx.x + 1; e < xLen; e += blockDim.x) {
            auto xIndex = shape::getIndexOffset(e, inputShape, xLen);
            auto yIndex = shape::getIndexOffset(e, indicesShape, xLen);
            if (y[yIndex] == segment) {
                nd4j::math::atomics::nd4j_atomicMax(&z[zIndex], x[xIndex]);
            }
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static __global__ void segmentMinLinearKernel(void* input, Nd4jLong* inputShape, int* starts, int* lengths, Nd4jLong numOfClasses, void* output, Nd4jLong* outputShape) {
        __shared__ T* val;
        __shared__ Nd4jLong xLen, zLen, segment, zIndex;
        __shared__ T* x;
        __shared__ T* z;
        __shared__ int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
            threadsPerSegment = (gridDim.x + numOfClasses - 1) / numOfClasses;
            segment = blockIdx.x / threadsPerSegment;
            x = reinterpret_cast<T*>(input);
            z = reinterpret_cast<T*>(output);
            extern __shared__ unsigned char shmem[];
            val = reinterpret_cast<T*>(shmem);
            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);

            if (segment < numOfClasses) {
                zIndex = shape::getIndexOffset(segment, outputShape, zLen);
                start = starts[segment];
                finish = start + lengths[segment];
                z[zIndex] = x[shape::getIndexOffset(start, inputShape, xLen)];
                val[segment] = z[zIndex];
            }

        }
        __syncthreads();

        for (auto e = start + threadIdx.x + 1; e < finish; e += blockDim.x) {
            auto xIndex = shape::getIndexOffset(e, inputShape, xLen);
           nd4j::math::atomics::nd4j_atomicMin(&z[zIndex], x[xIndex]);
        }

    }
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static __global__ void unsortedSegmentMinLinearKernel(void* input, Nd4jLong* inputShape, void* indices, Nd4jLong* indicesShape, int* starts, int* lengths, Nd4jLong numOfClasses, void* output, Nd4jLong* outputShape) {
        __shared__ T* val;
        __shared__ Nd4jLong xLen, zLen, segment, zIndex;
        __shared__ T* x;
        __shared__ T* z;
        __shared__ I* y; //int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
//            threadsPerSegment = (gridDim.x + numOfClasses - 1) / numOfClasses;
            segment = blockIdx.x;// / threadsPerSegment;
            x = reinterpret_cast<T*>(input);
            z = reinterpret_cast<T*>(output);
            y = reinterpret_cast<I*>(indices);
//            extern __shared__ unsigned char shmem[];
//            val = reinterpret_cast<T*>(shmem);
            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);

//            if (segment < numOfClasses) {
            zIndex = shape::getIndexOffset(segment, outputShape, zLen);
            //start = starts[segment];
            //finish = start + lengths[segment];
            if (lengths[segment] > 0)
                z[zIndex] = x[shape::getIndexOffset(starts[segment], inputShape, xLen)];
            else
                z[zIndex] = DataTypeUtils::max<T>();
//                val[segment] = z[zIndex];
//            }

        }
        __syncthreads();
        if (lengths[segment] > 0)
            for (auto e = threadIdx.x + 1; e < xLen; e += blockDim.x) {
                auto xIndex = shape::getIndexOffset(e, inputShape, xLen);
                auto yIndex = shape::getIndexOffset(e, indicesShape, xLen);
                if (y[yIndex] == segment) {
                    nd4j::math::atomics::nd4j_atomicMin(&z[zIndex], x[xIndex]);
                }
            }
    }
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static __global__ void segmentSumLinearKernel(void* input, Nd4jLong* inputShape, int* starts, int* lengths, Nd4jLong numOfClasses, void* output, Nd4jLong* outputShape) {
        __shared__ T* val;
        __shared__ Nd4jLong xLen, zLen, segment, zIndex;
        __shared__ T* x;
        __shared__ T* z;
        __shared__ int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
            threadsPerSegment = (gridDim.x + numOfClasses - 1) / numOfClasses;
            segment = blockIdx.x / threadsPerSegment;
            x = reinterpret_cast<T*>(input);
            z = reinterpret_cast<T*>(output);

            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);


            if (segment < numOfClasses) {
                zIndex = shape::getIndexOffset(segment, outputShape, zLen);
                start = starts[segment];
                finish = start + lengths[segment];
                //val[segment] = ;
                z[zIndex] = x[shape::getIndexOffset(start, inputShape, xLen)];
//                val[segment] = z[zIndex];
            }

        }
        __syncthreads();

        for (auto e = start + threadIdx.x + 1; e < finish; e += blockDim.x) {
            auto xIndex = shape::getIndexOffset(e, inputShape, xLen);
            nd4j::math::atomics::nd4j_atomicAdd(&z[zIndex], x[xIndex]);
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static __global__ void unsortedSegmentSumLinearKernel(void* input, Nd4jLong* inputShape, void* indices, Nd4jLong* indicesShape, int* starts, int* lengths, Nd4jLong numOfClasses, void* output, Nd4jLong* outputShape) {
        __shared__ T* val;
        __shared__ Nd4jLong xLen, zLen, segment, zIndex;
        __shared__ T* x;
        __shared__ T* z;
        __shared__ I* y; //int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
//            threadsPerSegment = (gridDim.x + numOfClasses - 1) / numOfClasses;
            segment = blockIdx.x;// / threadsPerSegment;
            x = reinterpret_cast<T*>(input);
            z = reinterpret_cast<T*>(output);
            y = reinterpret_cast<I*>(indices);
//            extern __shared__ unsigned char shmem[];
//            val = reinterpret_cast<T*>(shmem);
            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);

//            if (segment < numOfClasses) {
            zIndex = shape::getIndexOffset(segment, outputShape, zLen);
            //start = starts[segment];
            //finish = start + lengths[segment];
            if (lengths[segment] > 0)
                z[zIndex] = x[shape::getIndexOffset(starts[segment], inputShape, xLen)];
            else
                z[zIndex] = 0; //DataTypeUtils::max<T>();
//                val[segment] = z[zIndex];
//            }

        }
        __syncthreads();
        if (lengths[segment] > 0)
            for (auto e = threadIdx.x; e < xLen; e += blockDim.x) {
                auto xIndex = shape::getIndexOffset(e, inputShape, xLen);
                auto yIndex = shape::getIndexOffset(e, indicesShape, xLen);
                if (y[yIndex] == segment && e != starts[segment]) {
                    nd4j::math::atomics::nd4j_atomicAdd(&z[zIndex], x[xIndex]);
                }
            }
    }
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static __global__ void segmentMeanLinearKernel(void* input, Nd4jLong* inputShape, int* starts, int* lengths, Nd4jLong numOfClasses, void* output, Nd4jLong* outputShape) {
        __shared__ T* val;
        __shared__ Nd4jLong xLen, zLen, segment, zIndex;
        __shared__ T* x;
        __shared__ T* z;
        __shared__ int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
            threadsPerSegment = (gridDim.x + numOfClasses - 1) / numOfClasses;
            segment = blockIdx.x / threadsPerSegment;
            x = reinterpret_cast<T*>(input);
            z = reinterpret_cast<T*>(output);
//            extern __shared__ unsigned char shmem[];
//            val = reinterpret_cast<T*>(shmem);
            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);

            //[zIndex] =
            if (segment < numOfClasses) {
                zIndex = shape::getIndexOffset(segment, outputShape, zLen);
                start = starts[segment];
                finish = start + lengths[segment];
                //val[segment] = ;
                z[zIndex] = T(x[shape::getIndexOffset(start, inputShape, xLen)] / lengths[segment]);
//                val[segment] = z[zIndex];
            }

        }
        __syncthreads();

        for (auto e = start + threadIdx.x + 1; e < finish; e += blockDim.x) {
            auto xIndex = shape::getIndexOffset(e, inputShape, xLen);
            if (lengths[segment])
            nd4j::math::atomics::nd4j_atomicAdd(&z[zIndex], T(x[xIndex] / lengths[segment]));
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    static __global__ void unsortedSegmentMeanLinearKernel(void* input, Nd4jLong* inputShape, void* indices, Nd4jLong* indicesShape, int* starts, int* lengths, Nd4jLong numOfClasses, void* output, Nd4jLong* outputShape) {
        __shared__ T* val;
        __shared__ Nd4jLong xLen, zLen, segment, zIndex;
        __shared__ T* x;
        __shared__ T* z;
        __shared__ I* y; //int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
//            threadsPerSegment = (gridDim.x + numOfClasses - 1) / numOfClasses;
            segment = blockIdx.x;// / threadsPerSegment;
            x = reinterpret_cast<T*>(input);
            z = reinterpret_cast<T*>(output);
            y = reinterpret_cast<I*>(indices);
//            extern __shared__ unsigned char shmem[];
//            val = reinterpret_cast<T*>(shmem);
            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);

//            if (segment < numOfClasses) {
            zIndex = shape::getIndexOffset(segment, outputShape, zLen);
            //start = starts[segment];
            //finish = start + lengths[segment];
            if (lengths[segment] > 0)
                z[zIndex] = T(x[shape::getIndexOffset(starts[segment], inputShape, xLen)] / T(lengths[segment]));
            else
                z[zIndex] = 0; //DataTypeUtils::max<T>();
//                val[segment] = z[zIndex];
//            }

        }
        __syncthreads();
        if (lengths[segment] > 0)
            for (auto e = threadIdx.x; e < xLen; e += blockDim.x) {
                auto xIndex = shape::getIndexOffset(e, inputShape, xLen);
                auto yIndex = shape::getIndexOffset(e, indicesShape, xLen);
                if (y[yIndex] == segment && e != starts[segment]) {
                    nd4j::math::atomics::nd4j_atomicAdd(&z[zIndex], T(x[xIndex]/T(lengths[segment])));
                }
            }
    }
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static __global__ void segmentProdLinearKernel(void* input, Nd4jLong* inputShape, int* starts, int* lengths, Nd4jLong numOfClasses, void* output, Nd4jLong* outputShape) {
        __shared__ T* val;
        __shared__ Nd4jLong xLen, zLen, segment, zIndex;
        __shared__ T* x;
        __shared__ T* z;
        __shared__ int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
            threadsPerSegment = (gridDim.x + numOfClasses - 1) / numOfClasses;
            segment = blockIdx.x / threadsPerSegment;
            x = reinterpret_cast<T*>(input);
            z = reinterpret_cast<T*>(output);
            extern __shared__ unsigned char shmem[];
            val = reinterpret_cast<T*>(shmem);
            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);

            if (segment < numOfClasses) {
                zIndex = shape::getIndexOffset(segment, outputShape, zLen);
                start = starts[segment];
                finish = start + lengths[segment];
                //val[segment] = ;
                z[zIndex] = x[shape::getIndexOffset(start, inputShape, xLen)];
                val[segment] = z[zIndex];
            }

        }
        __syncthreads();
//         auto tid = threadIdx.x + blockIdx.x * blockDim.x;
//         auto step = blockDim.x * gridDim.x;

        for (auto e = start + threadIdx.x + 1; e < finish; e += blockDim.x) {
            auto xIndex = shape::getIndexOffset(e, inputShape, xLen);
            nd4j::math::atomics::nd4j_atomicMul(&val[segment], x[xIndex]);
        }
        __syncthreads();

        if (threadIdx.x == 0) {
            z[zIndex] = val[segment];
        }

    }
    template <typename T, typename I>
    static __global__ void unsortedSegmentProdLinearKernel(void* input, Nd4jLong* inputShape, void* indices, Nd4jLong* indicesShape, int* starts, int* lengths, Nd4jLong numOfClasses, void* output, Nd4jLong* outputShape) {
        __shared__ T* val;
        __shared__ Nd4jLong xLen, zLen, segment, zIndex;
        __shared__ T* x;
        __shared__ T* z;
        __shared__ I* y; //int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
//            threadsPerSegment = (gridDim.x + numOfClasses - 1) / numOfClasses;
            segment = blockIdx.x;// / threadsPerSegment;
            x = reinterpret_cast<T*>(input);
            z = reinterpret_cast<T*>(output);
            y = reinterpret_cast<I*>(indices);
//            extern __shared__ unsigned char shmem[];
//            val = reinterpret_cast<T*>(shmem);
            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);

//            if (segment < numOfClasses) {
            zIndex = shape::getIndexOffset(segment, outputShape, zLen);
            //start = starts[segment];
            //finish = start + lengths[segment];
            if (lengths[segment] > 0)
                z[zIndex] = x[shape::getIndexOffset(starts[segment], inputShape, xLen)];
            else
                z[zIndex] = 0; //DataTypeUtils::max<T>();
//                val[segment] = z[zIndex];
//            }

        }
        __syncthreads();
        if (lengths[segment] > 0)
            for (auto e = threadIdx.x; e < xLen; e += blockDim.x) {
                auto xIndex = shape::getIndexOffset(e, inputShape, xLen);
                auto yIndex = shape::getIndexOffset(e, indicesShape, xLen);
                if (y[yIndex] == segment && e != starts[segment]) {
                    nd4j::math::atomics::nd4j_atomicMul(&z[zIndex], x[xIndex]);
                }
            }
    }
    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    static __global__ void unsortedSegmentSqrtNLinearKernel(void* input, Nd4jLong* inputShape, void* indices, Nd4jLong* indicesShape, int* starts, int* lengths, Nd4jLong numOfClasses, void* output, Nd4jLong* outputShape) {
        __shared__ T* val;
        __shared__ Nd4jLong xLen, zLen, segment, zIndex;
        __shared__ T* x;
        __shared__ T* z;
        __shared__ I* y; //int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
//            threadsPerSegment = (gridDim.x + numOfClasses - 1) / numOfClasses;
            segment = blockIdx.x;// / threadsPerSegment;
            x = reinterpret_cast<T*>(input);
            z = reinterpret_cast<T*>(output);
            y = reinterpret_cast<I*>(indices);
//            extern __shared__ unsigned char shmem[];
//            val = reinterpret_cast<T*>(shmem);
            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);

//            if (segment < numOfClasses) {
            zIndex = shape::getIndexOffset(segment, outputShape, zLen);
            //start = starts[segment];
            //finish = start + lengths[segment];
            if (lengths[segment] > 0)
                z[zIndex] = x[shape::getIndexOffset(starts[segment], inputShape, xLen)] / nd4j::math::nd4j_sqrt<int, T>(lengths[segment]);
            else
                z[zIndex] = 0; //DataTypeUtils::max<T>();
//                val[segment] = z[zIndex];
//            }

        }
        __syncthreads();
        if (lengths[segment] > 0)
            for (auto e = threadIdx.x + 1; e < xLen; e += blockDim.x) {
                auto xIndex = shape::getIndexOffset(e, inputShape, xLen);
                auto yIndex = shape::getIndexOffset(e, indicesShape, xLen);
                if (y[yIndex] == segment && e != starts[segment]) {
                    nd4j::math::atomics::nd4j_atomicAdd(&z[zIndex], x[xIndex] / nd4j::math::nd4j_sqrt<int, T>(lengths[segment]));
                }
            }
    }
    // -------------------------------------------------------------------------------------------------------------- //
    // fill up segments starts and ends - splitted ordered case
    template <typename I>
    static __global__ void fillUpSegmentsKernel(void* indices, Nd4jLong* indexShape, int numClasses, int* classesRangesStart, int* classesRangesLenghts) {
        __shared__ I* idxBuf;
        __shared__ Nd4jLong idxLen;
        __shared__ int* result;
        if (threadIdx.x == 0) {
            idxBuf = reinterpret_cast<I*>(indices);
            idxLen = shape::length(indexShape);
        }
        __syncthreads();

        auto tid = threadIdx.x + blockDim.x * blockIdx.x;
        auto step = blockDim.x * gridDim.x;

        for (auto j = tid; j < idxLen; j += step) {
            auto pos = idxBuf[j];
            nd4j::math::atomics::nd4j_atomicMin(&classesRangesStart[pos], (int)j);
            nd4j::math::atomics::nd4j_atomicAdd(&classesRangesLenghts[pos], 1);
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //

    // -------------------------------------------------------------------------------------------------------------- //
    // fill up segments starts and counts - cumulative  case
    template <typename I>
    static __global__ void fillUpUnsortedSegmentsKernel(void* indices, Nd4jLong* indexShape, int numClasses, int* classes) {
        __shared__ I* idxBuf;
        __shared__ Nd4jLong idxLen;
        __shared__ int* result;
        if (threadIdx.x == 0) {
            idxBuf = reinterpret_cast<I*>(indices);
            idxLen = shape::length(indexShape);
        }
        __syncthreads();

        auto tid = threadIdx.x + blockDim.x * blockIdx.x;
        auto step = blockDim.x * gridDim.x;

        for (auto j = tid; j < idxLen; j += step) {
            auto k = idxBuf[j];
            auto beginPos = 2 * k;
            auto sizePos = beginPos + 1;
            printf("%d, %d\n", beginPos, sizePos);
            nd4j::math::atomics::nd4j_atomicMin(&classes[beginPos], (int)j);
            nd4j::math::atomics::nd4j_atomicAdd(&classes[sizePos], 1);
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //

    // -------------------------------------------------------------------------------------------------------------- //
    // segment ops multidimentional cases
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static __global__ void segmentMaxTadKernel(void* inputBuf, Nd4jLong* inputShape, Nd4jLong* inputTads,
            Nd4jLong* inputTadOffsets, I* indices, int* starts, int* lengths, Nd4jLong numOfClasses, void* outputBuf,
            Nd4jLong* outputShape, Nd4jLong* outputTads, Nd4jLong* outputTadOffsets, T filler = 0) {

        __shared__ T* val;
        __shared__ Nd4jLong len, segment, zIndex, total;
        __shared__ T* z;
        __shared__ int start, finish;

        if (threadIdx.x == 0) {
            segment = indices[blockIdx.x]; // / threadsPerSegment;
            z = reinterpret_cast<T*>(outputBuf) + outputTadOffsets[segment];
            len = shape::length(inputTads);

            start = starts[segment];
            finish = start + lengths[segment];
            total = shape::sizeAt(inputShape, 0);
        }
        __syncthreads();

        auto idx = blockIdx.x;
        if (blockIdx.x <= total) {
            auto x = reinterpret_cast<T *>(inputBuf) + inputTadOffsets[idx];
            if (blockIdx.x == start) {
                for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                    auto xIndex = shape::getIndexOffset(e, inputTads, len);
                    auto zIndex = shape::getIndexOffset(e, outputTads, len);
                    z[zIndex] = x[xIndex];
                }
            }
            else {
                for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                    auto xIndex = shape::getIndexOffset(e, inputTads, len);
                    auto zIndex = shape::getIndexOffset(e, outputTads, len);
                    nd4j::math::atomics::nd4j_atomicMax(&z[zIndex], x[xIndex]);
                }
            }
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //

    // SegmentMin kernel
    template <typename T, typename I>
    static __global__ void segmentMinTadKernel(void* inputBuf, Nd4jLong* inputShape, Nd4jLong* inputTads, Nd4jLong* inputTadOffsets, I* indices, int* starts, int* lengths, Nd4jLong numOfClasses, void* outputBuf, Nd4jLong* outputShape, Nd4jLong* outputTads, Nd4jLong* outputTadOffsets) {
        __shared__ T* val;
        __shared__ Nd4jLong len, segment, zIndex, total;
        __shared__ T* z;
        __shared__ int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
            segment = indices[blockIdx.x]; // / threadsPerSegment;
            z = reinterpret_cast<T*>(outputBuf) + outputTadOffsets[segment];
            len = shape::length(inputTads);
            start = starts[segment];
            finish = start + lengths[segment];
            total = shape::sizeAt(inputShape, 0);

        }
        __syncthreads();

        auto idx = blockIdx.x;
        if (blockIdx.x <= total) {
            auto x = reinterpret_cast<T *>(inputBuf) + inputTadOffsets[idx];
            if (blockIdx.x == start) {
                for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                    auto xIndex = shape::getIndexOffset(e, inputTads, len);
                    auto zIndex = shape::getIndexOffset(e, outputTads, len);
                    z[zIndex] = x[xIndex];
                }
            }
            else {
                for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                    auto xIndex = shape::getIndexOffset(e, inputTads, len);
                    auto zIndex = shape::getIndexOffset(e, outputTads, len);
                    nd4j::math::atomics::nd4j_atomicMin(&z[zIndex], x[xIndex]);
                }
            }
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //

    // SegmentSum kernel
    template <typename T, typename I>
    static __global__ void segmentSumTadKernel(void* inputBuf, Nd4jLong* inputShape, Nd4jLong* inputTads, Nd4jLong* inputTadOffsets, I* indices, int* starts, int* lengths, Nd4jLong numOfClasses, void* outputBuf, Nd4jLong* outputShape, Nd4jLong* outputTads, Nd4jLong* outputTadOffsets) {
        __shared__ T* val;
        __shared__ Nd4jLong len, segment, zIndex, total;
        __shared__ T* z;
        __shared__ int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
            segment = indices[blockIdx.x]; // / threadsPerSegment;
            z = reinterpret_cast<T*>(outputBuf) + outputTadOffsets[segment];
            len = shape::length(inputTads);
            start = starts[segment];
            finish = start + lengths[segment];
            total = shape::sizeAt(inputShape, 0);

        }
        __syncthreads();

        auto idx = blockIdx.x;
        if (blockIdx.x <= total) {
            auto x = reinterpret_cast<T *>(inputBuf) + inputTadOffsets[idx];
            if (blockIdx.x == start) {
                for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                    auto xIndex = shape::getIndexOffset(e, inputTads, len);
                    auto zIndex = shape::getIndexOffset(e, outputTads, len);
                    z[zIndex] = x[xIndex];
                }
            }
            else {
                for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                    auto xIndex = shape::getIndexOffset(e, inputTads, len);
                    auto zIndex = shape::getIndexOffset(e, outputTads, len);
                    if (lengths[segment])
                    nd4j::math::atomics::nd4j_atomicAdd(&z[zIndex], x[xIndex]);
                }
            }
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //

    // SegmentMean kernel
    template <typename T, typename I>
    static __global__ void segmentMeanTadKernel(void* inputBuf, Nd4jLong* inputShape, Nd4jLong* inputTads, Nd4jLong* inputTadOffsets, I* indices, int* starts, int* lengths, Nd4jLong numOfClasses, void* outputBuf, Nd4jLong* outputShape, Nd4jLong* outputTads, Nd4jLong* outputTadOffsets) {
        __shared__ T* val;
        __shared__ Nd4jLong len, segment, zIndex, total;
        __shared__ T* z;
        __shared__ int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
            segment = indices[blockIdx.x]; // / threadsPerSegment;
            z = reinterpret_cast<T*>(outputBuf) + outputTadOffsets[segment];
            len = shape::length(inputTads);
            start = starts[segment];
            finish = start + lengths[segment];
            total = shape::sizeAt(inputShape, 0);

        }
        __syncthreads();

        auto idx = blockIdx.x;
        if (blockIdx.x <= total) {
            auto x = reinterpret_cast<T *>(inputBuf) + inputTadOffsets[idx];
            if (blockIdx.x == start) {
                for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                    auto xIndex = shape::getIndexOffset(e, inputTads, len);
                    auto zIndex = shape::getIndexOffset(e, outputTads, len);
                    z[zIndex] = T(x[xIndex]/lengths[segment]);
                }
            }
            else {
                for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                    auto xIndex = shape::getIndexOffset(e, inputTads, len);
                    auto zIndex = shape::getIndexOffset(e, outputTads, len);
                    if (lengths[segment])
                        nd4j::math::atomics::nd4j_atomicAdd(&z[zIndex], T(x[xIndex]/lengths[segment]));
                }
            }
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //

    // SegmentProd kernel
    template <typename T, typename I>
    static __global__ void segmentProdTadKernel(void* inputBuf, Nd4jLong* inputShape, Nd4jLong* inputTads, Nd4jLong* inputTadOffsets, I* indices, int* starts, int* lengths, Nd4jLong numOfClasses, void* outputBuf, Nd4jLong* outputShape, Nd4jLong* outputTads, Nd4jLong* outputTadOffsets) {
        __shared__ T* val;
        __shared__ Nd4jLong len, segment, zIndex, total;
        __shared__ T* z;
        __shared__ int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
            segment = indices[blockIdx.x]; // / threadsPerSegment;
            z = reinterpret_cast<T*>(outputBuf) + outputTadOffsets[segment];
            len = shape::length(inputTads);
            start = starts[segment];
            finish = start + lengths[segment];
            total = shape::sizeAt(inputShape, 0);

        }
        __syncthreads();

        auto idx = blockIdx.x;
        if (blockIdx.x <= total) {
            auto x = reinterpret_cast<T *>(inputBuf) + inputTadOffsets[idx];
            if (blockIdx.x == start) {
                for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                    auto xIndex = shape::getIndexOffset(e, inputTads, len);
                    auto zIndex = shape::getIndexOffset(e, outputTads, len);
                    z[zIndex] = x[xIndex];
                }
            }
            else {
                for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                    auto xIndex = shape::getIndexOffset(e, inputTads, len);
                    auto zIndex = shape::getIndexOffset(e, outputTads, len);
                    nd4j::math::atomics::nd4j_atomicMul(&z[zIndex], x[xIndex]);
                }
            }
        }
    }
    // SegmentSqrtN kernel
    template <typename T, typename I>
    static __global__ void segmentSqrtNTadKernel(void* inputBuf, Nd4jLong* inputShape, Nd4jLong* inputTads, Nd4jLong* inputTadOffsets, I* indices, int* starts, int* lengths, Nd4jLong numOfClasses, void* outputBuf, Nd4jLong* outputShape, Nd4jLong* outputTads, Nd4jLong* outputTadOffsets) {
        __shared__ T* val;
        __shared__ Nd4jLong len, segment, zIndex, total;
        __shared__ T* z;
        __shared__ int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
            segment = indices[blockIdx.x]; // / threadsPerSegment;
            z = reinterpret_cast<T*>(outputBuf) + outputTadOffsets[segment];
            len = shape::length(inputTads);
            start = starts[segment];
            finish = start + lengths[segment];
            total = shape::sizeAt(inputShape, 0);

        }
        __syncthreads();

        auto idx = blockIdx.x;
        if (blockIdx.x <= total) {
            auto x = reinterpret_cast<T *>(inputBuf) + inputTadOffsets[idx];
            if (blockIdx.x == start) {
                for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                    auto xIndex = shape::getIndexOffset(e, inputTads, len);
                    auto zIndex = shape::getIndexOffset(e, outputTads, len);
                    z[zIndex] = x[xIndex] / nd4j::math::nd4j_sqrt<int, T>(lengths[segment]);
                }
            }
            else {
                for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                    auto xIndex = shape::getIndexOffset(e, inputTads, len);
                    auto zIndex = shape::getIndexOffset(e, outputTads, len);
                    nd4j::math::atomics::nd4j_atomicAdd(&z[zIndex], x[xIndex] / nd4j::math::nd4j_sqrt<int, T>(lengths[segment]));
                }
            }
        }
    }

    // -------------------------------------------------------------------------------------------------------------- //
    // Sorted segments ops implementations
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static void segmentMaxFunctor_(LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output) {
        //int numClasses = output->sizeAt(0);
        // if input is a vector: (as if in doc sample)
        //Nd4jLong idx = indices->e<Nd4jLong>(0);
        auto stream = context->getCudaStream();
        Nd4jLong numClasses = indices->e<Nd4jLong>(indices->lengthOf() - 1) + 1;
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numClasses});
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numClasses});

        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);

        dim3 dims(256, 512, 256);
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());
        fillUpSegmentsKernel<I><<<dims.x, dims.y, dims.z, *stream>>>(indices->specialBuffer(), indices->specialShapeInfo(), numClasses, begins, lengths);

        if (input->isVector()) {
            segmentMaxLinearKernel<T,I><<<numClasses, input->lengthOf(), numClasses * 32 + 32, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
            auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
            Nd4jLong* inputTads = packX.specialShapeInfo();
            Nd4jLong* inputTadOffsets = packX.specialOffsets();
            Nd4jLong* outputTads = packZ.specialShapeInfo();
            Nd4jLong* outputTadOffsets = packZ.specialOffsets();
            segmentMaxTadKernel<T,I><<<input->sizeAt(0), 512, 2048, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);
        }
    }

    // segmen min 
    template <typename T, typename I>
    static void segmentMinFunctor_(LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output) {
        auto stream = context->getCudaStream();
        Nd4jLong numClasses = indices->e<Nd4jLong>(indices->lengthOf() - 1) + 1;
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numClasses});
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numClasses});

        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);

        dim3 dims(numClasses, indices->lengthOf(), numClasses * 32 + 32);
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());
        fillUpSegmentsKernel<I><<<dims.x, dims.y, dims.z, *stream>>>(indices->specialBuffer(), indices->specialShapeInfo(), numClasses, begins, lengths);

        if (input->isVector()) {
            segmentMinLinearKernel<T,I><<<numClasses, input->lengthOf(), numClasses * 32 + 32, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
            auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
            Nd4jLong* inputTads = packX.specialShapeInfo();
            Nd4jLong* inputTadOffsets = packX.specialOffsets();
            Nd4jLong* outputTads = packZ.specialShapeInfo();
            Nd4jLong* outputTadOffsets = packZ.specialOffsets();
            segmentMinTadKernel<T,I><<<input->sizeAt(0), 512, 2048, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);

        }
    }

    // segmen mean
    template <typename T, typename I>
    static void segmentMeanFunctor_(LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output) {
        auto stream = context->getCudaStream();
        Nd4jLong numClasses = indices->e<Nd4jLong>(indices->lengthOf() - 1) + 1;
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numClasses});
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numClasses});

        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);

        dim3 dims(numClasses, indices->lengthOf(), numClasses * 32 + 32);
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());
        fillUpSegmentsKernel<I><<<dims.x, dims.y, dims.z, *stream>>>(indices->specialBuffer(), indices->specialShapeInfo(), numClasses, begins, lengths);

        if (input->isVector()) {
            segmentMeanLinearKernel<T,I><<<numClasses, input->lengthOf(), numClasses * 32 + 32, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
            auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
            Nd4jLong* inputTads = packX.specialShapeInfo();
            Nd4jLong* inputTadOffsets = packX.specialOffsets();
            Nd4jLong* outputTads = packZ.specialShapeInfo();
            Nd4jLong* outputTadOffsets = packZ.specialOffsets();
            segmentMeanTadKernel<T,I><<<input->sizeAt(0), 512, 2048, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);
        }

    }

    template <typename T, typename I>
    static void segmentSumFunctor_(nd4j::LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output) {
        auto stream = context->getCudaStream();
        Nd4jLong numClasses = indices->e<Nd4jLong>(indices->lengthOf() - 1) + 1;
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numClasses});
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numClasses});

        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);

        dim3 dims(numClasses, indices->lengthOf(), numClasses * 32 + 32);
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());
        fillUpSegmentsKernel<I><<<dims.x, dims.y, dims.z, *stream>>>(indices->specialBuffer(), indices->specialShapeInfo(), numClasses, begins, lengths);

        if (input->isVector()) {
            segmentSumLinearKernel<T,I><<<numClasses, input->lengthOf(), numClasses * 32 + 32, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
            auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
            Nd4jLong* inputTads = packX.specialShapeInfo();
            Nd4jLong* inputTadOffsets = packX.specialOffsets();
            Nd4jLong* outputTads = packZ.specialShapeInfo();
            Nd4jLong* outputTadOffsets = packZ.specialOffsets();
            segmentSumTadKernel<T,I><<<input->sizeAt(0), 512, 2048, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);
        }

    }

    template <typename T, typename I>
    static void segmentProdFunctor_(nd4j::LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output) {
        auto stream = context->getCudaStream();
        Nd4jLong numClasses = indices->e<Nd4jLong>(indices->lengthOf() - 1) + 1;
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numClasses});
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numClasses});

        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);

        dim3 dims(numClasses, indices->lengthOf(), numClasses * 32 + 32);
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());
        fillUpSegmentsKernel<I><<<dims.x, dims.y, dims.z, *stream>>>(indices->specialBuffer(), indices->specialShapeInfo(), numClasses, begins, lengths);

        if (input->isVector()) {
            segmentProdLinearKernel<T,I><<<numClasses, input->lengthOf(), numClasses * 32 + 32, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
            auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
            Nd4jLong* inputTads = packX.specialShapeInfo();
            Nd4jLong* inputTadOffsets = packX.specialOffsets();
            Nd4jLong* outputTads = packZ.specialShapeInfo();
            Nd4jLong* outputTadOffsets = packZ.specialOffsets();
            segmentProdTadKernel<T,I><<<input->sizeAt(0), 512, 2048, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);
        }

    }

    template <typename T, typename I>
    static bool segmentIndicesValidate_(NDArray* indices, NDArray& aexpected, NDArray& aoutput) {
        return true;
    }

    void segmentMaxFunctor(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), segmentMaxFunctor_, (context, input, indices, output), NUMERIC_TYPES, INTEGER_TYPES);
    }

    void segmentMinFunctor(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), segmentMinFunctor_, (context, input, indices, output), NUMERIC_TYPES, INTEGER_TYPES);
    }

    void segmentMeanFunctor(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), segmentMeanFunctor_, (context, input, indices, output), FLOAT_TYPES, INTEGER_TYPES);
    }

    void segmentSumFunctor(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), segmentSumFunctor_, (context, input, indices, output), NUMERIC_TYPES, INTEGER_TYPES);
    }

    void segmentProdFunctor(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), segmentProdFunctor_, (context, input, indices, output), FLOAT_TYPES, INTEGER_TYPES);
    }

    bool segmentIndicesValidate(nd4j::LaunchContext * context, NDArray* indices, NDArray& expected, NDArray& output) {
        BUILD_DOUBLE_SELECTOR(output.dataType(), indices->dataType(), return segmentIndicesValidate_, (indices, expected, output), NUMERIC_TYPES, INTEGER_TYPES);
    }

    BUILD_DOUBLE_TEMPLATE(template bool segmentIndicesValidate_, (NDArray*, NDArray&, NDArray&), NUMERIC_TYPES, INTEGER_TYPES);
    BUILD_DOUBLE_TEMPLATE(template void segmentProdFunctor_, (nd4j::LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output), FLOAT_TYPES, INTEGER_TYPES);
    BUILD_DOUBLE_TEMPLATE(template void segmentSumFunctor_, (nd4j::LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output), NUMERIC_TYPES, INTEGER_TYPES);
    BUILD_DOUBLE_TEMPLATE(template void segmentMeanFunctor_, (nd4j::LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output), FLOAT_TYPES, INTEGER_TYPES);
    BUILD_DOUBLE_TEMPLATE(template void segmentMinFunctor_, (nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* output), NUMERIC_TYPES, INTEGER_TYPES);
    BUILD_DOUBLE_TEMPLATE(template void segmentMaxFunctor_, (LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output), NUMERIC_TYPES, INTEGER_TYPES);

    // -------------------------------------------------------------------------------------------------------------- //

    // -------------------------------------------------------------------------------------------------------------- //
    // Unsorted segment ops functors implementation
    // -------------------------------------------------------------------------------------------------------------- //

    bool unsortedSegmentIndicesValidate(nd4j::LaunchContext * context, NDArray* indices, Nd4jLong expected, Nd4jLong& output) {
        return true;
    }
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static void unsortedSegmentMaxFunctor_(nd4j::LaunchContext* context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        auto stream = context->getCudaStream();
//        NDArray classes = NDArrayFactory::create<int>('c', {numOfClasses, 2});
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numOfClasses});
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numOfClasses});
//        NDArray row = NDArrayFactory::create<int>('c', {1, 2}, {(int)indices->lengthOf(), (int)0});
//        classes.applyTrueBroadcast(nd4j::BroadcastOpsTuple::Assign(), &row, &classes);
        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);
        dim3 dims(numOfClasses, indices->lengthOf(), numOfClasses * 32 + 32);
//        int* classesBuf = reinterpret_cast<int*>(classes.specialBuffer());
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());
        fillUpSegmentsKernel<I><<<dims.x, dims.y, dims.z, *stream>>>(indices->specialBuffer(), indices->specialShapeInfo(), numOfClasses, begins, lengths);
        classesRangesBegs.syncToHost();
        classesRangesLens.syncToHost();

        if (input->isVector()) {
            unsortedSegmentMaxLinearKernel<T,I><<<dims.x, dims.y, dims.z, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), indices->specialBuffer(), indices->specialShapeInfo(), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
            auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
            Nd4jLong* inputTads = packX.specialShapeInfo();
            Nd4jLong* inputTadOffsets = packX.specialOffsets();
            Nd4jLong* outputTads = packZ.specialShapeInfo();
            Nd4jLong* outputTadOffsets = packZ.specialOffsets();
            dims.x = input->sizeAt(0);
            output->assign(-DataTypeUtils::max<T>());
            segmentMaxTadKernel<T,I><<<dims.x, dims.y, dims.z, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);
        }

    }
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static void unsortedSegmentMinFunctor_(nd4j::LaunchContext* context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        auto stream = context->getCudaStream();
//        NDArray classes = NDArrayFactory::create<int>('c', {numOfClasses, 2});
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numOfClasses});
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numOfClasses});
//        NDArray row = NDArrayFactory::create<int>('c', {1, 2}, {(int)indices->lengthOf(), (int)0});
//        classes.applyTrueBroadcast(nd4j::BroadcastOpsTuple::Assign(), &row, &classes);
        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);
        dim3 dims(numOfClasses, indices->lengthOf(), numOfClasses * 32 + 32);
//        int* classesBuf = reinterpret_cast<int*>(classes.specialBuffer());
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());
        fillUpSegmentsKernel<I><<<dims.x, dims.y, dims.z, *stream>>>(indices->specialBuffer(), indices->specialShapeInfo(), numOfClasses, begins, lengths);

        if (input->isVector()) {
            unsortedSegmentMinLinearKernel<T,I><<<dims.x, dims.y, dims.z, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), indices->specialBuffer(), indices->specialShapeInfo(), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            output->assign(DataTypeUtils::max<T>());
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
            auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
            Nd4jLong* inputTads = packX.specialShapeInfo();
            Nd4jLong* inputTadOffsets = packX.specialOffsets();
            Nd4jLong* outputTads = packZ.specialShapeInfo();
            Nd4jLong* outputTadOffsets = packZ.specialOffsets();
            dims.x = input->sizeAt(0);
            segmentMinTadKernel<T,I><<<dims.x, dims.y, dims.z, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);
        }

    }
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static void unsortedSegmentMeanFunctor_(nd4j::LaunchContext* context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        auto stream = context->getCudaStream();
//        NDArray classes = NDArrayFactory::create<int>('c', {numOfClasses, 2});
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numOfClasses});
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numOfClasses});
//        NDArray row = NDArrayFactory::create<int>('c', {1, 2}, {(int)indices->lengthOf(), (int)0});
//        classes.applyTrueBroadcast(nd4j::BroadcastOpsTuple::Assign(), &row, &classes);
        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);
        dim3 dims(numOfClasses, indices->lengthOf(), numOfClasses * 32 + 32);
//        int* classesBuf = reinterpret_cast<int*>(classes.specialBuffer());
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());
        fillUpSegmentsKernel<I><<<dims.x, dims.y, dims.z, *stream>>>(indices->specialBuffer(), indices->specialShapeInfo(), numOfClasses, begins, lengths);

        if (input->isVector()) {
            unsortedSegmentMeanLinearKernel<T,I><<<dims.x, dims.y, dims.z, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), indices->specialBuffer(), indices->specialShapeInfo(), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            output->assign(0);
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
            auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
            Nd4jLong* inputTads = packX.specialShapeInfo();
            Nd4jLong* inputTadOffsets = packX.specialOffsets();
            Nd4jLong* outputTads = packZ.specialShapeInfo();
            Nd4jLong* outputTadOffsets = packZ.specialOffsets();
            dims.x = input->sizeAt(0);
            segmentMeanTadKernel<T,I><<<dims.x, dims.y, dims.z, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);
        }

    }
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static void unsortedSegmentSumFunctor_(nd4j::LaunchContext* context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        auto stream = context->getCudaStream();
//        NDArray classes = NDArrayFactory::create<int>('c', {numOfClasses, 2});
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numOfClasses});
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numOfClasses});
//        NDArray row = NDArrayFactory::create<int>('c', {1, 2}, {(int)indices->lengthOf(), (int)0});
//        classes.applyTrueBroadcast(nd4j::BroadcastOpsTuple::Assign(), &row, &classes);
        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);
        dim3 dims(numOfClasses, indices->lengthOf(), (numOfClasses + 1) * 64);
//        int* classesBuf = reinterpret_cast<int*>(classes.specialBuffer());
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());
        fillUpSegmentsKernel<I><<<dims.x, dims.y, dims.z, *stream>>>(indices->specialBuffer(), indices->specialShapeInfo(), numOfClasses, begins, lengths);

        if (input->isVector()) {
            unsortedSegmentSumLinearKernel<T,I><<<dims.x, dims.y, dims.z, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), indices->specialBuffer(), indices->specialShapeInfo(), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            output->assign(0);
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
            auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
            Nd4jLong* inputTads = packX.specialShapeInfo();
            Nd4jLong* inputTadOffsets = packX.specialOffsets();
            Nd4jLong* outputTads = packZ.specialShapeInfo();
            Nd4jLong* outputTadOffsets = packZ.specialOffsets();
            dims.x = input->sizeAt(0);
            segmentSumTadKernel<T,I><<<dims.x, dims.y, dims.z, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);
        }

    }
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static void unsortedSegmentProdFunctor_(nd4j::LaunchContext* context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        auto stream = context->getCudaStream();
//        NDArray classes = NDArrayFactory::create<int>('c', {numOfClasses, 2});
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numOfClasses});
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numOfClasses});
//        NDArray row = NDArrayFactory::create<int>('c', {1, 2}, {(int)indices->lengthOf(), (int)0});
//        classes.applyTrueBroadcast(nd4j::BroadcastOpsTuple::Assign(), &row, &classes);
        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);
        dim3 dims(numOfClasses, indices->lengthOf(), numOfClasses * 32 + 32);
//        int* classesBuf = reinterpret_cast<int*>(classes.specialBuffer());
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());
        fillUpSegmentsKernel<I><<<dims.x, dims.y, dims.z, *stream>>>(indices->specialBuffer(), indices->specialShapeInfo(), numOfClasses, begins, lengths);

        if (input->isVector()) {
            unsortedSegmentProdLinearKernel<T,I><<<dims.x, dims.y, dims.z, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), indices->specialBuffer(), indices->specialShapeInfo(), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            output->assign(1);
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
            auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
            Nd4jLong* inputTads = packX.specialShapeInfo();
            Nd4jLong* inputTadOffsets = packX.specialOffsets();
            Nd4jLong* outputTads = packZ.specialShapeInfo();
            Nd4jLong* outputTadOffsets = packZ.specialOffsets();
            dims.x = input->sizeAt(0);
            segmentProdTadKernel<T,I><<<dims.x, dims.y, dims.z, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);
        }

    }
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static void unsortedSegmentSqrtNFunctor_(nd4j::LaunchContext* context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        auto stream = context->getCudaStream();
//        NDArray classes = NDArrayFactory::create<int>('c', {numOfClasses, 2});
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numOfClasses});
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numOfClasses});
//        NDArray row = NDArrayFactory::create<int>('c', {1, 2}, {(int)indices->lengthOf(), (int)0});
//        classes.applyTrueBroadcast(nd4j::BroadcastOpsTuple::Assign(), &row, &classes);
        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);
        dim3 dims(numOfClasses, indices->lengthOf(), numOfClasses * 32 + 32);
//        int* classesBuf = reinterpret_cast<int*>(classes.specialBuffer());
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());
        fillUpSegmentsKernel<I><<<dims.x, dims.y, dims.z, *stream>>>(indices->specialBuffer(), indices->specialShapeInfo(), numOfClasses, begins, lengths);

        if (input->isVector()) {
            unsortedSegmentSqrtNLinearKernel<T,I><<<dims.x, dims.y, dims.z, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), indices->specialBuffer(), indices->specialShapeInfo(), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            output->assign(0);
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
            auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
            Nd4jLong* inputTads = packX.specialShapeInfo();
            Nd4jLong* inputTadOffsets = packX.specialOffsets();
            Nd4jLong* outputTads = packZ.specialShapeInfo();
            Nd4jLong* outputTadOffsets = packZ.specialOffsets();
            dims.x = input->sizeAt(0);
            segmentSqrtNTadKernel<T,I><<<dims.x, dims.y, dims.z, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //
    // -------------------------------------------------------------------------------------------------------------- //
    // unsorted ops functors
    // -------------------------------------------------------------------------------------------------------------- //

    void unsortedSegmentMaxFunctor(nd4j::LaunchContext* context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), unsortedSegmentMaxFunctor_, (context, input, indices, numOfClasses, output), NUMERIC_TYPES, INTEGER_TYPES);
    }
    // -------------------------------------------------------------------------------------------------------------- //

    void unsortedSegmentMinFunctor(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), unsortedSegmentMinFunctor_, (context, input, indices, numOfClasses, output),
                              NUMERIC_TYPES, INTEGER_TYPES);
    }
    // -------------------------------------------------------------------------------------------------------------- //

    void unsortedSegmentMeanFunctor(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), unsortedSegmentMeanFunctor_, (context, input, indices, numOfClasses, output),
                              FLOAT_TYPES, INTEGER_TYPES);
    }
    // -------------------------------------------------------------------------------------------------------------- //

    void unsortedSegmentSumFunctor(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), unsortedSegmentSumFunctor_, (context, input, indices, numOfClasses, output),
                              NUMERIC_TYPES, INTEGER_TYPES);

    }
    // -------------------------------------------------------------------------------------------------------------- //

    void unsortedSegmentProdFunctor(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), unsortedSegmentProdFunctor_, (context, input, indices, numOfClasses, output),
                              FLOAT_TYPES, INTEGER_TYPES);
    }
    // -------------------------------------------------------------------------------------------------------------- //

    void unsortedSegmentSqrtNFunctor(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), unsortedSegmentSqrtNFunctor_, (context, input, indices, numOfClasses, output),
                              FLOAT_TYPES, INTEGER_TYPES);
    }
    // -------------------------------------------------------------------------------------------------------------- //

    BUILD_DOUBLE_TEMPLATE(template void unsortedSegmentMaxFunctor_, (nd4j::LaunchContext* context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output), NUMERIC_TYPES, INTEGER_TYPES);
    BUILD_DOUBLE_TEMPLATE(template void unsortedSegmentMinFunctor_, (nd4j::LaunchContext * context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output), NUMERIC_TYPES, INTEGER_TYPES);
    BUILD_DOUBLE_TEMPLATE(template void unsortedSegmentMeanFunctor_, (nd4j::LaunchContext * context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output), FLOAT_TYPES, INTEGER_TYPES);
    BUILD_DOUBLE_TEMPLATE(template void unsortedSegmentSumFunctor_, (nd4j::LaunchContext * context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output), NUMERIC_TYPES, INTEGER_TYPES);
    BUILD_DOUBLE_TEMPLATE(template void unsortedSegmentProdFunctor_, (nd4j::LaunchContext * context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output), FLOAT_TYPES, INTEGER_TYPES);
    BUILD_DOUBLE_TEMPLATE(template void unsortedSegmentSqrtNFunctor_, (nd4j::LaunchContext * context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output), FLOAT_TYPES, INTEGER_TYPES);
    // -------------------------------------------------------------------------------------------------------------- //
    // -------------------------------------------------------------------------------------------------------------- //
    // Backpropagate ops helpers
    // -------------------------------------------------------------------------------------------------------------- //
    // Sorted backpropagate ops
    //

    // segment max
    template <typename T>
    int segmentMaxFunctorBP_(NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {

        return Status::OK();
    }

    int segmentMaxFunctorBP(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        BUILD_SINGLE_SELECTOR(output->dataType(), return segmentMaxFunctorBP_, (input, indices, gradOut, output), NUMERIC_TYPES);
    }
    BUILD_SINGLE_TEMPLATE(template int segmentMaxFunctorBP_, (NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output), NUMERIC_TYPES);

    // segmen min
    int segmentMinFunctorBP(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        return Status::OK();
    }

    // segmen mean
    int segmentMeanFunctorBP(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        return Status::OK();
    }

    int segmentSumFunctorBP(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        return Status::OK();
    }

    // -------------------------------------------------------------------------------------------------------------- //
    // Unsorted backpropagate segment ops
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T>
    static int unsortedSegmentMaxFunctorBP_(NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        return Status::OK();
    }

    int unsortedSegmentMaxFunctorBP(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        BUILD_SINGLE_SELECTOR(output->dataType(), return unsortedSegmentMaxFunctorBP_, (input, indices, gradOut, numOfClasses, output), NUMERIC_TYPES);
    }
    BUILD_SINGLE_TEMPLATE(template int unsortedSegmentMaxFunctorBP_, (NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output), NUMERIC_TYPES);

    template <typename T>
    static int unsortedSegmentMinFunctorBP_(NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        return Status::OK();
    }

    int unsortedSegmentMinFunctorBP(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        BUILD_SINGLE_SELECTOR(output->dataType(), return unsortedSegmentMinFunctorBP_, (input, indices, gradOut, numOfClasses, output), NUMERIC_TYPES);
    }
    BUILD_SINGLE_TEMPLATE(template int unsortedSegmentMinFunctorBP_, (NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output), NUMERIC_TYPES);

    int unsortedSegmentMeanFunctorBP(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        return Status::OK();
    }

    int unsortedSegmentSumFunctorBP(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        return Status::OK();
    }

    int unsortedSegmentProdFunctorBP(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        return Status::OK();
    }

//    template <typename T>
    int unsortedSegmentSqrtNFunctorBP(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        return Status::OK();
    }

//    int unsortedSegmentSqrtNFunctorBP(NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
//        BUILD_SINGLE_SELECTOR(output->dataType(), return unsortedSegmentSqrtNFunctorBP_, (input, indices, gradOut, numOfClasses, output), FLOAT_TYPES);
//    }
//    BUILD_SINGLE_TEMPLATE(template int unsortedSegmentSqrtNFunctorBP_, (NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output), FLOAT_TYPES);
}
}
}