#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author GS <sgazeos@gmail.com>
//

#include <ops/declarable/helpers/segment.h>
#include <NDArrayFactory.h>
#include <helpers/ShapeUtils.h>
#include <helpers/TAD.h>
#include <exceptions/cuda_exception.h>
#include <PointersManager.h>
#include <ConstantTadHelper.h>

namespace nd4j {
namespace ops {
namespace helpers {

    // -------------------------------------------------------------------------------------------------------------- //
    // Segment ops linear kernels
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static __global__ void segmentMaxLinearKernel(void* input, Nd4jLong* inputShape, int* starts, int* lengths, Nd4jLong numOfClasses, void* output, Nd4jLong* outputShape) {
         __shared__ T* val;
         __shared__ Nd4jLong xLen, zLen, segment, zIndex;
         __shared__ T* x;
         __shared__ T* z;
         __shared__ int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
            threadsPerSegment = (gridDim.x + numOfClasses - 1) / numOfClasses;
            segment = blockIdx.x / threadsPerSegment;
            x = reinterpret_cast<T*>(input);
            z = reinterpret_cast<T*>(output);
            extern __shared__ unsigned char shmem[];
            val = reinterpret_cast<T*>(shmem);
            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);

            if (segment < numOfClasses) {
                zIndex = shape::getIndexOffset(segment, outputShape, zLen);
                start = starts[segment];
                finish = start + lengths[segment];
                z[zIndex] = x[shape::getIndexOffset(start, inputShape, xLen)];
                val[segment] = z[zIndex];
            }

        }
        __syncthreads();

         for (auto e = start + threadIdx.x + 1; e < finish; e += blockDim.x) {
             auto xIndex = shape::getIndexOffset(e, inputShape, xLen);
             nd4j::math::atomics::nd4j_atomicMax(&z[zIndex], x[xIndex]);
         }
    }
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static __global__ void unsortedSegmentMaxLinearKernel(void* input, Nd4jLong* inputShape, void* indices, Nd4jLong* indicesShape, int* starts, int* lengths, Nd4jLong numOfClasses, void* output, Nd4jLong* outputShape) {
        __shared__ T* val;
        __shared__ Nd4jLong xLen, zLen, segment, zIndex;
        __shared__ T* x;
        __shared__ T* z;
        __shared__ I* y; //int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
            segment = blockIdx.x;
            x = reinterpret_cast<T*>(input);
            z = reinterpret_cast<T*>(output);
            y = reinterpret_cast<I*>(indices);
            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);

            zIndex = shape::getIndexOffset(segment, outputShape, zLen);
            //start = starts[segment];
            //finish = start + lengths[segment];
            if (lengths[segment] > 0)
                z[zIndex] = x[shape::getIndexOffset(starts[segment], inputShape, xLen)];
            else
                z[zIndex] = -DataTypeUtils::max<T>();
        }
        __syncthreads();
        if (lengths[segment] > 0)
        for (auto e = threadIdx.x + 1; e < xLen; e += blockDim.x) {
            auto xIndex = shape::getIndexOffset(e, inputShape, xLen);
            auto yIndex = shape::getIndexOffset(e, indicesShape, xLen);
            if (y[yIndex] == segment) {
                nd4j::math::atomics::nd4j_atomicMax(&z[zIndex], x[xIndex]);
            }
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static __global__ void segmentMinLinearKernel(void* input, Nd4jLong* inputShape, int* starts, int* lengths, Nd4jLong numOfClasses, void* output, Nd4jLong* outputShape) {
        __shared__ T* val;
        __shared__ Nd4jLong xLen, zLen, segment, zIndex;
        __shared__ T* x;
        __shared__ T* z;
        __shared__ int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
            threadsPerSegment = (gridDim.x + numOfClasses - 1) / numOfClasses;
            segment = blockIdx.x / threadsPerSegment;
            x = reinterpret_cast<T*>(input);
            z = reinterpret_cast<T*>(output);
            extern __shared__ unsigned char shmem[];
            val = reinterpret_cast<T*>(shmem);
            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);

            if (segment < numOfClasses) {
                zIndex = shape::getIndexOffset(segment, outputShape, zLen);
                start = starts[segment];
                finish = start + lengths[segment];
                z[zIndex] = x[shape::getIndexOffset(start, inputShape, xLen)];
                val[segment] = z[zIndex];
            }

        }
        __syncthreads();

        for (auto e = start + threadIdx.x + 1; e < finish; e += blockDim.x) {
            auto xIndex = shape::getIndexOffset(e, inputShape, xLen);
           nd4j::math::atomics::nd4j_atomicMin(&z[zIndex], x[xIndex]);
        }

    }
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static __global__ void unsortedSegmentMinLinearKernel(void* input, Nd4jLong* inputShape, void* indices, Nd4jLong* indicesShape, int* starts, int* lengths, Nd4jLong numOfClasses, void* output, Nd4jLong* outputShape) {
        __shared__ T* val;
        __shared__ Nd4jLong xLen, zLen, segment, zIndex;
        __shared__ T* x;
        __shared__ T* z;
        __shared__ I* y; //int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
            segment = blockIdx.x;
            x = reinterpret_cast<T*>(input);
            z = reinterpret_cast<T*>(output);
            y = reinterpret_cast<I*>(indices);
            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);

            zIndex = shape::getIndexOffset(segment, outputShape, zLen);
            if (lengths[segment] > 0)
                z[zIndex] = x[shape::getIndexOffset(starts[segment], inputShape, xLen)];
            else
                z[zIndex] = DataTypeUtils::max<T>();

        }
        __syncthreads();
        if (lengths[segment] > 0)
            for (auto e = threadIdx.x + 1; e < xLen; e += blockDim.x) {
                auto xIndex = shape::getIndexOffset(e, inputShape, xLen);
                auto yIndex = shape::getIndexOffset(e, indicesShape, xLen);
                if (y[yIndex] == segment) {
                    nd4j::math::atomics::nd4j_atomicMin(&z[zIndex], x[xIndex]);
                }
            }
    }
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static __global__ void segmentSumLinearKernel(void* input, Nd4jLong* inputShape, int* starts, int* lengths, Nd4jLong numOfClasses, void* output, Nd4jLong* outputShape) {
        __shared__ T* val;
        __shared__ Nd4jLong xLen, zLen, segment, zIndex;
        __shared__ T* x;
        __shared__ T* z;
        __shared__ int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
            threadsPerSegment = (gridDim.x + numOfClasses - 1) / numOfClasses;
            segment = blockIdx.x / threadsPerSegment;
            x = reinterpret_cast<T*>(input);
            z = reinterpret_cast<T*>(output);

            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);


            if (segment < numOfClasses) {
                zIndex = shape::getIndexOffset(segment, outputShape, zLen);
                start = starts[segment];
                finish = start + lengths[segment];
                //val[segment] = ;
                z[zIndex] = x[shape::getIndexOffset(start, inputShape, xLen)];
            }

        }
        __syncthreads();

        for (auto e = start + threadIdx.x + 1; e < finish; e += blockDim.x) {
            auto xIndex = shape::getIndexOffset(e, inputShape, xLen);
            nd4j::math::atomics::nd4j_atomicAdd(&z[zIndex], x[xIndex]);
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static __global__ void unsortedSegmentSumLinearKernel(void* input, Nd4jLong* inputShape, void* indices, Nd4jLong* indicesShape, int* starts, int* lengths, Nd4jLong numOfClasses, void* output, Nd4jLong* outputShape) {
        __shared__ T* val;
        __shared__ Nd4jLong xLen, zLen, segment, zIndex;
        __shared__ T* x;
        __shared__ T* z;
        __shared__ I* y; //int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
            segment = blockIdx.x;
            x = reinterpret_cast<T*>(input);
            z = reinterpret_cast<T*>(output);
            y = reinterpret_cast<I*>(indices);
            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);

            zIndex = shape::getIndexOffset(segment, outputShape, zLen);
            if (lengths[segment] > 0)
                z[zIndex] = x[shape::getIndexOffset(starts[segment], inputShape, xLen)];
            else
                z[zIndex] = 0; //DataTypeUtils::max<T>();
        }
        __syncthreads();

        if (lengths[segment] > 0)
            for (auto e = threadIdx.x; e < xLen; e += blockDim.x) {
                auto xIndex = shape::getIndexOffset(e, inputShape, xLen);
                auto yIndex = shape::getIndexOffset(e, indicesShape, xLen);
                if (y[yIndex] == segment && e != starts[segment]) {
                    nd4j::math::atomics::nd4j_atomicAdd(&z[zIndex], x[xIndex]);
                }
            }
    }
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static __global__ void segmentMeanLinearKernel(void* input, Nd4jLong* inputShape, int* starts, int* lengths, Nd4jLong numOfClasses, void* output, Nd4jLong* outputShape) {
        __shared__ T* val;
        __shared__ Nd4jLong xLen, zLen, segment, zIndex;
        __shared__ T* x;
        __shared__ T* z;
        __shared__ int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
            threadsPerSegment = (gridDim.x + numOfClasses - 1) / numOfClasses;
            segment = blockIdx.x / threadsPerSegment;
            x = reinterpret_cast<T*>(input);
            z = reinterpret_cast<T*>(output);
//            extern __shared__ unsigned char shmem[];
//            val = reinterpret_cast<T*>(shmem);
            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);

            //[zIndex] =
            if (segment < numOfClasses) {
                zIndex = shape::getIndexOffset(segment, outputShape, zLen);
                start = starts[segment];
                finish = start + lengths[segment];
                //val[segment] = ;
                z[zIndex] = T(x[shape::getIndexOffset(start, inputShape, xLen)] / lengths[segment]);
//                val[segment] = z[zIndex];
            }

        }
        __syncthreads();

        for (auto e = start + threadIdx.x + 1; e < finish; e += blockDim.x) {
            auto xIndex = shape::getIndexOffset(e, inputShape, xLen);
            if (lengths[segment])
            nd4j::math::atomics::nd4j_atomicAdd(&z[zIndex], T(x[xIndex] / lengths[segment]));
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    static __global__ void unsortedSegmentMeanLinearKernel(void* input, Nd4jLong* inputShape, void* indices, Nd4jLong* indicesShape, int* starts, int* lengths, Nd4jLong numOfClasses, void* output, Nd4jLong* outputShape) {
        __shared__ T* val;
        __shared__ Nd4jLong xLen, zLen, segment, zIndex;
        __shared__ T* x;
        __shared__ T* z;
        __shared__ I* y; //int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
//            threadsPerSegment = (gridDim.x + numOfClasses - 1) / numOfClasses;
            segment = blockIdx.x;// / threadsPerSegment;
            x = reinterpret_cast<T*>(input);
            z = reinterpret_cast<T*>(output);
            y = reinterpret_cast<I*>(indices);
//            extern __shared__ unsigned char shmem[];
//            val = reinterpret_cast<T*>(shmem);
            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);

//            if (segment < numOfClasses) {
            zIndex = shape::getIndexOffset(segment, outputShape, zLen);
            //start = starts[segment];
            //finish = start + lengths[segment];
            if (lengths[segment] > 0)
                z[zIndex] = T(x[shape::getIndexOffset(starts[segment], inputShape, xLen)] / T(lengths[segment]));
            else
                z[zIndex] = 0; //DataTypeUtils::max<T>();
//                val[segment] = z[zIndex];
//            }

        }
        __syncthreads();
        if (lengths[segment] > 0)
            for (auto e = threadIdx.x; e < xLen; e += blockDim.x) {
                auto xIndex = shape::getIndexOffset(e, inputShape, xLen);
                auto yIndex = shape::getIndexOffset(e, indicesShape, xLen);
                if (y[yIndex] == segment && e != starts[segment]) {
                    nd4j::math::atomics::nd4j_atomicAdd(&z[zIndex], T(x[xIndex]/T(lengths[segment])));
                }
            }
    }
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static __global__ void segmentProdLinearKernel(void* input, Nd4jLong* inputShape, int* starts, int* lengths, Nd4jLong numOfClasses, void* output, Nd4jLong* outputShape) {
        __shared__ T* val;
        __shared__ Nd4jLong xLen, zLen, segment, zIndex;
        __shared__ T* x;
        __shared__ T* z;
        __shared__ int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
            threadsPerSegment = (gridDim.x + numOfClasses - 1) / numOfClasses;
            segment = blockIdx.x / threadsPerSegment;
            x = reinterpret_cast<T*>(input);
            z = reinterpret_cast<T*>(output);
            extern __shared__ unsigned char shmem[];
            val = reinterpret_cast<T*>(shmem);
            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);

            if (segment < numOfClasses) {
                zIndex = shape::getIndexOffset(segment, outputShape, zLen);
                start = starts[segment];
                finish = start + lengths[segment];
                //val[segment] = ;
                z[zIndex] = x[shape::getIndexOffset(start, inputShape, xLen)];
                val[segment] = z[zIndex];
            }

        }
        __syncthreads();
//         auto tid = threadIdx.x + blockIdx.x * blockDim.x;
//         auto step = blockDim.x * gridDim.x;

        for (auto e = start + threadIdx.x + 1; e < finish; e += blockDim.x) {
            auto xIndex = shape::getIndexOffset(e, inputShape, xLen);
            nd4j::math::atomics::nd4j_atomicMul(&val[segment], x[xIndex]);
        }
        __syncthreads();

        if (threadIdx.x == 0) {
            z[zIndex] = val[segment];
        }

    }
    template <typename T, typename I>
    static __global__ void unsortedSegmentProdLinearKernel(void* input, Nd4jLong* inputShape, void* indices, Nd4jLong* indicesShape, int* starts, int* lengths, Nd4jLong numOfClasses, void* output, Nd4jLong* outputShape) {
        __shared__ T* val;
        __shared__ Nd4jLong xLen, zLen, segment, zIndex;
        __shared__ T* x;
        __shared__ T* z;
        __shared__ I* y; //int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
//            threadsPerSegment = (gridDim.x + numOfClasses - 1) / numOfClasses;
            segment = blockIdx.x;// / threadsPerSegment;
            x = reinterpret_cast<T*>(input);
            z = reinterpret_cast<T*>(output);
            y = reinterpret_cast<I*>(indices);
//            extern __shared__ unsigned char shmem[];
//            val = reinterpret_cast<T*>(shmem);
            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);

//            if (segment < numOfClasses) {
            zIndex = shape::getIndexOffset(segment, outputShape, zLen);
            //start = starts[segment];
            //finish = start + lengths[segment];
            if (lengths[segment] > 0)
                z[zIndex] = x[shape::getIndexOffset(starts[segment], inputShape, xLen)];
            else
                z[zIndex] = 0; //DataTypeUtils::max<T>();
//                val[segment] = z[zIndex];
//            }

        }
        __syncthreads();
        if (lengths[segment] > 0)
            for (auto e = threadIdx.x; e < xLen; e += blockDim.x) {
                auto xIndex = shape::getIndexOffset(e, inputShape, xLen);
                auto yIndex = shape::getIndexOffset(e, indicesShape, xLen);
                if (y[yIndex] == segment && e != starts[segment]) {
                    nd4j::math::atomics::nd4j_atomicMul(&z[zIndex], x[xIndex]);
                }
            }
    }
    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    static __global__ void unsortedSegmentSqrtNLinearKernel(void* input, Nd4jLong* inputShape, void* indices, Nd4jLong* indicesShape, int* starts, int* lengths, Nd4jLong numOfClasses, void* output, Nd4jLong* outputShape) {
        __shared__ T* val;
        __shared__ Nd4jLong xLen, zLen, segment, zIndex;
        __shared__ T* x;
        __shared__ T* z;
        __shared__ I* y; //int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
//            threadsPerSegment = (gridDim.x + numOfClasses - 1) / numOfClasses;
            segment = blockIdx.x;// / threadsPerSegment;
            x = reinterpret_cast<T*>(input);
            z = reinterpret_cast<T*>(output);
            y = reinterpret_cast<I*>(indices);
//            extern __shared__ unsigned char shmem[];
//            val = reinterpret_cast<T*>(shmem);
            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);

//            if (segment < numOfClasses) {
            zIndex = shape::getIndexOffset(segment, outputShape, zLen);
            //start = starts[segment];
            //finish = start + lengths[segment];
            if (lengths[segment] > 0)
                z[zIndex] = x[shape::getIndexOffset(starts[segment], inputShape, xLen)] / nd4j::math::nd4j_sqrt<int, T>(lengths[segment]);
            else
                z[zIndex] = 0; //DataTypeUtils::max<T>();
//                val[segment] = z[zIndex];
//            }

        }
        __syncthreads();
        if (lengths[segment] > 0)
            for (auto e = threadIdx.x + 1; e < xLen; e += blockDim.x) {
                auto xIndex = shape::getIndexOffset(e, inputShape, xLen);
                auto yIndex = shape::getIndexOffset(e, indicesShape, xLen);
                if (y[yIndex] == segment && e != starts[segment]) {
                    nd4j::math::atomics::nd4j_atomicAdd(&z[zIndex], x[xIndex] / nd4j::math::nd4j_sqrt<int, T>(lengths[segment]));
                }
            }
    }
    // -------------------------------------------------------------------------------------------------------------- //
    // fill up segments starts and ends - splitted ordered case
    template <typename I>
    static __global__ void fillUpSegmentsKernel(void* indices, Nd4jLong* indexShape, int numClasses, int* classesRangesStart, int* classesRangesLenghts) {
        __shared__ I* idxBuf;
        __shared__ Nd4jLong idxLen;
        __shared__ int* result;
        if (threadIdx.x == 0) {
            idxBuf = reinterpret_cast<I*>(indices);
            idxLen = shape::length(indexShape);
        }
        __syncthreads();

        auto tid = threadIdx.x + blockDim.x * blockIdx.x;
        auto step = blockDim.x * gridDim.x;

        for (auto j = tid; j < idxLen; j += step) {
            auto pos = idxBuf[j];
            nd4j::math::atomics::nd4j_atomicMin(&classesRangesStart[pos], (int)j);
            nd4j::math::atomics::nd4j_atomicAdd(&classesRangesLenghts[pos], 1);
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //

    // -------------------------------------------------------------------------------------------------------------- //
    // fill up segments starts and counts - cumulative  case
    template <typename I>
    static __global__ void fillUpUnsortedSegmentsKernel(void* indices, Nd4jLong* indexShape, int numClasses, int* classes) {
        __shared__ I* idxBuf;
        __shared__ Nd4jLong idxLen;
        __shared__ int* result;
        if (threadIdx.x == 0) {
            idxBuf = reinterpret_cast<I*>(indices);
            idxLen = shape::length(indexShape);
        }
        __syncthreads();

        auto tid = threadIdx.x + blockDim.x * blockIdx.x;
        auto step = blockDim.x * gridDim.x;

        for (auto j = tid; j < idxLen; j += step) {
            auto k = idxBuf[j];
            auto beginPos = 2 * k;
            auto sizePos = beginPos + 1;
            printf("%d, %d\n", beginPos, sizePos);
            nd4j::math::atomics::nd4j_atomicMin(&classes[beginPos], (int)j);
            nd4j::math::atomics::nd4j_atomicAdd(&classes[sizePos], 1);
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //

    // -------------------------------------------------------------------------------------------------------------- //
    // segment ops multidimentional cases
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static __global__ void segmentMaxTadKernel(void* inputBuf, Nd4jLong* inputShape, Nd4jLong* inputTads,
            Nd4jLong* inputTadOffsets, I* indices, int* starts, int* lengths, Nd4jLong numOfClasses, void* outputBuf,
            Nd4jLong* outputShape, Nd4jLong* outputTads, Nd4jLong* outputTadOffsets, T filler = 0) {

        __shared__ T* val;
        __shared__ Nd4jLong len, segment, zIndex, total;
        __shared__ T* z;
        __shared__ int start, finish;

        if (threadIdx.x == 0) {
            segment = indices[blockIdx.x]; // / threadsPerSegment;
            z = reinterpret_cast<T*>(outputBuf) + outputTadOffsets[segment];
            len = shape::length(inputTads);

            start = starts[segment];
            finish = start + lengths[segment];
            total = shape::sizeAt(inputShape, 0);
        }
        __syncthreads();

        auto idx = blockIdx.x;
        if (blockIdx.x <= total) {
            auto x = reinterpret_cast<T *>(inputBuf) + inputTadOffsets[idx];
            if (blockIdx.x == start) {
                for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                    auto xIndex = shape::getIndexOffset(e, inputTads, len);
                    auto zIndex = shape::getIndexOffset(e, outputTads, len);
                    z[zIndex] = x[xIndex];
                }
            }
            else {
                for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                    auto xIndex = shape::getIndexOffset(e, inputTads, len);
                    auto zIndex = shape::getIndexOffset(e, outputTads, len);
                    nd4j::math::atomics::nd4j_atomicMax(&z[zIndex], x[xIndex]);
                }
            }
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //

    // SegmentMin kernel
    template <typename T, typename I>
    static __global__ void segmentMinTadKernel(void* inputBuf, Nd4jLong* inputShape, Nd4jLong* inputTads, Nd4jLong* inputTadOffsets, I* indices, int* starts, int* lengths, Nd4jLong numOfClasses, void* outputBuf, Nd4jLong* outputShape, Nd4jLong* outputTads, Nd4jLong* outputTadOffsets) {
        __shared__ T* val;
        __shared__ Nd4jLong len, segment, zIndex, total;
        __shared__ T* z;
        __shared__ int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
            segment = indices[blockIdx.x]; // / threadsPerSegment;
            z = reinterpret_cast<T*>(outputBuf) + outputTadOffsets[segment];
            len = shape::length(inputTads);
            start = starts[segment];
            finish = start + lengths[segment];
            total = shape::sizeAt(inputShape, 0);

        }
        __syncthreads();

        auto idx = blockIdx.x;
        if (blockIdx.x <= total) {
            auto x = reinterpret_cast<T *>(inputBuf) + inputTadOffsets[idx];
            if (blockIdx.x == start) {
                for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                    auto xIndex = shape::getIndexOffset(e, inputTads, len);
                    auto zIndex = shape::getIndexOffset(e, outputTads, len);
                    z[zIndex] = x[xIndex];
                }
            }
            else {
                for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                    auto xIndex = shape::getIndexOffset(e, inputTads, len);
                    auto zIndex = shape::getIndexOffset(e, outputTads, len);
                    nd4j::math::atomics::nd4j_atomicMin(&z[zIndex], x[xIndex]);
                }
            }
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //

    // SegmentSum kernel
    template <typename T, typename I>
    static __global__ void segmentSumTadKernel(void* inputBuf, Nd4jLong* inputShape, Nd4jLong* inputTads, Nd4jLong* inputTadOffsets, I* indices, int* starts, int* lengths, Nd4jLong numOfClasses, void* outputBuf, Nd4jLong* outputShape, Nd4jLong* outputTads, Nd4jLong* outputTadOffsets) {
        __shared__ T* val;
        __shared__ Nd4jLong len, segment, zIndex, total;
        __shared__ T* z;
        __shared__ int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
            segment = indices[blockIdx.x]; // / threadsPerSegment;
            z = reinterpret_cast<T*>(outputBuf) + outputTadOffsets[segment];
            len = shape::length(inputTads);
            start = starts[segment];
            finish = start + lengths[segment];
            total = shape::sizeAt(inputShape, 0);

        }
        __syncthreads();

        auto idx = blockIdx.x;
        if (blockIdx.x <= total) {
            auto x = reinterpret_cast<T *>(inputBuf) + inputTadOffsets[idx];
            if (blockIdx.x == start) {
                for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                    auto xIndex = shape::getIndexOffset(e, inputTads, len);
                    auto zIndex = shape::getIndexOffset(e, outputTads, len);
                    z[zIndex] = x[xIndex];
                }
            }
            else {
                for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                    auto xIndex = shape::getIndexOffset(e, inputTads, len);
                    auto zIndex = shape::getIndexOffset(e, outputTads, len);
                    if (lengths[segment])
                    nd4j::math::atomics::nd4j_atomicAdd(&z[zIndex], x[xIndex]);
                }
            }
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //

    // SegmentMean kernel
    template <typename T, typename I>
    static __global__ void segmentMeanTadKernel(void* inputBuf, Nd4jLong* inputShape, Nd4jLong* inputTads, Nd4jLong* inputTadOffsets, I* indices, int* starts, int* lengths, Nd4jLong numOfClasses, void* outputBuf, Nd4jLong* outputShape, Nd4jLong* outputTads, Nd4jLong* outputTadOffsets) {
        __shared__ T* val;
        __shared__ Nd4jLong len, segment, zIndex, total;
        __shared__ T* z;
        __shared__ int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
            segment = indices[blockIdx.x]; // / threadsPerSegment;
            z = reinterpret_cast<T*>(outputBuf) + outputTadOffsets[segment];
            len = shape::length(inputTads);
            start = starts[segment];
            finish = start + lengths[segment];
            total = shape::sizeAt(inputShape, 0);

        }
        __syncthreads();

        auto idx = blockIdx.x;
        if (blockIdx.x <= total) {
            auto x = reinterpret_cast<T *>(inputBuf) + inputTadOffsets[idx];
            if (blockIdx.x == start) {
                for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                    auto xIndex = shape::getIndexOffset(e, inputTads, len);
                    auto zIndex = shape::getIndexOffset(e, outputTads, len);
                    z[zIndex] = T(x[xIndex]/lengths[segment]);
                }
            }
            else {
                for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                    auto xIndex = shape::getIndexOffset(e, inputTads, len);
                    auto zIndex = shape::getIndexOffset(e, outputTads, len);
                    if (lengths[segment])
                        nd4j::math::atomics::nd4j_atomicAdd(&z[zIndex], T(x[xIndex]/lengths[segment]));
                }
            }
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //

    // SegmentProd kernel
    template <typename T, typename I>
    static __global__ void segmentProdTadKernel(void* inputBuf, Nd4jLong* inputShape, Nd4jLong* inputTads, Nd4jLong* inputTadOffsets, I* indices, int* starts, int* lengths, Nd4jLong numOfClasses, void* outputBuf, Nd4jLong* outputShape, Nd4jLong* outputTads, Nd4jLong* outputTadOffsets) {
        __shared__ T* val;
        __shared__ Nd4jLong len, segment, zIndex, total;
        __shared__ T* z;
        __shared__ int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
            segment = indices[blockIdx.x]; // / threadsPerSegment;
            z = reinterpret_cast<T*>(outputBuf) + outputTadOffsets[segment];
            len = shape::length(inputTads);
            start = starts[segment];
            finish = start + lengths[segment];
            total = shape::sizeAt(inputShape, 0);

        }
        __syncthreads();

        auto idx = blockIdx.x;
        if (blockIdx.x <= total) {
            auto x = reinterpret_cast<T *>(inputBuf) + inputTadOffsets[idx];
            if (blockIdx.x == start) {
                for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                    auto xIndex = shape::getIndexOffset(e, inputTads, len);
                    auto zIndex = shape::getIndexOffset(e, outputTads, len);
                    z[zIndex] = x[xIndex];
                }
            }
            else {
                for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                    auto xIndex = shape::getIndexOffset(e, inputTads, len);
                    auto zIndex = shape::getIndexOffset(e, outputTads, len);
                    nd4j::math::atomics::nd4j_atomicMul(&z[zIndex], x[xIndex]);
                }
            }
        }
    }
    // SegmentSqrtN kernel
    template <typename T, typename I>
    static __global__ void segmentSqrtNTadKernel(void* inputBuf, Nd4jLong* inputShape, Nd4jLong* inputTads, Nd4jLong* inputTadOffsets, I* indices, int* starts, int* lengths, Nd4jLong numOfClasses, void* outputBuf, Nd4jLong* outputShape, Nd4jLong* outputTads, Nd4jLong* outputTadOffsets) {
        __shared__ T* val;
        __shared__ Nd4jLong len, segment, zIndex, total;
        __shared__ T* z;
        __shared__ int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
            segment = indices[blockIdx.x]; // / threadsPerSegment;
            z = reinterpret_cast<T*>(outputBuf) + outputTadOffsets[segment];
            len = shape::length(inputTads);
            start = starts[segment];
            finish = start + lengths[segment];
            total = shape::sizeAt(inputShape, 0);

        }
        __syncthreads();

        auto idx = blockIdx.x;
        if (blockIdx.x <= total) {
            auto x = reinterpret_cast<T *>(inputBuf) + inputTadOffsets[idx];
            if (blockIdx.x == start) {
                for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                    auto xIndex = shape::getIndexOffset(e, inputTads, len);
                    auto zIndex = shape::getIndexOffset(e, outputTads, len);
                    z[zIndex] = x[xIndex] / nd4j::math::nd4j_sqrt<int, T>(lengths[segment]);
                }
            }
            else {
                for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                    auto xIndex = shape::getIndexOffset(e, inputTads, len);
                    auto zIndex = shape::getIndexOffset(e, outputTads, len);
                    nd4j::math::atomics::nd4j_atomicAdd(&z[zIndex], x[xIndex] / nd4j::math::nd4j_sqrt<int, T>(lengths[segment]));
                }
            }
        }
    }

    // -------------------------------------------------------------------------------------------------------------- //
    // Sorted segments ops implementations
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static void segmentMaxFunctor_(LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output) {
        //int numClasses = output->sizeAt(0);
        // if input is a vector: (as if in doc sample)
        //Nd4jLong idx = indices->e<Nd4jLong>(0);
        auto stream = context->getCudaStream();
        Nd4jLong numClasses = indices->e<Nd4jLong>(indices->lengthOf() - 1) + 1;
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numClasses});
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numClasses});

        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);
        dim3 dims(256, 512, 256);
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());
        fillUpSegmentsKernel<I><<<dims.x, dims.y, dims.z, *stream>>>(indices->specialBuffer(), indices->specialShapeInfo(), numClasses, begins, lengths);

        NDArray::prepareSpecialUse({output}, {input, indices, &classesRangesBegs, &classesRangesLens});

        if (input->isVector()) {
            segmentMaxLinearKernel<T,I><<<numClasses, input->lengthOf(), numClasses * 32 + 32, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
            auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
            Nd4jLong* inputTads = packX.specialShapeInfo();
            Nd4jLong* inputTadOffsets = packX.specialOffsets();
            Nd4jLong* outputTads = packZ.specialShapeInfo();
            Nd4jLong* outputTadOffsets = packZ.specialOffsets();
            segmentMaxTadKernel<T,I><<<input->sizeAt(0), 512, 2048, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);
        }
        NDArray::registerSpecialUse({output}, {input, indices, &classesRangesBegs, &classesRangesLens});
    }

    // segmen min 
    template <typename T, typename I>
    static void segmentMinFunctor_(LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output) {
        auto stream = context->getCudaStream();
        Nd4jLong numClasses = indices->e<Nd4jLong>(indices->lengthOf() - 1) + 1;
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numClasses});
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numClasses});

        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);

        dim3 dims(numClasses, indices->lengthOf(), numClasses * 32 + 32);
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());
        fillUpSegmentsKernel<I><<<dims.x, dims.y, dims.z, *stream>>>(indices->specialBuffer(), indices->specialShapeInfo(), numClasses, begins, lengths);
        NDArray::prepareSpecialUse({output}, {input, indices, &classesRangesBegs, &classesRangesLens});

        if (input->isVector()) {
            segmentMinLinearKernel<T,I><<<numClasses, input->lengthOf(), numClasses * 32 + 32, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
            auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
            Nd4jLong* inputTads = packX.specialShapeInfo();
            Nd4jLong* inputTadOffsets = packX.specialOffsets();
            Nd4jLong* outputTads = packZ.specialShapeInfo();
            Nd4jLong* outputTadOffsets = packZ.specialOffsets();
            segmentMinTadKernel<T,I><<<input->sizeAt(0), 512, 2048, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);

        }
        NDArray::registerSpecialUse({output}, {input, indices, &classesRangesBegs, &classesRangesLens});

    }

    // segmen mean
    template <typename T, typename I>
    static void segmentMeanFunctor_(LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output) {
        auto stream = context->getCudaStream();
        Nd4jLong numClasses = indices->e<Nd4jLong>(indices->lengthOf() - 1) + 1;
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numClasses});
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numClasses});

        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);

        dim3 dims(numClasses, indices->lengthOf(), numClasses * 32 + 32);
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());
        fillUpSegmentsKernel<I><<<dims.x, dims.y, dims.z, *stream>>>(indices->specialBuffer(), indices->specialShapeInfo(), numClasses, begins, lengths);

        if (input->isVector()) {
            segmentMeanLinearKernel<T,I><<<numClasses, input->lengthOf(), numClasses * 32 + 32, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
            auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
            Nd4jLong* inputTads = packX.specialShapeInfo();
            Nd4jLong* inputTadOffsets = packX.specialOffsets();
            Nd4jLong* outputTads = packZ.specialShapeInfo();
            Nd4jLong* outputTadOffsets = packZ.specialOffsets();
            segmentMeanTadKernel<T,I><<<input->sizeAt(0), 512, 2048, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);
        }

    }

    template <typename T, typename I>
    static void segmentSumFunctor_(nd4j::LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output) {
        auto stream = context->getCudaStream();
        Nd4jLong numClasses = indices->e<Nd4jLong>(indices->lengthOf() - 1) + 1;
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numClasses});
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numClasses});

        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);

        dim3 dims(numClasses, indices->lengthOf(), numClasses * 32 + 32);
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());
        fillUpSegmentsKernel<I><<<dims.x, dims.y, dims.z, *stream>>>(indices->specialBuffer(), indices->specialShapeInfo(), numClasses, begins, lengths);

        if (input->isVector()) {
            segmentSumLinearKernel<T,I><<<numClasses, input->lengthOf(), numClasses * 32 + 32, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
            auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
            Nd4jLong* inputTads = packX.specialShapeInfo();
            Nd4jLong* inputTadOffsets = packX.specialOffsets();
            Nd4jLong* outputTads = packZ.specialShapeInfo();
            Nd4jLong* outputTadOffsets = packZ.specialOffsets();
            segmentSumTadKernel<T,I><<<input->sizeAt(0), 512, 2048, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);
        }

    }

    template <typename T, typename I>
    static void segmentProdFunctor_(nd4j::LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output) {
        auto stream = context->getCudaStream();
        Nd4jLong numClasses = indices->e<Nd4jLong>(indices->lengthOf() - 1) + 1;
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numClasses});
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numClasses});

        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);

        dim3 dims(numClasses, indices->lengthOf(), numClasses * 32 + 32);
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());
        fillUpSegmentsKernel<I><<<dims.x, dims.y, dims.z, *stream>>>(indices->specialBuffer(), indices->specialShapeInfo(), numClasses, begins, lengths);

        if (input->isVector()) {
            segmentProdLinearKernel<T,I><<<numClasses, input->lengthOf(), numClasses * 32 + 32, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
            auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
            Nd4jLong* inputTads = packX.specialShapeInfo();
            Nd4jLong* inputTadOffsets = packX.specialOffsets();
            Nd4jLong* outputTads = packZ.specialShapeInfo();
            Nd4jLong* outputTadOffsets = packZ.specialOffsets();
            segmentProdTadKernel<T,I><<<input->sizeAt(0), 512, 2048, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);
        }

    }

    template <typename T, typename I>
    static bool segmentIndicesValidate_(NDArray* indices, NDArray& aexpected, NDArray& aoutput) {
        return true;
    }

    void segmentMaxFunctor(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), segmentMaxFunctor_, (context, input, indices, output), NUMERIC_TYPES, INTEGER_TYPES);
    }

    void segmentMinFunctor(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), segmentMinFunctor_, (context, input, indices, output), NUMERIC_TYPES, INTEGER_TYPES);
    }

    void segmentMeanFunctor(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), segmentMeanFunctor_, (context, input, indices, output), FLOAT_TYPES, INTEGER_TYPES);
    }

    void segmentSumFunctor(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), segmentSumFunctor_, (context, input, indices, output), NUMERIC_TYPES, INTEGER_TYPES);
    }

    void segmentProdFunctor(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), segmentProdFunctor_, (context, input, indices, output), NUMERIC_TYPES, INTEGER_TYPES);
    }

    bool segmentIndicesValidate(nd4j::LaunchContext* context , NDArray* indices, NDArray& expected, NDArray& output) {
        BUILD_DOUBLE_SELECTOR(output.dataType(), indices->dataType(), return segmentIndicesValidate_, (indices, expected, output), NUMERIC_TYPES, INTEGER_TYPES);
    }

    BUILD_DOUBLE_TEMPLATE(template bool segmentIndicesValidate_, (NDArray*, NDArray&, NDArray&), NUMERIC_TYPES, INTEGER_TYPES);
    BUILD_DOUBLE_TEMPLATE(template void segmentProdFunctor_, (nd4j::LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output), FLOAT_TYPES, INTEGER_TYPES);
    BUILD_DOUBLE_TEMPLATE(template void segmentSumFunctor_, (nd4j::LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output), NUMERIC_TYPES, INTEGER_TYPES);
    BUILD_DOUBLE_TEMPLATE(template void segmentMeanFunctor_, (nd4j::LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output), FLOAT_TYPES, INTEGER_TYPES);
    BUILD_DOUBLE_TEMPLATE(template void segmentMinFunctor_, (nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* output), NUMERIC_TYPES, INTEGER_TYPES);
    BUILD_DOUBLE_TEMPLATE(template void segmentMaxFunctor_, (LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output), NUMERIC_TYPES, INTEGER_TYPES);

    // -------------------------------------------------------------------------------------------------------------- //

    // -------------------------------------------------------------------------------------------------------------- //
    // Unsorted segment ops functors implementation
    // -------------------------------------------------------------------------------------------------------------- //
    template <typename I>
    static __global__ void unsortedSegmentIndexValidateKernel(I* indices, Nd4jLong* indicesShape, I expected, I* found) {
        __shared__ bool onlyTrue;
        __shared__ Nd4jLong len;

        if (threadIdx.x == 0) {
            onlyTrue = true;
            len = shape::length(indicesShape);
        }
        __syncthreads();
        auto start = threadIdx.x + blockIdx.x * blockDim.x;
        auto step = gridDim.x * blockDim.x;
        for (int e = start; e < len && onlyTrue; e += step) {
            nd4j::math::atomics::nd4j_atomicMax(found, indices[e]);
            if (expected < *found)
                onlyTrue = false;
        }
    }

    template <typename I>
    static bool unsortedSegmentIndicesValidate_(nd4j::LaunchContext* context , NDArray* indices, Nd4jLong expected, Nd4jLong& output) {
        output = expected;
        I found = output;
        I exp = expected;
        auto stream = context->getCudaStream();
        I* devFound;
        hipMalloc(&devFound, sizeof(I));
        hipMemcpy(devFound, &found, sizeof(I), hipMemcpyHostToDevice);
        unsortedSegmentIndexValidateKernel<I><<<1, indices->lengthOf(), 128, *stream>>>(reinterpret_cast<I*>(indices->specialBuffer()), indices->specialShapeInfo(), exp, devFound);
        hipMemcpy(&found, devFound, sizeof(I), hipMemcpyDeviceToHost);
        hipFree(devFound);
        output = found;
        return expected == output;
    }

    bool unsortedSegmentIndicesValidate(nd4j::LaunchContext* context , NDArray* indices, Nd4jLong expected, Nd4jLong& output) {
        BUILD_SINGLE_SELECTOR(indices->dataType(), return unsortedSegmentIndicesValidate_, (context, indices, expected, output), INTEGER_TYPES);
    }
    BUILD_SINGLE_TEMPLATE(template bool unsortedSegmentIndicesValidate_, (nd4j::LaunchContext* context , NDArray* indices, Nd4jLong expected, Nd4jLong& output), INTEGER_TYPES);
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static void unsortedSegmentMaxFunctor_(nd4j::LaunchContext* context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        auto stream = context->getCudaStream();
//        NDArray classes = NDArrayFactory::create<int>('c', {numOfClasses, 2});
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numOfClasses});
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numOfClasses});
//        NDArray row = NDArrayFactory::create<int>('c', {1, 2}, {(int)indices->lengthOf(), (int)0});
//        classes.applyTrueBroadcast(nd4j::BroadcastOpsTuple::Assign(), &row, &classes);
        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);
        dim3 dims(numOfClasses, indices->lengthOf(), numOfClasses * 32 + 32);
//        int* classesBuf = reinterpret_cast<int*>(classes.specialBuffer());
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());
        fillUpSegmentsKernel<I><<<dims.x, dims.y, dims.z, *stream>>>(indices->specialBuffer(), indices->specialShapeInfo(), numOfClasses, begins, lengths);
        classesRangesBegs.syncToHost();
        classesRangesLens.syncToHost();

        if (input->isVector()) {
            unsortedSegmentMaxLinearKernel<T,I><<<dims.x, dims.y, dims.z, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), indices->specialBuffer(), indices->specialShapeInfo(), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
            auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
            Nd4jLong* inputTads = packX.specialShapeInfo();
            Nd4jLong* inputTadOffsets = packX.specialOffsets();
            Nd4jLong* outputTads = packZ.specialShapeInfo();
            Nd4jLong* outputTadOffsets = packZ.specialOffsets();
            dims.x = input->sizeAt(0);
            output->assign(-DataTypeUtils::max<T>());
            segmentMaxTadKernel<T,I><<<dims.x, dims.y, dims.z, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);
        }

    }
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static void unsortedSegmentMinFunctor_(nd4j::LaunchContext* context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        auto stream = context->getCudaStream();
//        NDArray classes = NDArrayFactory::create<int>('c', {numOfClasses, 2});
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numOfClasses});
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numOfClasses});
//        NDArray row = NDArrayFactory::create<int>('c', {1, 2}, {(int)indices->lengthOf(), (int)0});
//        classes.applyTrueBroadcast(nd4j::BroadcastOpsTuple::Assign(), &row, &classes);
        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);
        dim3 dims(numOfClasses, indices->lengthOf(), numOfClasses * 32 + 32);
//        int* classesBuf = reinterpret_cast<int*>(classes.specialBuffer());
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());
        fillUpSegmentsKernel<I><<<dims.x, dims.y, dims.z, *stream>>>(indices->specialBuffer(), indices->specialShapeInfo(), numOfClasses, begins, lengths);

        if (input->isVector()) {
            unsortedSegmentMinLinearKernel<T,I><<<dims.x, dims.y, dims.z, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), indices->specialBuffer(), indices->specialShapeInfo(), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            output->assign(DataTypeUtils::max<T>());
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
            auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
            Nd4jLong* inputTads = packX.specialShapeInfo();
            Nd4jLong* inputTadOffsets = packX.specialOffsets();
            Nd4jLong* outputTads = packZ.specialShapeInfo();
            Nd4jLong* outputTadOffsets = packZ.specialOffsets();
            dims.x = input->sizeAt(0);
            segmentMinTadKernel<T,I><<<dims.x, dims.y, dims.z, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);
        }

    }
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static void unsortedSegmentMeanFunctor_(nd4j::LaunchContext* context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        auto stream = context->getCudaStream();
//        NDArray classes = NDArrayFactory::create<int>('c', {numOfClasses, 2});
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numOfClasses});
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numOfClasses});
//        NDArray row = NDArrayFactory::create<int>('c', {1, 2}, {(int)indices->lengthOf(), (int)0});
//        classes.applyTrueBroadcast(nd4j::BroadcastOpsTuple::Assign(), &row, &classes);
        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);
        dim3 dims(numOfClasses, indices->lengthOf(), numOfClasses * 32 + 32);
//        int* classesBuf = reinterpret_cast<int*>(classes.specialBuffer());
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());
        fillUpSegmentsKernel<I><<<dims.x, dims.y, dims.z, *stream>>>(indices->specialBuffer(), indices->specialShapeInfo(), numOfClasses, begins, lengths);

        if (input->isVector()) {
            unsortedSegmentMeanLinearKernel<T,I><<<dims.x, dims.y, dims.z, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), indices->specialBuffer(), indices->specialShapeInfo(), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            output->assign(0);
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
            auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
            Nd4jLong* inputTads = packX.specialShapeInfo();
            Nd4jLong* inputTadOffsets = packX.specialOffsets();
            Nd4jLong* outputTads = packZ.specialShapeInfo();
            Nd4jLong* outputTadOffsets = packZ.specialOffsets();
            dims.x = input->sizeAt(0);
            segmentMeanTadKernel<T,I><<<dims.x, dims.y, dims.z, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);
        }

    }
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static void unsortedSegmentSumFunctor_(nd4j::LaunchContext* context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        auto stream = context->getCudaStream();
//        NDArray classes = NDArrayFactory::create<int>('c', {numOfClasses, 2});
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numOfClasses});
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numOfClasses});
//        NDArray row = NDArrayFactory::create<int>('c', {1, 2}, {(int)indices->lengthOf(), (int)0});
//        classes.applyTrueBroadcast(nd4j::BroadcastOpsTuple::Assign(), &row, &classes);
        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);
        dim3 dims(numOfClasses, indices->lengthOf(), (numOfClasses + 1) * 64);
//        int* classesBuf = reinterpret_cast<int*>(classes.specialBuffer());
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());
        fillUpSegmentsKernel<I><<<dims.x, dims.y, dims.z, *stream>>>(indices->specialBuffer(), indices->specialShapeInfo(), numOfClasses, begins, lengths);

        if (input->isVector()) {
            unsortedSegmentSumLinearKernel<T,I><<<dims.x, dims.y, dims.z, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), indices->specialBuffer(), indices->specialShapeInfo(), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            output->assign(0);
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
            auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
            Nd4jLong* inputTads = packX.specialShapeInfo();
            Nd4jLong* inputTadOffsets = packX.specialOffsets();
            Nd4jLong* outputTads = packZ.specialShapeInfo();
            Nd4jLong* outputTadOffsets = packZ.specialOffsets();
            dims.x = input->sizeAt(0);
            segmentSumTadKernel<T,I><<<dims.x, dims.y, dims.z, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);
        }

    }
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static void unsortedSegmentProdFunctor_(nd4j::LaunchContext* context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        auto stream = context->getCudaStream();
//        NDArray classes = NDArrayFactory::create<int>('c', {numOfClasses, 2});
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numOfClasses});
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numOfClasses});
//        NDArray row = NDArrayFactory::create<int>('c', {1, 2}, {(int)indices->lengthOf(), (int)0});
//        classes.applyTrueBroadcast(nd4j::BroadcastOpsTuple::Assign(), &row, &classes);
        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);
        dim3 dims(numOfClasses, indices->lengthOf(), numOfClasses * 32 + 32);
//        int* classesBuf = reinterpret_cast<int*>(classes.specialBuffer());
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());
        fillUpSegmentsKernel<I><<<dims.x, dims.y, dims.z, *stream>>>(indices->specialBuffer(), indices->specialShapeInfo(), numOfClasses, begins, lengths);

        if (input->isVector()) {
            unsortedSegmentProdLinearKernel<T,I><<<dims.x, dims.y, dims.z, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), indices->specialBuffer(), indices->specialShapeInfo(), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            output->assign(1);
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
            auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
            Nd4jLong* inputTads = packX.specialShapeInfo();
            Nd4jLong* inputTadOffsets = packX.specialOffsets();
            Nd4jLong* outputTads = packZ.specialShapeInfo();
            Nd4jLong* outputTadOffsets = packZ.specialOffsets();
            dims.x = input->sizeAt(0);
            segmentProdTadKernel<T,I><<<dims.x, dims.y, dims.z, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);
        }

    }
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static void unsortedSegmentSqrtNFunctor_(nd4j::LaunchContext* context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        auto stream = context->getCudaStream();
//        NDArray classes = NDArrayFactory::create<int>('c', {numOfClasses, 2});
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numOfClasses});
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numOfClasses});
//        NDArray row = NDArrayFactory::create<int>('c', {1, 2}, {(int)indices->lengthOf(), (int)0});
//        classes.applyTrueBroadcast(nd4j::BroadcastOpsTuple::Assign(), &row, &classes);
        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);
        dim3 dims(numOfClasses, indices->lengthOf(), numOfClasses * 32 + 32);
//        int* classesBuf = reinterpret_cast<int*>(classes.specialBuffer());
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());
        fillUpSegmentsKernel<I><<<dims.x, dims.y, dims.z, *stream>>>(indices->specialBuffer(), indices->specialShapeInfo(), numOfClasses, begins, lengths);

        if (input->isVector()) {
            unsortedSegmentSqrtNLinearKernel<T,I><<<dims.x, dims.y, dims.z, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), indices->specialBuffer(), indices->specialShapeInfo(), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            output->assign(0);
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
            auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
            Nd4jLong* inputTads = packX.specialShapeInfo();
            Nd4jLong* inputTadOffsets = packX.specialOffsets();
            Nd4jLong* outputTads = packZ.specialShapeInfo();
            Nd4jLong* outputTadOffsets = packZ.specialOffsets();
            dims.x = input->sizeAt(0);
            segmentSqrtNTadKernel<T,I><<<dims.x, dims.y, dims.z, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //
    // -------------------------------------------------------------------------------------------------------------- //
    // unsorted ops functors
    // -------------------------------------------------------------------------------------------------------------- //

    void unsortedSegmentMaxFunctor(nd4j::LaunchContext* context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), unsortedSegmentMaxFunctor_, (context, input, indices, numOfClasses, output), NUMERIC_TYPES, INTEGER_TYPES);
    }
    // -------------------------------------------------------------------------------------------------------------- //

    void unsortedSegmentMinFunctor(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), unsortedSegmentMinFunctor_, (context, input, indices, numOfClasses, output),
                              NUMERIC_TYPES, INTEGER_TYPES);
    }
    // -------------------------------------------------------------------------------------------------------------- //

    void unsortedSegmentMeanFunctor(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), unsortedSegmentMeanFunctor_, (context, input, indices, numOfClasses, output),
                              FLOAT_TYPES, INTEGER_TYPES);
    }
    // -------------------------------------------------------------------------------------------------------------- //

    void unsortedSegmentSumFunctor(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), unsortedSegmentSumFunctor_, (context, input, indices, numOfClasses, output),
                              NUMERIC_TYPES, INTEGER_TYPES);

    }
    // -------------------------------------------------------------------------------------------------------------- //

    void unsortedSegmentProdFunctor(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), unsortedSegmentProdFunctor_, (context, input, indices, numOfClasses, output),
                              FLOAT_TYPES, INTEGER_TYPES);
    }
    // -------------------------------------------------------------------------------------------------------------- //

    void unsortedSegmentSqrtNFunctor(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), unsortedSegmentSqrtNFunctor_, (context, input, indices, numOfClasses, output),
                              FLOAT_TYPES, INTEGER_TYPES);
    }
    // -------------------------------------------------------------------------------------------------------------- //

    BUILD_DOUBLE_TEMPLATE(template void unsortedSegmentMaxFunctor_, (nd4j::LaunchContext* context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output), NUMERIC_TYPES, INTEGER_TYPES);
    BUILD_DOUBLE_TEMPLATE(template void unsortedSegmentMinFunctor_, (nd4j::LaunchContext* context , NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output), NUMERIC_TYPES, INTEGER_TYPES);
    BUILD_DOUBLE_TEMPLATE(template void unsortedSegmentMeanFunctor_, (nd4j::LaunchContext* context , NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output), FLOAT_TYPES, INTEGER_TYPES);
    BUILD_DOUBLE_TEMPLATE(template void unsortedSegmentSumFunctor_, (nd4j::LaunchContext* context , NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output), NUMERIC_TYPES, INTEGER_TYPES);
    BUILD_DOUBLE_TEMPLATE(template void unsortedSegmentProdFunctor_, (nd4j::LaunchContext* context , NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output), FLOAT_TYPES, INTEGER_TYPES);
    BUILD_DOUBLE_TEMPLATE(template void unsortedSegmentSqrtNFunctor_, (nd4j::LaunchContext* context , NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output), FLOAT_TYPES, INTEGER_TYPES);
    // -------------------------------------------------------------------------------------------------------------- //

    // -------------------------------------------------------------------------------------------------------------- //
    // Backpropagate ops helpers
    // -------------------------------------------------------------------------------------------------------------- //
    // Sorted backpropagate ops
    // -------------------------------------------------------------------------------------------------------------- //
    // segment max
    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    static __global__ void segmentMaxBPLinearKernel(void* inputBuf, Nd4jLong* inputShape, void* forwardOutput,
            Nd4jLong* forwardShape, void* eps, Nd4jLong* epsShape, void* indicesBuf, Nd4jLong* indicesShape,
            void* outputBuf, Nd4jLong* outputShape) {
        __shared__ T* x;
        __shared__ T* gradIn;
        __shared__ T* gradOut;
        __shared__ I* y;
        __shared__ T* z;
        __shared__ Nd4jLong xLen, gradLen;

        if (threadIdx.x == 0) {
            xLen = shape::length(inputShape);
            x = reinterpret_cast<T*>(inputBuf);
            y = reinterpret_cast<I*>(indicesBuf);
            z = reinterpret_cast<T*>(outputBuf);
            gradIn = reinterpret_cast<T*>(forwardOutput);
            gradOut = reinterpret_cast<T*>(eps);
            gradLen = shape::length(epsShape);
        }

        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = gridDim.x * blockDim.x;

        for (auto e = start; e < xLen; e += step) {

            auto zOffset = shape::getIndexOffset(e, outputShape, xLen);
            auto xOffset = shape::getIndexOffset(e, inputShape, xLen);
            auto yOffset = shape::getIndexOffset(e, indicesShape, xLen);
            auto classIndex = y[yOffset];
            auto gradOffsetI = shape::getIndexOffset(classIndex, forwardShape, gradLen);
            auto gradOffsetO = shape::getIndexOffset(classIndex, epsShape, gradLen);

            if (nd4j::math::nd4j_abs(gradIn[gradOffsetI] - x[xOffset]) <= T(1.e-6)) {
                z[zOffset] = gradOut[gradOffsetO];
            }
        }
    }
    template <typename T, typename I>
    static __global__ void segmentSumBPLinearKernel(void* inputBuf, Nd4jLong* inputShape, void* eps, Nd4jLong* epsShape,
            void* indicesBuf, Nd4jLong* indicesShape, void* outputBuf, Nd4jLong* outputShape) {
        __shared__ T* x;
        __shared__ T* gradIn;
        __shared__ T* gradOut;
        __shared__ I* y;
        __shared__ T* z;
        __shared__ Nd4jLong xLen, gradLen;

        if (threadIdx.x == 0) {
            xLen = shape::length(inputShape);
            x = reinterpret_cast<T*>(inputBuf);
            y = reinterpret_cast<I*>(indicesBuf);
            z = reinterpret_cast<T*>(outputBuf);
            gradOut = reinterpret_cast<T*>(eps);
            gradLen = shape::length(epsShape);
        }

        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = gridDim.x * blockDim.x;

        for (auto e = start; e < xLen; e += step) {

            auto zOffset = shape::getIndexOffset(e, outputShape, xLen);
            auto xOffset = shape::getIndexOffset(e, inputShape, xLen);
            auto yOffset = shape::getIndexOffset(e, indicesShape, xLen);
            auto classIndex = y[yOffset];
            auto gradOffsetO = shape::getIndexOffset(classIndex, epsShape, gradLen);

            z[zOffset] = gradOut[gradOffsetO];
        }
    }

    template <typename T, typename I>
    static __global__ void segmentProdBPLinearKernel(void* inputBuf, Nd4jLong* inputShape, void* forwardOutput,
                                                    Nd4jLong* forwardShape, void* eps, Nd4jLong* epsShape, void* indicesBuf, Nd4jLong* indicesShape,
                                                    void* outputBuf, Nd4jLong* outputShape) {
        __shared__ T* x;
        __shared__ T* gradIn;
        __shared__ T* gradOut;
        __shared__ I* y;
        __shared__ T* z;
        __shared__ Nd4jLong xLen, gradLen;

        if (threadIdx.x == 0) {
            xLen = shape::length(inputShape);
            x = reinterpret_cast<T*>(inputBuf);
            y = reinterpret_cast<I*>(indicesBuf);
            z = reinterpret_cast<T*>(outputBuf);
            gradIn = reinterpret_cast<T*>(forwardOutput);
            gradOut = reinterpret_cast<T*>(eps);
            gradLen = shape::length(epsShape);
        }

        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = gridDim.x * blockDim.x;

        for (auto e = start; e < xLen; e += step) {

            auto zOffset = shape::getIndexOffset(e, outputShape, xLen);
            auto xOffset = shape::getIndexOffset(e, inputShape, xLen);
            auto yOffset = shape::getIndexOffset(e, indicesShape, xLen);
            auto classIndex = y[yOffset];
            auto gradOffsetI = shape::getIndexOffset(classIndex, forwardShape, gradLen);
            auto gradOffsetO = shape::getIndexOffset(classIndex, epsShape, gradLen);

            z[zOffset] = gradOut[gradOffsetO]  * gradIn[gradOffsetI] / x[xOffset];
        }
    }

    template <typename T, typename I>
    static __global__ void segmentMeanBPLinearKernel(void* inputBuf, Nd4jLong* inputShape, void* eps, Nd4jLong* epsShape, void* indicesBuf, Nd4jLong* indicesShape,
                                                     int* lengths, void* outputBuf, Nd4jLong* outputShape) {
        __shared__ T* x;
        __shared__ T* gradIn;
        __shared__ T* gradOut;
        __shared__ I* y;
        __shared__ T* z;
        __shared__ Nd4jLong xLen, gradLen;

        if (threadIdx.x == 0) {
            xLen = shape::length(inputShape);
            x = reinterpret_cast<T*>(inputBuf);
            y = reinterpret_cast<I*>(indicesBuf);
            z = reinterpret_cast<T*>(outputBuf);
            gradOut = reinterpret_cast<T*>(eps);
            gradLen = shape::length(epsShape);
        }

        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = gridDim.x * blockDim.x;

        for (auto e = start; e < xLen; e += step) {

            auto zOffset = shape::getIndexOffset(e, outputShape, xLen);
            auto xOffset = shape::getIndexOffset(e, inputShape, xLen);
            auto yOffset = shape::getIndexOffset(e, indicesShape, xLen);
            auto classIndex = y[yOffset];
            auto gradOffsetO = shape::getIndexOffset(classIndex, epsShape, gradLen);

            z[zOffset] = T(gradOut[gradOffsetO] / float(lengths[classIndex]));
        }
    }

    template <typename T, typename I>
    static __global__ void segmentSqrtNBPLinearKernel(void* inputBuf, Nd4jLong* inputShape, void* eps, Nd4jLong* epsShape, void* indicesBuf, Nd4jLong* indicesShape,
                                                     int* lengths, void* outputBuf, Nd4jLong* outputShape) {
        __shared__ T* x;
        __shared__ T* gradIn;
        __shared__ T* gradOut;
        __shared__ I* y;
        __shared__ T* z;
        __shared__ Nd4jLong xLen, gradLen;

        if (threadIdx.x == 0) {
            xLen = shape::length(inputShape);
            x = reinterpret_cast<T*>(inputBuf);
            y = reinterpret_cast<I*>(indicesBuf);
            z = reinterpret_cast<T*>(outputBuf);
            gradOut = reinterpret_cast<T*>(eps);
            gradLen = shape::length(epsShape);
        }

        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = gridDim.x * blockDim.x;

        for (auto e = start; e < xLen; e += step) {

            auto zOffset = shape::getIndexOffset(e, outputShape, xLen);
            auto xOffset = shape::getIndexOffset(e, inputShape, xLen);
            auto yOffset = shape::getIndexOffset(e, indicesShape, xLen);
            auto classIndex = y[yOffset];
            auto gradOffsetO = shape::getIndexOffset(classIndex, epsShape, gradLen);

            z[zOffset] = T(gradOut[gradOffsetO] / math::nd4j_sqrt<int, float>(lengths[classIndex]));
        }
    }

    template <typename T, typename I>
    static __global__ void segmentMaxBPTadKernel(void* inputBuf, Nd4jLong* inputShape, void* forwardOutput,
                                                    Nd4jLong* forwardShape, void* eps, Nd4jLong* epsShape, void* indicesBuf, Nd4jLong* indicesShape,
                                                    void* outputBuf, Nd4jLong* outputShape,Nd4jLong* inputTad,
                                                    Nd4jLong* inputOffsets, Nd4jLong* gradInTad, Nd4jLong* gradInOffsets,
                                                    Nd4jLong* gradOutTad, Nd4jLong* gradOutOffsets, Nd4jLong* outTad,
                                                    Nd4jLong* outOffsets) {
        __shared__ T* x;
        __shared__ T* gradIn;
        __shared__ T* gradOut;
        __shared__ I* y;
        __shared__ T* z;
        __shared__ Nd4jLong xLen, yLen, gradLen, currentLen;

        if (threadIdx.x == 0) {
            xLen = shape::length(inputShape);
            x = reinterpret_cast<T*>(inputBuf);
            y = reinterpret_cast<I*>(indicesBuf);
            z = reinterpret_cast<T*>(outputBuf);
            yLen = shape::length(indicesShape);
            gradOut = reinterpret_cast<T*>(eps);
            gradIn = reinterpret_cast<T*>(forwardOutput);
            gradLen = shape::length(epsShape);
            currentLen = shape::length(outTad);
        }

        for (auto i = blockIdx.x; i < yLen; i += gridDim.x) {
            auto yIndex = shape::getIndexOffset(i, indicesShape, yLen);
            auto segment = y[yIndex];
            T* current = x + inputOffsets[i];
            T* currentOut = z + outOffsets[i];
            T* in = gradIn + gradInOffsets[segment];
            T* outGrad = gradOut + gradOutOffsets[segment];

            for (auto e = threadIdx.x; e < currentLen; e += blockDim.x) {
                if (nd4j::math::nd4j_abs(in[e] - current[e]) <= T(1.e-6))
                    currentOut[e] = outGrad[e];
            }
        }
    }

    template <typename T, typename I>
    static __global__ void segmentSumBPTadKernel(void* inputBuf, Nd4jLong* inputShape, void* eps, Nd4jLong* epsShape,
            void* indicesBuf, Nd4jLong* indicesShape, void* outputBuf, Nd4jLong* outputShape, Nd4jLong* inputTad,
            Nd4jLong* inputOffsets, Nd4jLong* gradOutTad, Nd4jLong* gradOutOffsets, Nd4jLong* outTad, Nd4jLong* outOffsets) {
        __shared__ T* x;
        __shared__ T* gradOut;
        __shared__ I* y;
        __shared__ T* z;
        __shared__ Nd4jLong xLen, yLen, gradLen, currentLen;

        if (threadIdx.x == 0) {
            xLen = shape::length(inputShape);
            x = reinterpret_cast<T*>(inputBuf);
            y = reinterpret_cast<I*>(indicesBuf);
            z = reinterpret_cast<T*>(outputBuf);
            yLen = shape::length(indicesShape);
            gradOut = reinterpret_cast<T*>(eps);
            gradLen = shape::length(epsShape);
            currentLen = shape::length(outTad);
        }

        for (auto i = blockIdx.x; i < yLen; i += gridDim.x) {
            auto yIndex = shape::getIndexOffset(i, indicesShape, yLen);
            auto segment = y[yIndex];
            T* currentOut = z + outOffsets[i];
            T* outGrad = gradOut + gradOutOffsets[segment];

            for (auto e = threadIdx.x; e < currentLen; e += blockDim.x) {
                currentOut[e] = outGrad[e];
            }
        }

    }
    template <typename T, typename I>
    static __global__ void segmentMeanBPTadKernel(void* inputBuf, Nd4jLong* inputShape, void* eps, Nd4jLong* epsShape,
            void* indicesBuf, Nd4jLong* indicesShape, int* lengths, void* outputBuf, Nd4jLong* outputShape,Nd4jLong* inputTad,
            Nd4jLong* inputOffsets, Nd4jLong* gradOutTad, Nd4jLong* gradOutOffsets, Nd4jLong* outTad, Nd4jLong* outOffsets) {
        __shared__ T* x;
        __shared__ T* gradOut;
        __shared__ I* y;
        __shared__ T* z;
        __shared__ Nd4jLong xLen, yLen, gradLen, currentLen;

        if (threadIdx.x == 0) {
            xLen = shape::length(inputShape);
            x = reinterpret_cast<T*>(inputBuf);
            y = reinterpret_cast<I*>(indicesBuf);
            z = reinterpret_cast<T*>(outputBuf);
            yLen = shape::length(indicesShape);
            gradOut = reinterpret_cast<T*>(eps);
            gradLen = shape::length(epsShape);
            currentLen = shape::length(outTad);
        }
        __syncthreads();

        for (auto i = blockIdx.x; i < yLen; i += gridDim.x) {
//            auto yIndex = shape::getIndexOffset(i, indicesShape, yLen);
            auto segment = y[i]; //yIndex];
            T* currentOut = z + outOffsets[i];
            T* outGrad = gradOut + gradOutOffsets[segment];

            for (auto e = threadIdx.x; e < currentLen; e += blockDim.x) {
                auto zIndex = shape::getIndexOffset(e, outTad, currentLen);
                auto gradIndex = shape::getIndexOffset(e, gradOutTad, gradLen);
                if (lengths[segment] > 0)
                currentOut[zIndex] = T(outGrad[gradIndex] / float(lengths[segment]));
            }
        }
    }
    template <typename T, typename I>
    static __global__ void segmentSqrtNBPTadKernel(void* inputBuf, Nd4jLong* inputShape, void* eps, Nd4jLong* epsShape,
                                                  void* indicesBuf, Nd4jLong* indicesShape, int* lengths, void* outputBuf, Nd4jLong* outputShape,Nd4jLong* inputTad,
                                                  Nd4jLong* inputOffsets, Nd4jLong* gradOutTad, Nd4jLong* gradOutOffsets, Nd4jLong* outTad, Nd4jLong* outOffsets) {
        __shared__ T* x;
        __shared__ T* gradOut;
        __shared__ I* y;
        __shared__ T* z;
        __shared__ Nd4jLong xLen, yLen, gradLen, currentLen;

        if (threadIdx.x == 0) {
            xLen = shape::length(inputShape);
            x = reinterpret_cast<T*>(inputBuf);
            y = reinterpret_cast<I*>(indicesBuf);
            z = reinterpret_cast<T*>(outputBuf);
            yLen = shape::length(indicesShape);
            gradOut = reinterpret_cast<T*>(eps);
            gradLen = shape::length(epsShape);
            currentLen = shape::length(outTad);
        }
        __syncthreads();

        for (auto i = blockIdx.x; i < yLen; i += gridDim.x) {
//            auto yIndex = shape::getIndexOffset(i, indicesShape, yLen);
            auto segment = y[i]; //yIndex];
            T* currentOut = z + outOffsets[i];
            T* outGrad = gradOut + gradOutOffsets[segment];

            for (auto e = threadIdx.x; e < currentLen; e += blockDim.x) {
                auto zIndex = shape::getIndexOffset(e, outTad, currentLen);
                auto gradIndex = shape::getIndexOffset(e, gradOutTad, gradLen);
                if (lengths[segment] > 0)
                    currentOut[zIndex] = T(outGrad[gradIndex] / math::nd4j_sqrt<int, float>(lengths[segment]));
            }
        }
    }

    template <typename T, typename I>
    static __global__ void segmentProdBPTadKernel(void* inputBuf, Nd4jLong* inputShape, void* forwardOutput,
                                                 Nd4jLong* forwardShape, void* eps, Nd4jLong* epsShape, void* indicesBuf, Nd4jLong* indicesShape,
                                                 void* outputBuf, Nd4jLong* outputShape,Nd4jLong* inputTad,
                                                 Nd4jLong* inputOffsets, Nd4jLong* gradInTad, Nd4jLong* gradInOffsets,
                                                 Nd4jLong* gradOutTad, Nd4jLong* gradOutOffsets, Nd4jLong* outTad,
                                                 Nd4jLong* outOffsets) {
        __shared__ T* x;
        __shared__ T* gradIn;
        __shared__ T* gradOut;
        __shared__ I* y;
        __shared__ T* z;
        __shared__ Nd4jLong xLen, yLen, gradLen, currentLen;

        if (threadIdx.x == 0) {
            xLen = shape::length(inputShape);
            x = reinterpret_cast<T*>(inputBuf);
            y = reinterpret_cast<I*>(indicesBuf);
            z = reinterpret_cast<T*>(outputBuf);
            yLen = shape::length(indicesShape);
            gradOut = reinterpret_cast<T*>(eps);
            gradIn = reinterpret_cast<T*>(forwardOutput);
            gradLen = shape::length(epsShape);
            currentLen = shape::length(outTad);
        }

        for (auto i = blockIdx.x; i < yLen; i += gridDim.x) {
            auto yIndex = shape::getIndexOffset(i, indicesShape, yLen);
            auto segment = y[yIndex];
            T* current = x + inputOffsets[i];
            T* currentOut = z + outOffsets[i];
            T* in = gradIn + gradInOffsets[segment];
            T* outGrad = gradOut + gradOutOffsets[segment];

            for (auto e = threadIdx.x; e < currentLen; e += blockDim.x) {
                    currentOut[e] = outGrad[e] * in[e] / current[e];
            }
        }

    }

    template <typename T, typename I>
    int segmentMaxFunctorBP_(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        //int numOfClasses = gradOut->sizeAt(0);
        // if input is a vector: (as if in doc sample)
        auto stream = context->getCudaStream();
        NDArray tempRes(gradOut->ordering(), gradOut->getShapeAsVector(), DataTypeUtils::fromT<T>(), context);//->shapeInfo(), context);
        segmentMaxFunctor_<T, I>(context, input, indices, &tempRes);
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut, &tempRes});
        if (input->isVector()) {
            Nd4jLong loop_size = input->lengthOf();
            auto numOfClasses = gradOut->lengthOf(); //indices->e<Nd4jLong>(loop_size - 1);
            segmentMaxBPLinearKernel<T,I><<<1 + gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                    tempRes.specialBuffer(), tempRes.specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
            auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
            auto packGradIn = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(tempRes.getShapeInfo(), dimensions);
            auto packGradOut = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(gradOut->getShapeInfo(), dimensions);
            Nd4jLong* inputTads = packX.specialShapeInfo();
            Nd4jLong* inputTadOffsets = packX.specialOffsets();
            Nd4jLong* outputTads = packZ.specialShapeInfo();
            Nd4jLong* outputTadOffsets = packZ.specialOffsets();
            Nd4jLong* gradInTads = packGradIn.specialShapeInfo();
            Nd4jLong* gradInTadOffsets = packGradIn.specialOffsets();
            Nd4jLong* gradOutTads = packGradOut.specialShapeInfo();
            Nd4jLong* gradOutTadOffsets = packGradOut.specialOffsets();

            segmentMaxBPTadKernel<T,I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                    tempRes.specialBuffer(), tempRes.specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(),
                    inputTads, inputTadOffsets, gradInTads, gradInTadOffsets, gradOutTads, gradOutTadOffsets,
                    outputTads, outputTadOffsets);
        }
        NDArray::registerSpecialUse({output}, {input, indices, gradOut, &tempRes});
        return Status::OK();
    }
    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    int segmentMinFunctorBP_(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        //int numOfClasses = gradOut->sizeAt(0);
        // if input is a vector: (as if in doc sample)
        auto stream = context->getCudaStream();
        NDArray tempRes(gradOut->ordering(), gradOut->getShapeAsVector(), DataTypeUtils::fromT<T>(), context);//->shapeInfo(), context);
        segmentMinFunctor_<T, I>(context, input, indices, &tempRes);
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut, &tempRes});
        if (input->isVector()) {
            Nd4jLong loop_size = input->lengthOf();
            auto numOfClasses = gradOut->lengthOf(); //indices->e<Nd4jLong>(loop_size - 1);
            segmentMaxBPLinearKernel<T,I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                    tempRes.specialBuffer(), tempRes.specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
            auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
            auto packGradIn = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(tempRes.getShapeInfo(), dimensions);
            auto packGradOut = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(gradOut->getShapeInfo(), dimensions);
            Nd4jLong* inputTads = packX.specialShapeInfo();
            Nd4jLong* inputTadOffsets = packX.specialOffsets();
            Nd4jLong* outputTads = packZ.specialShapeInfo();
            Nd4jLong* outputTadOffsets = packZ.specialOffsets();
            Nd4jLong* gradInTads = packGradIn.specialShapeInfo();
            Nd4jLong* gradInTadOffsets = packGradIn.specialOffsets();
            Nd4jLong* gradOutTads = packGradOut.specialShapeInfo();
            Nd4jLong* gradOutTadOffsets = packGradOut.specialOffsets();

            segmentMaxBPTadKernel<T,I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                    tempRes.specialBuffer(), tempRes.specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(),
                    inputTads, inputTadOffsets, gradInTads, gradInTadOffsets, gradOutTads, gradOutTadOffsets,
                    outputTads, outputTadOffsets);
        }
        NDArray::registerSpecialUse({output}, {input, indices, gradOut, &tempRes});
        return Status::OK();
    }
    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    int segmentSumFunctorBP_(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        auto stream = context->getCudaStream();
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
        if (input->isVector()) {
            Nd4jLong loop_size = input->lengthOf();
            auto numOfClasses = gradOut->lengthOf(); //indices->e<Nd4jLong>(loop_size - 1);
            segmentSumBPLinearKernel<T,I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(),
                    input->specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
            auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
            auto packGradOut = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(gradOut->getShapeInfo(), dimensions);
            Nd4jLong* inputTads = packX.specialShapeInfo();
            Nd4jLong* inputTadOffsets = packX.specialOffsets();
            Nd4jLong* outputTads = packZ.specialShapeInfo();
            Nd4jLong* outputTadOffsets = packZ.specialOffsets();
            Nd4jLong* gradOutTads = packGradOut.specialShapeInfo();
            Nd4jLong* gradOutTadOffsets = packGradOut.specialOffsets();

            segmentSumBPTadKernel<T,I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                    gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(),
                    inputTads, inputTadOffsets, gradOutTads, gradOutTadOffsets,
                    outputTads, outputTadOffsets);
        }
        NDArray::registerSpecialUse({output}, {input, indices, gradOut});
        return Status::OK();
    }
    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    int segmentMeanFunctorBP_(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        auto stream = context->getCudaStream();
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
        auto numClasses = indices->e<int>(indices->lengthOf() - 1) + 1;
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numClasses});
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numClasses});

        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);
        dim3 dims(numClasses, indices->lengthOf(), numClasses * 32 + 32);
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());
        fillUpSegmentsKernel<I><<<dims.x, dims.y, dims.z, *stream>>>(indices->specialBuffer(), indices->specialShapeInfo(), numClasses, begins, lengths);

        if (input->isVector()) {
            Nd4jLong loop_size = input->lengthOf();
            auto numOfClasses = gradOut->lengthOf(); //indices->e<Nd4jLong>(loop_size - 1);
            segmentMeanBPLinearKernel<T,I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(),
                    input->specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), lengths, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
            auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
//            auto packGradIn = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(tempRes.getShapeInfo(), dimensions);
            auto packGradOut = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(gradOut->getShapeInfo(), dimensions);
            Nd4jLong* inputTads = packX.specialShapeInfo();
            Nd4jLong* inputTadOffsets = packX.specialOffsets();
            Nd4jLong* outputTads = packZ.specialShapeInfo();
            Nd4jLong* outputTadOffsets = packZ.specialOffsets();
            Nd4jLong* gradOutTads = packGradOut.specialShapeInfo();
            Nd4jLong* gradOutTadOffsets = packGradOut.specialOffsets();

            segmentMeanBPTadKernel<T,I><<<indices->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                    gradOut->specialBuffer(), gradOut->specialShapeInfo(), indices->specialBuffer(), indices->specialShapeInfo(), lengths,
                    output->specialBuffer(), output->specialShapeInfo(), inputTads, inputTadOffsets, gradOutTads, gradOutTadOffsets,
                    outputTads, outputTadOffsets);
        }
        NDArray::registerSpecialUse({output}, {input, indices, gradOut});
        return Status::OK();
    }
    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    int segmentProdFunctorBP_(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        auto stream = context->getCudaStream();
        NDArray tempRes(gradOut->ordering(), gradOut->getShapeAsVector(), DataTypeUtils::fromT<T>(), context);//->shapeInfo(), context);
        segmentProdFunctor_<T, I>(context, input, indices, &tempRes);
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
        if (input->isVector()) {
            Nd4jLong loopSize = input->lengthOf();
            auto numOfClasses = gradOut->lengthOf(); //indices->e<Nd4jLong>(loop_size - 1);
            segmentProdBPLinearKernel<T,I><<<gradOut->lengthOf(), loopSize, 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                    tempRes.specialBuffer(), tempRes.specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
            auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
            auto packGradIn = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(tempRes.getShapeInfo(), dimensions);
            auto packGradOut = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(gradOut->getShapeInfo(), dimensions);
            Nd4jLong* inputTads = packX.specialShapeInfo();
            Nd4jLong* inputTadOffsets = packX.specialOffsets();
            Nd4jLong* outputTads = packZ.specialShapeInfo();
            Nd4jLong* outputTadOffsets = packZ.specialOffsets();
            Nd4jLong* gradInTads = packGradIn.specialShapeInfo();
            Nd4jLong* gradInTadOffsets = packGradIn.specialOffsets();
            Nd4jLong* gradOutTads = packGradOut.specialShapeInfo();
            Nd4jLong* gradOutTadOffsets = packGradOut.specialOffsets();

            segmentProdBPTadKernel<T,I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                    tempRes.specialBuffer(), tempRes.specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(),
                    inputTads, inputTadOffsets, gradInTads, gradInTadOffsets, gradOutTads, gradOutTadOffsets,
                    outputTads, outputTadOffsets);
        }
        NDArray::registerSpecialUse({output}, {input, indices, gradOut});
        return Status::OK();
    }

    // -------------------------------------------------------------------------------------------------------------- //
    int segmentMaxFunctorBP(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), return segmentMaxFunctorBP_, (context, input,
                indices, gradOut, output), NUMERIC_TYPES, INTEGER_TYPES);
    }

    // -------------------------------------------------------------------------------------------------------------- //
    // segmen min
    int segmentMinFunctorBP(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), return segmentMinFunctorBP_, (context, input,
                indices, gradOut, output), NUMERIC_TYPES, INTEGER_TYPES);
    }

    // -------------------------------------------------------------------------------------------------------------- //
    // segmen mean
    int segmentMeanFunctorBP(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), return segmentMeanFunctorBP_, (context, input,
                indices, gradOut, output), NUMERIC_TYPES, INTEGER_TYPES);
    }
    // -------------------------------------------------------------------------------------------------------------- //

    int segmentSumFunctorBP(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), return segmentSumFunctorBP_, (context, input,
                indices, gradOut, output), NUMERIC_TYPES, INTEGER_TYPES);
    }
    // -------------------------------------------------------------------------------------------------------------- //

    int segmentProdFunctorBP(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), return segmentProdFunctorBP_, (context, input,
                indices, gradOut, output), FLOAT_TYPES, INTEGER_TYPES);
    }
    // -------------------------------------------------------------------------------------------------------------- //

    BUILD_DOUBLE_TEMPLATE(template int segmentMaxFunctorBP_, (nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output), NUMERIC_TYPES, INTEGER_TYPES);
    BUILD_DOUBLE_TEMPLATE(template int segmentMinFunctorBP_, (nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output), NUMERIC_TYPES, INTEGER_TYPES);
    BUILD_DOUBLE_TEMPLATE(template int segmentSumFunctorBP_, (nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output), NUMERIC_TYPES, INTEGER_TYPES);
    BUILD_DOUBLE_TEMPLATE(template int segmentMeanFunctorBP_, (nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output), FLOAT_TYPES, INTEGER_TYPES);
    BUILD_DOUBLE_TEMPLATE(template int segmentProdFunctorBP_, (nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output), FLOAT_TYPES, INTEGER_TYPES);

    // -------------------------------------------------------------------------------------------------------------- //
    // Unsorted backpropagate segment ops
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static int unsortedSegmentMaxFunctorBP_(nd4j::LaunchContext* context, NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        //int numOfClasses = gradOut->sizeAt(0);
        // if input is a vector: (as if in doc sample)
        auto stream = context->getCudaStream();
        NDArray tempRes(gradOut->ordering(), gradOut->getShapeAsVector(), DataTypeUtils::fromT<T>(), context);//->shapeInfo(), context);
        unsortedSegmentMaxFunctor_<T, I>(context, input, indices, numOfClasses, &tempRes);
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut, &tempRes});
        if (input->isVector()) {
            Nd4jLong loop_size = input->lengthOf();
            auto numOfClasses = gradOut->lengthOf(); //indices->e<Nd4jLong>(loop_size - 1);
            segmentMaxBPLinearKernel<T,I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                    tempRes.specialBuffer(), tempRes.specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
            auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
            auto packGradIn = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(tempRes.getShapeInfo(), dimensions);
            auto packGradOut = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(gradOut->getShapeInfo(), dimensions);
            Nd4jLong* inputTads = packX.specialShapeInfo();
            Nd4jLong* inputTadOffsets = packX.specialOffsets();
            Nd4jLong* outputTads = packZ.specialShapeInfo();
            Nd4jLong* outputTadOffsets = packZ.specialOffsets();
            Nd4jLong* gradInTads = packGradIn.specialShapeInfo();
            Nd4jLong* gradInTadOffsets = packGradIn.specialOffsets();
            Nd4jLong* gradOutTads = packGradOut.specialShapeInfo();
            Nd4jLong* gradOutTadOffsets = packGradOut.specialOffsets();

            segmentMaxBPTadKernel<T,I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                    tempRes.specialBuffer(), tempRes.specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(),
                    inputTads, inputTadOffsets, gradInTads, gradInTadOffsets, gradOutTads, gradOutTadOffsets,
                    outputTads, outputTadOffsets);
        }
        NDArray::registerSpecialUse({output}, {input, indices, gradOut, &tempRes});
        return Status::OK();
    }
    // -------------------------------------------------------------------------------------------------------------- //


    template <typename T, typename I>
    static int unsortedSegmentMinFunctorBP_(nd4j::LaunchContext* context, NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        //int numOfClasses = gradOut->sizeAt(0);
        // if input is a vector: (as if in doc sample)
        auto stream = context->getCudaStream();
        NDArray tempRes(gradOut->ordering(), gradOut->getShapeAsVector(), DataTypeUtils::fromT<T>(), context);//->shapeInfo(), context);
        unsortedSegmentMinFunctor_<T, I>(context, input, indices, numOfClasses, &tempRes);
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut, &tempRes});
        if (input->isVector()) {
            Nd4jLong loop_size = input->lengthOf();
            auto numOfClasses = gradOut->lengthOf(); //indices->e<Nd4jLong>(loop_size - 1);
            segmentMaxBPLinearKernel<T,I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                    tempRes.specialBuffer(), tempRes.specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
            auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
            auto packGradIn = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(tempRes.getShapeInfo(), dimensions);
            auto packGradOut = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(gradOut->getShapeInfo(), dimensions);
            Nd4jLong* inputTads = packX.specialShapeInfo();
            Nd4jLong* inputTadOffsets = packX.specialOffsets();
            Nd4jLong* outputTads = packZ.specialShapeInfo();
            Nd4jLong* outputTadOffsets = packZ.specialOffsets();
            Nd4jLong* gradInTads = packGradIn.specialShapeInfo();
            Nd4jLong* gradInTadOffsets = packGradIn.specialOffsets();
            Nd4jLong* gradOutTads = packGradOut.specialShapeInfo();
            Nd4jLong* gradOutTadOffsets = packGradOut.specialOffsets();

            segmentMaxBPTadKernel<T,I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                    tempRes.specialBuffer(), tempRes.specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(),
                    inputTads, inputTadOffsets, gradInTads, gradInTadOffsets, gradOutTads, gradOutTadOffsets,
                    outputTads, outputTadOffsets);
        }
        NDArray::registerSpecialUse({output}, {input, indices, gradOut, &tempRes});
        return Status::OK();
    }
    // -------------------------------------------------------------------------------------------------------------- //


    template <typename T, typename I>
    static int unsortedSegmentMeanFunctorBP_(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        auto stream = context->getCudaStream();
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
        auto numClasses = indices->e<int>(indices->lengthOf() - 1) + 1;
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numClasses});
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numClasses});

        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);
        dim3 dims(numClasses, indices->lengthOf(), numClasses * 32 + 32);
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());
        fillUpSegmentsKernel<I><<<dims.x, dims.y, dims.z, *stream>>>(indices->specialBuffer(), indices->specialShapeInfo(), numClasses, begins, lengths);

        if (input->isVector()) {
            Nd4jLong loop_size = input->lengthOf();
            auto numOfClasses = gradOut->lengthOf(); //indices->e<Nd4jLong>(loop_size - 1);
            segmentMeanBPLinearKernel<T,I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(),
                    input->specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), lengths, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
            auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
//            auto packGradIn = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(tempRes.getShapeInfo(), dimensions);
            auto packGradOut = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(gradOut->getShapeInfo(), dimensions);
            Nd4jLong* inputTads = packX.specialShapeInfo();
            Nd4jLong* inputTadOffsets = packX.specialOffsets();
            Nd4jLong* outputTads = packZ.specialShapeInfo();
            Nd4jLong* outputTadOffsets = packZ.specialOffsets();
            Nd4jLong* gradOutTads = packGradOut.specialShapeInfo();
            Nd4jLong* gradOutTadOffsets = packGradOut.specialOffsets();

            segmentMeanBPTadKernel<T,I><<<indices->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                    gradOut->specialBuffer(), gradOut->specialShapeInfo(), indices->specialBuffer(), indices->specialShapeInfo(), lengths,
                    output->specialBuffer(), output->specialShapeInfo(), inputTads, inputTadOffsets, gradOutTads, gradOutTadOffsets,
                    outputTads, outputTadOffsets);
        }
        NDArray::registerSpecialUse({output}, {input, indices, gradOut});
        return Status::OK();
    }
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static int unsortedSegmentSumFunctorBP_(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        auto stream = context->getCudaStream();
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
        if (input->isVector()) {
            Nd4jLong loop_size = input->lengthOf();
            auto numOfClasses = gradOut->lengthOf(); //indices->e<Nd4jLong>(loop_size - 1);
            segmentSumBPLinearKernel<T,I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(),
                    input->specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
            auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
            auto packGradOut = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(gradOut->getShapeInfo(), dimensions);
            Nd4jLong* inputTads = packX.specialShapeInfo();
            Nd4jLong* inputTadOffsets = packX.specialOffsets();
            Nd4jLong* outputTads = packZ.specialShapeInfo();
            Nd4jLong* outputTadOffsets = packZ.specialOffsets();
            Nd4jLong* gradOutTads = packGradOut.specialShapeInfo();
            Nd4jLong* gradOutTadOffsets = packGradOut.specialOffsets();

            segmentSumBPTadKernel<T,I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                    gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(),
                    inputTads, inputTadOffsets, gradOutTads, gradOutTadOffsets,
                    outputTads, outputTadOffsets);
        }
        NDArray::registerSpecialUse({output}, {input, indices, gradOut});
        return Status::OK();
    }
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static int unsortedSegmentProdFunctorBP_(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        auto stream = context->getCudaStream();
        NDArray tempRes(gradOut->ordering(), gradOut->getShapeAsVector(), DataTypeUtils::fromT<T>(), context);//->shapeInfo(), context);
        unsortedSegmentProdFunctor_<T, I>(context, input, indices, numOfClasses, &tempRes);
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
        if (input->isVector()) {
            Nd4jLong loopSize = input->lengthOf();
            auto numOfClasses = gradOut->lengthOf(); //indices->e<Nd4jLong>(loop_size - 1);
            segmentProdBPLinearKernel<T,I><<<gradOut->lengthOf(), loopSize, 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                    tempRes.specialBuffer(), tempRes.specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
            auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
            auto packGradIn = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(tempRes.getShapeInfo(), dimensions);
            auto packGradOut = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(gradOut->getShapeInfo(), dimensions);
            Nd4jLong* inputTads = packX.specialShapeInfo();
            Nd4jLong* inputTadOffsets = packX.specialOffsets();
            Nd4jLong* outputTads = packZ.specialShapeInfo();
            Nd4jLong* outputTadOffsets = packZ.specialOffsets();
            Nd4jLong* gradInTads = packGradIn.specialShapeInfo();
            Nd4jLong* gradInTadOffsets = packGradIn.specialOffsets();
            Nd4jLong* gradOutTads = packGradOut.specialShapeInfo();
            Nd4jLong* gradOutTadOffsets = packGradOut.specialOffsets();

            segmentProdBPTadKernel<T,I><<<indices->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                    tempRes.specialBuffer(), tempRes.specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(),
                    inputTads, inputTadOffsets, gradInTads, gradInTadOffsets, gradOutTads, gradOutTadOffsets,
                    outputTads, outputTadOffsets);
        }
        NDArray::registerSpecialUse({output}, {input, indices, gradOut});
        return Status::OK();
    }
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static int unsortedSegmentSqrtNFunctorBP_(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        auto stream = context->getCudaStream();
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
        auto numClasses = indices->e<int>(indices->lengthOf() - 1) + 1;
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numClasses});
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numClasses});

        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);
        dim3 dims(numClasses, indices->lengthOf(), numClasses * 32 + 32);
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());
        fillUpSegmentsKernel<I><<<dims.x, dims.y, dims.z, *stream>>>(indices->specialBuffer(), indices->specialShapeInfo(), numClasses, begins, lengths);

        if (input->isVector()) {
            Nd4jLong loop_size = input->lengthOf();
            auto numOfClasses = gradOut->lengthOf(); //indices->e<Nd4jLong>(loop_size - 1);
            segmentSqrtNBPLinearKernel<T,I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(),
                    input->specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), lengths, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
            auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
//            auto packGradIn = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(tempRes.getShapeInfo(), dimensions);
            auto packGradOut = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(gradOut->getShapeInfo(), dimensions);
            Nd4jLong* inputTads = packX.specialShapeInfo();
            Nd4jLong* inputTadOffsets = packX.specialOffsets();
            Nd4jLong* outputTads = packZ.specialShapeInfo();
            Nd4jLong* outputTadOffsets = packZ.specialOffsets();
            Nd4jLong* gradOutTads = packGradOut.specialShapeInfo();
            Nd4jLong* gradOutTadOffsets = packGradOut.specialOffsets();

            segmentSqrtNBPTadKernel<T,I><<<indices->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                    gradOut->specialBuffer(), gradOut->specialShapeInfo(), indices->specialBuffer(), indices->specialShapeInfo(), lengths,
                    output->specialBuffer(), output->specialShapeInfo(), inputTads, inputTadOffsets, gradOutTads, gradOutTadOffsets,
                    outputTads, outputTadOffsets);
        }
        NDArray::registerSpecialUse({output}, {input, indices, gradOut});

        return Status::OK();
    }
    // ============================================================================================================== //
    int unsortedSegmentMaxFunctorBP(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), return unsortedSegmentMaxFunctorBP_, (context, input, indices, gradOut, numOfClasses, output), NUMERIC_TYPES, INTEGER_TYPES);
    }
    // -------------------------------------------------------------------------------------------------------------- //

    int unsortedSegmentMinFunctorBP(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), return unsortedSegmentMinFunctorBP_, (context, input, indices, gradOut, numOfClasses, output), NUMERIC_TYPES, INTEGER_TYPES);
    }
    // -------------------------------------------------------------------------------------------------------------- //

    int unsortedSegmentSumFunctorBP(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), return unsortedSegmentSumFunctorBP_, (context, input, indices, gradOut, numOfClasses, output), NUMERIC_TYPES, INTEGER_TYPES);
    }
    // -------------------------------------------------------------------------------------------------------------- //

    int unsortedSegmentMeanFunctorBP(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), return unsortedSegmentMeanFunctorBP_, (context, input, indices, gradOut, numOfClasses, output), FLOAT_TYPES, INTEGER_TYPES);
    }
    // -------------------------------------------------------------------------------------------------------------- //

    int unsortedSegmentProdFunctorBP(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), return unsortedSegmentProdFunctorBP_, (context, input, indices, gradOut, numOfClasses, output), FLOAT_TYPES, INTEGER_TYPES);
    }
    // -------------------------------------------------------------------------------------------------------------- //

    int unsortedSegmentSqrtNFunctorBP(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), return unsortedSegmentSqrtNFunctorBP_, (context, input, indices, gradOut, numOfClasses, output), FLOAT_TYPES, INTEGER_TYPES);
    }
    // -------------------------------------------------------------------------------------------------------------- //

    BUILD_DOUBLE_TEMPLATE(template int unsortedSegmentMaxFunctorBP_, (nd4j::LaunchContext* context, NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output), NUMERIC_TYPES, INTEGER_TYPES);
    BUILD_DOUBLE_TEMPLATE(template int unsortedSegmentMinFunctorBP_, (nd4j::LaunchContext* context, NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output), NUMERIC_TYPES, INTEGER_TYPES);
    BUILD_DOUBLE_TEMPLATE(template int unsortedSegmentSumFunctorBP_, (nd4j::LaunchContext* context, NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output), NUMERIC_TYPES, INTEGER_TYPES);
    BUILD_DOUBLE_TEMPLATE(template int unsortedSegmentMeanFunctorBP_, (nd4j::LaunchContext* context, NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output), FLOAT_TYPES, INTEGER_TYPES);
    BUILD_DOUBLE_TEMPLATE(template int unsortedSegmentProdFunctorBP_, (nd4j::LaunchContext* context, NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output), FLOAT_TYPES, INTEGER_TYPES);
    BUILD_DOUBLE_TEMPLATE(template int unsortedSegmentSqrtNFunctorBP_, (nd4j::LaunchContext* context, NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output), FLOAT_TYPES, INTEGER_TYPES);
    // -------------------------------------------------------------------------------------------------------------- //

}
}
}
// -------------------------------------------------------------------------------------------------------------- //
// -------------------------------------------------------------------------------------------------------------- //
