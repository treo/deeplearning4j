#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author GS <sgazeos@gmail.com>
//

#include <ops/declarable/helpers/legacy_helpers.h>
#include <NDArrayFactory.h>
#include <op_boilerplate.h>

namespace nd4j {
namespace ops {
namespace helpers {
    template <typename T>
    linkage void reluDerivative__(NDArray* theFirst, NDArray* theSecond) {
        auto functor = LAMBDA_TT(x, y){
            return x > (T) 0.f ? y : T(0.f);
        };

        theFirst->applyPairwiseLambda(theSecond, functor, nullptr);
    }
    BUILD_SINGLE_TEMPLATE(template void reluDerivative__, (NDArray* input, NDArray* epsilon), FLOAT_TYPES);

    void reluDerivative(nd4j::LaunchContext * context, NDArray* theFirst, NDArray* theSecond) {
        BUILD_SINGLE_SELECTOR(theFirst->dataType(), reluDerivative__, (theFirst, theSecond), FLOAT_TYPES);
    }

    template <typename T>
    linkage void reluDerivative_(NDArray* input, NDArray* epsilon, NDArray* output) {
        auto functor = LAMBDA_TT(x, y){
            return x > (T)0.f ? y : T(0.f);
        };

        input->applyPairwiseLambda(epsilon, functor, output);
    }
    BUILD_SINGLE_TEMPLATE(template void reluDerivative_, (NDArray* input, NDArray* epsilon, NDArray*output);, FLOAT_TYPES);

    void reluDerivative(nd4j::LaunchContext * context, NDArray* theFirst, NDArray* theSecond, NDArray* theOutput) {
        BUILD_SINGLE_SELECTOR(theFirst->dataType(), reluDerivative_, (theFirst, theSecond, theOutput), FLOAT_TYPES);
    }

    template <typename T>
    linkage void relu6Derivative_(NDArray* input, NDArray* epsilon, NDArray* output) {
        auto functor = LAMBDA_TT(x, y){
            return x > (T)0.f && x < (T)6.f? y : T(0.f);
        };

        input->applyPairwiseLambda(epsilon, functor, output);
    }

    BUILD_SINGLE_TEMPLATE(template void relu6Derivative_, (NDArray* input, NDArray* epsilon, NDArray*output);, FLOAT_TYPES);

    void relu6Derivative(nd4j::LaunchContext * context, NDArray* theFirst, NDArray* theSecond, NDArray* theOutput) {
        BUILD_SINGLE_SELECTOR(theFirst->dataType(), relu6Derivative_, (theFirst, theSecond, theOutput), FLOAT_TYPES);
    }

    template <typename T>
    linkage void leakyReluDerivative_(NDArray* input, NDArray* epsilon, NDArray* output) {
        auto functor = LAMBDA_TT(x, y){
            return x >= (T)0.f? T(1.f) : T(0.f);
        };

        input->applyPairwiseLambda(epsilon, functor, output);
    }

    BUILD_SINGLE_TEMPLATE(template void leakyReluDerivative_, (NDArray* input, NDArray* epsilon, NDArray*output);, FLOAT_TYPES);

    void leakyReluDerivative(nd4j::LaunchContext * context, NDArray* theFirst, NDArray* theSecond, NDArray* theOutput) {
        BUILD_SINGLE_SELECTOR(theFirst->dataType(), leakyReluDerivative_, (theFirst, theSecond, theOutput), FLOAT_TYPES);
    }

    template <typename T>
    linkage void eluDerivative_(NDArray* input, NDArray* epsilon, NDArray* output) {
        auto functor = LAMBDA_TT(x, y){
            return y * nd4j::math::nd4j_eluderivative<T,T>(x);
        };

        input->applyPairwiseLambda(epsilon, functor, output);
    }

    BUILD_SINGLE_TEMPLATE(template void eluDerivative_, (NDArray* input, NDArray* epsilon, NDArray*output);, FLOAT_TYPES);

    void eluDerivative(nd4j::LaunchContext * context, NDArray* theFirst, NDArray* theSecond, NDArray* theOutput) {
        BUILD_SINGLE_SELECTOR(theFirst->dataType(), eluDerivative_, (theFirst, theSecond, theOutput), FLOAT_TYPES);
    }

    template <typename T>
    linkage void seluDerivative_(NDArray* input, NDArray* epsilon, NDArray* output) {
        auto functor = LAMBDA_TT(x, y){
            return y * simdOps::SELUDerivative<T>::op(x, nullptr);
        };

        input->applyPairwiseLambda(epsilon, functor, output);
    }

    BUILD_SINGLE_TEMPLATE(template void seluDerivative_, (NDArray* input, NDArray* epsilon, NDArray*output);, FLOAT_TYPES);

    void seluDerivative(nd4j::LaunchContext * context, NDArray* theFirst, NDArray* theSecond, NDArray* theOutput) {
        BUILD_SINGLE_SELECTOR(theFirst->dataType(), seluDerivative_, (theFirst, theSecond, theOutput), FLOAT_TYPES);
    }

    template <typename T>
    linkage void cubeDerivative_(NDArray* input, NDArray* epsilon, NDArray* output) {
        auto functor = LAMBDA_TT(x, y){
            return y * (3 * x * x);
        };

        input->applyPairwiseLambda(epsilon, functor, output);
    }

    BUILD_SINGLE_TEMPLATE(template void cubeDerivative_, (NDArray* input, NDArray* epsilon, NDArray*output);, FLOAT_TYPES);

    void cubeDerivative(nd4j::LaunchContext * context, NDArray* theFirst, NDArray* theSecond, NDArray* theOutput) {
        BUILD_SINGLE_SELECTOR(theFirst->dataType(), cubeDerivative_, (theFirst, theSecond, theOutput), FLOAT_TYPES);
    }

    //return (x >= X(0.f) ? y: -y);
    template <typename T>
    linkage void reduceNorm1_(NDArray* input, NDArray* epsilon, NDArray* output) {
        auto functor = LAMBDA_TT(x, y){
            return x > T(0.f)? y : -y;
        };

        input->applyPairwiseLambda(epsilon, functor, output);
    }

    BUILD_SINGLE_TEMPLATE(template void reduceNorm1_, (NDArray* input, NDArray* epsilon, NDArray*output);, FLOAT_TYPES);

    void reduceNorm1(nd4j::LaunchContext * context, NDArray* theFirst, NDArray* theSecond, NDArray* theOutput) {
        BUILD_SINGLE_SELECTOR(theFirst->dataType(), reduceNorm1_, (theFirst, theSecond, theOutput), FLOAT_TYPES);
    }

    ////////////////////////////////////////////////////////////////////////
    template <typename T>
    linkage void sigmCrossEntropy_(NDArray* logits, NDArray* labels, NDArray* output) {
        auto functor = LAMBDA_TT(x, y){
            return nd4j::math::nd4j_max<T>(x, (T)0.f) - x * y + nd4j::math::nd4j_log<T,T>((T)1.f + nd4j::math::nd4j_exp<T,T>(-nd4j::math::nd4j_abs(x)));
        };

        logits->applyPairwiseLambda(labels, functor, output);
    }

    BUILD_SINGLE_TEMPLATE(template void sigmCrossEntropy_, (NDArray* logits, NDArray* labels, NDArray* output);, FLOAT_TYPES);

    void sigmCrossEntropy(nd4j::LaunchContext * context, NDArray* logits, NDArray* labels, NDArray* output) {
        BUILD_SINGLE_SELECTOR(logits->dataType(), sigmCrossEntropy_, (logits, labels, output), FLOAT_TYPES);
    }

    ////////////////////////////////////////////////////////////////////////
    template <typename T>
    linkage void sigmCrossEntropyGrad_(NDArray* logits, NDArray* labels, NDArray* output) {
        // 1 - labels - 1 / (1 + exp(logits))
        auto functor = LAMBDA_TT(x, y) {
            if(x <= 0)
                return static_cast<T>(1.) - y - static_cast<T>(1.) / (static_cast<T>(1.) + nd4j::math::nd4j_exp<T,T>(x));
            auto e = nd4j::math::nd4j_exp<T,T>(-x);
            return static_cast<T>(1.) - y - e / (static_cast<T>(1.) + e);
        };

        logits->applyPairwiseLambda(labels, functor, output);
    }

    BUILD_SINGLE_TEMPLATE(template void sigmCrossEntropyGrad_, (NDArray* logits, NDArray* labels, NDArray*output);, FLOAT_TYPES);

    void sigmCrossEntropyGrad(nd4j::LaunchContext * context, NDArray* logits, NDArray* labels, NDArray* output) {
        BUILD_SINGLE_SELECTOR(logits->dataType(), sigmCrossEntropyGrad_, (logits, labels, output), FLOAT_TYPES);
    }

    ////////////////////////////////////////////////////////////////////////
    template <typename T>
    linkage void tanhDerivative_(NDArray* input, NDArray* epsilon, NDArray* output) {
        auto functor = LAMBDA_TT(x, y){
            T th = nd4j::math::nd4j_tanh<T,T>(x);
            return y * ((T)1.0f - (th * th));
        };

        input->applyPairwiseLambda(epsilon, functor, output);
    }

    BUILD_SINGLE_TEMPLATE(template void tanhDerivative_, (NDArray* input, NDArray* epsilon, NDArray*output);, FLOAT_TYPES);

    void tanhDerivative(nd4j::LaunchContext * context, NDArray* theFirst, NDArray* theSecond, NDArray* theOutput) {
        BUILD_SINGLE_SELECTOR(theFirst->dataType(), tanhDerivative_, (theFirst, theSecond, theOutput), FLOAT_TYPES);
    }

    // return static_cast<X>(d2) * simdOps::HardTanhDerivative<X>::op(d1, nullptr);
    template <typename T>
    linkage void hardTanhDerivative_(NDArray* input, NDArray* epsilon, NDArray* output) {
        auto functor = LAMBDA_TT(x, y){
            T th = nd4j::math::nd4j_tanh<T,T>(x);
            return y * simdOps::HardTanhDerivative<T>::op(x, nullptr);
        };

        input->applyPairwiseLambda(epsilon, functor, output);
    }

    BUILD_SINGLE_TEMPLATE(template void hardTanhDerivative_, (NDArray* input, NDArray* epsilon, NDArray*output);, FLOAT_TYPES);

    void hardTanhDerivative(nd4j::LaunchContext * context, NDArray* theFirst, NDArray* theSecond, NDArray* theOutput) {
        BUILD_SINGLE_SELECTOR(theFirst->dataType(), hardTanhDerivative_, (theFirst, theSecond, theOutput), FLOAT_TYPES);
    }

    template <typename T>
    linkage void rationalTanhDerivative_(NDArray* input, NDArray* epsilon, NDArray* output) {
        auto functor = LAMBDA_TT(x, y){
            return y * simdOps::RationalTanhDerivative<T>::op(x, nullptr);
        };

        input->applyPairwiseLambda(epsilon, functor, output);
    }

    BUILD_SINGLE_TEMPLATE(template void rationalTanhDerivative_, (NDArray* input, NDArray* epsilon, NDArray*output);, FLOAT_TYPES);

    void rationalTanhDerivative(nd4j::LaunchContext * context, NDArray* theFirst, NDArray* theSecond, NDArray* theOutput) {
        BUILD_SINGLE_SELECTOR(theFirst->dataType(), rationalTanhDerivative_, (theFirst, theSecond, theOutput), FLOAT_TYPES);
    }

    template <typename T>
    linkage void rectifiedTanhDerivative_(NDArray* input, NDArray* epsilon, NDArray* output) {
        auto functor = LAMBDA_TT(x, y){
            return x > (T) 0.0f ? y * (nd4j::math::nd4j_tanhderivative<T,T>(x)) : (T) 0.0f;
        };

        input->applyPairwiseLambda(epsilon, functor, output);
    }

    BUILD_SINGLE_TEMPLATE(template void rectifiedTanhDerivative_, (NDArray* input, NDArray* epsilon, NDArray*output);, FLOAT_TYPES);

    void rectifiedTanhDerivative(nd4j::LaunchContext * context, NDArray* theFirst, NDArray* theSecond, NDArray* theOutput) {
        BUILD_SINGLE_SELECTOR(theFirst->dataType(), rectifiedTanhDerivative_, (theFirst, theSecond, theOutput), FLOAT_TYPES);
    }

    //            X f = (X) 1.0f + nd4j::math::nd4j_abs<X>(d1);
    //            return (X) d2 * ((X) 1.0f / (f * f));

    template <typename T>
    linkage void softSignDerivative_(NDArray* input, NDArray* epsilon, NDArray* output) {
        auto functor = LAMBDA_TT(x, y){
            T ss = (T)1.f + nd4j::math::nd4j_abs<T>(x);
            return y * ((T) 1.0f  / (ss * ss));
        };

        input->applyPairwiseLambda(epsilon, functor, output);
    }

    BUILD_SINGLE_TEMPLATE(template void softSignDerivative_, (NDArray* input, NDArray* epsilon, NDArray*output);, FLOAT_TYPES);

    void softSignDerivative(nd4j::LaunchContext * context, NDArray* theFirst, NDArray* theSecond, NDArray* theOutput) {
        BUILD_SINGLE_SELECTOR(theFirst->dataType(), softSignDerivative_, (theFirst, theSecond, theOutput), FLOAT_TYPES);
    }

    template <typename T>
    linkage void softPlusDerivative_(NDArray* input, NDArray* epsilon, NDArray* output) {
        auto functor = LAMBDA_TT(x, y){
            T p = nd4j::math::nd4j_pow<T, T, T>(static_cast<T>(M_E), x);
            return y * (p / (p + 1.));
        };

        input->applyPairwiseLambda(epsilon, functor, output);
    }

    BUILD_SINGLE_TEMPLATE(template void softPlusDerivative_, (NDArray* input, NDArray* epsilon, NDArray*output);, FLOAT_TYPES);

    void softPlusDerivative(nd4j::LaunchContext * context, NDArray* theFirst, NDArray* theSecond, NDArray* theOutput) {
        BUILD_SINGLE_SELECTOR(theFirst->dataType(), softPlusDerivative_, (theFirst, theSecond, theOutput), FLOAT_TYPES);
    }
///
/// \param theFirst
/// \param theSecond
/// \param theOutput
    template <typename T>
    linkage void sigmoidDerivative_(NDArray* input, NDArray* epsilon, NDArray* output) {
        auto functor = LAMBDA_TT(x, y){
            T s = nd4j::math::nd4j_sigmoid<T,T>(x);
            return y * (s * ((T) 1.0f - s));
        };

        input->applyPairwiseLambda(epsilon, functor, output);
    }

    BUILD_SINGLE_TEMPLATE(template void sigmoidDerivative_, (NDArray* input, NDArray* epsilon, NDArray*output);, FLOAT_TYPES);

    void sigmoidDerivative(nd4j::LaunchContext * context, NDArray* theFirst, NDArray* theSecond, NDArray* theOutput) {
        BUILD_SINGLE_SELECTOR(theFirst->dataType(), sigmoidDerivative_, (theFirst, theSecond, theOutput), FLOAT_TYPES);
    }

    template <typename T>
    linkage void hardSigmoidDerivative_(NDArray* input, NDArray* epsilon, NDArray* output) {
        auto functor = LAMBDA_TT(x, y){
            return y * simdOps::HardSigmoidDerivative<T>::op(x, nullptr);
        };

        input->applyPairwiseLambda(epsilon, functor, output);
    }

    BUILD_SINGLE_TEMPLATE(template void hardSigmoidDerivative_, (NDArray* input, NDArray* epsilon, NDArray*output);, FLOAT_TYPES);

    void hardSigmoidDerivative(nd4j::LaunchContext * context, NDArray* theFirst, NDArray* theSecond, NDArray* theOutput) {
        BUILD_SINGLE_SELECTOR(theFirst->dataType(), hardSigmoidDerivative_, (theFirst, theSecond, theOutput), FLOAT_TYPES);
    }

    template <typename T>
    linkage void logSumExp_(NDArray* input, NDArray* axis, NDArray* output) {
        // reduce along axis with
        std::unique_ptr<NDArray> tempInput(input->dup());
        input->applyTransform(transform::Exp, tempInput.get());
        std::vector<int> axisVector;
        if (axis != nullptr) {
            axisVector.resize(axis->lengthOf());
            for (size_t i = 0; i < axisVector.size(); ++i)
                axisVector[i] = axis->e<int>(i);
        }
        tempInput->reduceAlongDimension(reduce::Sum, output, axisVector);
        output->applyTransform(transform::Log, nullptr, nullptr);
    }

    template <typename T>
    linkage void logSumExp_(NDArray* input, NDArray* subtrah, NDArray* axis, NDArray* output) {
        // reduce along axis with
        std::unique_ptr<NDArray> tempInput(input->dup());
        input->applyPairwiseTransform(pairwise::Subtract, subtrah, tempInput.get());
        tempInput->applyTransform(transform::Exp, nullptr, nullptr);

        std::vector<int> axisVector;
        if (axis != nullptr) {
            axisVector.resize(axis->lengthOf());
            for (size_t i = 0; i < axisVector.size(); ++i)
                axisVector[i] = axis->e<int>(i);
        }
        tempInput->reduceAlongDimension(reduce::Sum, output, axisVector);
        output->applyTransform(transform::Log, nullptr, nullptr);
    }

    void logSumExp(nd4j::LaunchContext * context, NDArray* input, NDArray* axis, NDArray* output) {
        BUILD_SINGLE_SELECTOR(input->dataType(), logSumExp_, (input, axis, output), FLOAT_TYPES);
    }
    BUILD_SINGLE_TEMPLATE(template void logSumExp_, (NDArray* input, NDArray* axis, NDArray*output);, FLOAT_TYPES);

    void logSumExp(nd4j::LaunchContext * context, NDArray* input, NDArray* subtrah, NDArray* axis, NDArray* output) {
        BUILD_SINGLE_SELECTOR(input->dataType(), logSumExp_, (input, subtrah, axis, output), FLOAT_TYPES);
    }
    BUILD_SINGLE_TEMPLATE(template void logSumExp_, (NDArray* input, NDArray* subtrah, NDArray* axis, NDArray*output);, FLOAT_TYPES);

//////////////////////////////////////////////////////////////////////////
template <typename T>
static void weightedCrossEntropyWithLogitsFunctor_(NDArray const* targets, NDArray const* input, NDArray const* weights, NDArray* output) {

}

void weightedCrossEntropyWithLogitsFunctor(nd4j::LaunchContext * context, NDArray const* targets, NDArray const* input, NDArray const* weights, NDArray* output) {
    BUILD_SINGLE_SELECTOR(targets->dataType(), weightedCrossEntropyWithLogitsFunctor_, (targets, input, weights, output), FLOAT_TYPES);
}
BUILD_SINGLE_TEMPLATE(template void weightedCrossEntropyWithLogitsFunctor_, (NDArray const* targets, NDArray const* input, NDArray const* weights, NDArray* output), FLOAT_TYPES);

}
}
}