#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com), created on 17.05.2018
// @author raver119@gmail.com
//

#include <ops/declarable/helpers/percentile.h>
#include <NDArrayFactory.h>
#include <helpers/ConstantTadHelper.h>
#include <helpers/DebugHelper.h>
#include "ResultSet.h"

namespace nd4j    {
namespace ops     {
namespace helpers {

    template <typename X>
    static _CUDA_G void percentileKernel(void *vx, Nd4jLong *xTadShapeInfo, Nd4jLong *xTadOffsets, const Nd4jLong numTads, const Nd4jLong tadLength, void *vz, Nd4jLong *zShapeInfo, const Nd4jLong zLength, const Nd4jLong position) {
        for (int t = blockIdx.x; t < numTads; t += gridDim.x) {
            auto x = reinterpret_cast<X*>(vx) + xTadOffsets[t];
            auto z = reinterpret_cast<X*>(vz);


            // sort tad
            if (tadLength > 1) {
                for (int m = 0; m < tadLength; m++) {
                    if (m % 2 == 0) {
                        for (int tid = threadIdx.x; tid < tadLength; tid += blockDim.x) {
                            auto top = 2 * tid + 1;
                            if (top < tadLength) {
                                auto t0 = shape::getIndexOffset(top - 1, xTadShapeInfo, tadLength);
                                auto t1 = shape::getIndexOffset(top, xTadShapeInfo, tadLength);

                                if (x[t0] > x[t1]) {
                                    //swap values
                                    X dz0 = x[t0];
                                    x[t0] = x[t1];
                                    x[t1] = dz0;
                                }
                            }
                        }
                    } else {
                        for (int tid = threadIdx.x; tid < tadLength; tid += blockDim.x) {
                            auto top = 2 * tid + 2;
                            if (top < tadLength) {
                                auto t0 = shape::getIndexOffset(top - 1, xTadShapeInfo, tadLength);
                                auto t1 = shape::getIndexOffset(top, xTadShapeInfo, tadLength);

                                if (x[t0] > x[t1]) {
                                    //swap values
                                    X dz0 = x[t0];
                                    x[t0] = x[t1];
                                    x[t1] = dz0;
                                }
                            }
                        }
                    }
                    __syncthreads();
                }
            }

            // saving final value
            if (threadIdx.x == 0)
                z[shape::getIndexOffset(t, zShapeInfo, zLength)] = x[shape::getIndexOffset(position, xTadShapeInfo, tadLength)];
        }
    }



    template <typename T>
    static void _percentile(nd4j::LaunchContext * context, const NDArray& input, NDArray& output, std::vector<int>& axis, const float q, const int interpolation) {
        const int inputRank = input.rankOf();

        if(axis.empty())
            for(int i=0; i<inputRank; ++i)
                axis.push_back(i);
        else
            shape::checkDimensions(inputRank, axis);

        auto tempArray = input.dup(input.ordering());
        auto packX = ConstantTadHelper::getInstance()->tadForDimensions(tempArray->getShapeInfo(), axis);

        auto tadLength = shape::length(packX.primaryShapeInfo());

        const float fraction = 1.f - q / 100.;
        Nd4jLong position = 0;

        switch(interpolation) {
            case 0: // lower
                position = static_cast<Nd4jLong>(math::nd4j_ceil<float,T>((tadLength - 1) * fraction));
                break;
            case 1: // higher
                position = static_cast<Nd4jLong>(math::nd4j_floor<float,T>((tadLength - 1) * fraction));
                break;
            case 2: // nearest
                position = static_cast<Nd4jLong>(math::nd4j_round<float,T>((tadLength - 1) * fraction));
                break;
        }
        position = tadLength - position - 1;

        percentileKernel<T><<<256, 512, 1024, *context->getCudaStream()>>>(tempArray->specialBuffer(), packX.platformShapeInfo(), packX.platformOffsets(), packX.numberOfTads(), tadLength, output.specialBuffer(), output.specialShapeInfo(), output.lengthOf(), position);

        nd4j::DebugHelper::checkErrorCode(context->getCudaStream(), "percentile");

        delete tempArray;
    }

    void percentile(nd4j::LaunchContext * context, const NDArray& input, NDArray& output, std::vector<int>& axises, const float q, const int interpolation) {
        NDArray::prepareSpecialUse({&output}, {&input});

        BUILD_SINGLE_SELECTOR(input.dataType(), _percentile, (context, input, output, axises, q, interpolation), LIBND4J_TYPES);

        NDArray::registerSpecialUse({&output}, {&input});
    }

    BUILD_SINGLE_TEMPLATE(template void _percentile, (nd4j::LaunchContext * context, const NDArray& input, NDArray& output, std::vector<int>& axises, const float q, const int interpolation), LIBND4J_TYPES);

}
}
}