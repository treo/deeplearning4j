#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author sgazeos@gmail.com
//

#include <ops/declarable/helpers/nth_element.h>
#include <TAD.h>
#include <ShapeUtils.h>
#include <PointersManager.h>
#include <NativeOps.h>
#include <helpers/ConstantTadHelper.h>

namespace nd4j {
namespace ops {
namespace helpers {

    template <typename T>
    static __global__ void fillUpElementKernel(void* outputBuffer, Nd4jLong* outputShapeInfo, void* inputBuffer, Nd4jLong* inputShapeInfo, Nd4jLong* pTadShape, Nd4jLong* pTadOffsets, Nd4jLong n) {
        __shared__ T *z, *x;
        __shared__ Nd4jLong bufferLength, arrLen;

        if (threadIdx.x == 0) {
            z = reinterpret_cast<T*>(outputBuffer);
            x = reinterpret_cast<T*>(inputBuffer);
            arrLen = shape::length(pTadShape);
            bufferLength = shape::length(outputShapeInfo);
        }
        __syncthreads();

        const auto tid = blockIdx.x * gridDim.x + threadIdx.x;
        const auto step = gridDim.x * blockDim.x;
        for (int t = tid; t < bufferLength; t += step) {
            auto tX = x + pTadOffsets[t];
            z[shape::getIndexOffset(t, outputShapeInfo, bufferLength)] = tX[shape::getIndexOffset(n, pTadShape, arrLen)]; //tX];
        }
    }

    template <typename T>
    void nthElementFunctor_(nd4j::LaunchContext * context, NDArray* input, Nd4jLong n, NDArray* output, bool reverse) {

        NDArray::prepareSpecialUse({output}, {input});
        NDArray sortedVals(*input);
        Nd4jPointer params[2];
        params[0] = nullptr;
        params[1] = context->getCudaStream();

        if (input->isVector()) {
            NativeOps ops;
            ops.sort(params, nullptr, sortedVals.shapeInfo(), sortedVals.specialBuffer(), sortedVals.specialShapeInfo(), reverse);

            hipMemcpy(reinterpret_cast<T*>(output->specialBuffer()), reinterpret_cast<T*>(sortedVals.specialBuffer()) + n, sizeof(T), hipMemcpyDeviceToDevice);
        }
        else { // rank greater than 1
            std::vector<int> lastDims({input->rankOf() - 1}); // = ShapeUtils::evalDimsToExclude(input->rankOf(), {input->rankOf() - 1});

            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(sortedVals.getShapeInfo(), lastDims);

            //PointersManager manager(context, "helpers::nth_element");
            auto pTadShape = packX.specialShapeInfo();
            auto pTadOffsets = packX.specialOffsets();
            //auto pLastDimData = (int*) manager.replicatePointer(lastDims.data(), lastDims.size() * sizeof(int));

            NativeOps ops;
            ops.sortTad(params, sortedVals.buffer(), sortedVals.shapeInfo(), sortedVals.specialBuffer(), sortedVals.specialShapeInfo(), lastDims.data(), lastDims.size(), pTadShape, pTadOffsets, reverse);
            sortedVals.tickWriteDevice();
            sortedVals.syncToHost();
            sortedVals.printIndexedBuffer("Hello");
            sortedVals.printBuffer("Hello line");
            auto stream = context->getCudaStream();
            fillUpElementKernel<T><<<32, 64, 1024, *stream>>>(output->specialBuffer(), output->specialShapeInfo(), sortedVals.specialBuffer(), sortedVals.specialShapeInfo(), pTadShape, pTadOffsets, n);
            //manager.synchronize();
        }
        NDArray::registerSpecialUse({output}, {input});
    }
    void nthElementFunctor(nd4j::LaunchContext * context, NDArray* input, Nd4jLong n, NDArray* output, bool reverse) {
    BUILD_SINGLE_SELECTOR(input->dataType(), nthElementFunctor_, (context, input, n, output, reverse), LIBND4J_TYPES);

    }
    BUILD_SINGLE_TEMPLATE(template void nthElementFunctor_, (nd4j::LaunchContext * context, NDArray* input, Nd4jLong n, NDArray* output, bool reverse), LIBND4J_TYPES);
    
}
}
}
