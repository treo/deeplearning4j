#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com), created on 20.04.2018
//


#include<ops/declarable/helpers/transforms.h>
#include <array/ResultSet.h>
#include <helpers/ShapeUtils.h>
#include <numeric>
#include <NDArrayFactory.h>
#include <helpers/TAD.h>
#include <exceptions/cuda_exception.h>
#include <PointersManager.h>
#include <ConstantTadHelper.h>

namespace nd4j 	  {
namespace ops 	  {
namespace helpers {

///////////////////////////////////////////////////////////////////
template<typename T>
__global__ static void concatCuda(const int numOfArrs, void* pVx,  void* pxShapeInfo, void* pVz, void* pzShapeInfo) {

    __shared__ int arrIdx, blocksPerArr;
    __shared__ T *x, *z;
    __shared__ Nd4jLong *zShapeInfo, *xShapeInfo, arrLen, arrLenPerBlock, start, end;

    if (threadIdx.x == 0) {

        blocksPerArr = (gridDim.x + numOfArrs - 1) / numOfArrs;     // ceil
        arrIdx = blockIdx.x / blocksPerArr;

        x = reinterpret_cast<T*>(reinterpret_cast<void**>(pVx)[arrIdx]);
        z = reinterpret_cast<T*>(reinterpret_cast<void**>(pVz)[arrIdx]);
        xShapeInfo = reinterpret_cast<Nd4jLong**>(pxShapeInfo)[arrIdx];
        zShapeInfo = reinterpret_cast<Nd4jLong**>(pzShapeInfo)[arrIdx];
        arrLen = shape::length(xShapeInfo);

        arrLenPerBlock = (arrLen + blocksPerArr - 1) / blocksPerArr;  // ceil

        start = (blockIdx.x % blocksPerArr) * arrLenPerBlock;
        end   = (start + arrLenPerBlock) > arrLen ? arrLen : (start + arrLenPerBlock);
    }

    __syncthreads();

    for (Nd4jLong i = start + threadIdx.x; i < end; i += blockDim.x)
        z[shape::getIndexOffset(i, zShapeInfo, arrLen)] = x[shape::getIndexOffset(i, xShapeInfo, arrLen)];
}

///////////////////////////////////////////////////////////////////
template<typename T>
__host__ static void concatCudaLauncher(const int numOfArrs, const hipStream_t *stream,  void* pVx, void* pxShapeInfo, void* pVz, void* pzShapeInfo) {

    concatCuda<T><<<512, 256, 1024, *stream>>>(numOfArrs, pVx, pxShapeInfo, pVz, pzShapeInfo);
}

///////////////////////////////////////////////////////////////////
// x - input, y - paddings, z - output
template<typename X, typename Y>
__global__ static void padCuda(const int mode,
                               const void *vx, const Nd4jLong *xShapeInfo,
                               const void *vy, const Nd4jLong *yShapeInfo,
                                     void *vz, const Nd4jLong *zShapeInfo,
                               const void *vPadVal) {

    const X padVal = *reinterpret_cast<const X*>(vPadVal);

    const auto x = reinterpret_cast<const X*>(vx);
    const auto y = reinterpret_cast<const Y*>(vy);
          auto z = reinterpret_cast<X*>(vz);

    __shared__ int rank, rankMinusOne;
    __shared__ Nd4jLong zLen, yLen, totalThreads, *coords, *xShape, *zShape, *xStride, *zStride, shift1, shift2, yStride0;

    if (threadIdx.x == 0) {

        extern __shared__ unsigned char shmem[];
        coords    = reinterpret_cast<Nd4jLong*>(shmem);
        zLen     = shape::length(zShapeInfo);
        xShape   = shape::shapeOf(const_cast<Nd4jLong*>(xShapeInfo));
        zShape   = shape::shapeOf(const_cast<Nd4jLong*>(zShapeInfo));
        xStride  = shape::stride(const_cast<Nd4jLong*>(xShapeInfo));
        zStride  = shape::stride(const_cast<Nd4jLong*>(zShapeInfo));
        yStride0 = shape::stride(const_cast<Nd4jLong*>(yShapeInfo))[0];
        rank     = shape::rank(xShapeInfo);
        zLen     = shape::length(zShapeInfo);
        yLen     = 2 * rank;
        rankMinusOne = rank - 1;
        totalThreads = gridDim.x * blockDim.x;
        shift1 = mode == 1 ? 0 : 1;         // REFLECT : SYMMETRIC
        shift2 = mode == 1 ? 2 : 1;         // REFLECT : SYMMETRIC
    }

    __syncthreads();

    auto xzCoord = coords + threadIdx.x * rank;       // we use xzCoord storage both for x and z arrays

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    if(mode == 0) { // CONSTANT case

        for (Nd4jLong i = tid; i < zLen; i += totalThreads) {

            shape::index2coords(rank, zShape, i, zLen, xzCoord);
            const auto zOffset = shape::getOffset(0, zShape, zStride, xzCoord, rank);

            bool within = true;
            for(int j = rankMinusOne; j >= 0; --j) {
                if(xShape[j] == zShape[j]) continue;
                const auto left = y[shape::getIndexOffset(yStride0 * j, yShapeInfo, yLen)];
                if(xzCoord[j] < left || xzCoord[j] >= left + xShape[j]) {within = false; break;}
                else                                                    {xzCoord[j] = xzCoord[j] - left;}
            }

            if(within)
                z[zOffset] = x[shape::getOffset(0, xShape, xStride, xzCoord, rank)];
            else
                z[zOffset] = padVal;
        }
    }
    else {  // REFLECT and SYMMETRIC cases

        for (Nd4jLong i = tid; i < zLen; i += totalThreads) {

            shape::index2coords(rank, zShape, i, zLen, xzCoord);
            const auto zOffset = shape::getOffset(0, zShape, zStride, xzCoord, rank);

            for(int j = rankMinusOne; j >= 0; --j) {

                if(xShape[j] == zShape[j]) continue;
                xzCoord[j] = xzCoord[j] - y[shape::getIndexOffset(yStride0 * j, yShapeInfo, yLen)];    // are ready to fill middle (within input dimension range)
                if(xzCoord[j] < 0)               xzCoord[j] = -xzCoord[j] - shift1;                // means fill from left
                else if(xzCoord[j] >= xShape[j]) xzCoord[j] = 2 * xShape[j] - xzCoord[j] - shift2; // means fill from right
            }

            const auto xOffset = shape::getOffset(0, xShape, xStride, xzCoord, rank);
            z[zOffset] = x[xOffset];
        }
    }
}

///////////////////////////////////////////////////////////////////
template<typename X, typename Y>
static void padCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                                const int mode,
                                const void *vx, const Nd4jLong *xShapeInfo,
                                const void *vy, const Nd4jLong *yShapeInfo,
                                      void *vz, const Nd4jLong *zShapeInfo,
                                const void* padVal) {

    padCuda<X,Y><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(mode, vx, xShapeInfo, vy, yShapeInfo, vz, zShapeInfo, padVal);
}

///////////////////////////////////////////////////////////////////
void pad(nd4j::LaunchContext * context, const int mode, const NDArray& input, const NDArray& paddings, NDArray& output, const NDArray& padValue) {

    PointersManager manager(context, "pad");

    NDArray::prepareSpecialUse({&output}, {&input, &paddings, &padValue});

    const int threadsPerBlock = MAX_NUM_THREADS / 4;
    const int blocksPerGrid = (output.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
    const int sharedMem = 8 * threadsPerBlock * output.rankOf() + 128;

    const auto xType = input.dataType();
    const auto yType = paddings.dataType();

    BUILD_DOUBLE_SELECTOR(xType, yType, padCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), mode, input.getSpecialBuffer(), input.getSpecialShapeInfo(), paddings.getSpecialBuffer(), paddings.getSpecialShapeInfo(), output.getSpecialBuffer(), output.getSpecialShapeInfo(), padValue.getSpecialBuffer()), LIBND4J_TYPES, INTEGER_TYPES);

    NDArray::registerSpecialUse({&output}, {&input, &paddings, &padValue});
    manager.synchronize();
}

///////////////////////////////////////////////////////////////////
template<typename T>
__global__ static void invertPermutationCuda(const void* vx, const Nd4jLong* xShapeInfo, void* vz, const Nd4jLong* zShapeInfo) {

    const T* x = reinterpret_cast<const T*>(vx);
          T* z = reinterpret_cast<T*>(vz);

    __shared__ Nd4jLong len, totalThreads;

    if (threadIdx.x == 0) {

        len  = shape::length(xShapeInfo);
        totalThreads = gridDim.x * blockDim.x;
    }

    __syncthreads();

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (Nd4jLong i = tid; i < len; i += totalThreads) {

        const auto xOffset = shape::getIndexOffset(i, xShapeInfo, len);
        const Nd4jLong index = x[xOffset];
        const auto zOffset = shape::getIndexOffset(index, zShapeInfo, len);
        z[zOffset] = i;
    }
}

///////////////////////////////////////////////////////////////////
template<typename T>
__host__ static void invertPermutationCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const hipStream_t *stream,
                                                   const void* vx, const Nd4jLong* xShapeInfo, void* vz, const Nd4jLong* zShapeInfo) {

    invertPermutationCuda<T><<<blocksPerGrid, threadsPerBlock, 1024, *stream>>>(vx, xShapeInfo, vz, zShapeInfo);
}
BUILD_SINGLE_TEMPLATE(template void invertPermutationCudaLauncher, (const int blocksPerGrid, const int threadsPerBlock, const hipStream_t *stream, const void* vx, const Nd4jLong* xShapeInfo, void* vz, const Nd4jLong* zShapeInfo), LIBND4J_TYPES);

////////////////////////////////////////////////////////////////////////
void invertPermutation(nd4j::LaunchContext* context, const NDArray& input, NDArray& output) {

    const int threadsPerBlock = MAX_NUM_THREADS;
    const int blocksPerGrid = (input.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;

    PointersManager manager(context, "invertPermutation");

    NDArray::prepareSpecialUse({&output}, {&input});
    BUILD_SINGLE_SELECTOR(input.dataType(), invertPermutationCudaLauncher, (blocksPerGrid, threadsPerBlock, context->getCudaStream(), input.getSpecialBuffer(), input.getSpecialShapeInfo(), output.getSpecialBuffer(), output.getSpecialShapeInfo()), LIBND4J_TYPES);
    NDArray::registerSpecialUse({&output}, {&input});

    manager.synchronize();
}

//////////////////////////////////////////////////////////////////////////
template<typename T>
__global__ static void traceCuda(const void* vx, const Nd4jLong* xShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const uint diagLen) {

    const auto x = reinterpret_cast<const T*>(vx);
          auto z = reinterpret_cast<T*>(vz);

    __shared__ T* sharedMem;
    __shared__ int xRank, zRank;        // xRank = zRank + 2
    __shared__ Nd4jLong xLen, zLen, *coordsMem;

    if (threadIdx.x == 0) {
        extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<T*>(shmem);
        coordsMem = reinterpret_cast<Nd4jLong*>(shmem + blockDim.x * sizeof(T));

        xRank = shape::rank(xShapeInfo);
        zRank = shape::rank(zShapeInfo);
        xLen = shape::length(xShapeInfo);
        zLen = shape::length(zShapeInfo);   // corresponds to number of matrices

    }
    __syncthreads();

    Nd4jLong* coords = coordsMem + threadIdx.x * xRank;

    for (uint m = blockIdx.x; m < zLen; m += gridDim.x) {   // one block per each element of z, that is per each matrix

        shape::index2coords(zRank, shape::shapeOf(const_cast<Nd4jLong*>(zShapeInfo)), m, zLen, coords);
        const auto zOffset = shape::getOffset(0, shape::shapeOf(const_cast<Nd4jLong*>(zShapeInfo)), shape::stride(const_cast<Nd4jLong*>(zShapeInfo)), coords, zRank);

        sharedMem[threadIdx.x] = 0;

          for (uint i = threadIdx.x; i < diagLen; i += blockDim.x) {

            coords[zRank] = coords[zRank + 1] = i;
            const auto xOffset = shape::getOffset(0, shape::shapeOf(const_cast<Nd4jLong*>(xShapeInfo)), shape::stride(const_cast<Nd4jLong*>(xShapeInfo)), coords, xRank);
            sharedMem[threadIdx.x] += x[xOffset];
          }

          __syncthreads();

        // aggregate sum
        for (Nd4jLong activeThreads = blockDim.x / 2; activeThreads > 0; activeThreads /= 2) {
            if (threadIdx.x < activeThreads)
                sharedMem[threadIdx.x] += sharedMem[threadIdx.x + activeThreads];
            __syncthreads();
        }

        if (threadIdx.x == 0)
            z[zOffset] = *sharedMem;
    }

}

///////////////////////////////////////////////////////////////////
template<typename T>
static void traceCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                              const void *vx, const Nd4jLong *xShapeInfo,
                                    void *vz, const Nd4jLong *zShapeInfo,
                                    const uint diagLen) {

    traceCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vz, zShapeInfo, diagLen);
}
BUILD_SINGLE_TEMPLATE(template void traceCudaLauncher, (const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream, const void* vx, const Nd4jLong* xShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const uint diagLen), LIBND4J_TYPES);

///////////////////////////////////////////////////////////////////
void trace(nd4j::LaunchContext* context, const NDArray& input, NDArray& output) {

    PointersManager manager(context, "trace");

    const uint diagLen = input.sizeAt(-1) < input.sizeAt(-2) ? input.sizeAt(-1) : input.sizeAt(-2);
    const int threadsPerBlock = MAX_NUM_THREADS / 4;
    const int blocksPerGrid = (output.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
    const int sharedMem = threadsPerBlock * (sizeof(Nd4jLong) * input.rankOf() + input.sizeOfT()) + 128;

    NDArray::prepareSpecialUse({&output}, {&input});
    BUILD_SINGLE_SELECTOR(input.dataType(), traceCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), input.getSpecialBuffer(), input.getSpecialShapeInfo(), output.specialBuffer(), output.specialShapeInfo(), diagLen), LIBND4J_TYPES);
    NDArray::registerSpecialUse({&output}, {&input});

    manager.synchronize();
}
























    //////////////////////////////////////////////////////////////////////////
    template <typename T>
    static void triuBP_(nd4j::LaunchContext * context, const NDArray& input, const NDArray& gradO, NDArray& gradI, const int diagonal) {

    }

    void triuBP(nd4j::LaunchContext * context, const NDArray& input, const NDArray& gradO, NDArray& gradI, const int diagonal) {
        BUILD_SINGLE_SELECTOR(gradO.dataType(), triuBP_, (context, input, gradO, gradI, diagonal), LIBND4J_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template void triuBP_, (nd4j::LaunchContext * context, const NDArray& input, const NDArray& gradO, NDArray& gradI, const int diagonal), LIBND4J_TYPES);

    //////////////////////////////////////////////////////////////////////////
    template <typename T>
    void randomShuffle_(nd4j::LaunchContext * context, NDArray& input, NDArray& output, nd4j::random::RandomBuffer& rng, const bool isInplace) {

    }

    void randomShuffle(nd4j::LaunchContext * context, NDArray& input, NDArray& output, nd4j::random::RandomBuffer& rng, const bool isInplace) {
        BUILD_SINGLE_SELECTOR(input.dataType(), randomShuffle_, (context, input, output, rng, isInplace), LIBND4J_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template void randomShuffle_, (nd4j::LaunchContext * context, NDArray& input, NDArray& output, nd4j::random::RandomBuffer& rng, const bool isInplace), LIBND4J_TYPES);

    ////////////////////////////////////////////////////////////////////////
    template<typename T>
    static void gatherND_(nd4j::LaunchContext * context, NDArray& input, NDArray& indices, NDArray& output) {

    }

    void gatherND(nd4j::LaunchContext * context, NDArray& input, NDArray& indices, NDArray& output) {
        BUILD_SINGLE_SELECTOR(input.dataType(), gatherND_, (context, input, indices, output), LIBND4J_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template void gatherND_, (nd4j::LaunchContext * context, NDArray& input, NDArray& indices, NDArray& output), LIBND4J_TYPES);



//////////////////////////////////////////////////////////////////////////
void eye(nd4j::LaunchContext * context, NDArray& output) {

    output.setIdentity();
}

    //////////////////////////////////////////////////////////////////////////
    void scatterUpdate(nd4j::LaunchContext * context, NDArray& operand, NDArray& updates, const std::vector<int>* intArgs) {

    }

    //////////////////////////////////////////////////////////////////////////
    template <typename T, typename Z>
    static __global__ void global_mergeMaxIndex_(void **inArrs, void **inShapes, const int numArrays, void *voutput, Nd4jLong *outputShape, Nd4jLong length) {
        auto output = reinterpret_cast<Z*>(voutput);

        const auto tid = blockIdx.x * gridDim.x + threadIdx.x;
        const auto step = gridDim.x * blockDim.x;

        for (Nd4jLong e = tid; e < length; e += step) {
            T mVal = -DataTypeUtils::max<T>();
            Z mIdx(0);

            for (int i = 0; i < numArrays; i++) {
                auto x = reinterpret_cast<T*>(inArrs[i]);
                auto xShape = reinterpret_cast<Nd4jLong *>(inShapes[i]);
                auto val = x[shape::getIndexOffset(e, xShape, length)];;
                if (mVal < val)
                    mIdx = static_cast<Z>(e);
            }
            __syncthreads();

            output[shape::getIndexOffset(e, outputShape, length)] = mIdx;
        }
    }

    template <typename T, typename Z>
    static void mergeMaxIndex_(nd4j::LaunchContext * context, const std::vector<NDArray*>& inArrs, NDArray& output) {
        std::vector<void *> inBuffers(inArrs.size());
        std::vector<void *> inShapes(inArrs.size());

        for (int e = 0; e < inArrs.size(); e++) {
            inBuffers[e] = inArrs[e]->getSpecialBuffer();
            inShapes[e] = inArrs[e]->getSpecialShapeInfo();
        }

        PointersManager manager(context, "mergeMaxIndex");

        auto pInBuffers = reinterpret_cast<void **>(manager.replicatePointer(inBuffers.data(), inBuffers.size() * sizeof(void *)));
        auto pInShapes = reinterpret_cast<void **>(manager.replicatePointer(inShapes.data(), inShapes.size() * sizeof(void *)));
        auto length = output.lengthOf();

        global_mergeMaxIndex_<T,Z><<<512, 512, 512, *context->getCudaStream()>>>(pInBuffers, pInShapes, (int) inArrs.size(), output.getSpecialBuffer(), output.getSpecialShapeInfo(), length);

        manager.synchronize();
    }

    void mergeMaxIndex(nd4j::LaunchContext * context, const std::vector<NDArray*>& inArrs, NDArray& output) {
        BUILD_DOUBLE_SELECTOR(inArrs[0]->dataType(), output.dataType(), mergeMaxIndex_, (context, inArrs, output), LIBND4J_TYPES, INTEGER_TYPES);
    }

    BUILD_DOUBLE_TEMPLATE(template void mergeMaxIndex_, (nd4j::LaunchContext * context, const std::vector<NDArray*>& inArrs, NDArray& output), LIBND4J_TYPES, INTEGER_TYPES);

    //////////////////////////////////////////////////////////////////////////
    template <typename T>
    static __global__ void global_mergeMax_(void **inArrs, void **inShapes, const int numArrays, void *voutput, Nd4jLong *outputShape, Nd4jLong length) {
        auto output = reinterpret_cast<T*>(voutput);

        const auto tid = blockIdx.x * gridDim.x + threadIdx.x;
        const auto step = gridDim.x * blockDim.x;

        for (Nd4jLong e = tid; e < length; e += step) {
            T mVal = -DataTypeUtils::max<T>();

            for (int i = 0; i < numArrays; i++) {
                auto x = reinterpret_cast<T*>(inArrs[i]);
                auto xShape = reinterpret_cast<Nd4jLong *>(inShapes[i]);
                auto val = x[shape::getIndexOffset(e, xShape, length)];;
                if (mVal < val)
                    mVal = val;
            }
            __syncthreads();

            output[shape::getIndexOffset(e, outputShape, length)] = mVal;
        }
    }

    template<typename T>
    static void mergeMax_(nd4j::LaunchContext * context, const std::vector<NDArray*>& inArrs, NDArray& output) {
        std::vector<void *> inBuffers(inArrs.size());
        std::vector<void *> inShapes(inArrs.size());

        for (int e = 0; e < inArrs.size(); e++) {
            inBuffers[e] = inArrs[e]->getSpecialBuffer();
            inShapes[e] = inArrs[e]->getSpecialShapeInfo();
        }

        PointersManager manager(context, "mergeMax");

        auto pInBuffers = reinterpret_cast<void **>(manager.replicatePointer(inBuffers.data(), inBuffers.size() * sizeof(void *)));
        auto pInShapes = reinterpret_cast<void **>(manager.replicatePointer(inShapes.data(), inShapes.size() * sizeof(void *)));
        auto length = output.lengthOf();

        global_mergeMax_<T><<<512, 512, 512, *context->getCudaStream()>>>(pInBuffers, pInShapes, (int) inArrs.size(), output.getSpecialBuffer(), output.getSpecialShapeInfo(), length);

        manager.synchronize();
    }
    BUILD_SINGLE_TEMPLATE(template void mergeMax_, (nd4j::LaunchContext * context, const std::vector<NDArray*>& inArrs, NDArray& output), LIBND4J_TYPES);

    void mergeMax(nd4j::LaunchContext * context, const std::vector<NDArray*>& inArrs, NDArray& output) {
        BUILD_SINGLE_SELECTOR(output.dataType(), mergeMax_, (context, inArrs, output), LIBND4J_TYPES);
    }

    //////////////////////////////////////////////////////////////////////////
    template <typename T>
    static __global__ void global_mergeAvg_(void **inArrs, void **inShapes, const int numArrays, void *voutput, Nd4jLong *outputShape, Nd4jLong length) {
        auto output = reinterpret_cast<T*>(voutput);

        const auto tid = blockIdx.x * gridDim.x + threadIdx.x;
        const auto step = gridDim.x * blockDim.x;

        for (Nd4jLong e = tid; e < length; e += step) {
            T sum(0.0f);

            for (int i = 0; i < numArrays; i++) {
                auto x = reinterpret_cast<T*>(inArrs[i]);
                auto xShape = reinterpret_cast<Nd4jLong *>(inShapes[i]);

                sum += x[shape::getIndexOffset(e, xShape, length)];
            }

            output[shape::getIndexOffset(e, outputShape, length)] = sum / numArrays;
        }
    }

    template<typename T>
    static void mergeAvg_(nd4j::LaunchContext * context, const std::vector<NDArray*>& inArrs, NDArray& output) {
        std::vector<void *> inBuffers(inArrs.size());
        std::vector<void *> inShapes(inArrs.size());

        for (int e = 0; e < inArrs.size(); e++) {
            inBuffers[e] = inArrs[e]->getSpecialBuffer();
            inShapes[e] = inArrs[e]->getSpecialShapeInfo();
        }

        PointersManager manager(context, "mergeAvg");

        auto pInBuffers = reinterpret_cast<void **>(manager.replicatePointer(inBuffers.data(), inBuffers.size() * sizeof(void *)));
        auto pInShapes = reinterpret_cast<void **>(manager.replicatePointer(inShapes.data(), inShapes.size() * sizeof(void *)));
        auto length = output.lengthOf();

        global_mergeAvg_<T><<<512, 512, 512, *context->getCudaStream()>>>(pInBuffers, pInShapes, (int) inArrs.size(), output.getSpecialBuffer(), output.getSpecialShapeInfo(), length);

        manager.synchronize();
    }
    BUILD_SINGLE_TEMPLATE(template void mergeAvg_, (nd4j::LaunchContext * context, const std::vector<NDArray*>& inArrs, NDArray& output), LIBND4J_TYPES);

    void mergeAvg(nd4j::LaunchContext * context, const std::vector<NDArray*>& inArrs, NDArray& output) {
        BUILD_SINGLE_SELECTOR(output.dataType(), mergeAvg_, (context, inArrs, output), LIBND4J_TYPES);
    }

    //////////////////////////////////////////////////////////////////////////
    template <typename T>
    static __global__ void global_mergeAdd_(void **inArrs, void **inShapes, const int numArrays, void *voutput, Nd4jLong *outputShape, Nd4jLong length) {
        auto output = reinterpret_cast<T*>(voutput);

        const auto tid = blockIdx.x * gridDim.x + threadIdx.x;
        const auto step = gridDim.x * blockDim.x;

        for (Nd4jLong e = tid; e < length; e += step) {
            T sum(0.0f);

            for (int i = 0; i < numArrays; i++) {
                auto x = reinterpret_cast<T*>(inArrs[i]);
                auto xShape = reinterpret_cast<Nd4jLong *>(inShapes[i]);

                sum += x[shape::getIndexOffset(e, xShape, length)];
            }

            output[shape::getIndexOffset(e, outputShape, length)] = sum;
        }
    }

    template<typename T>
    static void mergeAdd_(nd4j::LaunchContext * context, const std::vector<NDArray*>& inArrs, NDArray& output) {
        std::vector<void *> inBuffers(inArrs.size());
        std::vector<void *> inShapes(inArrs.size());

        for (int e = 0; e < inArrs.size(); e++) {
            inBuffers[e] = inArrs[e]->getSpecialBuffer();
            inShapes[e] = inArrs[e]->getSpecialShapeInfo();
        }

        PointersManager manager(context, "mergeAdd");

        auto pInBuffers = reinterpret_cast<void **>(manager.replicatePointer(inBuffers.data(), inBuffers.size() * sizeof(void *)));
        auto pInShapes = reinterpret_cast<void **>(manager.replicatePointer(inShapes.data(), inShapes.size() * sizeof(void *)));
        auto length = output.lengthOf();

        global_mergeAdd_<T><<<512, 512, 512, *context->getCudaStream()>>>(pInBuffers, pInShapes, (int) inArrs.size(), output.getSpecialBuffer(), output.getSpecialShapeInfo(), length);

        manager.synchronize();
    }
    BUILD_SINGLE_TEMPLATE(template void mergeAdd_, (nd4j::LaunchContext * context, const std::vector<NDArray*>& inArrs, NDArray& output), LIBND4J_TYPES);

    void mergeAdd(nd4j::LaunchContext * context, const std::vector<NDArray*>& inArrs, NDArray& output) {
        BUILD_SINGLE_SELECTOR(output.dataType(), mergeAdd_, (context, inArrs, output), LIBND4J_TYPES);
    }

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    template <typename T>
    static __global__ void clipByNormInplaceKernel(Nd4jLong numOfSubArrs, T* inputBuffer, Nd4jLong* shape, Nd4jLong* inputOffsets, T* norm2Buf, Nd4jLong* norm2shape, T clipNorm) {
        for (int arr = blockIdx.x; arr < numOfSubArrs; arr += gridDim.x) {
            __shared__ T* z;
            __shared__ Nd4jLong len;
            if (threadIdx.x == 0) {
                len = shape::length(shape);
                z = inputBuffer + inputOffsets[arr];
            }
            __syncthreads();
            for (int j = threadIdx.x; j < len; j+= blockDim.x) {
                auto xIndex = shape::getIndexOffset(j, shape, len);

                if(norm2Buf[arr] > clipNorm)
                z[xIndex] *= clipNorm / norm2Buf[arr]; // case with ews = 1 and ordering is 'c'
            }
        }
    }
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    template <typename T>
    static __global__ void clipByNormKernel(Nd4jLong numOfSubArrs, T* inputBuffer, Nd4jLong* shape, Nd4jLong* inputOffsets, T* outputBuffer, Nd4jLong* outputShape, Nd4jLong* outputOffsets, T* norm2Buf, Nd4jLong* norm2shape, T clipNorm) {
        for (Nd4jLong arr = blockIdx.x; arr < numOfSubArrs; arr += gridDim.x) {
            __shared__ T* x, *z;
            __shared__ Nd4jLong lenX, lenZ;
            __shared__ T norm2;

            if (threadIdx.x == 0) {
                lenX = shape::length(shape);
                x = inputBuffer + inputOffsets[arr];
                z = outputBuffer + outputOffsets[arr];
                lenZ = shape::length(outputShape);
                norm2 = norm2Buf[shape::getIndexOffset(arr, norm2shape, numOfSubArrs)];
                //printf("%d: %lf (vs %lf) %lld %lld\n", arr, norm2, clipNorm, lenX, lenZ);
            }
            __syncthreads();
            for (Nd4jLong j = threadIdx.x; j < lenZ; j+= blockDim.x) {
                auto xIndex = shape::getIndexOffset(j, shape, lenX);
                auto zIndex = shape::getIndexOffset(j, outputShape, lenZ);
                if(norm2 > clipNorm) {
                    z[zIndex] = x[xIndex] * clipNorm / norm2; // case with ews = 1 and ordering is 'c'
                } else {
                    z[zIndex] = x[xIndex];
                }
                //printf("%lld: %lf %lf\n", j, z[zIndex], x[xIndex]);
            }
            __syncthreads();
        }
    }

    //////////////////////////////////////////////////////////////////////////
    template<typename T>
    static void clipByNorm_(nd4j::LaunchContext * context, NDArray& input, NDArray& output, const std::vector<int>& dimensions, NDArray const& clipNormA, const bool isInplace) {
        const int rank = input.rankOf();
        auto norm2 = input.reduceAlongDims(reduce::Norm2, dimensions);
        clipNormA.syncToHost();
        //norm2.printBuffer("Norm2");
        T const clipNorm = clipNormA.e<T>(0);
        //clipNormA.printBuffer("ClipNorm");
        auto stream = context->getCudaStream();
        if (isInplace) {
            if(norm2.lengthOf() == 1) {
                norm2.syncToHost();
                T norm2Val = norm2.e<T>(0);
                if(norm2Val > clipNorm)
                    input *= clipNorm / norm2Val;
            }
            else {

                std::vector<int> dimsToExclude = ShapeUtils::evalDimsToExclude(rank, dimensions);
                const Nd4jLong numOfSubArrs = ShapeUtils::getNumOfSubArrs(input.getShapeInfo(), dimsToExclude);
                auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input.getShapeInfo(), dimensions);
                //auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output.getShapeInfo(), dimsToExclude);
                T* inputBuffer = reinterpret_cast<T*>(input.specialBuffer());
                T* norm2buf = reinterpret_cast<T*>(norm2.specialBuffer());

                clipByNormInplaceKernel<T><<<256, 512, 1024, *stream>>>(numOfSubArrs, inputBuffer, packX.specialShapeInfo(), packX.specialOffsets(), norm2buf, norm2.specialShapeInfo(), clipNorm);
            }
        }
        else {

            if(norm2.lengthOf() == 1) {
                norm2.syncToHost();
                T norm2Val = norm2.e<T>(0);

                if(norm2Val > clipNorm)
                    output.assign( input * (clipNorm / norm2Val));
                else
                    output.assign( input );
            }
            else {

                std::vector<int> dimsToExclude = ShapeUtils::evalDimsToExclude(rank, dimensions);
                const Nd4jLong numOfSubArrs = ShapeUtils::getNumOfSubArrs(input.getShapeInfo(), dimsToExclude);
                auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input.getShapeInfo(), dimensions);
                auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output.getShapeInfo(), dimensions);
                T* inputBuffer = reinterpret_cast<T*>(input.specialBuffer());
                T* norm2buf = reinterpret_cast<T*>(norm2.specialBuffer());
                T* outputBuffer = reinterpret_cast<T*>(output.specialBuffer());

                clipByNormKernel<T><<<256, 512, 1024, *stream>>>(numOfSubArrs, inputBuffer, packX.specialShapeInfo(), packX.specialOffsets(), outputBuffer, packZ.specialShapeInfo(), packZ.specialOffsets(), norm2buf, norm2.specialShapeInfo(), clipNorm);
            }
        }
    }

    void clipByNorm(nd4j::LaunchContext * context, NDArray& input, NDArray& output, const std::vector<int>& dimensions, const NDArray& clipNorm, const bool isInplace) {
        BUILD_SINGLE_SELECTOR(output.dataType(), clipByNorm_, (context, input, output, dimensions, clipNorm, isInplace), FLOAT_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template void clipByNorm_, (nd4j::LaunchContext * context, NDArray& input, NDArray& output, const std::vector<int>& dimensions, const NDArray& clipNorm, const bool isInplace), FLOAT_TYPES);

    template <typename T>
    static void clipByGlobalNorm_(nd4j::LaunchContext * context, std::vector<NDArray*> const& inputs, double clipNorm, nd4j::memory::Workspace* workspace, std::vector<NDArray*>& outputs, bool isInplace) {

    }

    void clipByGlobalNorm(nd4j::LaunchContext * context, std::vector<NDArray*> const& inputs, double clipNorm, nd4j::memory::Workspace* workspace, std::vector<NDArray*>& outputs, bool isInplace) {
        BUILD_SINGLE_SELECTOR(outputs[0]->dataType(), clipByGlobalNorm_, (context, inputs, clipNorm, workspace, outputs, isInplace), FLOAT_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template void clipByGlobalNorm_, (nd4j::LaunchContext * context, std::vector<NDArray*> const& inputs, double clipNorm, nd4j::memory::Workspace* workspace, std::vector<NDArray*>& outputs, bool isInplace), FLOAT_TYPES);

    //////////////////////////////////////////////////////////////////////////
    template<typename T>
    static void clipByNormBP_(nd4j::LaunchContext * context, const NDArray& input, const NDArray& gradO, NDArray& gradI /*output*/, const std::vector<int>& dimensions, const NDArray& clipNorm) {

    }

    void clipByNormBP(nd4j::LaunchContext * context, const NDArray& input, const NDArray& gradO, NDArray& gradI /*output*/, const std::vector<int>& dimensions, const NDArray& clipNorm) {
        BUILD_SINGLE_SELECTOR(gradI.dataType(), clipByNormBP_, (context, input, gradO, gradI, dimensions, clipNorm), FLOAT_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template void clipByNormBP_, (nd4j::LaunchContext * context, const NDArray& input, const NDArray& gradO, NDArray& gradI /*output*/, const std::vector<int>& dimensions, const NDArray& clipNorm), FLOAT_TYPES);


    //////////////////////////////////////////////////////////////////////////
    template<typename T>
    static void clipByAveraged_(nd4j::LaunchContext * context, NDArray& input, NDArray& output, const std::vector<int>& dimensions, const NDArray& clipNorm, const bool isInplace) {
        auto cn = clipNorm.e<T>(0);
        if (dimensions.size() == 0) {
            // all-reduce
            T n2 = input.reduceNumber(reduce::Norm2).e<T>(0) / input.lengthOf();
            if (n2 <= cn) {
                if (!isInplace)
                    output.assign(input);
            }
            else {
                const T factor = cn / n2;
                //auto lambda = LAMBDA_T(_x, factor) { return _x * factor; };
                //input.applyLambda<T>(lambda, &output);
                output.assign(input * factor);
            }
        }
        else {
            // along dimension
            auto norm2 = input.reduceAlongDims(reduce::Norm2, dimensions, false);
            if (!isInplace)
                output.assign(input);
            auto tads = output.allTensorsAlongDimension(dimensions);
            auto outTads = output.allTensorsAlongDimension(dimensions);
            // TODO: make this CUDA-compliant somehow
            for (int e = 0; e < tads->size(); e++) {
                T n2 = norm2.e<T>(e) / tads->at(e)->lengthOf();
                const T factor = cn / n2;
                if (n2 > cn) {
                    //auto lambda = LAMBDA_T(_x, factor) {return _x * factor;};
                    tads->at(e)->applyScalar(scalar::Multiply, factor, outTads->at(e));//applyLambda<T>(lambda, &output);
                }
            }
            delete tads;
            delete outTads;
        }
    }

    void clipByAveraged(nd4j::LaunchContext * context, NDArray& input, NDArray& output, const std::vector<int>& dimensions, const NDArray& clipNorm, const bool isInplace) {
        BUILD_SINGLE_SELECTOR(input.dataType(), clipByAveraged_, (context, input, output, dimensions, clipNorm, isInplace), FLOAT_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template void clipByAveraged_, (nd4j::LaunchContext * context, NDArray& input, NDArray& output, const std::vector<int>& dimensions, const NDArray& clipNorm, const bool isInplace), FLOAT_TYPES);

/*
    if (d1 > params[1])
    return params[1];
    else if (d1 < params[0])
    return params[0];
    else return d1;
*/
    template <typename T>
    static void __global__ clipByValueKernel(void* input, Nd4jLong* inputShape, void* output, Nd4jLong* outputShape, double leftBound, double rightBound) {
        __shared__ T* outputBuf;
        __shared__ T* inputBuf;
        __shared__ Nd4jLong length;
        __shared__ bool linearBuffers;
        if (threadIdx.x == 0) {
            outputBuf = reinterpret_cast<T *>(output);
            inputBuf = reinterpret_cast<T *>(input);
            length = shape::length(inputShape);
            linearBuffers = shape::elementWiseStride(inputShape) == shape::elementWiseStride(outputShape) && shape::elementWiseStride(inputShape) == 1;
        }
        __syncthreads();
        const auto tid = blockIdx.x * gridDim.x + threadIdx.x;
        const auto step = gridDim.x * blockDim.x;

        for (Nd4jLong e = tid; e < length; e += step) {
            if (linearBuffers) {
                if (inputBuf[e] > rightBound) outputBuf[e] = (T) rightBound;
                else if (inputBuf[e] < leftBound) outputBuf[e] = (T) leftBound;
                else outputBuf[e] = inputBuf[e];
            }
            else {
                auto inputOffset = shape::getIndexOffset(e, inputShape, length);
                auto outputOffset = shape::getIndexOffset(e, outputShape, length);
                if (inputBuf[inputOffset] > rightBound) outputBuf[outputOffset] = (T) rightBound;
                else if (inputBuf[inputOffset] < leftBound) outputBuf[outputOffset] = (T) leftBound;
                else outputBuf[outputOffset] = inputBuf[outputOffset];
            }
        }
    }

    template <typename T>
    static void clipByValue_(nd4j::LaunchContext * context, NDArray& input, double leftBound, double rightBound, NDArray& output) {
        auto stream = context->getCudaStream();
        if (!input.isActualOnDeviceSide())
            input.syncToDevice();
        NDArray::prepareSpecialUse({&output}, {&input});
        clipByValueKernel<T><<<256, 512, 8192, *stream>>>(input.specialBuffer(), input.specialShapeInfo(), output.specialBuffer(), output.specialShapeInfo(), leftBound, rightBound);
        NDArray::registerSpecialUse({&output}, {&input});
    }

    void clipByValue(nd4j::LaunchContext * context, NDArray& input, double leftBound, double rightBound, NDArray& output) {
        BUILD_SINGLE_SELECTOR(input.dataType(), clipByValue_, (context, input, leftBound, rightBound, output), FLOAT_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template void clipByValue_, (nd4j::LaunchContext * context, NDArray& input, double leftBound, double rightBound, NDArray& output);, FLOAT_TYPES);

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    template <typename T>
    static __global__ void mirrorPadLinearKernel(void const* vx, Nd4jLong* xShape, void* vz, Nd4jLong* zShape, Nd4jLong leftSide, Nd4jLong leftSideCorrected, Nd4jLong xLen, Nd4jLong len, Nd4jLong zLen) {

        __shared__ T const* x;
        __shared__ T* z;
        if (threadIdx.x == 0) {
            x = reinterpret_cast<T const*>(vx);
            z = reinterpret_cast<T*>(vz);
        }
        __syncthreads();
        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = blockDim.x * gridDim.x;

        for(int i = start; i < zLen; i+= step) {
            auto zIndex = shape::getIndexOffset(i, zShape, zLen);
            auto xIndex = shape::getIndexOffset(len - i, xShape, xLen);

            if (i < leftSide)                                   // left side
                xIndex = shape::getIndexOffset(leftSideCorrected - i, xShape, xLen);

            else if(i >= leftSide && i < leftSide + xLen)       // middle
                xIndex = shape::getIndexOffset(i - leftSide, xShape, xLen);

//            else                                                // right side
//                z[i] = x[len - i];
            z[zIndex] = x[xIndex];
        }

    }

    template <typename F, typename I>
    static __global__ void mirrorPadKernel(void const* vx, Nd4jLong* xShape, void* vz, Nd4jLong* zShape, Nd4jLong outLen, void const* paddings, Nd4jLong* paddingShape, int reflBorder) {

        __shared__ F const* x;
        __shared__ I const* pads;
        __shared__ F* z;
        __shared__ Nd4jLong zRank, rank;
        __shared__ Nd4jLong* xShapeOf, *xStrideOf, *padsShapeOf, *padsStrideOf;
        __shared__ Nd4jLong* zShapeOf, *zStrideOf;
        __shared__ Nd4jLong* xIdx;
        if (threadIdx.x == 0) {
            extern __shared__ unsigned char shmem[];
            xIdx    = reinterpret_cast<Nd4jLong*>(shmem);
            rank = shape::rank(xShape);

            x = reinterpret_cast<F const*>(vx);//
            pads = reinterpret_cast<I const*>(paddings);
            z = reinterpret_cast<F*>(vz);
            xShapeOf = shape::shapeOf(xShape);
            xStrideOf = shape::stride(xShape);
            zShapeOf = shape::shapeOf(zShape);
            zRank = shape::rank(zShape);
            zStrideOf = shape::stride(zShape);
            padsShapeOf = shape::shapeOf(paddingShape);
            padsStrideOf = shape::stride(paddingShape);
        }
        __syncthreads();
        auto start = threadIdx.x + blockIdx.x * blockDim.x;
        auto step = blockDim.x * gridDim.x;

            for(Nd4jLong i = start; i < outLen; i+= step) {
                auto xzCoord = xIdx + threadIdx.x * rank;
                //auto zxCoord = xIdx + (threadIdx.x + threadIdx.x % 2 + 1) * rank;

                shape::index2coords(rank, zShapeOf, i, xzCoord);
                auto outOffset = shape::getOffset(0, zShapeOf, zStrideOf, xzCoord, rank);
//                auto intStep = blockDim.y * gridDim.y;
                for(int j = 0; j < rank; j++) {

                    const Nd4jLong inLen         = shape::sizeAt(xShape, j);
                    Nd4jLong coords[2] = {j, 0};
                    auto padOffset = shape::getOffset(0, padsShapeOf, padsStrideOf, coords, 2); // padding already has rank 2
                    const auto leftSide          = pads[padOffset];
                    const auto leftSideCorrected = leftSide - reflBorder;
                    const Nd4jLong len           = 2 * (inLen - 1) + leftSide + reflBorder;

                    if(xzCoord[j] < leftSide)                                        // left side
                        xzCoord[j] = leftSideCorrected - xzCoord[j];

                    else if(xzCoord[j] >= leftSide && xzCoord[j] < leftSide + inLen)  // middle
                        xzCoord[j] = xzCoord[j] - leftSide;

                    else if (len > xzCoord[j])                                                           // right side
                        xzCoord[j] = len - xzCoord[j];
                    else
                        xzCoord[j] = xzCoord[j] - len;
                }

                auto inOffset  = shape::getOffset(0, xShapeOf, xStrideOf,  xzCoord,  rank);
                z[outOffset] = x[inOffset];
            }
    }

    template<typename F, typename I>
    static void mirrorPad_(nd4j::LaunchContext * context, const NDArray& input, const NDArray& paddings, NDArray& output, const int mode) {
        // mode:  0 - REFLECT, else - SYMMETRIC
        const int reflBorder = (bool)mode ? 1 : 0;
        const int rank        = input.rankOf();
        const Nd4jLong outLen = output.lengthOf();
        auto stream = context->getCudaStream();
        NDArray::prepareSpecialUse({&output}, {&input, &paddings});

        if(rank <= 1) {

            const Nd4jLong inLen         = input.lengthOf();
            const auto leftSide          = paddings.e<Nd4jLong>(0);
            const auto leftSideCorrected = leftSide - reflBorder;
            const Nd4jLong len           = 2*(inLen-1) + leftSide + reflBorder;

            mirrorPadLinearKernel<F><<<256, 512, 256, *stream>>>(input.getSpecialBuffer(), input.getSpecialShapeInfo(), output.specialBuffer(), output.specialShapeInfo(), leftSide, leftSideCorrected, inLen, len, outLen);
            nd4j::DebugHelper::checkErrorCode(stream, "helpers::mirrorPadLinearKernel(...) failed");
        }
        else {
            mirrorPadKernel<F, I><<<256, 256, 8192, *stream>>>(input.getSpecialBuffer(), input.getSpecialShapeInfo(), output.specialBuffer(), output.specialShapeInfo(), outLen, paddings.getSpecialBuffer(), paddings.getSpecialShapeInfo(), reflBorder);
            nd4j::DebugHelper::checkErrorCode(stream, "helpers::mirrorPadKernel(...) failed");
        }
        NDArray::registerSpecialUse({&output}, {&input, &paddings});
    }

    void mirrorPad(nd4j::LaunchContext * context, const NDArray& input, const NDArray& paddings, NDArray& output, const int mode) {
        BUILD_DOUBLE_SELECTOR(input.dataType(), paddings.dataType(), mirrorPad_, (context, input, paddings, output, mode), LIBND4J_TYPES, INTEGER_TYPES);
    }

    BUILD_DOUBLE_TEMPLATE(template void mirrorPad_, (nd4j::LaunchContext * context, const NDArray& input, const NDArray& paddings, NDArray& output, const int mode), LIBND4J_TYPES, INTEGER_TYPES);

//////////////////////////////////////////////////////////////////////////
void concat(nd4j::LaunchContext * context, const std::vector<NDArray*>& inArrs, NDArray& output, const int axis) {

    const int numOfArrs = inArrs.size();
    for(int i = 0; i < numOfArrs; ++i)
        if(!inArrs[i]->isActualOnDeviceSide()) inArrs[i]->syncToDevice();

    const int rank  = inArrs[0]->rankOf();
    const int rank2 = 2*rank;
    std::vector<std::vector<Nd4jLong>> indices(numOfArrs, std::vector<Nd4jLong>(rank2,0));

    // take into account indices for first array
    indices[0][2 * axis + 1] = inArrs[0]->sizeAt(axis);

    // loop through the rest of input arrays
    for(int i = 1; i < numOfArrs; ++i) {
        indices[i][2 * axis]     = indices[i-1][2 * axis + 1];                                // index start from
        indices[i][2 * axis + 1] = indices[i-1][2 * axis + 1] + inArrs[i]->sizeAt(axis);      // index end with (excluding)
    }

    std::vector<NDArray*> outSubArrs(numOfArrs);
    for(int i = 0; i < numOfArrs; ++i)
        outSubArrs[i] = new NDArray(output(indices[i], true));

    // prepare arrays of pointers on buffers and shapes
    std::vector<void*>     hOutBuffers(numOfArrs), hInBuffers(numOfArrs);
    std::vector<Nd4jLong*> hOutShapeInfo(numOfArrs), hInShapeInfo(numOfArrs);
    for(int i = 0; i < numOfArrs; ++i) {
        hOutBuffers[i]   = outSubArrs[i]->getSpecialBuffer();
        hInBuffers[i]    =     inArrs[i]->getSpecialBuffer();
        hOutShapeInfo[i] = outSubArrs[i]->getSpecialShapeInfo();
        hInShapeInfo[i]  =     inArrs[i]->getSpecialShapeInfo();
    }

    // allocate and copy all buffers and shapes arrays to global memory
    PointersManager manager(context, "helpers::concat");
    void* dOutBuffers	= manager.replicatePointer(hOutBuffers.data(),   hOutBuffers.size() * sizeof(void*));
    void* dInBuffers	= manager.replicatePointer(hInBuffers.data(),    hInBuffers.size() * sizeof(void*));
    void* dInShapeInfo  = manager.replicatePointer(hInShapeInfo.data(),  hInShapeInfo.size() * sizeof(Nd4jLong*));
    void* dOutShapeInfo = manager.replicatePointer(hOutShapeInfo.data(), hOutShapeInfo.size() * sizeof(Nd4jLong*));

    BUILD_SINGLE_SELECTOR(inArrs[0]->dataType(), concatCudaLauncher, (numOfArrs, context->getCudaStream(), dInBuffers, dInShapeInfo, dOutBuffers, dOutShapeInfo), LIBND4J_TYPES);

    manager.synchronize();

    for(int i = 0; i < numOfArrs; ++i)
        delete outSubArrs[i];

    for(int i = 0; i < numOfArrs; ++i)
        inArrs[i]->tickReadHost();

    output.tickWriteDevice();
}




    //////////////////////////////////////////////////////////////////////////
    template <typename T>
    static void tileBP_(nd4j::LaunchContext * context, const NDArray& gradO /*input*/, NDArray& gradI /*output*/, const std::vector<Nd4jLong> reps) {

    }

    void tileBP(nd4j::LaunchContext * context, const NDArray& gradO /*input*/, NDArray& gradI /*output*/, const std::vector<Nd4jLong> reps) {
        BUILD_SINGLE_SELECTOR(gradI.dataType(), tileBP_, (context, gradO, gradI, reps), FLOAT_TYPES);
    }


    BUILD_SINGLE_TEMPLATE(template void tileBP_, (nd4j::LaunchContext * context, const NDArray& gradO /*input*/, NDArray& gradI /*output*/, const std::vector<Nd4jLong> reps), FLOAT_TYPES);

    void scatterSimple(const int opId, NDArray& input, const NDArray& updates, const NDArray& indices, const std::vector<int>& dimensions) {

    }


BUILD_SINGLE_TEMPLATE(template void concatCudaLauncher,  (const int numOfArrs, const hipStream_t *stream, void* pVx, void* pxShapeInfo, void* pVz, void* pzShapeInfo), LIBND4J_TYPES);
BUILD_DOUBLE_TEMPLATE(template void padCudaLauncher,     (const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream, const int mode, const void *vx, const Nd4jLong *xShapeInfo, const void *vy, const Nd4jLong *yShapeInfo, void *vz, const Nd4jLong *zShapeInfo, const void* vPadVal), LIBND4J_TYPES, INTEGER_TYPES);

}
}
}

