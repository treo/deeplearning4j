#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com)
//

#include <ops/declarable/helpers/top_k.h>
#include <PointersManager.h>
#include <ConstantTadHelper.h>

namespace nd4j    {
namespace ops     {
namespace helpers {

//////////////////////////////////////////////////////////////////////////
template<typename X, typename Y>
__global__ static void inTopKCuda(const void* vx, const Nd4jLong* xShapeInfo,
                                  const void* vy, const Nd4jLong* yShapeInfo,
                                        void* vz, const Nd4jLong* zShapeInfo,
                                  const Nd4jLong* xTadShapeInfo, const Nd4jLong* xTadOffsets,
                                  const uint k) {


    const auto y = reinterpret_cast<const Y*>(vy);
          auto z = reinterpret_cast<bool*>(vz);

    __shared__ uint* sharedMem;
    __shared__ X elemToCompare;
    __shared__ const X* xTad;
    __shared__ Nd4jLong idx, xTadLen;

    if (threadIdx.x == 0) {
        extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<uint*>(shmem);

        xTadLen = shape::length(xTadShapeInfo);

        xTad = reinterpret_cast<const X*>(vx) + xTadOffsets[blockIdx.x];
        idx = y[shape::getIndexOffset(blockIdx.x, yShapeInfo, shape::length(yShapeInfo))]; // shape::length(yShapeInfo) == numTads
        elemToCompare = xTad[shape::getIndexOffset(idx, xTadShapeInfo, xTadLen)];
    }

    __syncthreads();

    sharedMem[threadIdx.x] = 0;
    for (Nd4jLong i = threadIdx.x; i < xTadLen; i += blockDim.x)
        if(elemToCompare < xTad[shape::getIndexOffset(i, xTadShapeInfo, xTadLen)])
            ++sharedMem[threadIdx.x];

    __syncthreads();

    // aggregate sum
    for (uint activeThreads = blockDim.x / 2; activeThreads > 0; activeThreads /= 2) {
        if (threadIdx.x < activeThreads)
            sharedMem[threadIdx.x] += sharedMem[threadIdx.x + activeThreads];
        __syncthreads();
    }

    if (threadIdx.x == 0)
        z[shape::getIndexOffset(blockIdx.x, zShapeInfo, shape::length(zShapeInfo))] = *sharedMem < k;
}

///////////////////////////////////////////////////////////////////
template<typename X, typename Y>
static void inTopKCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                               const void *vx, const Nd4jLong *xShapeInfo,
                               const void *vy, const Nd4jLong *yShapeInfo,
                                     void *vz, const Nd4jLong *zShapeInfo,
                               const Nd4jLong* xTadShapeInfo, const Nd4jLong* xTadOffsets,
                               const uint k) {

    inTopKCuda<X,Y><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vy, yShapeInfo, vz, zShapeInfo, xTadShapeInfo, xTadOffsets, k);
}

///////////////////////////////////////////////////////////////////
int inTopKFunctor(nd4j::LaunchContext * context, const NDArray* predictions, const NDArray* targets, NDArray* output, const uint k) {

    PointersManager manager(context, "in_top_k");

    const auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(predictions->getShapeInfo(), {1});

    const int threadsPerBlock = MAX_NUM_THREADS;
    const int blocksPerGrid = static_cast<int>(packX.numberOfTads());
    const int sharedMem = sizeof(uint) * threadsPerBlock + 128;

    const auto xType = predictions->dataType();
    const auto yType = targets->dataType();

    NDArray::prepareSpecialUse({output}, {predictions, targets});
    BUILD_DOUBLE_SELECTOR(xType, yType, inTopKCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), predictions->getSpecialBuffer(), predictions->getSpecialShapeInfo(), targets->getSpecialBuffer(), targets->getSpecialShapeInfo(), output->getSpecialBuffer(), output->getSpecialShapeInfo(), packX.specialShapeInfo(), packX.specialOffsets(), k), FLOAT_TYPES, INTEGER_TYPES);
    NDArray::registerSpecialUse({output}, {predictions, targets});

    manager.synchronize();

    return Status::OK();
}

    template <typename X, typename Y>
    static _CUDA_G void topValuesMover(void *vx, Nd4jLong *xTadShapeInfo, Nd4jLong *xTadOffsets, void *vi, Nd4jLong *iTadShapeInfo, Nd4jLong *iTadOffsets, void *vz, Nd4jLong *zTadShapeInfo, Nd4jLong *zTadOffsets, Nd4jLong tadLength, int numTads, int k) {
        for (int t = blockIdx.x; t < numTads; t += gridDim.x) {
            auto x = reinterpret_cast<X*>(vx) + xTadOffsets[t];
            auto i = reinterpret_cast<Y*>(vi) + iTadOffsets[t];
            auto z = reinterpret_cast<X*>(vz) + zTadOffsets[t];

            for (int e = threadIdx.x; e < k; e += blockDim.x) {
                auto idx = i[shape::getIndexOffset(e, iTadShapeInfo, k)];

                z[shape::getIndexOffset(e, zTadShapeInfo, k)] = x[shape::getIndexOffset(idx, xTadShapeInfo, tadLength)];
            }
        }
    }


    template <typename X, typename Y>
    static _CUDA_G void indicesAlongDimension(void *vx, Nd4jLong *xTadShapeInfo, Nd4jLong *xTadOffsets, void *vi, Nd4jLong *iTadShapeInfo, Nd4jLong *iTadOffsets, void *vz, Nd4jLong *zTadShapeInfo, Nd4jLong *zTadOffsets, Nd4jLong tadLength, int numTads, int k, int scanWidth) {
        extern __shared__ char _shmem[];

        X* tempValues = reinterpret_cast<X*>(_shmem) + threadIdx.x * scanWidth;
        Y* tempIndices = reinterpret_cast<Y*>(reinterpret_cast<X*>(_shmem) + blockDim.x * scanWidth) + threadIdx.x * scanWidth;

        __shared__ X localMaximum;
        if (threadIdx.x == 0)
            localMaximum = -DataTypeUtils::max<X>();
        __syncthreads();

        for (int t = blockIdx.x; t < numTads; t += gridDim.x) {
            auto x = reinterpret_cast<X *>(vx) + xTadOffsets[t];
            auto i = reinterpret_cast<Y *>(vi) + iTadOffsets[t];
            auto z = reinterpret_cast<X *>(vz) + zTadOffsets[t];

            // we'll do multiple reads here
            for (int p = 0; p < k; p += scanWidth) {

                // resetting temporary storage
                for (int p = 0; p < scanWidth; p++) {
                    tempValues[p] = -DataTypeUtils::max<X>();
                    tempIndices[p] = DataTypeUtils::max<Y>();
                }

                // local max values/indices
                for (int e = threadIdx.x; e < tadLength; e++) {
                    auto value = x[shape::getIndexOffset(e, xTadShapeInfo, tadLength)];

                    // we'll compare this value to current stored ones
                    for (int f = 0; f < scanWidth; f++) {
                        if (value > tempValues[f] && (p == 0 || value < localMaximum)) {
                            tempValues[f] = value;
                            tempIndices[f] = e;
                        }
                    }
                }
                __syncthreads();

                // at this point we have local part ready for merge and define global maximum for this iteration, and local maximum for next iteration
                for (uint activeThreads = blockDim.x / 2; activeThreads > 0; activeThreads /= 2) {
                    if (threadIdx.x < activeThreads) {
                        if (tempValues[0] < tempValues[0 + activeThreads * scanWidth]) {
                            tempValues[0] = tempValues[0 + activeThreads * scanWidth];
                            tempIndices[0] = tempIndices[0 + activeThreads * scanWidth];
                        }
                    }
                    __syncthreads();
                }
                __syncthreads();

                // at this point we know local minimum for next iteration
                if (threadIdx.x == 0) {
                    localMaximum = tempValues[scanWidth - 1];
                    z[shape::getIndexOffset(p, zTadShapeInfo, k)] = tempValues[scanWidth - 1];
                    i[shape::getIndexOffset(p, iTadShapeInfo, k)] = tempIndices[scanWidth - 1];
                }
                __syncthreads();
            }
        }
    }


    template <typename X, typename Y>
    static int topKFunctor_(nd4j::LaunchContext * context, const NDArray* input, NDArray* values, NDArray* indices, const uint k, bool needSort) {

        auto packX = ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), {input->rankOf() - 1});
        auto packI = ConstantTadHelper::getInstance()->tadForDimensions(indices->shapeInfo(), {input->rankOf() - 1});
        auto packZ = ConstantTadHelper::getInstance()->tadForDimensions(values->shapeInfo(), {input->rankOf() - 1});

        auto tadLength = shape::length(packX.primaryShapeInfo());

        // we get top K values first
        if (k == 1) {
            input->applyIndexReduce(indexreduce::IndexMax, indices, {input->rankOf() - 1});

            // copy values on specified indices
            topValuesMover<X,Y><<<256, 256, 1024, *context->getCudaStream()>>>(input->getSpecialBuffer(), packX.platformShapeInfo(), packX.platformOffsets(), indices->specialBuffer(), packI.platformShapeInfo(), packI.platformOffsets(), values->specialBuffer(), packZ.platformShapeInfo(), packZ.platformOffsets(), tadLength, packX.numberOfTads(), k);
        } else {
            int scanWidth = 1;
            int numTreads = 256;
            int shMemSize = (numTreads * sizeof(X) * scanWidth) + (numTreads * sizeof(Y) * scanWidth) + 512;

            indicesAlongDimension<X,Y><<<256, numTreads, shMemSize, *context->getCudaStream()>>>(input->getSpecialBuffer(), packX.platformShapeInfo(), packX.platformOffsets(), indices->specialBuffer(), packI.platformShapeInfo(), packI.platformOffsets(), values->specialBuffer(), packZ.platformShapeInfo(), packZ.platformOffsets(), tadLength, packX.numberOfTads(), k, scanWidth);
        }


        // optional sort
        if (k > 1 && needSort) {
            //
        }

        return Status::OK();
    }

    int topKFunctor(nd4j::LaunchContext * context, const NDArray* input, NDArray* values, NDArray* indices, const uint k, bool needSort) {
        input->syncToDevice();

        BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), topKFunctor_, (context, input, values, indices, k, needSort), LIBND4J_TYPES, INTEGER_TYPES);

        values->tickWriteDevice();
        indices->tickWriteDevice();

        return Status::OK();
    }


    BUILD_DOUBLE_TEMPLATE(template int topKFunctor_, (nd4j::LaunchContext * context, const NDArray* input, NDArray* values, NDArray* indices, const uint k, bool needSort), LIBND4J_TYPES, INTEGER_TYPES);

}
}
}