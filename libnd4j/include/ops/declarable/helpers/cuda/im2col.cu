#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// Created by raver119 on 30.11.17.
//

#include <ops/declarable/helpers/im2col.h>
#include <PointersManager.h>

namespace nd4j {
namespace ops {
namespace helpers {


//////////////////////////////////////////////////////////////////////////
// input [bS, iC, iH, iW] is convoluted to output [bS, iC, kH, kW, oH, oW]
template <typename T>
__global__ static void im2colCuda(const void *image, void *columns,
                                  const Nd4jLong *imShapeInfo, const Nd4jLong *colShapeInfo,
                                  const int sH, const int sW,
                                  const int pH, const int pW,
                                  const int dH, const int dW,
                                  const double zeroPadValD) {

    T zeroPadVal = static_cast<T>(zeroPadValD); //Value to use when value is padding. Usually 0 but not always
    const auto im  = reinterpret_cast<const T*>(image);
          auto col = reinterpret_cast<T*>(columns);

    __shared__ Nd4jLong colLen, *sharedMem, iH, iW;
    __shared__ int imRank, colRank;

    if (threadIdx.x == 0) {

        extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<Nd4jLong*>(shmem);

        colRank = 6;
        imRank  = 4;

        colLen = shape::length(colShapeInfo);

        iH = imShapeInfo[3];
        iW = imShapeInfo[4];
    }

    __syncthreads();

    const auto colInd = threadIdx.x + blockIdx.x * blockDim.x;

    if(colInd >= colLen)
        return;

    auto coords = sharedMem + threadIdx.x * colRank;

    shape::index2coords(colRank, colShapeInfo + 1, colInd, colLen, coords);

    const auto colOffset = shape::getOffset(0, colShapeInfo + 1, colShapeInfo + colRank + 1, coords, colRank);

    coords[2] = (-pH + coords[2] * dH) + coords[4] * sH;   // imH
    coords[3] = (-pW + coords[3] * dW) + coords[5] * sW;   // imW

    if (static_cast<unsigned>(coords[2]) >= static_cast<unsigned>(iH) || static_cast<unsigned>(coords[3]) >= static_cast<unsigned>(iW))
        col[colOffset] = zeroPadVal;
    else
        col[colOffset] = im[shape::getOffset(0, imShapeInfo + 1, imShapeInfo + imRank + 1, coords, imRank)];
}


//////////////////////////////////////////////////////////////////////////
template <typename T>
static void im2colCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, nd4j::LaunchContext & context, const void *image, void *columns, const Nd4jLong *imShapeInfo, const Nd4jLong *colShapeInfo, int sH, int sW, int pH, int pW, int dH, int dW, double zeroPadVal) {
    im2colCuda<T><<<blocksPerGrid, threadsPerBlock, threadsPerBlock * sizeof(Nd4jLong) * 6 /* rank of columns = 6 */, *context.getCudaStream()>>>(image, columns, imShapeInfo, colShapeInfo, sH, sW, pH, pW, dH, dW, zeroPadVal);
}
BUILD_SINGLE_TEMPLATE(template void im2colCudaLauncher, (const int blocksPerGrid, const int threadsPerBlock, nd4j::LaunchContext& context, const void *image, void *columns, const Nd4jLong *imShapeInfo, const Nd4jLong *colShapeInfo, const int sH, const int sW, const int pH, const int pW, const int dH, const int dW, const double zeroPadVal), FLOAT_TYPES);

//////////////////////////////////////////////////////////////////////////
void im2col(nd4j::LaunchContext& context, const NDArray& image, NDArray& columns, const int kH, const int kW, const int sH, const int sW, const int pH, const int pW, const int dH, const int dW, const NDArray& arrZeroPadVal) {

    PointersManager manager(&context, "im2col");

    const int threadsPerBlock = 512;
    const int blocksPerGrid = (columns.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;

    NDArray::prepareSpecialUse({&columns}, {&image});
    BUILD_SINGLE_SELECTOR(columns.dataType(), im2colCudaLauncher, (blocksPerGrid, threadsPerBlock, context, image.getSpecialBuffer(), columns.getSpecialBuffer(), image.getSpecialShapeInfo(), columns.getSpecialShapeInfo(), sH, sW, pH, pW, dH, dW, arrZeroPadVal.e<double>(0)), FLOAT_TYPES);
    NDArray::registerSpecialUse({&columns}, {&image});

    manager.synchronize();
}





}
}
}