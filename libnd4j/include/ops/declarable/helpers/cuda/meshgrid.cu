#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
//


#include<ops/declarable/helpers/meshgrid.h>
#include <helpers/PointersManager.h>
#include <helpers/ConstantTadHelper.h>
#include <array/ResultSet.h>
#include <numeric>

namespace nd4j 	  {
namespace ops 	  {
namespace helpers {

    template <typename T>
    static _CUDA_D void assign_(void *vx, Nd4jLong *xShapeInfo, void *vz, Nd4jLong *zShapeInfo) {
        auto x = reinterpret_cast<T*>(vx);
        auto z = reinterpret_cast<T*>(vz);

        auto tid = threadIdx.x + blockIdx.x * blockDim.x;

        auto xEws = shape::elementWiseStride(xShapeInfo);
        auto zEws = shape::elementWiseStride(zShapeInfo);

        auto xOrder = shape::order(xShapeInfo);
        auto zOrder = shape::order(zShapeInfo);

        __shared__ Nd4jLong length;

        if (threadIdx.x == 0) {
            length = shape::length(xShapeInfo);
        }
        __syncthreads();

        if (xEws > 0 && zEws > 0 && xOrder == zOrder) {
            for (int i = threadIdx.x; i < length; i += blockDim.x) {
                z[i * zEws] = x[i * xEws];
            }
        } else {
            for (int i = threadIdx.x; i < length; i += blockDim.x) {
                auto xOffset = shape::getIndexOffset(i, xShapeInfo, length);
                auto zOffset = shape::getIndexOffset(i, zShapeInfo, length);

                z[zOffset] = x[xOffset];
            }
        }

    }

    template <typename T>
    static _CUDA_G void meshgridKernel(int rank, void **outBuffers, Nd4jLong **tadShapes, Nd4jLong **tadOffsets, Nd4jLong *numTads, void **inBuffers, Nd4jLong **inShapes) {
        // for all arrays
        for (int i = blockIdx.x; i < rank; i += gridDim.x) {

            // for all tads in this array
            for(Nd4jLong j = 0; j < numTads[i]; j++) {
                assign_<T>(inBuffers[i], inShapes[i], reinterpret_cast<T*>(outBuffers[i]) + tadOffsets[i][j], tadShapes[i]);
            }
            __syncthreads();
        }
    }

    template <typename T>
    static void meshgrid_(nd4j::LaunchContext * context, const std::vector<NDArray*>& inArrs, const std::vector<NDArray*>& outArrs, const bool swapFirst2Dims) {
        const int rank = inArrs.size();
        int inIndices[MAX_RANK];
        std::iota(inIndices, inIndices + rank, 0);
        if(swapFirst2Dims && rank > 1) {
            inIndices[0] = 1;
            inIndices[1] = 0;
        }

        PointersManager pm(context, "meshgrid");
        std::vector<void *> hInBuffers(rank);
        std::vector<void *> hOutBuffers(rank);
        std::vector<Nd4jLong *> hInShapes(rank);

        std::vector<Nd4jLong *> hOutTadShapes(rank);
        std::vector<Nd4jLong *> hOutTadOffsets(rank);

        std::vector<Nd4jLong> hNumTads(rank);

        for(int i = 0; i < rank; ++i) {
            hInBuffers[i] = inArrs[i]->specialBuffer();
            hInShapes[i] = inArrs[i]->specialShapeInfo();

            hOutBuffers[i] = outArrs[i]->specialBuffer();


            auto pack = ConstantTadHelper::getInstance()->tadForDimensions(outArrs[i]->shapeInfo(), {inIndices[i]});
            hOutTadShapes[i] = pack.specialShapeInfo();
            hOutTadOffsets[i] = pack.specialOffsets();
            hNumTads[i] = pack.numberOfTads();


            //auto list = outArrs[i]->allTensorsAlongDimension({inIndices[i]});
            //for(int j = 0; j < list->size(); ++j)
            //    list->at(j)->assign(inArrs[i]);

            //delete list;
        }

        auto dInBuffers = reinterpret_cast<void **>(pm.replicatePointer(hInBuffers.data(), hInBuffers.size() * sizeof(void *)));
        auto dOutBuffers = reinterpret_cast<void **>(pm.replicatePointer(hOutBuffers.data(), hOutBuffers.size() * sizeof(void *)));


        auto dInShapes = reinterpret_cast<Nd4jLong **>(pm.replicatePointer(hInShapes.data(), hInShapes.size() * sizeof(Nd4jLong *)));
        auto dOutTadShapes = reinterpret_cast<Nd4jLong **>(pm.replicatePointer(hOutTadShapes.data(), hOutTadShapes.size() * sizeof(Nd4jLong *)));
        auto dOutTadOffsets = reinterpret_cast<Nd4jLong **>(pm.replicatePointer(hOutTadOffsets.data(), hOutTadOffsets.size() * sizeof(Nd4jLong *)));

        auto dNumTads = reinterpret_cast<Nd4jLong *>(pm.replicatePointer(hNumTads.data(), hNumTads.size() * sizeof(Nd4jLong)));


        meshgridKernel<T><<<256, 256, 1024, *context->getCudaStream()>>>(rank, dOutBuffers, dOutTadShapes, dOutTadOffsets, dNumTads, dInBuffers, dInShapes);

        pm.synchronize();
    }

    //////////////////////////////////////////////////////////////////////////
    void meshgrid(nd4j::LaunchContext * context, const std::vector<NDArray*>& inArrs, const std::vector<NDArray*>& outArrs, const bool swapFirst2Dims) {

        BUILD_SINGLE_SELECTOR(inArrs.at(0)->dataType(), meshgrid_, (context, inArrs, outArrs, swapFirst2Dims), LIBND4J_TYPES);

        for (auto v:outArrs)
            v->tickWriteDevice();
    }

}
}
}

