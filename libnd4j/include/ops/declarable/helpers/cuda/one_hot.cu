#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com), created on 30.05.2019
//


#include <ops/declarable/helpers/one_hot.h>
#include <array/ResultSet.h>
#include <helpers/ShapeUtils.h>
#include <numeric>
#include <NDArrayFactory.h>
#include <helpers/TAD.h>
#include <exceptions/cuda_exception.h>
#include <PointersManager.h>
#include <ConstantTadHelper.h>


namespace nd4j 		{
namespace ops		{
namespace helpers 	{

///////////////////////////////////////////////////////////////////
// x - indices, z - output
template<typename X, typename Z>
__global__ static void onehotCuda(const void *vx, const Nd4jLong *xShapeInfo, void *vz, const Nd4jLong *zShapeInfo, const uint axis, const uint depth, const Z on, const Z off) {

    const auto x = reinterpret_cast<const X*>(vx);
          auto z = reinterpret_cast<Z*>(vz);

    __shared__ int xRank, zRank;
    __shared__ Nd4jLong zLen, totalThreads, *sharedMem;

    if (threadIdx.x == 0) {

        extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<Nd4jLong*>(shmem);
        xRank = shape::rank(xShapeInfo);
        zRank = shape::rank(zShapeInfo);
        zLen  = shape::length(zShapeInfo);
        totalThreads = gridDim.x * blockDim.x;
    }

    auto coord = sharedMem + threadIdx.x * zRank;

    __syncthreads();

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (Nd4jLong i = tid; i < zLen; i += totalThreads) {

        shape::index2coords(zRank, shape::shapeOf(const_cast<Nd4jLong*>(zShapeInfo)), i, zLen, coord);
        const auto zOffset = shape::getOffset(0, shape::shapeOf(const_cast<Nd4jLong*>(zShapeInfo)), shape::stride(const_cast<Nd4jLong*>(zShapeInfo)), coord, zRank);
        const auto depthCoord = coord[axis];

        for (uint j = axis; j < zRank - 1; ++j)
          coord[j] = coord[j + 1];

        const auto xOffset = shape::getOffset(0, shape::shapeOf(const_cast<Nd4jLong*>(xShapeInfo)), shape::stride(const_cast<Nd4jLong*>(xShapeInfo)), coord, xRank);
        const Nd4jLong idx = x[xOffset];
        z[zOffset] = depthCoord == idx ? on : off;
    }
}

///////////////////////////////////////////////////////////////////
template<typename X, typename Y>
static void onehotCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                               const void *vx, const Nd4jLong *xShapeInfo,
                                     void *vz, const Nd4jLong *zShapeInfo,
                               const uint axis, const uint depth,
                               const double on, const double off) {

    onehotCuda<X,Y><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vz, zShapeInfo, axis, depth, static_cast<Y>(on), static_cast<Y>(off));
}

///////////////////////////////////////////////////////////////////
void onehot(const nd4j::LaunchContext* context, const NDArray *indices, NDArray *output, const uint axis, const uint depth, const double on, const double off) {

	const auto xType = indices->dataType();
	const auto zType = output->dataType();

	const int threadsPerBlock = MAX_NUM_THREADS / 4;
    const int blocksPerGrid = (output->lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
	const int sharedMem = threadsPerBlock * sizeof(decltype(*output->getShapeInfo())) * output->rankOf() + 128;

	PointersManager manager(context, "onehot");

    NDArray::prepareSpecialUse({output}, {indices});
  	BUILD_DOUBLE_SELECTOR(xType, zType, onehotCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), indices->getSpecialBuffer(), indices->getSpecialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(), axis, depth, on, off), LIBND4J_TYPES, LIBND4J_TYPES);
  	NDArray::registerSpecialUse({output}, {indices});

    manager.synchronize();
}


}
}
}