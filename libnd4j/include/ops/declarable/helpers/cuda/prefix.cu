#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com), created on 12.06.2019
//

#include <ops/ops.h>
#include <ConstantTadHelper.h>
#include <PointersManager.h>
#include <ShapeUtils.h>
#include <ops/declarable/helpers/prefix.h>

namespace nd4j {
namespace ops {
namespace helpers {

///////////////////////////////////////////////////////////////////
template <typename T>
__global__ static void prefixPerBlockCuda(scalar::Ops op,
                                        const void* vx, const Nd4jLong* xTadShapeInfo, const Nd4jLong* xTadOffsets,
                                              void* vz, const Nd4jLong* zTadShapeInfo, const Nd4jLong* zTadOffsets,
                                        const Nd4jLong numTads, const Nd4jLong tadLen,
                                        const bool exclusive, const bool reverse) {

    __shared__ T *shared, lastElemInChunk;
    __shared__ uint numTadChunks, blockDim2;

    if (threadIdx.x == 0) {
        extern __shared__ unsigned char shmem[];
        shared       = reinterpret_cast<T*>(shmem);
        blockDim2    = 2 * blockDim.x;
        numTadChunks = (tadLen +  blockDim2 - 1) / blockDim2;  // ceil
    }
    __syncthreads();

    const auto xTad = reinterpret_cast<const T*>(vx) + xTadOffsets[blockIdx.x];
          auto zTad = reinterpret_cast<T*>(vz)       + zTadOffsets[blockIdx.x];

    Nd4jLong sharedInd(2 * threadIdx.x), leftArrInd, rightArrInd, step;
    T xLeft, xRight;

    for (uint i = 0; i < numTadChunks; ++i) {

        leftArrInd  = sharedInd + i * blockDim2;
        rightArrInd = leftArrInd + 1;

        if(reverse) {
            if(rightArrInd < tadLen) {
                rightArrInd = tadLen - 1 - rightArrInd;
                leftArrInd  = tadLen - 1 - leftArrInd;
            }
            else if(leftArrInd < tadLen)
                leftArrInd = tadLen - 1 - leftArrInd;
        }

        if(leftArrInd < tadLen)
            shared[sharedInd] = xLeft = xTad[shape::getIndexOffset(leftArrInd, xTadShapeInfo, tadLen)];
        // else
        //     shared[sharedInd] = (op == scalar::Add) ? 0 : 1;

        if(rightArrInd < tadLen)
            shared[sharedInd + 1] = xRight = xTad[shape::getIndexOffset(rightArrInd, xTadShapeInfo, tadLen)];
        // else
        //     shared[sharedInd + 1] = (op == scalar::Add) ? 0 : 1;


        step = 1;

        for (uint d = blockDim.x; d > 0; d /= 2) {

            __syncthreads();
            if(threadIdx.x < d) {
                uint left  = step * (sharedInd + 1) - 1;
                uint right = step * (sharedInd + 2) - 1;
                shared[right] = (op == scalar::Add) ? (shared[right] + shared[left]) : (shared[right] * shared[left]);
            }
            step *= 2;
        }

        if (threadIdx.x == 0)
            shared[blockDim2 - 1] = (op == scalar::Add) ? 0 : 1;

        for (uint d = 1; d < blockDim2; d *= 2) {

            step /= 2;

            __syncthreads();
            if(threadIdx.x < d) {
                uint left  = step * (sharedInd + 1) - 1;
                uint right = step * (sharedInd + 2) - 1;
                T temp = shared[left];
                shared[left]  = shared[right];
                shared[right] = (op == scalar::Add) ? (shared[right] + temp) : (shared[right] * temp);
            }
        }

        __syncthreads();

        if(leftArrInd < tadLen) {
            T result = shared[sharedInd];
            if(!exclusive)
                result = (op == scalar::Add) ? result + xLeft : result * xLeft;
            if(i > 0)
                result = (op == scalar::Add) ? result + lastElemInChunk : result * lastElemInChunk;
            zTad[shape::getIndexOffset(leftArrInd, zTadShapeInfo, tadLen)] = result;
        }

        if(rightArrInd < tadLen) {
            T result = shared[sharedInd + 1];
            if(!exclusive)
                result = (op == scalar::Add) ? result + xRight : result * xRight;
            if(i > 0)
                result = (op == scalar::Add) ? result + lastElemInChunk : result * lastElemInChunk;
            if(i < numTadChunks - 1 && threadIdx.x == blockDim.x - 1)    // last element in chunk
                lastElemInChunk = !exclusive ? result : (op == scalar::Add) ? result + xRight : result * xRight;
            zTad[shape::getIndexOffset(rightArrInd, zTadShapeInfo, tadLen)] = result;
        }
    }
}

///////////////////////////////////////////////////////////////////
template<typename X>
static void prefixPerBlockCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                                        scalar::Ops op,
                                        const void* vx, const Nd4jLong* xTadShapeInfo, const Nd4jLong* xTadOffsets,
                                              void* vz, const Nd4jLong* zTadShapeInfo, const Nd4jLong* zTadOffsets,
                                        const Nd4jLong numTads, const Nd4jLong tadLen,
                                        const bool exclusive, const bool reverse) {

    prefixPerBlockCuda<X><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(op, vx, xTadShapeInfo, xTadOffsets, vz, zTadShapeInfo, zTadOffsets, numTads, tadLen, exclusive, reverse);
}

///////////////////////////////////////////////////////////////////
void prefix(nd4j::LaunchContext * context, scalar::Ops op, const NDArray* x, NDArray* z, const std::vector<int>& dims, bool exclusive, bool reverse) {

    auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(x->getShapeInfo(), dims);
    auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(z->getShapeInfo(), dims);

    const Nd4jLong numTads = packX.numberOfTads();
    const Nd4jLong tadLen = x->lengthOf() / numTads;

    const int threadsPerBlock = MAX_NUM_THREADS / 2;
    const int blocksPerGrid = numTads;
    const int sharedMem = 2 * threadsPerBlock * x->sizeOfT() + 128;

    PointersManager manager(context, "prefix");

    NDArray::prepareSpecialUse({z}, {x});
    BUILD_SINGLE_SELECTOR(x->dataType(), prefixPerBlockCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), op, x->getSpecialBuffer(), packX.platformShapeInfo(), packX.platformOffsets(), z->specialBuffer(), packZ.platformShapeInfo(), packZ.platformOffsets(), numTads, tadLen, exclusive, reverse), LIBND4J_TYPES);
    NDArray::registerSpecialUse({z}, {x});

    manager.synchronize();
}

///////////////////////////////////////////////////////////////////
void prefix(nd4j::LaunchContext * context, scalar::Ops op, const NDArray* x, NDArray* z, bool exclusive, bool reverse) {
    prefix(context, op, x, z, {}, exclusive, reverse);
}

}
}
}