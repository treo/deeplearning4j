#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com), created on 15.02.2018
//

// implementation of gated Recurrent Unit cell
// (cf. http://arxiv.org/abs/1406.1078).
// Kyunghyun Cho, Bart van Merrienboer, Caglar Gulcehre, Dzmitry Bahdanau, Fethi Bougares, Holger Schwenk, Yoshua Bengio
// "Learning Phrase Representations using RNN Encoder-Decoder for Statistical Machine Translation"


#include<ops/declarable/helpers/gru.h>
#include <ops/declarable/CustomOperations.h>
#include<ops/declarable/helpers/transforms.h>
#include <MmulHelper.h>

namespace nd4j 	  {
namespace ops 	  {
namespace helpers {


//////////////////////////////////////////////////////////////////////////
void gruCell(nd4j::LaunchContext * context, const NDArray* x, const NDArray* hLast, const NDArray* Wru, const NDArray* Wc,
             const NDArray* bru, const NDArray* bc,
             NDArray* r, NDArray* u, NDArray* c, NDArray* h) {

    //Inputs:
    // x        input [bS x inSize]
    // hLast    previous cell output [bS x numUnits],  that is at previous time step t-1
    // Wru      RU weights - [bS, 2*numUnits] - reset and update gates
    // Wc       C weights - [bS, numUnits] - cell gate
    // bru      r and u biases, [2*numUnits] - reset and update gates
    // bc       c biases, [numUnits] - cell gate

    //Outputs:
    // r        Reset gate output [bS, numUnits]
    // u        Update gate output [bS, numUnits]
    // c        Cell gate output [bS, numUnits]
    // h        current cell output [bS, numUnits]

    const int nIn = x->sizeAt(1);
    const int nU = hLast->sizeAt(1);                // number of units

    //Concat inputs: [x, yt-1]: concat([bs,nIn],[bs,nOut]) -> [bs, (nIn+nOut)]
    nd4j::ops::concat concatOp;
    std::vector<NDArray*> inputs;
    std::vector<double> targs;
    std::vector<Nd4jLong> iargs({1});   //Axis = 1
    std::vector<bool> bargs;
    inputs.emplace_back(const_cast<NDArray*>(x));
    inputs.emplace_back(const_cast<NDArray*>(hLast));

    auto result = concatOp.execute(inputs, targs, iargs, bargs);
    auto concatOut = result->at(0);

    //mmul/z for reset and update gates: (x * weight_ux + hLast * weight_xr + b_u)
    auto m = mmul(*concatOut, *Wru);    //mmul: [bs, (nIn+numUnits)]* [(inSize+numUnits), 2*numUnits] = [bs, 4*numUnits]
    m += (*bru);

    sigmoidInplace(m);  //sigmoid(rz) and sigmoid(uz)
    auto mr = m({0,0, 0, nU});
    auto mu = m({0,0, nU, 2*nU});

    r->assign(&mr);
    u->assign(&mu);

    //Concatenated inputs: [x, yt-1 .* r]
    auto yr = (*concatOut)({0,0, nIn, nIn+nU});
    yr *= (*r);

    //c = tanh(x * weight_cx + (hLast .* r) * weight_cr + b_c)
    MmulHelper::mmul(concatOut, const_cast<NDArray*>(Wc), c, 1.0, 0.0);       //c = 1.0 * concatOut * Wc + 0.0 * c
    *c += *bc;
    tanhInplace(*c);

    //Output: h = (1-u).*c + u .* hPrev
    //auto hResult = (*u) * (*hLast) + (1.0f - *u) * (*c); const_cast<NDArray*>(h)->assign(&hResult);
    u->applyPairwiseTransform(pairwise::Multiply, hLast, h, nullptr);        //h = u * hLast
    auto temp = (1.0f - *u);
    temp *= (*c);
    (*h) += temp;

    delete result;
}

//////////////////////////////////////////////////////////////////////////
void gruTimeLoop(nd4j::LaunchContext * context, const NDArray* x, const NDArray* h0, const NDArray* Wx, const NDArray* Wh, const NDArray* b, NDArray* h) {

}

//////////////////////////////////////////////////////////////////////////
void gruCellBP(nd4j::LaunchContext * context, const NDArray* x, const NDArray* h0, const NDArray* Wx, const NDArray* Wh, const NDArray* b, const NDArray* dLdh, const NDArray* dLdWx0,
               const NDArray* dLdWh0, const NDArray* dLdb0, NDArray* dLdx, NDArray* dLdh0, NDArray* dLdWx, NDArray* dLdWh, NDArray* dLdb) {

}


}
}
}

