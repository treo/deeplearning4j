#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com, created on 30.11.17.
// @author Yurii Shyrma (iuriish@yahoo.com)
//

#include <ops/declarable/helpers/col2im.h>
#include <PointersManager.h>

namespace nd4j    {
namespace ops     {
namespace helpers {

//////////////////////////////////////////////////////////////////////////
// columns [bS, iC, kH, kW, oH, oW] to be de-convoluted to image [bS, iC, iH, iW]
template <typename T>
static __global__ void col2imCuda(const void* columns, const Nd4jLong* colShapeInfo, void* image, const Nd4jLong* imShapeInfo, const int sH, const int sW, const int pH, const int pW, const int dH, const int dW) {

    const T* col = reinterpret_cast<const T*>(columns);
          T* im = reinterpret_cast<T*>(image);

    __shared__ int colRank, imRank, kHeff, kWeff, oH, oW;
    __shared__ Nd4jLong *sharedMem, imLen;

    if (threadIdx.x == 0) {

        extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<Nd4jLong*>(shmem);

        oH = colShapeInfo[5];
        oW = colShapeInfo[6];

        kHeff = colShapeInfo[3] + (colShapeInfo[3] - 1) * (dH - 1);
        kWeff = colShapeInfo[4] + (colShapeInfo[4] - 1) * (dW - 1);

        imRank = 4;
        colRank = 6;

        imLen = shape::length(imShapeInfo);
    }

    __syncthreads();

    const auto imInd = threadIdx.x + blockIdx.x * blockDim.x;

    if(imInd >= imLen)
        return;

    auto coords = sharedMem + threadIdx.x * colRank;

    shape::index2coords(imRank, imShapeInfo + 1, imInd, imLen, coords);

    const auto imOffset = shape::getOffset(0, imShapeInfo + 1, imShapeInfo + imRank + 1, coords, imRank);

    const int imH = coords[2] + pH;
    const int imW = coords[3] + pW;

    const int colHstart = (imH < kHeff) ? 0 : (imH - kHeff) / sH + 1;
    const int colWstart = (imW < kWeff) ? 0 : (imW - kWeff) / sW + 1;

    const int colHend = nd4j::math::nd4j_min<int>(imH / sH + 1, oH);
    const int colWend = nd4j::math::nd4j_min<int>(imW / sW + 1, oW);

    T val = 0;

    for(coords[4] = colHstart; coords[4] < colHend; ++coords[4]) {
        coords[2] = imH - coords[4] * sH;

      for(coords[5] = colWstart; coords[5] < colWend; ++coords[5]) {
          coords[3] = imW - coords[5] * sW;

            if(coords[2] % dH == 0 && coords[3] % dW == 0) {
                coords[2] /= dH;
                coords[3] /= dW;

                val += col[shape::getOffset(0, colShapeInfo + 1, colShapeInfo + colRank + 1, coords, colRank)];
            }
        }
    }

    im[imOffset] = val;
}

////////////////////////////////////////////////////////////////////////
// columns [bS, iC, kH, kW, oH, oW] to be de-convoluted to image [bS, iC, iH, iW]
template<typename T>
__global__ static void col2imCuda2(const void *columns, void *image, const Nd4jLong *colShapeInfo, const Nd4jLong *imShapeInfo, const int sH, const int sW, const int pH, const int pW, const int dH, const int dW) {

          const auto col = reinterpret_cast<const T*>(columns);
          auto im = reinterpret_cast<T*>(image);

          auto colShape = shape::shapeOf(const_cast<Nd4jLong *>(colShapeInfo));
          auto colStride = shape::stride(const_cast<Nd4jLong *>(colShapeInfo));

          int colStride0 = colStride[0];
          int colStride1 = colStride[1];
          int colStride2 = colStride[2];
          int colStride3 = colStride[3];
          int colStride4 = colStride[4];
          int colStride5 = colStride[5];

          int kH = colShape[2];
          int kW = colShape[3];

          auto imShape  = shape::shapeOf(const_cast<Nd4jLong *>(imShapeInfo));
          auto imOrder  = shape::order(const_cast<Nd4jLong *>(imShapeInfo));
          auto imStride = shape::stride(const_cast<Nd4jLong *>(imShapeInfo));

          int bS = imShape[0];
          int iC = imShape[1];
          int iH = imShape[2];
          int iW = imShape[3];

          int oH = colShape[4];//(iH + 2 * pH - kH) / sW + 1;
          int oW = colShape[5];//(iW + 2 * pW - kW) / sH + 1;

          int n = bS * iC * iH * iW;

          //Effective kernel size, accounting for dilation
          int kHeff = kH + (kH - 1) * (dH - 1);
          int kWeff = kW + (kW - 1) * (dW - 1);

          for (int i = (blockDim.x * blockIdx.x) + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
              T val = 0;
              int w_im = i % iW + pW;
              int h_im = (i / iW) % iH + pH;
              int c_im = i / (iW * iH);

              int b = c_im / iC;
              int c = c_im % iC;

              // compute the start and end of the output
              // These are the indexes for dimensions ??? in the 6d col matrix
              int w_col_start = (w_im < kWeff) ? 0 : (w_im - kWeff) / sW + 1;
              int w_col_end = nd4j::math::nd4j_min<int>(w_im / sW + 1, oW);

              int h_col_start = (h_im < kHeff) ? 0 : (h_im - kHeff) / sH + 1;
              int h_col_end = nd4j::math::nd4j_min<int>(h_im / sH + 1, oH);

              //Iterate over col entries in the 6d array... these are added up
              for (int colH = h_col_start; colH < h_col_end; colH += 1) {
                  for (int colW = w_col_start; colW < w_col_end; colW += 1) {
                      int kRow = (h_im - colH * sH);
                      int kCol = (w_im - colW * sW);

                      if(kRow % dH == 0 && kCol % dW == 0){
                          kRow /= dH;
                          kCol /= dW;

                          int data_col_index = b * colStride0 + c * colStride1 + kRow * colStride2 + kCol * colStride3 + colH * colStride4 + colW * colStride5;
                          val += col[data_col_index];
                      }
                  }
              }

              int i_f = 0;
              int i_c = i;
              for (int dim = 3; dim >= 0; dim--) {
                  i_f += (i_c % imShape[dim])  * imStride[dim];
                  i_c = i_c / imShape[dim];
              }

              im[i_f] = val;
          }
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
static void col2imCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                                const void* columns, const Nd4jLong* colShapeInfo,
                                      void* image, const Nd4jLong* imShapeInfo,
                                const int sH, const int sW, const int pH, const int pW, const int dH, const int dW) {

    // col2imCuda2<T><<<512, 512, 1024, *stream>>>(columns, image, colShapeInfo, imShapeInfo, sH, sW, pH, pW, dH, dW);
    col2imCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(columns, colShapeInfo, image, imShapeInfo, sH, sW, pH, pW, dH, dW);
}
BUILD_SINGLE_TEMPLATE(template void col2imCudaLauncher, (const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t* stream, const void *col, const Nd4jLong *colShapeInfo, void *im, const Nd4jLong *imShapeInfo, const int sH, const int sW, const int pH, const int pW, const int dH, const int dW), FLOAT_TYPES);

//////////////////////////////////////////////////////////////////////////
void col2im(nd4j::LaunchContext& context, const NDArray& col, NDArray& im, const int sH, const int sW, const int pH, const int pW, const int iH, const int iW, const int dH, const int dW) {

    PointersManager manager(&context, "col2im");

    const int threadsPerBlock = MAX_NUM_THREADS / 2;
    const int blocksPerGrid = (im.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
    const int sharedMem = col.rankOf() * sizeof(Nd4jLong) * threadsPerBlock  + 128;

    NDArray::prepareSpecialUse({&im}, {&col});
    BUILD_SINGLE_SELECTOR(im.dataType(), col2imCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context.getCudaStream(), col.getSpecialBuffer(), col.getSpecialShapeInfo(), im.specialBuffer(), im.specialShapeInfo(), sH, sW, pH, pW, dH, dW), FLOAT_TYPES);
    NDArray::registerSpecialUse({&im}, {&col});

    manager.synchronize();
}



}
}
}