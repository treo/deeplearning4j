#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

#ifndef NDARRAY_CPP
#define NDARRAY_CPP

#include "../NDArray.h"
#include "../NDArrayFactory.h"
#include "NativeOpExecutioner.h"
#include <memory/Workspace.h>
#include <memory/MemoryRegistrator.h>
#include <ops.h>
#include <ops/gemm.h>
#include <pointercast.h>
#include <stdexcept>
#include <memory>
#include <helpers/logger.h>
#include <loops/pairwise_transform.h>
#include <loops/transform_same.h>
#include <loops/random.h>
#include <loops/broadcasting.h>
#include <indexing/NDIndex.h>
#include <indexing/IndicesList.h>
#include <helpers/ShapeUtils.h>
#include <sstream>
#include <helpers/ArrayUtils.h>
#include <MmulHelper.h>
#include <helpers/threshold.h>
#include <exceptions/datatype_exception.h>
#include <exceptions/cuda_exception.h>
#include <specials_cuda.h>
#include <loops/special_kernels.h>
#include <PointersManager.h>
#include "../NDArray.hpp"
#include <ConstantShapeHelper.h>

namespace nd4j {

void* NDArray::platformBuffer()             { return specialBuffer();    }
void* NDArray::getPlatformBuffer() const    { return getSpecialBuffer(); }

Nd4jLong* NDArray::getPlatformShapeInfo() const { return getSpecialShapeInfo(); }
Nd4jLong* NDArray::platformShapeInfo()          { return specialShapeInfo(); }

void NDArray::syncToDevice() const          { _buffer->syncToSpecial();  }
void NDArray::syncToHost() const            { _buffer->syncToPrimary(getContext()); }
void NDArray::tickWriteHost() const         { _buffer->writePrimary();   }
void NDArray::tickWriteDevice() const       { _buffer->writeSpecial();   }
void NDArray::tickReadHost() const          { _buffer->readPrimary();    }
void NDArray::tickReadDevice() const        { _buffer->readSpecial();    }
void NDArray::tickBothActual() const        { _buffer->writePrimary(); _buffer->readSpecial(); }
bool NDArray::isActualOnHostSide() const    { return _buffer->isPrimaryActual(); }
bool NDArray::isActualOnDeviceSide() const  { return _buffer->isSpecialActual(); }
void NDArray::makeBothBuffersActual() const { if(!isActualOnHostSide()) syncToHost(); if(!isActualOnDeviceSide()) syncToDevice(); }


///////////////////////////////////////////////////////////////////
template<typename T>
__global__ static void fillAsTriangularCuda(const void* vx, const Nd4jLong* xShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const T val, const int lower, const int upper) {

    const auto x = reinterpret_cast<const T*>(vx);
          auto z = reinterpret_cast<T*>(vz);

    __shared__ int zRank, xRank, areSameOffsets;        // xRank == zRank always, except when xRank = 1, in this case zRank = 2
    __shared__ Nd4jLong zLen, totalThreads, *sharedMem;  // xLen == zLen, except when xRank = 1, in this case zLen = 2*xLen

    if (threadIdx.x == 0) {

        extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<Nd4jLong*>(shmem);
        areSameOffsets = shape::haveSameShapeAndStrides(xShapeInfo, zShapeInfo);
        xRank = shape::rank(xShapeInfo);
        zRank = shape::rank(zShapeInfo);
        zLen  = shape::length(zShapeInfo);
        totalThreads = gridDim.x * blockDim.x;
    }

    __syncthreads();

    auto coords = sharedMem + threadIdx.x * zRank;

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (Nd4jLong i = tid; i < zLen; i += totalThreads) {

        shape::index2coords(zRank, shape::shapeOf(const_cast<Nd4jLong*>(zShapeInfo)), i, zLen, coords);
        const auto zOffset = shape::getOffset(0, shape::shapeOf(const_cast<Nd4jLong*>(zShapeInfo)), shape::stride(const_cast<Nd4jLong*>(zShapeInfo)), coords, zRank);

        // if( (row + upper < col) || (row + lower > col) )
        if((coords[zRank - 2] + upper < coords[zRank - 1]) || (coords[zRank - 2] + lower > coords[zRank - 1]))
            z[zOffset] = val;
        else if(vx != vz) {      // when x and z are different arrays
            if(xRank != zRank)
                coords[0] = coords[1];
            const auto xOffset = areSameOffsets ? zOffset : shape::getOffset(0, shape::shapeOf(const_cast<Nd4jLong*>(xShapeInfo)), shape::stride(const_cast<Nd4jLong*>(xShapeInfo)), coords, xRank);
            z[zOffset] = x[xOffset];
        }
    }
}

///////////////////////////////////////////////////////////////////
template<typename T>
void NDArray::fillAsTriangular(const float val, int lower, int upper, const char direction, NDArray* target) {

    if (isS())
        throw std::runtime_error("NDArray::fillAsTriangular: you can't use this method on String array!");

    if(target == nullptr)
        target = this;

    if(!isSameShape(target) && !(rankOf() == 1 && target->rankOf() == 2 && sizeAt(0) == target->sizeAt(0) && sizeAt(0) == target->sizeAt(1)))
        throw std::string("NDArray::fillAsTriangular method: wrong shape of target array !");

     if (direction == 'u')
        lower = -target->sizeAt(-2);
    else if (direction == 'l')
        upper = target->sizeAt(-1);

    const int threadsPerBlock = MAX_NUM_THREADS / 4;
    const int blocksPerGrid = (target->lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
    const int sharedMem = threadsPerBlock * sizeof(decltype(*target->getShapeInfo())) * target->rankOf() + 128;

    PointersManager manager(getContext(), "NDArray::fillAsTriangular");

    NDArray::prepareSpecialUse({target}, {this});
    fillAsTriangularCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *getContext()->getCudaStream()>>>(getPlatformBuffer(), getPlatformShapeInfo(), target->getPlatformBuffer(), target->getPlatformShapeInfo(), static_cast<T>(val), lower, upper);
    NDArray::registerSpecialUse({target}, {this});

    manager.synchronize();
}
BUILD_SINGLE_TEMPLATE(template void NDArray::fillAsTriangular, (const float val, int lower, int upper, const char direction, NDArray* target), LIBND4J_TYPES);

////////////////////////////////////////////////////////////////////////
template<typename T>
__global__ static void identityMatrixCuda(void* vx, const Nd4jLong* xShapeInfo, const T val) {

    auto x = reinterpret_cast<T*>(vx);

    __shared__ int rank;
    __shared__ Nd4jLong len, totalThreads, *sharedMem;  // xLen == zLen, except when xRank = 1, in this case zLen = 2*xLen

    if (threadIdx.x == 0) {

        extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<Nd4jLong*>(shmem);
        rank = shape::rank(xShapeInfo);
        len  = shape::length(xShapeInfo);
        totalThreads = gridDim.x * blockDim.x;
    }

    __syncthreads();

    auto coords = sharedMem + threadIdx.x * rank;

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (Nd4jLong i = tid; i < len; i += totalThreads) {

        shape::index2coords(rank, shape::shapeOf(const_cast<Nd4jLong*>(xShapeInfo)), i, len, coords);
        const auto offset = shape::getOffset(0, shape::shapeOf(const_cast<Nd4jLong*>(xShapeInfo)), shape::stride(const_cast<Nd4jLong*>(xShapeInfo)), coords, rank);

        if(coords[rank - 2] == coords[rank - 1]) // row == col -> on diagonal
            x[offset] = val;
        else
            x[offset] = static_cast<T>(0);
    }
}

///////////////////////////////////////////////////////////////////
template<typename T>
static void identityMatrixCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream, void* vx, const Nd4jLong *xShapeInfo, const float val) {

    identityMatrixCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, static_cast<T>(val));
}
BUILD_SINGLE_TEMPLATE(template void identityMatrixCudaLauncher, (const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream, void* vx, const Nd4jLong *xShapeInfo, const float val), LIBND4J_TYPES);

////////////////////////////////////////////////////////////////////////
void NDArray::setIdentity() {
    if (isS())
        throw std::runtime_error("NDArray::setIdentity: you can't use this method on String array!");

    // if (rankOf() != 2)
    //     throw std::runtime_error("NDArray::setIdentity: method should work only for 2D tensors. But " + toStringValue(rankOf()) + " was given.");

    const int threadsPerBlock = MAX_NUM_THREADS / 4;
    const int blocksPerGrid = (lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
    const int sharedMem = threadsPerBlock * sizeof(decltype(getShapeInfo())) * rankOf() + 128;

    PointersManager manager(getContext(), "NDArray::setIdentity");

    syncToDevice();
    BUILD_SINGLE_SELECTOR(dataType(), identityMatrixCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, getContext()->getCudaStream(), getPlatformBuffer(), getPlatformShapeInfo(), 1.f), LIBND4J_TYPES);
    tickWriteDevice();

    manager.synchronize();
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void NDArray::swapUnsafe(NDArray& other) {
    auto xType = this->dataType();

    if (xType != other.dataType())
        throw std::runtime_error("NDArray::swapUnsage method: both arrays must have the same data type");

    if(specialBuffer() == nullptr || other.specialBuffer() == nullptr)
        throw std::runtime_error("NDArray::swapUnsafe method: input array should not be empty!");

    if(lengthOf() != other.lengthOf())
        throw std::runtime_error("NDArray::swapUnsafe method: input arrays should have the same length!");

    BUILD_SINGLE_SELECTOR(xType, templatedSwapUnsafe, (specialBuffer(), specialShapeInfo(), other.specialBuffer(), other.specialShapeInfo(), getContext()->getCudaStream()), LIBND4J_TYPES);
}

////////////////////////////////////////////////////////////////////////
void NDArray::synchronize(const char* msg) const {
    auto res = hipStreamSynchronize(*(getContext()->getCudaStream()));
    if (res != 0)
        throw std::runtime_error(msg + std::string(": synchronization failed !"));
}
////////////////////////////////////////////////////////////////////////
void NDArray::prepareSpecialUse(const std::initializer_list<const NDArray*>& writeList, const std::initializer_list<const NDArray*>& readList, bool synchronizeWritables) {

    for (const auto& a : readList)
        if(a != nullptr)
            a->syncToDevice();

    for (const auto& a : writeList) {
        a->getDataBuffer()->allocateSpecial();
        if (synchronizeWritables)
            a->syncToDevice();
    }
}

////////////////////////////////////////////////////////////////////////
void NDArray::registerSpecialUse(const std::initializer_list<const NDArray*>& writeList, const std::initializer_list<const NDArray*>& readList) {

    for (const auto& p : readList)
        if(p != nullptr)
            p->tickReadDevice();

    for (const auto& p : writeList)
        p->tickWriteDevice();
}

////////////////////////////////////////////////////////////////////////
void NDArray::preparePrimaryUse(const std::initializer_list<const NDArray*>& writeList, const std::initializer_list<const NDArray*>& readList, bool synchronizeWritables) {

    for (const auto& a : readList)
        if(a != nullptr)
            a->syncToHost();

    for (const auto& a : writeList) {
        a->getDataBuffer()->allocatePrimary();
        if (synchronizeWritables)
            a->syncToHost();
    }
}

////////////////////////////////////////////////////////////////////////
void NDArray::registerPrimaryUse(const std::initializer_list<const NDArray*>& writeList, const std::initializer_list<const NDArray*>& readList) {

    for (const auto& p : readList)
        if(p != nullptr)
            p->tickReadHost();

    for (const auto& p : writeList)
        p->tickWriteHost();
}

//////////////////////////////////////////////////////////////////////////
void NDArray::syncShape() const {
    hipMemcpy(getSpecialShapeInfo(), getShapeInfo(), shape::shapeInfoByteLength(getShapeInfo()), hipMemcpyHostToDevice);
}

//////////////////////////////////////////////////////////////////////////
void* NDArray::specialBufferWithOffset(Nd4jLong offset) const {
    return getSpecialBuffer() != nullptr ? static_cast<int8_t*>(getSpecialBuffer()) + (offset * sizeOfT()) : nullptr;
}

//////////////////////////////////////////////////////////////////////////
// change an array by repeating it the number of times given by reps.
NDArray NDArray::tile(const std::vector<Nd4jLong>& reps) const {
    int dim = reps.size();
    int product = 1;
    for(const auto& item : reps)
        product *= item;
    if(product == 0)
        throw std::runtime_error("NDArray::tile method: one of the elements in reps array is zero !");

    int rankOld = rankOf();
    int diff = rankOld - dim;
    if(product==1) {        // in this case 2 possibilities are present: just reshape or nothing to do
        NDArray result(*this);
        if(diff < 0) {      // reshape to higher dimension
            std::vector<Nd4jLong> shapeNew = reps;               // need to have unities at first "diff" positions of new shape
            memcpy(&shapeNew[-diff], result.getShapeInfo()+1, rankOld * sizeof(Nd4jLong));   // put old shape numbers at rest of positions
            result.reshapei(ordering(), shapeNew);
        }
        return result;             // nothing to do, if diff >= 0 -> identity tile
    }

    // evaluate shapeInfo for resulting array
    auto newShapeInfo = ShapeUtils::evalTileShapeInfo(*this, reps, getContext()->getWorkspace());
    // create new buffer, in any case the memory amount new buffer points to is bigger then those for old _buffer
    std::shared_ptr<DataBuffer> newBuff = std::make_shared<DataBuffer>(shape::length(newShapeInfo) * sizeOfT(), dataType(), getContext()->getWorkspace(), true);
    // assign new shape and new buffer to resulting array
    NDArray result(newBuff, ShapeDescriptor(newShapeInfo), getContext());

    // fill newBuff, loop through all elements of newBuff
    // looping through getBuffer() goes automatically by means of getSubArrayIndex applying
    const auto resultLen = result.lengthOf();
    auto xType = this->dataType();
    auto stream = getContext()->getCudaStream();

    prepareSpecialUse({&result}, {this});
    BUILD_SINGLE_SELECTOR(xType, tileKernelH, (this->getSpecialBuffer(), this->getSpecialShapeInfo(), result.getSpecialBuffer(), result.getSpecialShapeInfo(), resultLen, stream), LIBND4J_TYPES);
    registerSpecialUse({&result}, {this});

    return result;
}

//////////////////////////////////////////////////////////////////////////
// change an array by repeating it the number of times given by reps.
void NDArray::tile(const std::vector<Nd4jLong>& reps, NDArray& target) const {

    // evaluate true tile shapeInfo for comparison with target shapeInfo
    auto newShapeInfo = ShapeUtils::evalTileShapeInfo(*this, reps, getContext()->getWorkspace());
    if(!shape::equalsSoft(newShapeInfo, target.getShapeInfo()))  {
        throw std::runtime_error("NDArray::tile method - shapeInfo of target array is not suitable for tile operation !");
    }

    // fill newBuff, loop through all elements of newBuff
    // looping through getBuffer() goes automatically by means of getSubArrayIndex applying
    const int ews = target.ews();
    const int targetLen = target.lengthOf();
    auto stream = getContext()->getCudaStream();

    prepareSpecialUse({&target}, {this});
    BUILD_DOUBLE_SELECTOR(target.dataType(), dataType(), tileKernelHH, (getSpecialBuffer(), getSpecialShapeInfo(), target.getSpecialBuffer(), target.getSpecialShapeInfo(), targetLen, ews, stream), LIBND4J_TYPES, LIBND4J_TYPES);
    registerSpecialUse({&target}, {this});
}

//////////////////////////////////////////////////////////////////////////
void NDArray::tile(NDArray& target) const {
    if(rankOf() > target.rankOf())
        throw std::runtime_error("NDArray::tile method - rank of target array must be bigger or equal to the rank of this array !");

    if(!ShapeUtils::areShapesBroadcastable(*this, target))
        throw std::runtime_error("NDArray::tile method - shapeInfo of target array is not suitable for tile operation !");

    // fill newBuff, loop through all elements of newBuff
    // looping through getBuffer() goes automatically by means of getSubArrayIndex applying
    const auto ews = target.ews();
    const auto targetLen = target.lengthOf();
    auto stream = getContext()->getCudaStream();

    prepareSpecialUse({&target}, {this});
    BUILD_DOUBLE_SELECTOR(target.dataType(), dataType(), tileKernelHH, (getSpecialBuffer(), getSpecialShapeInfo(), target.getSpecialBuffer(), target.getSpecialShapeInfo(), targetLen, ews, stream), LIBND4J_TYPES, LIBND4J_TYPES);
    registerSpecialUse({&target}, {this});
}

//////////////////////////////////////////////////////////////////////////
// create new  array by repeating it the number of times given by reps
NDArray* NDArray::repeat(int dimension, const std::vector<Nd4jLong>& repeats) const {
    auto outShape = ShapeUtils::evalRepeatShape(dimension, repeats, *this);

    // the size of outShape == rank
    int rank = rankOf();            // = outShape.size()

    std::vector<Nd4jLong> newShape(rank);
    for (int i = 0; i < rank; i++)
        newShape[i] = outShape[i];

    auto ret = new NDArray('c', outShape, dataType(),  getContext());

    auto repeatDelta = shape::prodLong(newShape.data(), rank) / this->lengthOf();
    std::vector<int> dimsToExclude = ShapeUtils::evalDimsToExclude(rankOf(), {dimension});
    const Nd4jLong numTads = ShapeUtils::getNumOfSubArrs(getShapeInfo(), dimsToExclude); //this->tensorsAlongDimension({dimension});
    std::vector<int> copy({dimension});

    auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(this->getShapeInfo(), copy);
    auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(ret->getShapeInfo(), copy);

    prepareSpecialUse({ret}, {this});
    auto stream = getContext()->getCudaStream();
    BUILD_SINGLE_SELECTOR(dataType(), repeatKernelH, (getSpecialBuffer(), ret->getSpecialBuffer(), numTads, lengthOf(), ret->lengthOf(), packX.platformShapeInfo(), packX.platformOffsets(), packZ.platformShapeInfo(), packZ.platformOffsets(), *stream), LIBND4J_TYPES);
    registerSpecialUse({ret}, {this});

    return ret;
}

//////////////////////////////////////////////////////////////////////////
// fill array by repeating it the number of times given by reps
void NDArray::repeat(int dimension, NDArray& target) const {

    if(dimension < 0)
        dimension += rankOf();

    if(rankOf() != target.rankOf())
        throw std::invalid_argument("NDArray::repeat(int dimension, NDArray& target) method: wrong rank of target array it must be equal to this array rank!");

    Nd4jLong repeatDelta = target.sizeAt(dimension) / sizeAt(dimension);

    if(repeatDelta == 0)
        throw std::invalid_argument("NDArray::repeat(int dimension, NDArray& target) method: wrong shape of target array!");


    std::vector<int> dimsToExclude = ShapeUtils::evalDimsToExclude(rankOf(), {dimension});
    const Nd4jLong numTads = ShapeUtils::getNumOfSubArrs(getShapeInfo(), dimsToExclude);

    std::vector<int> copy({dimension});
    auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(this->getShapeInfo(), copy);
    auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(target.getShapeInfo(), copy);

    NDArray::prepareSpecialUse({&target}, {this});
    auto stream = getContext()->getCudaStream();
    BUILD_DOUBLE_SELECTOR(target.dataType(), dataType(), repeatKernelHH, (getSpecialBuffer(), target.getSpecialBuffer(), numTads, lengthOf(), packX.platformShapeInfo(), packX.platformOffsets(), packZ.platformShapeInfo(), packZ.platformOffsets(), *stream), LIBND4J_TYPES, LIBND4J_TYPES);
    NDArray::registerSpecialUse({&target}, {this});
}

////////////////////////////////////////////////////////////////////////
void* NDArray::specialBuffer() {

    if (_buffer->special() == nullptr)
        return getBuffer();
    // FIXME: this should be fixed once CUDA backend added
    return static_cast<int8_t*>(_buffer->special()) + (_offset * sizeOfT());
}

////////////////////////////////////////////////////////////////////////
void* NDArray::getSpecialBuffer() const {
    if (_buffer->special() == nullptr)
        return getBuffer();
    // FIXME: this should be fixed once CUDA backend added
    return static_cast<int8_t*>(_buffer->special()) + (_offset * sizeOfT());
}

//////////////////////////////////////////////////////////////////////////
template<typename T>
void NDArray::printCurrentBuffer(const bool host, const char* msg, const int precision) const {

    if(_length == 0)
            { printf("NDArray::printActualBuffer: array length is zero !\n"); return; }

    if(msg)
        printf("%s", msg);

    if(host) {
        if(getBuffer() == nullptr || _length == 0)
            { printf("NDArray::printActualBuffer: host buffer is nullptr !\n"); return; }

        const T* buff = bufferAsT<T>();
        for (uint i = 0; i < _length; i++)
            printf("%.*f, ", precision, (double)buff[getOffset(i)]);
        printf("\n");
    }
    else {
        if(getSpecialBuffer() == nullptr || _length == 0)
            { printf("NDArray::printSpecialBuffer: special buffer is nullptr !\n"); return; }

        void* pHost = operator new(sizeof(T) * _length);

        if (ews() != 1) {
            for (uint i = 0; i < _length; i++)
                hipMemcpyAsync(reinterpret_cast<T*>(pHost) + i, specialBufferWithOffset(i), sizeof(T), hipMemcpyDeviceToHost, *(getContext()->getCudaStream()));
        }
        else
            hipMemcpyAsync(pHost, getSpecialBuffer(), sizeOfT() * _length, hipMemcpyDeviceToHost, *getContext()->getCudaStream());

        hipError_t cudaResult = hipStreamSynchronize(*getContext()->getCudaStream());
        if(cudaResult != 0)
            throw std::runtime_error("NDArray::printSpecialBuffer: hipStreamSynchronize failed!");

        for (uint i = 0; i < _length; i++)
            printf("%.*f, ", precision, (double)reinterpret_cast<T*>(pHost)[i]);
        printf("\n");

        operator delete(pHost);
    }
}
template void NDArray::printCurrentBuffer<int>(const bool host,const char* msg, const int precision) const;
template void NDArray::printCurrentBuffer<float>(const bool host, const char* msg, const int precision) const;
template void NDArray::printCurrentBuffer<double>(const bool host, const char* msg, const int precision) const;


#if defined(__HIPCC__) && !defined(BUILD_TESTS)

#include <cpu/NDArrayLambda.hpp>

#endif

} // end namespace nd4j
#endif

