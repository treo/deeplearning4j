#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/


#include "NativeOpExecutioner.h"
#include "../NativeOps.h"
#include <hip/hip_runtime.h>

#include <buffer.h>


#include <loops/transform_any.h>
#include <loops/reduce_bool.h>
#include <loops/reduce_long.h>
#include <helpers/threshold.h>
#include <ops/specials_cuda.h>
#include <helpers/DebugHelper.h>

#include <exceptions/datatype_exception.h>
#include <helpers/CudaLaunchHelper.h>
// FIXME: we need cuda-specific implementations
#include <GraphExecutioner.h>
#include <graph/GraphHolder.h>
#include <ops/declarable/CustomOperations.h>
#include <PointersManager.h>


//#include <sys/time.h>

#include <hiprand.h>
#include <Status.h>
#include <helpers/DebugHelper.h>

using namespace nd4j;

#include <loops/special_kernels.h>

hipDeviceProp_t *deviceProperties;
hipFuncAttributes *funcAttributes = new hipFuncAttributes[64];
int blockLimit = 128;
int maxThreads = 512;
bool allowedP2P = false;
bool supportedP2P = false;
#ifdef __ND4J_EXPERIMENTAL__
bool experimentalSupport = true;
#else
bool experimentalSupport = false;
#endif

int minThreads = 32;

__constant__ char deviceConstantMemory[49152];

typedef struct {
    long streamId;
    long callId;
} __syncInfo;

typedef __syncInfo SyncInfo;


// this method isn't used, left here for legacy and caution purposes
// TLDR: don't use this way, it sucks
void CUDART_CB syncCallback(hipStream_t stream, hipError_t status, void *data){
    SyncInfo *sync = reinterpret_cast<SyncInfo *>(data);

    //printf("Finished stream: [%i], kernel call: [%i]\n", sync->streamId, sync->callId);
}

// this method just does type conversion in fancy way
int getDeviceId(Nd4jPointer ptrToDeviceId) {
    return (int)(Nd4jLong)ptrToDeviceId;
}

/*
 * Basic CUDA constants here: number of blocks per MP
 */
int getDeviceBlockThreshold(int deviceId) {
	int ccMinor = deviceProperties[deviceId].minor;
	int ccMajor = deviceProperties[deviceId].major;

	int blockThreshold = 8;

	if (ccMajor >= 5)
		blockThreshold = 32;
	else if (ccMajor == 3)
		blockThreshold = 16;
	else if (ccMajor < 3)
		blockThreshold = 8;

	return blockThreshold;
}


/*
 * This message returns shared memory threshold value. default overflow ratio is 0.3
 */
int getDeviceSharedThreshold(int deviceId) {
	int ccMinor = deviceProperties[deviceId].minor;
	int ccMajor = deviceProperties[deviceId].major;

	// please note threshold isn't multiple of 32, and that's NOT a mistake

	int shmemThreshold;
	if (ccMajor == 6 && ccMinor == 0)
		shmemThreshold = 65536;
	else if (ccMajor == 6 && ccMinor == 1)
		shmemThreshold = 49152;
	else if (ccMajor == 5 && ccMinor == 2)
		shmemThreshold = 98304;
	else if (ccMajor == 5)
		shmemThreshold = 65536;
	else if (ccMajor == 3 && ccMinor == 7)
		shmemThreshold = 114688;
	else shmemThreshold = 49152;

	return shmemThreshold / 0.3;
}



nd4j::buffer::Buffer<Nd4jLong> * createScalarBuffer(hipStream_t stream) {
	Nd4jLong *scalarShapeInfo = shape::createScalarShapeInfo();
	nd4j::buffer::Buffer<Nd4jLong> *buff = nd4j::buffer::createBuffer(scalarShapeInfo,shape::shapeInfoLength(2), stream);
	nd4j::buffer::copyDataToGpu(&buff, stream);
	return buff;
}


class ScalarShapeInformation {
private:
	nd4j::buffer::Buffer<Nd4jLong> *scalarDimension;
	nd4j::buffer::Buffer<Nd4jLong> *scalarShapeInfo;
//	std::thread::id threadId;

public:
	ScalarShapeInformation(hipStream_t stream) {
		auto scalarDimensionBuff = reinterpret_cast<Nd4jLong *>(malloc(sizeof(Nd4jLong)));

		CHECK_ALLOC(scalarDimensionBuff, "Failed to allocate ShapeInfoBuffer", sizeof(Nd4jLong));

		scalarDimensionBuff[0] = MAX_DIMENSION;
		scalarDimension = nd4j::buffer::createBuffer(scalarDimensionBuff,1, stream);
		scalarShapeInfo = createScalarBuffer(stream);
//		threadId = std::this_thread::get_id();

	}
	~ScalarShapeInformation() {
		nd4j::buffer::freeBuffer(&scalarShapeInfo);
		nd4j::buffer::freeBuffer(&scalarDimension);
	}


	Nd4jLong *getShapeInfoHostPointer() {
		return scalarShapeInfo->data;
	}

	Nd4jLong * getShapeInfoGpuPointer() {
		return scalarShapeInfo->gData;
	}

	Nd4jLong * getDimensionHostPointer() {
		return scalarDimension->data;
	}

	Nd4jLong  * getDimensionGpuPointer() {
		return scalarDimension->gData;
	}

};





template <typename T>
class ScalarInfo {
	nd4j::buffer::Buffer<T> *scalarData;
	ScalarShapeInformation *shapeInfo;
	T finalResult;
	hipStream_t streamRef;
public:
	ScalarInfo(hipStream_t stream) {
		T *scalarResult = reinterpret_cast<T*>(malloc(sizeof(T)));

		CHECK_ALLOC(scalarResult, "Failed to allocate new scalar buffer", sizeof(T));

		shapeInfo = new ScalarShapeInformation(stream);
		scalarData = nd4j::buffer::createBuffer(scalarResult,1, stream);
		streamRef = stream;
		nd4j::buffer::copyDataToGpu(&scalarData, stream);
	}

	T getFinalResultFromDevice() {
		nd4j::buffer::copyDataFromGpu(&scalarData, streamRef);
		return scalarData->data[0];
	}

	/**
	 * Get the device shape information
	 * representing a scalar
	 */
	 Nd4jLong *getDeviceShapeInfo() {
		return shapeInfo->getShapeInfoGpuPointer();
	}

	/**
	 * Get the dZ pointers
	 */
	 T *getDevicePointer() {
		 return scalarData->gData;
	 }

	 /**
	  * Get the infinite dimension device pointer
	  */
	  Nd4jLong *getDimensionDevicePointer() {
		 return shapeInfo->getDimensionGpuPointer();
	 }

	 ~ScalarInfo() {
		 nd4j::buffer::freeBuffer(&scalarData);
		 delete shapeInfo;
	 }
};

NativeOps::NativeOps() {
//
}

void NativeOps::execPairwiseTransform( Nd4jPointer *extraPointers,
        								int opNum,
        								void *hX, Nd4jLong *hXShapeInfo,
        								void *dX, Nd4jLong *dXShapeInfo,
        								void *hY, Nd4jLong *hYShapeInfo,
        								void *dY, Nd4jLong *dYShapeInfo,
        								void *hZ, Nd4jLong *hZShapeInfo,
        								void *dZ, Nd4jLong *dZShapeInfo,
        								void *extraParams) {

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
	NativeOpExecutioner::execPairwiseTransform(&lc, opNum, hX, hXShapeInfo, dX, dXShapeInfo, hY, hYShapeInfo, dY, dYShapeInfo, hZ, hZShapeInfo, dZ, dZShapeInfo, extraParams);
}

////////////////////////////////////////////////////////////////////////
void NativeOps::execPairwiseTransformBool(Nd4jPointer *extraPointers,
        								int opNum,
        								void *hX, Nd4jLong *hXShapeInfo,
        								void *dX, Nd4jLong *dXShapeInfo,
        								void *hY, Nd4jLong *hYShapeInfo,
        								void *dY, Nd4jLong *dYShapeInfo,
        								void *hZ, Nd4jLong *hZShapeInfo,
        								void *dZ, Nd4jLong *dZShapeInfo,
        								void *extraParams) {

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
	NativeOpExecutioner::execPairwiseBoolTransform(&lc, opNum, hX, hXShapeInfo, dX, dXShapeInfo, hY, hYShapeInfo, dY, dYShapeInfo, hZ, hZShapeInfo, dZ, dZShapeInfo, extraParams);
}

////////////////////////////////////////////////////////////////////////
void NativeOps::execSummaryStatsScalar(Nd4jPointer *extraPointers,
                                       int opNum,
                                       void *hX, Nd4jLong *hXShapeInfo,
                                       void *dX, Nd4jLong *dXShapeInfo,
                                       void *extraParams,
                                       void *hZ, Nd4jLong *hZShapeInfo,
                                       void *dZ, Nd4jLong *dZShapeInfo,
                                       bool biasCorrected) {

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
	NativeOpExecutioner::execSummaryStatsScalar(&lc, opNum, hX, hXShapeInfo, dX, dXShapeInfo, extraParams, hZ, hZShapeInfo, dZ, dZShapeInfo, biasCorrected);
}

////////////////////////////////////////////////////////////////////////
void NativeOps::execBroadcastBool(Nd4jPointer *extraPointers,
        						int opNum,
        						void *hX, Nd4jLong *hXShapeInfo,
        						void *dX, Nd4jLong *dXShapeInfo,
        						void *hY, Nd4jLong *hYShapeInfo,
        						void *dY, Nd4jLong *dYShapeInfo,
        						void *hZ, Nd4jLong *hZShapeInfo,
        						void *dZ, Nd4jLong *dZShapeInfo,
        						void *hDimension, Nd4jLong *hDimensionShape,
		void *dDimension, Nd4jLong *dDimensionShape) {

	//Nd4jLong *tadOnlyShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[0]);
    //Nd4jLong *tadOffsets = reinterpret_cast<Nd4jLong *>(extraPointers[1]);
    //Nd4jLong *tadOnlyShapeInfoZ = reinterpret_cast<Nd4jLong *>(extraPointers[2]);
    //Nd4jLong *tadOffsetsZ = reinterpret_cast<Nd4jLong *>(extraPointers[3]);

	auto dimension = reinterpret_cast<int *>(dDimension);
	int dimensionLength = static_cast<int>(shape::length(hDimensionShape));

	auto hTADShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[9]);
	auto tadOnlyShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[10]);
	auto tadOffsets = reinterpret_cast<Nd4jLong *>(extraPointers[11]);
	auto tadOnlyShapeInfoZ = reinterpret_cast<Nd4jLong *>(extraPointers[12]);
	auto tadOffsetsZ = reinterpret_cast<Nd4jLong *>(extraPointers[13]);

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
    NativeOpExecutioner::execBroadcastBool(&lc, opNum, hX, hXShapeInfo, dX, dXShapeInfo, hY, hYShapeInfo, dY, dYShapeInfo, hZ, hZShapeInfo, dZ, dZShapeInfo, dimension, dimensionLength, tadOnlyShapeInfo, tadOffsets, tadOnlyShapeInfoZ, tadOffsetsZ);

}

/**
 *
 * @param opNum
 * @param dX
 * @param dXShapeInfo
 * @param dY
 * @param dYShapeInfo
 * @param dZ
 * @param dZShapeInfo
 * @param dimension
 * @param dimensionLength
 */
void   NativeOps::execBroadcast(
		Nd4jPointer *extraPointers,
		int opNum,
		void *hX, Nd4jLong *hXShapeInfo,
		void *dX, Nd4jLong *dXShapeInfo,
		void *hY, Nd4jLong *hYShapeInfo,
		void *dY, Nd4jLong *dYShapeInfo,
		void *hZ, Nd4jLong *hZShapeInfo,
		void *dZ, Nd4jLong *dZShapeInfo,
		void *hDimension, Nd4jLong *hDimensionShape,
		void *dDimension, Nd4jLong *dDimensionShape) {
/*
    hipEvent_t start;
    hipEventCreateWithFlags(&start, hipEventDisableTiming);
    timespec tsX;
    timespec tsY;
    clock_gettime(CLOCK_REALTIME, &tsX);
*/
	auto dimension = reinterpret_cast<int *>(dDimension);
	int dimensionLength = static_cast<int>(shape::length(hDimensionShape));

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);

	auto hTADShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[9]);
	auto tadOnlyShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[10]);
	auto tadOffsets = reinterpret_cast<Nd4jLong *>(extraPointers[11]);
	auto tadOnlyShapeInfoZ = reinterpret_cast<Nd4jLong *>(extraPointers[12]);
	auto tadOffsetsZ = reinterpret_cast<Nd4jLong *>(extraPointers[13]);

	auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
	auto yType = nd4j::ArrayOptions::dataType(hYShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

	if (nd4j::Environment::getInstance()->isDebugAndVerbose())
		printf("F3 opNum:[%i]\n", opNum);

	//Nd4jLong *tadOnlyShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[0]);
    //Nd4jLong *tadOffsets = reinterpret_cast<Nd4jLong *>(extraPointers[1]);
    //Nd4jLong *tadOnlyShapeInfoZ = reinterpret_cast<Nd4jLong *>(extraPointers[2]);
    //Nd4jLong *tadOffsetsZ = reinterpret_cast<Nd4jLong *>(extraPointers[3]);

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
    NativeOpExecutioner::execBroadcast(&lc, opNum, hX, hXShapeInfo, dX, dXShapeInfo, hY, hYShapeInfo, dY, dYShapeInfo, hZ, hZShapeInfo, dZ, dZShapeInfo, dimension, dimensionLength, tadOnlyShapeInfo, tadOffsets, tadOnlyShapeInfoZ, tadOffsetsZ);
}


/**
 *
 * @param opNum
 * @param dX
 * @param dXShapeInfo
 * @param extraParams
 * @param dZ
 * @param dZShapeInfo
 */
////////////////////////////////////////////////////////////////////////
void NativeOps::execReduceFloat(Nd4jPointer *extraPointers,
							int opNum,
							void *hX, Nd4jLong *hXShapeInfo,
							void *dX, Nd4jLong *dXShapeInfo,
							void *extraParams,
							void *hZ, Nd4jLong *hZShapeInfo,
							void *dZ, Nd4jLong *dZShapeInfo) {

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
	NativeOpExecutioner::execReduceFloatScalar(&lc, opNum, hX, hXShapeInfo, dX, dXShapeInfo, extraParams, hZ, hZShapeInfo, dZ, dZShapeInfo);
}

////////////////////////////////////////////////////////////////////////
void NativeOps::execReduceSame(Nd4jPointer *extraPointers,
                                int opNum,
                                void *hX, Nd4jLong *hXShapeInfo,
                                void *dX, Nd4jLong *dXShapeInfo,
                                void *extraParams,
                                void *hZ, Nd4jLong *hZShapeInfo,
                                void *dZ, Nd4jLong *dZShapeInfo) {

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
    NativeOpExecutioner::execReduceSameScalar(&lc, opNum, hX, hXShapeInfo, dX, dXShapeInfo, extraParams, hZ, hZShapeInfo, dZ, dZShapeInfo);
}

////////////////////////////////////////////////////////////////////////
void NativeOps::execReduceSame(Nd4jPointer *extraPointers,
                            int opNum,
                            void *hX, Nd4jLong *hXShapeInfo,
                            void *dX, Nd4jLong *dXShapeInfo,
                            void *extraParams,
                            void *hZ, Nd4jLong *hZShapeInfo,
                            void *dZ, Nd4jLong *dZShapeInfo,
							   void *hDimension, Nd4jLong *hDimensionShape,
							   void *dDimension, Nd4jLong *dDimensionShape) {
	auto dimension = reinterpret_cast<int *>(dDimension);
	int dimensionLength = static_cast<int>(shape::length(hDimensionShape));

	auto tadPack = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(hXShapeInfo, reinterpret_cast<int*>(hDimension), shape::length(hDimensionShape));

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
    NativeOpExecutioner::execReduceSame(&lc, opNum, hX, hXShapeInfo, dX, dXShapeInfo, extraParams, hZ, hZShapeInfo, dZ, dZShapeInfo, dimension, dimensionLength, tadPack.specialShapeInfo(), tadPack.specialOffsets());
}

////////////////////////////////////////////////////////////////////////
void NativeOps::execReduceLong(Nd4jPointer *extraPointers,
                            int opNum,
                            void *hX, Nd4jLong *hXShapeInfo,
                            void *dX, Nd4jLong *dXShapeInfo,
                            void *extraParams,
                            void *hZ, Nd4jLong *hZShapeInfo,
                            void *dZ, Nd4jLong *dZShapeInfo,
							   void *hDimension, Nd4jLong *hDimensionShape,
							   void *dDimension, Nd4jLong *dDimensionShape) {
	auto dimension = reinterpret_cast<int *>(dDimension);
	int dimensionLength = static_cast<int>(shape::length(hDimensionShape));

    auto tadPack = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(hXShapeInfo, reinterpret_cast<int*>(hDimension), shape::length(hDimensionShape));

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
    NativeOpExecutioner::execReduceLong(&lc, opNum, hX, hXShapeInfo, dX, dXShapeInfo, extraParams, hZ, hZShapeInfo, dZ, dZShapeInfo, dimension, dimensionLength, tadPack.specialShapeInfo(), tadPack.specialOffsets());
}

////////////////////////////////////////////////////////////////////////
void   NativeOps::execReduceLong(Nd4jPointer *extraPointers,
                                int opNum,
                                void *hX, Nd4jLong *hXShapeInfo,
                                void *dX, Nd4jLong *dXShapeInfo,
                                void *extraParams,
                                void *hZ, Nd4jLong *hZShapeInfo,
                                void *dZ, Nd4jLong *dZShapeInfo) {

    auto stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);
    auto hTADShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[9]);
	auto dTADShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[10]);

    if (nd4j::Environment::getInstance()->isDebugAndVerbose())
        printf("LF7 opNum:[%i]\n", opNum);

    auto reductionPointer = reinterpret_cast<void *>(extraPointers[4]);

    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

    if (zType != nd4j::DataType::INT64)
        throw datatype_exception::build("NativeOps::execReduceLong wrong Z data type", nd4j::DataType::INT64, zType);

    auto xLength = shape::length(hXShapeInfo);
    auto blockWidth = 256;
    auto numBlocks = CudaLaunchHelper::getReductionBlocks(xLength, blockWidth);
    dim3 launchDims(numBlocks, blockWidth, 32768);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::reduce::ReduceLongFunction, ::execReduceScalar(launchDims, stream, opNum, dX, dXShapeInfo, extraParams, dZ, dZShapeInfo, nullptr, 0, reductionPointer, dTADShapeInfo), LIBND4J_TYPES, LONG_TYPES);

    nd4j::DebugHelper::checkErrorCode(stream, "execReduceLong(...) failed");
}

////////////////////////////////////////////////////////////////////////
void NativeOps::execReduceBool(Nd4jPointer *extraPointers,
                            int opNum,
                            void *hX, Nd4jLong *hXShapeInfo,
                            void *dX, Nd4jLong *dXShapeInfo,
                            void *extraParams,
                            void *hZ, Nd4jLong *hZShapeInfo,
                            void *dZ, Nd4jLong *dZShapeInfo,
							   void *hDimension, Nd4jLong *hDimensionShape,
							   void *dDimension, Nd4jLong *dDimensionShape) {
	auto dimension = reinterpret_cast<int *>(dDimension);
	int dimensionLength = static_cast<int>(shape::length(hDimensionShape));

    auto tadPack = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(hXShapeInfo, reinterpret_cast<int*>(hDimension), shape::length(hDimensionShape));

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
	NativeOpExecutioner::execReduceBool(&lc, opNum, hX, hXShapeInfo, dX, dXShapeInfo, extraParams, hZ, hZShapeInfo, dZ, dZShapeInfo, dimension, dimensionLength, tadPack.specialShapeInfo(), tadPack.specialOffsets());
}

////////////////////////////////////////////////////////////////////////
void   NativeOps::execReduceBool(Nd4jPointer *extraPointers,
                                int opNum,
                                void *hX, Nd4jLong *hXShapeInfo,
                                void *dX, Nd4jLong *dXShapeInfo,
                                void *extraParams,
                                void *hZ, Nd4jLong *hZShapeInfo,
                                void *dZ, Nd4jLong *dZShapeInfo) {

    auto stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);
    auto hTADShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[9]);
	auto dTADShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[10]);

    if (nd4j::Environment::getInstance()->isDebugAndVerbose())
        printf("BF7 opNum:[%i]\n", opNum);

    auto reductionPointer = reinterpret_cast<void *>(extraPointers[4]);

    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

    if (zType != nd4j::DataType::BOOL)
        throw std::runtime_error("NativeOps::execReduceBool requires Z operand to have BOOL type");

    auto xLength = shape::length(hXShapeInfo);
    auto blockWidth = 256;
    auto numBlocks = CudaLaunchHelper::getReductionBlocks(xLength, blockWidth);
    dim3 launchDims(numBlocks, blockWidth, 32768);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::reduce::ReduceBoolFunction, ::execReduceScalar(launchDims, stream, opNum, dX, dXShapeInfo, extraParams, dZ, dZShapeInfo, nullptr, 0, reductionPointer, dTADShapeInfo), LIBND4J_TYPES, BOOL_TYPES);

    nd4j::DebugHelper::checkErrorCode(stream, "execReduceBool(...) failed");
}

/**
 *
 * @param opNum
 * @param dX
 * @param dXShapeInfo
 * @param extraParams
 * @param dZ
 * @param dZShapeInfo
 * @param dimension
 * @param dimensionLength
 */
////////////////////////////////////////////////////////////////////////
void NativeOps::execIndexReduce(Nd4jPointer *extraPointers,
								 int opNum,
								 void *hX, Nd4jLong *hXShapeInfo,
        						 void *dX, Nd4jLong *dXShapeInfo,
        						 void *extraParams,
        						 void *hZ, Nd4jLong *hZShapeInfo,
        						 void *dZ, Nd4jLong *dZShapeInfo,
								 void *hDimension, Nd4jLong *hDimensionShape,
		void *dDimension, Nd4jLong *dDimensionShape) {
	auto dimension = reinterpret_cast<int *>(dDimension);
	int dimensionLength = static_cast<int>(shape::length(hDimensionShape));

    auto tadPack = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(hXShapeInfo, reinterpret_cast<int*>(hDimension), shape::length(hDimensionShape));

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
    NativeOpExecutioner::execIndexReduce(&lc, opNum, hX, hXShapeInfo, dX, dXShapeInfo, extraParams, hZ, hZShapeInfo, dZ, dZShapeInfo, dimension, dimensionLength, tadPack.specialShapeInfo(), tadPack.specialOffsets());
}

/**
 *
 * @param opNum
 * @param dX
 * @param dXShapeInfo
 * @param extraParams
 * @param dZ
 * @param dZShapeInfo
 */
////////////////////////////////////////////////////////////////////////
void NativeOps::execReduceFloat(Nd4jPointer *extraPointers,
								int opNum,
								void *hX, Nd4jLong *hXShapeInfo,
        						void *dX, Nd4jLong *dXShapeInfo,
        						void *extraParams,
        						void *hZ, Nd4jLong *hZShapeInfo,
								void *dZ, Nd4jLong *dZShapeInfo,
								void *hDimension, Nd4jLong *hDimensionShape,
		void *dDimension, Nd4jLong *dDimensionShape) {
	auto dimension = reinterpret_cast<int *>(dDimension);
	int dimensionLength = static_cast<int>(shape::length(hDimensionShape));

    auto tadPack = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(hXShapeInfo, reinterpret_cast<int*>(hDimension), shape::length(hDimensionShape));

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
    NativeOpExecutioner::execReduceFloat(&lc, opNum, hX, hXShapeInfo, dX, dXShapeInfo, extraParams, hZ, hZShapeInfo, dZ, dZShapeInfo, dimension, dimensionLength, tadPack.specialShapeInfo(), tadPack.specialOffsets());
}

/**
 *
 * @param opNum
 * @param dX
 * @param dXShapeInfo
 * @param extraParams
 */
////////////////////////////////////////////////////////////////////////
void NativeOps::execIndexReduceScalar(
		Nd4jPointer *extraPointers,
		int opNum,
		void *hX, Nd4jLong *hXShapeInfo,
        void *dX, Nd4jLong *dXShapeInfo,
        void *extraParams,
        void *hZ, Nd4jLong *hZShapeInfo,
		void *dZ, Nd4jLong *dZShapeInfo){

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
	NativeOpExecutioner::execIndexReduceScalar(&lc, opNum, hX, hXShapeInfo, dX, dXShapeInfo, extraParams, hZ, hZShapeInfo, dZ, dZShapeInfo);
}

////////////////////////////////////////////////////////////////////////
void NativeOps::execTransformSame(Nd4jPointer *extraPointers,int opNum,
                                   void *hX, Nd4jLong *hXShapeInfo,
                                   void *dX, Nd4jLong *dXShapeInfo,
                                   void *hZ, Nd4jLong *hZShapeInfo,
                                   void *dZ, Nd4jLong *dZShapeInfo,
                                   void *extraParams) {

    auto tadShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers != nullptr ? extraPointers[0] : nullptr);
    auto tadOffsets = reinterpret_cast<Nd4jLong *>(extraPointers != nullptr ? extraPointers[1] : nullptr);

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
    NativeOpExecutioner::execTransformSame(&lc, opNum, hX, hXShapeInfo, dX, dXShapeInfo, hZ, hZShapeInfo, dZ, dZShapeInfo, extraParams, tadShapeInfo, tadOffsets);
}

////////////////////////////////////////////////////////////////////////
void NativeOps::execTransformBool(Nd4jPointer *extraPointers,int opNum,
								  void *hX, Nd4jLong *hXShapeInfo,
								  void *dX, Nd4jLong *dXShapeInfo,
								  void *hZ, Nd4jLong *hZShapeInfo,
								  void *dZ, Nd4jLong *dZShapeInfo,
								  void *extraParams) {

	auto tadShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers != nullptr ? extraPointers[0] : nullptr);
    auto tadOffsets = reinterpret_cast<Nd4jLong *>(extraPointers != nullptr ? extraPointers[1] : nullptr);

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
    NativeOpExecutioner::execTransformBool(&lc, opNum, hX, hXShapeInfo, dX, dXShapeInfo, hZ, hZShapeInfo, dZ, dZShapeInfo, extraParams, tadShapeInfo, tadOffsets);
}

////////////////////////////////////////////////////////////////////////
void NativeOps::execTransformAny(Nd4jPointer *extraPointers,int opNum,
								  void *hX, Nd4jLong *hXShapeInfo,
								  void *dX, Nd4jLong *dXShapeInfo,
								  void *hZ, Nd4jLong *hZShapeInfo,
								  void *dZ, Nd4jLong *dZShapeInfo,
								  void *extraParams) {

    auto stream = reinterpret_cast<hipStream_t*>(extraPointers[1]);
    LaunchContext lc(stream, extraPointers[4], extraPointers[5], extraPointers[3]);

    // FIXME: remove this once all operations are enabled
    if (opNum == nd4j::transform::IsMax && extraParams != nullptr) {
        auto hostYShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[7]);
        auto hostTShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[19]);
        auto tadMaxShapeInfo = reinterpret_cast<Nd4jLong *> (extraPointers[10]);
        auto tadMaxOffsets = reinterpret_cast<Nd4jLong *> (extraPointers[11]);
        int *dimension = reinterpret_cast<int *> (extraPointers[15]);
        int *hDimension = reinterpret_cast<int *> (extraPointers[16]);
        int dimensionLength = getDeviceId(extraPointers[18]);
        auto special = reinterpret_cast<double *>(extraPointers[17]);

        auto cshape = ShapeBuilders::createVectorShapeInfo(nd4j::DataType::INT32, dimensionLength);

        // we call for IMax on specified dimension
        execIndexReduce(extraPointers, indexreduce::IndexMax, nullptr, hXShapeInfo, dX, dXShapeInfo, extraParams, nullptr, hostTShapeInfo, special, hostYShapeInfo, hDimension, cshape, dimension, nullptr);

        DEBUG_KERNEL(stream, opNum);

        dim3 launchDims(256, 256, 16384);
        auto zType = ArrayOptions::dataType(hZShapeInfo);

        // at this point, all IMax indexes are gathered, and we execute filler
        BUILD_SINGLE_SELECTOR(zType, fillDimensionalIsMaxGeneric, (launchDims, stream, special, dZ, dZShapeInfo, tadMaxShapeInfo, dimension, dimensionLength, tadMaxOffsets), LIBND4J_TYPES);

        nd4j::DebugHelper::checkErrorCode(stream, "Legacy IsMax(...) failed");

        delete[] cshape;
    } else {
        NativeOpExecutioner::execTransformAny(&lc, opNum, hX, hXShapeInfo, dX, dXShapeInfo, hZ, hZShapeInfo, dZ, dZShapeInfo, extraParams, nullptr, nullptr);
    }
}

////////////////////////////////////////////////////////////////////////
void NativeOps::execTransformStrict(Nd4jPointer *extraPointers,int opNum,
                                  void *hX, Nd4jLong *hXShapeInfo,
                                  void *dX, Nd4jLong *dXShapeInfo,
                                  void *hZ, Nd4jLong *hZShapeInfo,
                                  void *dZ, Nd4jLong *dZShapeInfo,
                                  void *extraParams) {

    auto tadShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers != nullptr ? extraPointers[10] : nullptr);
    auto tadOffsets = reinterpret_cast<Nd4jLong *>(extraPointers != nullptr ? extraPointers[11] : nullptr);

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
    NativeOpExecutioner::execTransformStrict(&lc, opNum, hX, hXShapeInfo, dX, dXShapeInfo, hZ, hZShapeInfo, dZ, dZShapeInfo, extraParams, tadShapeInfo, tadOffsets);
}

////////////////////////////////////////////////////////////////////////
void NativeOps::execTransformFloat(Nd4jPointer *extraPointers,int opNum,
                                    void *hX, Nd4jLong *hXShapeInfo,
                                    void *dX, Nd4jLong *dXShapeInfo,
                                    void *hZ, Nd4jLong *hZShapeInfo,
                                    void *dZ, Nd4jLong *dZShapeInfo,
                                    void *extraParams) {

    auto tadShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers != nullptr ? extraPointers[10] : nullptr);
    auto tadOffsets = reinterpret_cast<Nd4jLong *>(extraPointers != nullptr ? extraPointers[11] : nullptr);

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
    NativeOpExecutioner::execTransformFloat(&lc, opNum, hX, hXShapeInfo, dZ, dXShapeInfo, hZ, hZShapeInfo, dZ, dZShapeInfo, extraParams, tadShapeInfo, tadOffsets);
}


/**
 * Append an input array
 * to the end of a flat array
 * in a particular order
 * @param offset the offset of the array to start at
 * @param order the order
 * @param dZ the dZ array
 * @param dZShapeInfo the shape info for te array
 * @param input the input for the array
 * @param inputShapeInfo the shape information for that array
 */
void NativeOps::flatten(Nd4jPointer *extraPointers,
						int offset,
						char order,
						void *hZ, Nd4jLong *hZShapeInfo,
						void *dZ, Nd4jLong *dZShapeInfo,
						void *hInput, Nd4jLong *hInputShapeInfo,
						void *dInput, Nd4jLong *dInputShapeInfo) {

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);
	auto hYShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[7]);

	if (nd4j::Environment::getInstance()->isDebugAndVerbose())
		printf("F22 opNum:[7]\n");

	// int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);

	dim3 launchDims(256, 256, 2048);

	if (nd4j::Environment::getInstance()->isVerbose() && launchDims.x == 1)
		printf("AF222 opNum:[7]\n");

	auto type = nd4j::ArrayOptions::dataType(hInputShapeInfo);
    BUILD_SINGLE_SELECTOR(type, flattenKernelGeneric, (launchDims, stream, extraPointers, offset, order, dZ, dZShapeInfo, dInput, dInputShapeInfo), LIBND4J_TYPES);

	DEBUG_KERNEL(stream, -1);
}



void NativeOps::checkP2P() {
	int curDevice = 0;

	hipGetDevice(&curDevice);

	int devCnt = 0;
	hipGetDeviceCount(&devCnt);

	if (curDevice < 0 && curDevice > devCnt)
		curDevice = 0;

	bool tempSupport = true;

	if (devCnt > 1) {
		for (int dX = 0; dX < devCnt; dX++) {

			for (int dY = 0; dY < devCnt; dY++) {
				if (dX == dY)
					continue;

				int canAccess = 0;
				hipSetDevice(dX);

				hipDeviceCanAccessPeer(&canAccess, dX , dY);

				if (!canAccess) {
                    tempSupport = false;
                    break;
                }
			}
		}

		supportedP2P = tempSupport;

		hipSetDevice(curDevice);
	} else {
		// if we have only 1 device - we say that we support P2P, since all data will be on 1 device
		supportedP2P = true;
	}
}

void NativeOps::enableP2P(bool enable) {
    if (enable == allowedP2P)
        return;

    int curDevice = 0;

    hipGetDevice(&curDevice);

    int devCnt = 0;
    hipGetDeviceCount(&devCnt);

	if (curDevice < 0 && curDevice > devCnt)
		curDevice = 0;

    if (devCnt > 1) {
        for (int dX = 0; dX < devCnt; dX++) {

            for (int dY = 0; dY < devCnt; dY++) {
                if (dX == dY)
                    continue;

                int canAccess = 0;
                hipSetDevice(dX);

                hipDeviceCanAccessPeer(&canAccess, dX , dY);

                if (canAccess) {
                    if (enable) {
                        hipDeviceEnablePeerAccess(dY, 0);
                    } else {
                        hipDeviceDisablePeerAccess(dY);
                    }
                } else {
					if (nd4j::Environment::getInstance()->isVerbose()) printf("Peer access [%i] -> [%i] isn't possible\n", dX, dY);
				}
            }
        }

        hipSetDevice(curDevice);
    }

    allowedP2P = enable;

    hipSetDevice(curDevice);
}

bool NativeOps::isP2PAvailable() {
	return supportedP2P;
}


void NativeOps::initializeDevicesAndFunctions() {
	int devCnt = 0;
	hipGetDeviceCount(&devCnt);
	deviceProperties = new hipDeviceProp_t[devCnt];
	for (int i = 0; i < devCnt; i++) {
		hipSetDevice(i);
		hipGetDeviceProperties(&deviceProperties[i], i);

		hipDeviceSetLimit(hipLimitStackSize, 4096);
	}

	hipSetDevice(0);

	checkP2P();

	// enabling p2p gpu access if it's supported
	if (supportedP2P && devCnt > 1)
    	enableP2P(allowedP2P);
}

void NativeOps::initializeFunctions(Nd4jPointer *functions) {
    nd4j::BlasHelper::getInstance()->initializeDeviceFunctions(functions);
	/*
	this->hipblasSgemv = (CublasSgemv)functions[0];
    this->hipblasDgemv = (CublasDgemv)functions[1];
    this->hipblasHgemm = (CublasHgemm)functions[2];
    this->hipblasSgemm = (CublasSgemm)functions[3];
    this->hipblasDgemm = (CublasDgemm)functions[4];
    this->cublasSgemmEx = (CublasSgemmEx)functions[5];
    this->hipblasHgemmBatched = (CublasHgemmBatched)functions[6];
    this->hipblasSgemmBatched = (CublasSgemmBatched)functions[7];
    this->hipblasDgemmBatched = (CublasDgemmBatched)functions[8];
	*/
}


/**
 * This method acquires memory chunk of requested size on host side
 *
 * @param pointer pointer that'll be used for allocation
 * @param memorySize memory size, in bytes
 * @param flags optional parameter
 */
Nd4jPointer NativeOps::mallocHost(Nd4jLong memorySize, int flags) {
	Nd4jPointer pointer;
	// hipHostMallocMapped |hipHostMallocPortable
	hipError_t res = hipHostAlloc(reinterpret_cast<void **>(&pointer), memorySize, hipHostMallocDefault);
	if (res != 0)
		pointer = 0L;
	return pointer;
}

/**
 * This method acquires memory chunk of requested size on specified device
 *
 * @param pointer pointer that'll be used for allocation
 * @param memorySize memory size, in bytes
 * @param ptrToDeviceId pointer to deviceId. For cuda that's just and int, for OpenCL that's pointer to device_id, etc
 * @param flags optional parameter
 */
Nd4jPointer NativeOps::mallocDevice(Nd4jLong memorySize, int deviceId, int flags) {
	Nd4jPointer pointer;
	auto res = hipMalloc(reinterpret_cast<void **>(&pointer), memorySize);
	if (res != 0)
		pointer = 0L;
	return pointer;
}

/**
 * This method releases previously allocated host memory space
 *
 * @param pointer pointer that'll be freed
 */
int NativeOps::freeHost(Nd4jPointer pointer) {
	hipError_t res = hipHostFree(reinterpret_cast<void *>(pointer));
	if (res != 0)
		pointer = 0L;
	return 1L;
}

/**
 * This method releases previously allocated memory space on device
 *
 * @param pointer pointer that'll be freed
 * @param ptrToDeviceId pointer to deviceId.
 */
int NativeOps::freeDevice(Nd4jPointer pointer, int deviceId) {
	hipError_t res = hipFree(reinterpret_cast<void *>(pointer));
	if (res != 0)
		pointer = 0L;
	return 1L;
}


Nd4jPointer NativeOps::createContext() {
	return 0L;
}

Nd4jPointer NativeOps::createStream() {
    /*
	Nd4jPointer nativeStream = (Nd4jPointer) malloc(sizeof(hipStream_t));

	CHECK_ALLOC(nativeStream, "Failed to allocate memory for new CUDA stream", sizeof(hipStream_t));

	hipError_t dZ = hipStreamCreate(reinterpret_cast<hipStream_t *>(&nativeStream));
	checkCudaErrors(dZ);
	if (dZ != 0)
		throw std::runtime_error("hipStreamCreate(...) failed");

	return nativeStream;
     */
    auto stream = new hipStream_t();
    auto dZ = hipStreamCreate(stream);
    if (dZ != 0)
        throw std::runtime_error("hipStreamCreate(...) failed");

    return stream;
}

Nd4jPointer NativeOps::createEvent() {
	Nd4jPointer nativeEvent= (Nd4jPointer) malloc(sizeof(hipEvent_t));

	CHECK_ALLOC(nativeEvent, "Failed to allocate new CUDA event buffer", sizeof(hipEvent_t));

	hipError_t dZ = hipEventCreateWithFlags(reinterpret_cast<hipEvent_t *>(&nativeEvent), hipEventDisableTiming);
	checkCudaErrors(dZ);
	if (dZ != 0)
		throw std::runtime_error("hipEventCreateWithFlags(...) failed");


	return nativeEvent;
}

int NativeOps::registerEvent(Nd4jPointer event, Nd4jPointer stream) {
	hipEvent_t *pEvent = reinterpret_cast<hipEvent_t *>(&event);
	hipStream_t *pStream = reinterpret_cast<hipStream_t *>(stream);

	hipError_t dZ = hipEventRecord(*pEvent, *pStream);
	checkCudaErrors(dZ);
	if (dZ != 0)
		throw std::runtime_error("hipEventRecord(...) failed");

	return 1;
}

int NativeOps::setDevice(int deviceId) {
	auto dZ = hipSetDevice(deviceId);
	checkCudaErrors(dZ);
	if (dZ != 0)
		throw std::runtime_error("hipSetDevice(...) failed");

	return 1;
}

Nd4jLong NativeOps::getDeviceFreeMemory() {
    size_t memFree = 0;
    size_t memTotal = 0;

    hipMemGetInfo(&memFree, &memTotal);

    return (Nd4jLong) memFree;
}

Nd4jLong NativeOps::getDeviceFreeMemory(int device) {
	int orig = -1;

	hipGetDevice(&orig);

	if (device >= 0 && device != orig) {
		hipSetDevice(device);
	}

	size_t memFree = 0;
	size_t memTotal = 0;

	hipMemGetInfo(&memFree, &memTotal);

	if (device >= 0 && device != orig) {
		hipSetDevice(orig);
	}

	return (Nd4jLong) memFree;
}

Nd4jLong NativeOps::getDeviceTotalMemory(int device) {
	int orig = -1;

	hipGetDevice(&orig);

	if (device >= 0 && device != orig) {
		hipSetDevice(device);
	}
	size_t memFree = 0;
	size_t memTotal = 0;

	hipMemGetInfo(&memFree, &memTotal);

	if (device >= 0 && device != orig) {
		hipSetDevice(orig);
	}

	return (Nd4jLong) memTotal;
}

int NativeOps::memcpy(Nd4jPointer dst, Nd4jPointer src, Nd4jLong size, int flags, Nd4jPointer reserved) {

	return memcpyAsync(dst, src, size, flags, reserved);
}

int NativeOps::memcpyAsync(Nd4jPointer dst, Nd4jPointer src, Nd4jLong size, int flags, Nd4jPointer reserved) {
	hipStream_t *pStream = reinterpret_cast<hipStream_t *>(reserved);

	hipMemcpyKind 	kind;

	DEBUG_KERNEL(pStream, 0);

	switch (flags) {
		case 0: {
				kind = hipMemcpyHostToHost;
			}
			break;
		case 1: {
				kind = hipMemcpyHostToDevice;
			}
			break;
		case 2: {
				kind = hipMemcpyDeviceToHost;
			}
		case 3: {
			kind = hipMemcpyDeviceToDevice;
		}
			break;
		default: {

			printf("UNDEFINED MEMCPY!\n");
			break;
		}
	}

	hipError_t dZ = hipMemcpyAsync(reinterpret_cast<void *>(dst), const_cast<const void *>(reinterpret_cast<void *>(src)), static_cast<size_t>(size), kind, *pStream);
	if (dZ != 0) {
        checkCudaErrors(dZ);
		printf("Failed on [%lu] -> [%lu], size: [%i], direction: [%i], dZ: [%i]\n", src, dst, size, flags, static_cast<int>(dZ));
        fflush(stdout);
        fflush(stderr);
        throw std::runtime_error("hipMemcpyAsync(...) failed");
		//return 0L;
	}

	return 1;
}

int NativeOps::memset(Nd4jPointer dst, int value, Nd4jLong size, int flags, Nd4jPointer reserved) {
	hipError_t dZ = hipMemset(reinterpret_cast<void *>(dst), value, static_cast<size_t>(size));
	checkCudaErrors(dZ);
	if (dZ != 0)
		throw std::runtime_error("hipMemset(...) failed");

	return 1;
}

int NativeOps::memsetAsync(Nd4jPointer dst, int value, Nd4jLong size, int flags, Nd4jPointer reserved) {
	hipStream_t *pStream = reinterpret_cast<hipStream_t *>(reserved);

	hipError_t dZ = hipMemsetAsync(reinterpret_cast<void *>(dst), value, static_cast<size_t>(size), *pStream);
	checkCudaErrors(dZ);
	if (dZ != 0)
		throw std::runtime_error("hipMemsetAsync(...) failed");

	return 1;
}

int NativeOps::destroyEvent(Nd4jPointer event) {
	hipEvent_t *pEvent = reinterpret_cast<hipEvent_t *>(&event);
	hipError_t dZ = hipEventDestroy(*pEvent);
	checkCudaErrors(dZ);
	if (dZ != 0)
		throw std::runtime_error("cudaEvenDestroy(...) failed");

	return 1;
}

int NativeOps::streamSynchronize(Nd4jPointer stream) {
	hipStream_t *pStream = reinterpret_cast<hipStream_t *>(stream);

	hipError_t dZ = hipStreamSynchronize(*pStream);
	checkCudaErrors(dZ);
	if (dZ != 0)
        throw std::runtime_error("hipStreamSynchronize(...) failed");

	return 1L;
}

int NativeOps::eventSynchronize(Nd4jPointer event) {
	hipEvent_t *pEvent = reinterpret_cast<hipEvent_t *>(&event);

	hipError_t dZ = hipEventSynchronize(*pEvent);
	checkCudaErrors(dZ);
	if (dZ != 0)
        throw std::runtime_error("hipEventSynchronize(...) failed");

	return 1L;
}

int NativeOps::getAvailableDevices() {
	int devCnt = 0;
	hipGetDeviceCount(&devCnt);
	return devCnt;
}

void NativeOps::enableDebugMode(bool reallyEnable) {
	nd4j::Environment::getInstance()->setDebug(reallyEnable);
}

void NativeOps::setGridLimit(int gridSize) {
	if (gridSize > 8192)
		gridSize = 8192;
	if (gridSize < 1)
		gridSize = 1;
	blockLimit = gridSize;
}

int NativeOps::ompGetMaxThreads() {
	return maxThreads;
}

int NativeOps::ompGetNumThreads() {
	return maxThreads;
}

void NativeOps::setOmpNumThreads(int threads) {
	if (threads > 1024)
		threads = 1024;
	if (threads < 32)
		threads = 32;
	maxThreads = threads;
}

void NativeOps::enableVerboseMode(bool reallyEnable) {
	nd4j::Environment::getInstance()->setVerbose(reallyEnable);
}

int NativeOps::getDeviceMajor(int device) {
	return deviceProperties[device].major;
}

int NativeOps::getDeviceMinor(int device) {
	return deviceProperties[device].minor;
}


const char * NativeOps::getDeviceName(int device) {
    return deviceProperties[device].name;
}

///////////////////////////////////////////////////////////////////
template<typename T>
__global__ static void concatCuda(const int numOfArrs, void* pVx,  void* pxShapeInfo, void* pVz, void* pzShapeInfo) {

    __shared__ int arrIdx, blocksPerArr;
    __shared__ T *x, *z;
    __shared__ Nd4jLong *zShapeInfo, *xShapeInfo, arrLen, arrLenZ, arrLenPerBlock, start, end;

    if (threadIdx.x == 0) {

        blocksPerArr = (gridDim.x - gridDim.x % numOfArrs) / numOfArrs;     // floor
        arrIdx = blockIdx.x / blocksPerArr;
        if (arrIdx >= numOfArrs)
            arrIdx = numOfArrs - 1;
        x = reinterpret_cast<T*>(reinterpret_cast<void**>(pVx)[arrIdx]);
        z = reinterpret_cast<T*>(reinterpret_cast<void**>(pVz)[arrIdx]);
        xShapeInfo = reinterpret_cast<Nd4jLong**>(pxShapeInfo)[arrIdx];
        zShapeInfo = reinterpret_cast<Nd4jLong**>(pzShapeInfo)[arrIdx];

        arrLen = shape::length(xShapeInfo);
        arrLenZ = shape::length(zShapeInfo);
        arrLenPerBlock = (arrLen + blocksPerArr - arrLen % blocksPerArr) / blocksPerArr;  // ceil

        start = arrLenPerBlock * (blockIdx.x % blocksPerArr);
        end   = (start + arrLenPerBlock) > arrLen ? arrLen : (start + arrLenPerBlock);
    }

    __syncthreads();
    for (Nd4jLong i = threadIdx.x + start; i < end; i += blockDim.x) {
        auto zOffset = shape::getIndexOffset(i, zShapeInfo, arrLenZ);
        auto xOffset = shape::getIndexOffset(i, xShapeInfo, arrLen);
        //printf("z[%i][%lld] = x[%i][%lld]\n", arrIdx, zOffset, arrIdx, xOffset);
        z[zOffset] = x[xOffset];
    }
}
template<typename T>
__host__ static void concatCudaLauncher(const int numOfArrs, hipStream_t *stream,  void* pVx, void* pxShapeInfo, void* pVz, void* pzShapeInfo) {
    //int blocks = numOfArrs * 16; // >> 1 << 2);
    //nd4j_printf("gridDim.x is %i\n", blocks);
    //if (blocks > 8192)
    //    blocks = 8192; // restrict grid dims to 8K max
    concatCuda<T><<<numOfArrs, 128, 512, *stream>>>(numOfArrs, pVx, pxShapeInfo, pVz, pzShapeInfo);
    nd4j::DebugHelper::checkErrorCode(stream, "concat(...) failed");
}
BUILD_SINGLE_TEMPLATE(template void concatCudaLauncher, (const int numOfArrs, hipStream_t *stream,  void* pVx, void* pxShapeInfo, void* pVz, void* pzShapeInfo), LIBND4J_TYPES);

static void
specialBufferAndShapeWithOffset(void* vZ, Nd4jLong* hZShapeInfo, Nd4jLong* dZShapeInfo, std::vector<Nd4jLong> const& idx, void*& outBuffer, Nd4jLong*& outShape) {
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);
    const int rank = shape::rank(hZShapeInfo);
    Nd4jLong* newShape = new Nd4jLong[shape::shapeInfoLength(rank)];
    //ALLOCATE(newShape, nullptr, , Nd4jLong)
    auto shapeSize = shape::shapeInfoByteLength(rank);
    memcpy(newShape, hZShapeInfo, shapeSize);

    auto shapeOf = shape::shapeOf(newShape);
    auto stridesOf = shape::stride(newShape);

    Nd4jLong offset(0), subArrLen(1);
    int n(2), first, last, stride;

    for (int d = rank - 1; d >= 0; --d) {

        if (idx[n * d] != idx[n * d + 1]) {
            auto axeDim = shape::sizeAt(hZShapeInfo, d);
            first  = idx[n * d]     >= 0 ? idx[n * d]     : idx[n * d]     + axeDim + 1;
            last   = idx[n * d + 1] >= 0 ? idx[n * d + 1] : idx[n * d + 1] + axeDim + 1;
            stride = 1;

            shapeOf[d] = (last - first + stride - 1) / stride;      // ceil (last - first) / stride;
            offset += first * stridesOf[d];

            if(shapeOf[d] != 1)
                stridesOf[d] *= stride;
        }

        subArrLen *= shapeOf[d];
    }

    // check if there is possibility to set ews = 1
    //shape::setEws(newShape, subArrLen);

    //makeBothBuffersActual();
    outBuffer = (void*)((int8_t*)vZ + offset * DataTypeUtils::sizeOfElement(zType));
    hipError_t err = hipMalloc(&outShape, shapeSize);
    if (err != 0) {
        printf("Cannot allocate memory with error %d\n", err);
        throw std::runtime_error("Cannot allocate memory for shape");
    }
    hipMemcpy(outShape, newShape, shapeSize, hipMemcpyHostToDevice);
    delete [] newShape;
}

/**
  * Concatneate multi array of the same shape together
  * along a particular dimension
  */
void NativeOps::concat(
        Nd4jPointer *extraPointers,
        int dimension,
        int numArrays,
        Nd4jPointer *data, Nd4jPointer *inputShapeInfo,
        Nd4jPointer *ddata, Nd4jPointer *dinputShapeInfo,
        void *hZ, Nd4jLong *hZShapeInfo,
        void *dZ, Nd4jLong *dZShapeInfo,
        Nd4jPointer *tadPointers, Nd4jPointer *offsetPointers) {

    auto stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);
    auto hXShapeInfo = hZShapeInfo;
    auto hShapePointers = reinterpret_cast<Nd4jLong **>(inputShapeInfo);
    auto dShapePointers = reinterpret_cast<Nd4jLong **>(dinputShapeInfo);
    // numArrays will be used as number of TADs, so each block process 1 input
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);
    auto axis = dimension;

    const int rank  = shape::rank(hZShapeInfo); //reinterpret_cast<Nd4jLong*>(inputShapeInfo[0]));
    const int rank2 = 2 * rank;
    std::vector<std::vector<Nd4jLong>> indices(numArrays, std::vector<Nd4jLong>(rank2,0));

    // take into account indices for first array
    auto axisSize = shape::sizeAt(reinterpret_cast<Nd4jLong*>(inputShapeInfo[0]), axis);
    indices[0][2 * axis + 1] = axisSize;
    printf("The axe size is %lld\n", axisSize);
    // loop through the rest of input arrays
    for(int i = 1; i < numArrays; ++i) {
        indices[i][2 * axis]     = indices[i-1][2 * axis + 1];                                // index start from
        indices[i][2 * axis + 1] = indices[i-1][2 * axis + 1] + shape::sizeAt(reinterpret_cast<Nd4jLong*>(inputShapeInfo[i]), axis);      // index end with (excluding)
    }

    std::vector<void*> outSubArrsBuffs(numArrays);
    std::vector<Nd4jLong*> outSubArrsShapes(numArrays);
    for(int i = 0; i < numArrays; ++i) {
        specialBufferAndShapeWithOffset(dZ, hZShapeInfo, dZShapeInfo, indices[i], outSubArrsBuffs[i], outSubArrsShapes[i]);
    }

    // prepare arrays of pointers on buffers and shapes
    std::vector<void*>     hOutBuffers(numArrays), hInBuffers(numArrays);
    std::vector<Nd4jLong*> hOutShapeInfo(numArrays), hInShapeInfo(numArrays);
    for(int i = 0; i < numArrays; ++i) {
        hOutBuffers[i]   = outSubArrsBuffs[i];
        hInBuffers[i]    = ddata[i];//->getSpecialBuffer();
        hOutShapeInfo[i] = outSubArrsShapes[i];
        hInShapeInfo[i]  = (Nd4jLong*)(dShapePointers[i]);//->getSpecialShapeInfo();
//        nd4j_printf("X_%i shape ptr: %p; data ptr: %p;\n", i, hInShapeInfo[i], hInBuffers[i]);
    }

//    nd4j_printf(" done\n", "");
    LaunchContext context(stream);
    // allocate and copy all buffers and shapes arrays to global memory
    PointersManager manager(&context, "NativeOps::concat");
    void* dOutBuffers	= manager.replicatePointer(hOutBuffers.data(),   hOutBuffers.size() * sizeof(void*));
    void* dInBuffers	= manager.replicatePointer(hInBuffers.data(),    hInBuffers.size() * sizeof(void*));
    void* dInShapeInfo  = manager.replicatePointer(hInShapeInfo.data(),  hInShapeInfo.size() * sizeof(Nd4jLong*));
    void* dOutShapeInfo = manager.replicatePointer(hOutShapeInfo.data(), hOutShapeInfo.size() * sizeof(Nd4jLong*));

    BUILD_SINGLE_SELECTOR(zType, concatCudaLauncher, (numArrays, stream, dInBuffers, dInShapeInfo, dOutBuffers, dOutShapeInfo), LIBND4J_TYPES);
    manager.synchronize();

    hipError_t err;
    for(int i = 0; i < numArrays; ++i) {
        err = hipFree(outSubArrsShapes[i]);
        if (err != 0) {
            printf("Error %d occured when shape %i was deallocating.\n", err, i);
            throw std::runtime_error("Cannot deallocate memory for shapes.");
        }
    }
}

/**
  * Concatneate multi array of the same shape together
  * along a particular dimension
  */
// void NativeOps::concat(
//		Nd4jPointer *extraPointers,
//        int dimension,
//        int numArrays,
//        Nd4jPointer *data, Nd4jPointer *inputShapeInfo,
//		Nd4jPointer *ddata, Nd4jPointer *dinputShapeInfo,
//		void *hZ, Nd4jLong *hZShapeInfo,
//        void *dZ, Nd4jLong *dZShapeInfo,
//		Nd4jPointer *tadPointers, Nd4jPointer *offsetPointers) {
//
//	hipStream_t *stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);
//	auto hXShapeInfo = hZShapeInfo;
//	auto hShapePointers = reinterpret_cast<Nd4jLong **>(inputShapeInfo);
//	// numArrays will be used as number of TADs, so each block process 1 input
//
//	int smem = 8192;
//	bool isVstack = false;
//	bool isScalar = true;
//	bool isHstack = false;
//
//	for (int i = 0; i < numArrays; i++) {
//		if (!shape::isScalar(hShapePointers[i])) {
//			isScalar = false;
//			break;
//		}
//	}
//
//	if (!isScalar && dimension == 0 && shape::rank(hZShapeInfo) == 2 && shape::order(hZShapeInfo) == 'c' ) {
//		isVstack = true;
//        for (int i = 0; i < numArrays; i++) {
//			if (!shape::isVector(hShapePointers[i]) || shape::elementWiseStride(hShapePointers[i]) <= 0 ||
//				shape::order(hShapePointers[i]) != 'c') {
//				isVstack = false;
//				break;
//			}
//		}
//	}
//
//    // let's try to fit N-dimensional vstack
//    if (!isVstack && !isScalar && dimension == 0 && shape::order(hXShapeInfo) == 'c') {
//		auto length0 = shape::length(hShapePointers[0]);
//        isVstack = true;
//        for (int i = 0; i < numArrays; i++) {
//            if (shape::elementWiseStride(hShapePointers[i]) <= 0 || shape::order(hShapePointers[i]) != 'c' || length0 != shape::length(hShapePointers[i])) {
//                isVstack = false;
//                break;
//            }
//        }
//    }
//
//	if (!isScalar && !isVstack && dimension == 1 && shape::isVector(hZShapeInfo)) {
//		isHstack = true;
//		for (int i = 0; i < numArrays; i++) {
//			if (!shape::isVector(hShapePointers[i]) || shape::elementWiseStride(hShapePointers[i]) <= 0) {
//				isHstack = false;
//				break;
//			}
//		}
//	}
//
//	if (isScalar) {
//		if (nd4j::Environment::getInstance()->isDebugAndVerbose())
//			printf("Going scalar concat\n");
//
//		dim3 launchDims(128, 128, 16384);
//		auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);
//		BUILD_SINGLE_SELECTOR(zType, concatKernelScalarGeneric, (launchDims, stream, numArrays, reinterpret_cast<Nd4jPointer *>(ddata[0]), dZ), LIBND4J_TYPES);
//
//	} else if (isVstack) {
//		if (nd4j::Environment::getInstance()->isDebugAndVerbose())
//			printf("Going VStack concat\n");
//
//		dim3 launchDims(128, 512, 16384);
//		auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);
//		BUILD_SINGLE_SELECTOR(zType, concatKernelVStackGeneric, (launchDims, stream, numArrays, reinterpret_cast<Nd4jPointer *>(ddata[0]), reinterpret_cast<Nd4jPointer *>(dinputShapeInfo[0]), dZ, dZShapeInfo), LIBND4J_TYPES);
//
//	} else if (isHstack) {
//		if (nd4j::Environment::getInstance()->isDebugAndVerbose())
//			printf("Going HStack concat\n");
//
//		dim3 launchDims(128, 128, 16384);
//		auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);
//		BUILD_SINGLE_SELECTOR(zType, concatKernelHStackGeneric, (launchDims, stream, numArrays, reinterpret_cast<Nd4jPointer *>(ddata[0]), reinterpret_cast<Nd4jPointer *>(dinputShapeInfo[0]), dZ, dZShapeInfo), LIBND4J_TYPES);
//	} else {
//		if (nd4j::Environment::getInstance()->isDebugAndVerbose())
//			printf("Going generic concat\n");
//
//        auto devZTadShape = reinterpret_cast<Nd4jLong *>(extraPointers[10]);
//		auto devZOffsets = reinterpret_cast<Nd4jLong *>(extraPointers[11]);
//
//		dim3 launchDims(128, 128, 8192);
//		auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);
//		BUILD_SINGLE_SELECTOR(zType, concatKernelGeneric, (launchDims, stream, numArrays, reinterpret_cast<Nd4jPointer *>(ddata[0]), reinterpret_cast<Nd4jPointer *>(dinputShapeInfo[0]), dZ, dZShapeInfo,  reinterpret_cast<Nd4jPointer *>(tadPointers[0]), reinterpret_cast<Nd4jPointer *>(offsetPointers[0]), devZTadShape, devZOffsets), LIBND4J_TYPES);
//	}
//	if (nd4j::Environment::getInstance()->isDebugAndVerbose())
//		printf("sharedMemory requested for concatFloat: [%i], registers: [%i]\n", smem, funcAttributes[31].numRegs);
//
//    hipError_t res = hipStreamSynchronize(*stream);
//    checkCudaErrors(res);
//    nd4j::DebugHelper::checkErrorCode(stream, "Legacy ConcatFloat(...) failed");
//}



void NativeOps::specialConcat(
        Nd4jPointer *extraPointers,
        int dimension,
        int numArrays,
        Nd4jPointer *data,
        Nd4jPointer *inputShapeInfo,
        void *dZ,
        Nd4jLong *dZShapeInfo, Nd4jPointer *tadPointers, Nd4jPointer *offsetPointers) {
    nd4j::SpecialMethods<float>::concatCpuGeneric(
            dimension,
            numArrays,
            data,
            inputShapeInfo,
            dZ,
            dZShapeInfo);

}


/**
 * This method saves
 */
nd4j::TadPack* NativeOps::tadOnlyShapeInfo(Nd4jLong *dXShapeInfo, int *dimension, int dimensionLength) {
	/*shape::TAD tad;
	tad.init(dXShapeInfo, dimension, dimensionLength);
	//tad->setOutputBuffer(target);
	tad.createTadOnlyShapeInfo();
	tad.createOffsets();


	std::memcpy(reinterpret_cast<void *>(target), tad.tadOnlyShapeInfo, shape::shapeInfoByteLength(tad.tadOnlyShapeInfo));
	std::memcpy(reinterpret_cast<void *>(offsets), tad.tadOffsets, tad.numTads * sizeof(Nd4jLong));
	*/
	auto pack = new TadPack();
	*pack = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(dXShapeInfo, dimension, dimensionLength);
    return pack;
}

int NativeOps::memcpyConstantAsync(Nd4jLong dst, Nd4jPointer src, Nd4jLong size, int flags, Nd4jPointer reserved) {
	hipStream_t *pStream = reinterpret_cast<hipStream_t *>(reserved);

	hipMemcpyKind 	kind;

	DEBUG_KERNEL(pStream, -1);

	switch (flags) {
		case 0: {
			kind = hipMemcpyHostToHost;
		}
			break;
		case 1: {
			kind = hipMemcpyHostToDevice;
		}
			break;
		case 2: {
			kind = hipMemcpyDeviceToHost;
		}
		case 3: {
			kind = hipMemcpyDeviceToDevice;
		}
			break;
	}
	//hipError_t dZ = hipMemcpyAsync((void *) dst, (const void *) src, (size_t) size, kind, *pStream);
	hipError_t dZ = hipMemcpyToSymbolAsync(HIP_SYMBOL(deviceConstantMemory), const_cast<const void *>(src), size, dst, kind, *pStream);
	checkCudaErrors(dZ);
	if (dZ != 0)
        throw std::runtime_error("hipMemcpyToSymbolAsync(...) failed");

	return 1;
}

Nd4jPointer NativeOps::getConstantSpace() {
	Nd4jPointer dConstAddr;
	hipError_t dZ = hipGetSymbolAddress(reinterpret_cast<void **>(&dConstAddr), deviceConstantMemory);

	if (dZ != 0)
        throw std::runtime_error("hipGetSymbolAddress(...) failed");

	return dConstAddr;
}

void NativeOps::pullRows(Nd4jPointer *extraPointers,
						 void *x, Nd4jLong *xShapeInfo,
						 void *dX, Nd4jLong *dXShapeInfo,
						 void *z, Nd4jLong *zShapeInfo,
						 void *dZ, Nd4jLong *dZShapeInfo,
						 Nd4jLong n,
						 Nd4jLong *indexes,
						 Nd4jLong *tadShapeInfo,
						 Nd4jLong *tadOffsets,
						 Nd4jLong *zTadShapeInfo,
						 Nd4jLong *zTadOffsets) {

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);
	dim3 launchDims(64, 256, 1024);
	auto xType = nd4j::ArrayOptions::dataType(xShapeInfo);
    BUILD_SINGLE_SELECTOR(xType, pullRowsKernelGeneric, (launchDims, stream, dX, dZ, n, indexes, tadShapeInfo, tadOffsets,  zTadShapeInfo,  zTadOffsets), LIBND4J_TYPES);

	DEBUG_KERNEL(stream, -1);
}


void NativeOps::average(Nd4jPointer *extras,
						Nd4jPointer *x, Nd4jLong *xShapeInfo,
						Nd4jPointer *dx, Nd4jLong *dXShapeInfo,
						void *z, Nd4jLong *zShapeInfo,
						void *dz, Nd4jLong *dzShapeInfo,
						int n,
						Nd4jLong length,
						bool propagate) {

	hipStream_t * stream = reinterpret_cast<hipStream_t *>(extras[1]);
	int mode = getDeviceId(extras[3]);

	auto dX = reinterpret_cast<void **>(dx);

	if (nd4j::Environment::getInstance()->isDebugAndVerbose())
		printf("averageFloat called\n");

	auto xType = nd4j::ArrayOptions::dataType(xShapeInfo);
	// launching on gpu
	if (mode == 0) {
		dim3 launchDims(256, 256, 4096);
    	BUILD_SINGLE_SELECTOR(xType, averagingKernelGeneric, (launchDims, stream, dX, dz, n, length, propagate), LIBND4J_TYPES);
        nd4j::DebugHelper::checkErrorCode(stream, "AverageFloat(...) failed");
	} else {
		// launching on host memory
        BUILD_SINGLE_SELECTOR(xType, nd4j::SpecialMethods, ::averageGeneric(x, z, zShapeInfo, n, length, propagate), LIBND4J_TYPES);
	}
}

void NativeOps::accumulate(Nd4jPointer *extras,
						   Nd4jPointer *x, Nd4jLong *xShapeInfo,
						   Nd4jPointer *dx, Nd4jLong *dXShapeInfo,
						   void *z, Nd4jLong *zShapeInfo,
						   void *dz, Nd4jLong *dzShapeInfo,
						   int n,
						   Nd4jLong length) {

	auto stream = reinterpret_cast<hipStream_t *>(extras[1]);
	int mode = getDeviceId(extras[3]);

	auto dX = reinterpret_cast<void **>(dx);

	if (nd4j::Environment::getInstance()->isDebugAndVerbose())
		printf("accumulateFloat called\n");
	auto xType = nd4j::ArrayOptions::dataType(xShapeInfo);

	// launching on gpu
	if (mode == 0) {
		dim3 launchDims(n, 256, 16384);
        BUILD_SINGLE_SELECTOR(xType, accumulateKernelGeneric, (launchDims, stream, dX, dz, n,length), LIBND4J_TYPES);
        nd4j::DebugHelper::checkErrorCode(stream, "AccumulateFloat(...) failed");
	} else {
		// launching on host memory
        BUILD_SINGLE_SELECTOR(xType, nd4j::SpecialMethods, ::accumulateGeneric(x, z, zShapeInfo, n, length), LIBND4J_TYPES);
	}
}


void NativeOps::shuffle(Nd4jPointer *extras,
						Nd4jPointer *x, Nd4jPointer *xShapeInfo,
						Nd4jPointer *dx, Nd4jPointer *dXShapeInfo,
						Nd4jPointer *z, Nd4jPointer *zShapeInfo,
						Nd4jPointer *dz, Nd4jPointer *dZShapeInfo,
						int N,
						int *shuffleMap,
						Nd4jPointer *tadShapeInfo,
						Nd4jPointer *tadOffsets) {

    hipStream_t *stream = reinterpret_cast<hipStream_t *>(extras[1]);

    auto dX = reinterpret_cast<void **>(dx);
    auto dZ = reinterpret_cast<void **>(dz);
    auto xShape = reinterpret_cast<Nd4jLong **>(xShapeInfo);
    auto dxShape = reinterpret_cast<Nd4jLong **>(dXShapeInfo);
    auto tadOnlyShapeInfo = reinterpret_cast<Nd4jLong **>(tadShapeInfo);
    auto tadOffset = reinterpret_cast<Nd4jLong **>(tadOffsets);

    auto xType = nd4j::ArrayOptions::dataType(xShape[0]);
    dim3 launchDims(256, 512, 8192);
    BUILD_SINGLE_SELECTOR(xType, shuffleKernelGeneric, (launchDims, stream, dX, dxShape, dZ, N, shuffleMap,  tadOnlyShapeInfo, tadOffset), LIBND4J_TYPES);

    nd4j::DebugHelper::checkErrorCode(stream, "shuffle(...) failed");
}

/*
void NativeOps::execMetaPredicateShape(Nd4jPointer *extras,
	                                  const int opTypeA,
	                                  const int opNumA,
	                                  const int opTypeB,
	                                  const int opNumB,
	                                  Nd4jLong N,
	                                  void *hX, Nd4jLong *hXShapeInfo,
                                      void *dX, Nd4jLong *dXShapeInfo,
                                      void *hY, Nd4jLong *hYShapeInfo,
                                      void *dY, Nd4jLong *dYShapeInfo,
                                      void *hZ, Nd4jLong *hZShapeInfo,
                                      void *dZ, Nd4jLong *dZShapeInfo,
	                                  void *extraA,
	                                  void *extraB,
	                                  double scalarA,
	                                  double scalarB) {

    hipStream_t *stream = reinterpret_cast<hipStream_t *>(extras[1]);
    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    BUILD_SINGLE_SELECTOR(xType, functions::grid::GRIDShaped, ::execMetaPredicateShaped(stream, extras, opTypeA, opNumA, opTypeB, opNumB, N, dX, dXShapeInfo, dY, dYShapeInfo, dZ, dZShapeInfo, extraA, extraB, scalarA, scalarB), LIBND4J_TYPES);
    // functions::grid::GRIDShaped<float>::execMetaPredicateShaped(stream, extras, opTypeA, opNumA, opTypeB, opNumB, N, dX, dXShapeInfo, dy, dYShapeInfo, dz, zShapeInfo, extraA, extraB, scalarA, scalarB);

	DEBUG_KERNEL(stream, opNumA);
}
*/

bool NativeOps::isExperimentalEnabled() {
    return nd4j::Environment::getInstance()->isExperimentalBuild();
}

void NativeOps::setOmpMinThreads(int threads) {
    minThreads = nd4j::math::nd4j_max<int>(32, threads);
    minThreads = nd4j::math::nd4j_min<int>(maxThreads, minThreads);
}

int NativeOps::getDevice() {
    int curDevice = -1;

    hipGetDevice(&curDevice);

    return curDevice;
}

void NativeOps::setElementThreshold(int num) {
    // this is no-op for CUDA
}

void NativeOps::setTADThreshold(int num) {
    // this is no-op for CUDA
}

////////////////////////////////////////////////////////////////////////
void NativeOps::execSummaryStats(Nd4jPointer *extraPointers,
                                 int opNum,
                                 void *hX, Nd4jLong *hXShapeInfo,
                                 void *dX, Nd4jLong *dXShapeInfo,
                                 void *extraParams,
                                 void *hZ, Nd4jLong *hZShapeInfo,
                                 void *dZ, Nd4jLong *dZShapeInfo,
                                 bool biasCorrected) {

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
	NativeOpExecutioner::execSummaryStats(&lc, opNum, hX, hXShapeInfo, dX, dXShapeInfo, extraParams, hZ, hZShapeInfo, dZ, dZShapeInfo, biasCorrected);
}

////////////////////////////////////////////////////////////////////////
void NativeOps::execSummaryStats(Nd4jPointer *extraPointers,
                                 int opNum,
                                 void *hX, Nd4jLong *hXShapeInfo,
                                 void *dX, Nd4jLong *dXShapeInfo,
                                 void *extraParams,
                                 void *hZ, Nd4jLong *hZShapeInfo,
                                 void *dZ, Nd4jLong *dZShapeInfo,
								 void *hDimension, Nd4jLong *hDimensionShape, void *dDimension, Nd4jLong *dDimensionShape,
                                 bool biasCorrected,
								 Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets) {
	auto dimension = reinterpret_cast<int *>(dDimension);
	int dimensionLength = static_cast<int>(shape::length(hDimensionShape));

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
	NativeOpExecutioner::execSummaryStats(&lc, opNum, hX, hXShapeInfo, dX, dXShapeInfo, extraParams, hZ, hZShapeInfo, dZ, dZShapeInfo, dimension, dimensionLength, tadShapeInfo, tadOffsets, biasCorrected);
}

////////////////////////////////////////////////////////////////////////
void NativeOps::execReduce3(Nd4jPointer *extraPointers,
                            int opNum,
                            void *hX, Nd4jLong *hXShapeInfo,
                            void *dX, Nd4jLong *dXShapeInfo,
                            void *extraParams,
                            void *hY, Nd4jLong *hYShapeInfo,
                            void *dY, Nd4jLong *dYShapeInfo,
                            void *hZ, Nd4jLong *hZShapeInfo,
                            void *dZ, Nd4jLong *dZShapeInfo) {

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
	NativeOpExecutioner::execReduce3(&lc, opNum, hX, hXShapeInfo, dX, dXShapeInfo, extraParams, hY, hYShapeInfo, dY, dYShapeInfo, hZ, hZShapeInfo, dZ, dZShapeInfo);
}

////////////////////////////////////////////////////////////////////////
void NativeOps::execReduce3(Nd4jPointer *extraPointers,
                            int opNum,
                            void *hX, Nd4jLong *hXShapeInfo,
                            void *dX, Nd4jLong *dXShapeInfo,
                            void *extraParams,
                            void *hY, Nd4jLong *hYShapeInfo,
                            void *dY, Nd4jLong *dYShapeInfo,
                            void *hZ, Nd4jLong *hZShapeInfo,
                            void *dZ, Nd4jLong *dZShapeInfo,
							void *hDimension, Nd4jLong *hDimensionShape, void *dDimension, Nd4jLong *dDimensionShape,
                            Nd4jLong *tadOnlyShapeInfo, Nd4jLong *tadOffsets,
                            Nd4jLong *yTadOnlyShapeInfo, Nd4jLong *yTadOffsets) {
	auto dimension = reinterpret_cast<int *>(dDimension);
	int dimensionLength = static_cast<int>(shape::length(hDimensionShape));

	// if (extraPointers == nullptr || extraPointers[2] == 0)
 //        NativeOpExecutioner::execReduce3(nullptr, opNum, hX, hXShapeInfo, dX, dXShapeInfo, extraParams, hY, hYShapeInfo, dY, dYShapeInfo, hZ, hZShapeInfo, dZ, dZShapeInfo, dimension, dimensionLength, tadOnlyShapeInfo, tadOffsets, yTadOnlyShapeInfo, yTadOffsets);
 //    else {
 //        // going tad-ways
 //        auto tadShapeInfo = reinterpret_cast<Nd4jLong *> (extraPointers[0]);
 //        auto tadOffsets = reinterpret_cast<Nd4jLong *>(extraPointers[1]);

 //        NativeOpExecutioner::execReduce3TAD(nullptr, opNum, hX, hXShapeInfo, dX, dXShapeInfo, extraParams, hY, hYShapeInfo, dY, dYShapeInfo, hZ, hZShapeInfo, dZ, dZShapeInfo, dimension, dimensionLength, tadShapeInfo, tadOffsets);
 //    }

    nd4j_printf("Starting...\n","");

    auto tadPack = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(hXShapeInfo, reinterpret_cast<int*>(hDimension), shape::length(hDimensionShape));
    auto tadLength = shape::length(tadPack.primaryShapeInfo());
    auto yLength = shape::length(hYShapeInfo);
    auto xLength = shape::length(hXShapeInfo);

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);

    if (tadLength == yLength || tadLength == xLength) {
        nd4j_printf("== way\n","");
        NativeOpExecutioner::execReduce3(&lc, opNum, hX, hXShapeInfo, dX, dXShapeInfo, extraParams, hY, hYShapeInfo, dY,
                                         dYShapeInfo, hZ, hZShapeInfo, dZ, dZShapeInfo, dimension, dimensionLength,
                                         tadOnlyShapeInfo, tadOffsets, yTadOnlyShapeInfo, yTadOffsets);
    } else
        NativeOpExecutioner::execReduce3TAD(&lc, opNum, hX, hXShapeInfo, dX, dXShapeInfo, extraParams, hY, hYShapeInfo, dY, dYShapeInfo, hZ, hZShapeInfo, dZ, dZShapeInfo, dimension, dimensionLength, tadOnlyShapeInfo, yTadOffsets, yTadOnlyShapeInfo, yTadOffsets);
}

////////////////////////////////////////////////////////////////////////
void NativeOps::execReduce3Scalar(Nd4jPointer *extraPointers,int opNum,
                                  void *hX, Nd4jLong *hXShapeInfo,
                                  void *dX, Nd4jLong *dXShapeInfo,
                                  void *extraParams,
                                  void *hY, Nd4jLong *hYShapeInfo,
                                  void *dY, Nd4jLong *dYShapeInfo,
                                  void *hZ, Nd4jLong *hZShapeInfo,
                                  void *dZ, Nd4jLong *dZShapeInfo) {

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
	NativeOpExecutioner::execReduce3Scalar(&lc, opNum,hX,hXShapeInfo,dX, dXShapeInfo,extraParams,hY,hYShapeInfo,dY,dYShapeInfo, hZ, hZShapeInfo, dZ, dZShapeInfo);
}

////////////////////////////////////////////////////////////////////////
void NativeOps::execScalarBool(Nd4jPointer *extraPointers,
							int opNum,
							void *hX, Nd4jLong *hXShapeInfo,
							void *dX, Nd4jLong *dXShapeInfo,
							void *hZ, Nd4jLong *hZShapeInfo,
							void *dZ, Nd4jLong *dZShapeInfo,
							void *hScalar, Nd4jLong *hScalarShapeInfo,
							void *dScalar, Nd4jLong *dScalarShapeInfo,
							void *extraParams) {

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
	NativeOpExecutioner::execScalarBool(&lc, opNum, hX, hXShapeInfo, dX, dXShapeInfo, hZ, hZShapeInfo, dZ, dZShapeInfo, hScalar, hScalarShapeInfo, dScalar, dScalarShapeInfo, extraParams);
}

////////////////////////////////////////////////////////////////////////
void NativeOps::execScalarBool(Nd4jPointer *extraPointers,
						   int opNum,
						   void *hX, Nd4jLong *hXShapeInfo,
						   void *dX, Nd4jLong *dXShapeInfo,
						   void *hZ, Nd4jLong *hZShapeInfo,
						   void *dZ, Nd4jLong *dZShapeInfo,
						   void *hScalars, Nd4jLong *hScalarShapeInfo,
						   void *dScalars, Nd4jLong *dScalarShapeInfo,
						   void *extraParams,
							   void *hDimension, Nd4jLong *hDimensionShape, void *dDimension, Nd4jLong *dDimensionShape,
                           Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets,
                           Nd4jLong *tadShapeInfoZ, Nd4jLong *tadOffsetsZ) {
	auto dimension = reinterpret_cast<int *>(dDimension);
	int dimensionLength = static_cast<int>(shape::length(hDimensionShape));

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
	NativeOpExecutioner::execScalarBool(&lc, opNum, hX, hXShapeInfo, dX, dXShapeInfo, extraParams, hZ, hZShapeInfo, dZ, dZShapeInfo, hScalars, hScalarShapeInfo, dScalars, dScalarShapeInfo, dimension, dimensionLength, tadShapeInfo, tadOffsets, tadShapeInfoZ, tadOffsetsZ);
}

////////////////////////////////////////////////////////////////////////
void NativeOps::execScalar(Nd4jPointer *extraPointers,
						int opNum,
						void *hX, Nd4jLong *hXShapeInfo,
						void *dX, Nd4jLong *dXShapeInfo,
						void *hZ, Nd4jLong *hZShapeInfo,
						void *dZ, Nd4jLong *dZShapeInfo,
						void *hScalar, Nd4jLong *hScalarShapeInfo,
						void *dScalar, Nd4jLong *dScalarShapeInfo,
						void *extraParams) {

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
	NativeOpExecutioner::execScalar(&lc, opNum, hX, hXShapeInfo, dX, dXShapeInfo, hZ, hZShapeInfo, dZ, dZShapeInfo, hScalar, hScalarShapeInfo, dScalar, dScalarShapeInfo, extraParams);
}

////////////////////////////////////////////////////////////////////////
void NativeOps::execScalar(Nd4jPointer *extraPointers,
					 int opNum,
					 void *hX, Nd4jLong *hXShapeInfo,
                     void *dX, Nd4jLong *dXShapeInfo,
                     void *hZ, Nd4jLong *hZShapeInfo,
                     void *dZ, Nd4jLong *dZShapeInfo,
                     void *hScalars, Nd4jLong *hScalarShapeInfo,
                     void *dScalars, Nd4jLong *dScalarShapeInfo,
					 void *extraParams,
						   void *hDimension, Nd4jLong *hDimensionShape,
						   void *dDimension, Nd4jLong *dDimensionShape,
                     Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets,
                     Nd4jLong *tadShapeInfoZ, Nd4jLong *tadOffsetsZ) {
	auto dimension = reinterpret_cast<int *>(dDimension);
	int dimensionLength = static_cast<int>(shape::length(hDimensionShape));

    hipStream_t *stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);

    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto yType = nd4j::ArrayOptions::dataType(hScalarShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

	if (yType != xType && yType != nd4j::DataType::BOOL && !this->isExperimentalEnabled())
		throw nd4j::datatype_exception::build("NativeOps::execScalar both operands must have same data type", xType, yType);

	dim3 launchDims(256, 256, 16384);

#ifdef __ND4J_EXPERIMENTAL__
    BUILD_PAIRWISE_SELECTOR(xType, yType, zType, functions::scalar::ScalarTransform, ::executeCudaAlongDimension(launchDims, stream, opNum, dX, dXShapeInfo, dZ, dZShapeInfo, dScalars, extraParams, dimension, dimensionLength, tadShapeInfo, tadOffsets, tadShapeInfoZ, tadOffsetsZ), LIBND4J_TYPES, LIBND4J_TYPES);
#else
	BUILD_SINGLE_SELECTOR_THRICE(xType, functions::scalar::ScalarTransform, ::executeCudaAlongDimension(launchDims, stream, opNum, dX, dXShapeInfo, dZ, dZShapeInfo, dScalars, extraParams, dimension, dimensionLength, tadShapeInfo, tadOffsets, tadShapeInfoZ, tadOffsetsZ), LIBND4J_TYPES);
#endif

	DEBUG_KERNEL(stream, opNum);
}

void NativeOps::execAggregate(Nd4jPointer *extraPointers,
								   int opNum,
                                   void **arguments,
                                   int numArguments,
                                   Nd4jLong **shapes,
                                   int numShapes,
                                   int *indexArguments,
                                   int numIndexArguments,
                                   int **intArrays,
                                   int numIntArrays,
                                   void *realArguments,
                                   int numRealArguments,
                                   nd4j::DataType dtype) {

    hipStream_t *stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);
    int numBlocks = getDeviceId(extraPointers[2]);
    int numThreads = getDeviceId(extraPointers[3]);
    int shmem = getDeviceId(extraPointers[4]);

    dim3 launchDims = dim3(numBlocks, numThreads, shmem);

    BUILD_SINGLE_SELECTOR(dtype, functions::aggregate::AggregatedFunction, ::aggregateKernelGeneric(launchDims, stream, opNum, arguments, numArguments, shapes, numShapes, indexArguments, numIndexArguments, intArrays, numIntArrays, realArguments, numRealArguments), FLOAT_TYPES);
    nd4j::DebugHelper::checkErrorCode(stream, "execAggregateFloat(...) failed");
}

void NativeOps::execAggregateBatch(Nd4jPointer *extraPointers,
									int numAggregates, int opNum,
									int maxArgs, int maxShapes,
									int maxIntArrays, int maxIntArraySize,
									int maxIdx, int maxReals,
									void *ptrToArguments, nd4j::DataType dtype) {
    // not implemented yet
    hipStream_t *stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);
    int numBlocks = getDeviceId(extraPointers[2]);
    int numThreads = getDeviceId(extraPointers[3]);
    int shmem = getDeviceId(extraPointers[4]);

    dim3 launchDims = dim3(numAggregates, numThreads, shmem);

	BUILD_SINGLE_SELECTOR(dtype, functions::aggregate::AggregatedFunction, ::aggregateBatchKernelGeneric(launchDims, stream, opNum, numAggregates, maxArgs, maxShapes, maxIntArrays, maxIntArraySize, maxIdx, maxReals, ptrToArguments), FLOAT_TYPES);

	DEBUG_KERNEL(stream, opNum);
}

////////////////////////////////////////////////////////////////////////
void NativeOps::execRandom(Nd4jPointer *extraPointers,
						  int opNum,
                          Nd4jPointer stateHost,
                          void *hZ, Nd4jLong *hZShapeInfo,
                          void *dZ, Nd4jLong *dZShapeInfo,
                          void *extraArguments) {

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
    NativeOpExecutioner::execRandom(&lc, opNum, extraPointers, hZ, hZShapeInfo, dZ, dZShapeInfo, extraArguments);
}

////////////////////////////////////////////////////////////////////////
void NativeOps::execRandom(Nd4jPointer *extraPointers, int opNum, Nd4jPointer stateHost,
						   void *hX, Nd4jLong *hXShapeInfo,
						   void *dX, Nd4jLong *dXShapeInfo,
						   void *hZ, Nd4jLong *hZShapeInfo,
						   void *dZ, Nd4jLong *dZShapeInfo,
						   void *extraArguments) {

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
    NativeOpExecutioner::execRandom(&lc, opNum, extraPointers, hX, hXShapeInfo, dX, dXShapeInfo, hZ, hZShapeInfo, dZ, dZShapeInfo, extraArguments);
}

////////////////////////////////////////////////////////////////////////
void NativeOps::execRandom(Nd4jPointer *extraPointers, int opNum, Nd4jPointer stateHost,
							void *hX, Nd4jLong *hXShapeInfo,
							void *dX, Nd4jLong *dXShapeInfo,
							void *hY, Nd4jLong *hYShapeInfo,
							void *dY, Nd4jLong *dYShapeInfo,
							void *hZ, Nd4jLong *hZShapeInfo,
							void *dZ, Nd4jLong *dZShapeInfo,
							void *extraArguments) {

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
    NativeOpExecutioner::execRandom(&lc, opNum, extraPointers, hX, hXShapeInfo, dX, dXShapeInfo, hY, hYShapeInfo, dY, dYShapeInfo, hZ, hZShapeInfo, dZ, dZShapeInfo, extraArguments);
}


Nd4jPointer NativeOps::initRandom(Nd4jPointer *extraPointers, long seed, long bufferSize, Nd4jPointer ptrToBuffer) {

    unsigned long long *ptrHost = reinterpret_cast<unsigned long long *>(extraPointers[0]);
    hipStream_t *stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);

    // we don't synchronize at random initialization, it's safe to go unsync here
	// hipStreamSynchronize(*stream);

    auto ptrDev = reinterpret_cast<unsigned long long *>(ptrToBuffer);
    auto buffer = new nd4j::random::RandomBuffer(seed, bufferSize, reinterpret_cast<uint64_t *>(ptrHost), reinterpret_cast<uint64_t *>(ptrDev));
    buffer->propagateToDevice(buffer, *stream);

    nd4j::DebugHelper::checkErrorCode(stream, "initRandom(...) failed A");

	// we generate sequence in the host memory
    nd4j::random::Xoroshiro128 generator(buffer);
    generator.refreshBuffer();

	// and copy it to gpu
    hipMemcpyAsync(ptrDev, ptrHost, bufferSize * 8, hipMemcpyHostToDevice, *stream);
    nd4j::DebugHelper::checkErrorCode(stream, "initRandom(...) failed B");

    return buffer;
}


void NativeOps::destroyRandom(Nd4jPointer ptrBuffer) {

    nd4j::random::RandomBuffer *buffer = reinterpret_cast<nd4j::random::RandomBuffer *> (ptrBuffer);

    // FIXME: it's bad thing, but we can't know in advance, which stream(s) where using this generator in practice
    hipDeviceSynchronize();

    delete buffer;
}

void NativeOps::refreshBuffer(Nd4jPointer *extraPointers, long seed, Nd4jPointer ptrRandom) {

    nd4j::random::RandomBuffer *buffer = reinterpret_cast<nd4j::random::RandomBuffer *> (ptrRandom);

    unsigned long long *ptrHost = reinterpret_cast<unsigned long long *>(extraPointers[0]);
    hipStream_t *stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);
    hipStreamSynchronize(*stream);

    uint64_t *ptrDev = buffer->getDeviceBuffer();

	// update rng state
    buffer->setSeed(seed);
    buffer->setOffset(0);
    buffer->propagateToDevice(buffer, *stream);

	// refresh buffer on host size
    nd4j::random::Xoroshiro128 generator(buffer);
    generator.refreshBuffer();

	// copy back to gpu
    hipMemcpyAsync(ptrDev, ptrHost, buffer->getSize() * 8, hipMemcpyHostToDevice, *stream);
}

void NativeOps::reSeedBuffer(Nd4jPointer *extraPointers, long seed, Nd4jPointer ptrRandom) {

    nd4j::random::RandomBuffer *buffer = reinterpret_cast<nd4j::random::RandomBuffer *> (ptrRandom);

    hipStream_t *stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);
    hipStreamSynchronize(*stream);

	// update rng state
    buffer->reSeed(seed);
    buffer->setOffset(0);
    buffer->propagateToDevice(buffer, *stream);
}



/**
    * Return the length of a shape buffer
    * based on the pointer
    * @param buffer  the buffer pointer to check
    * @return
    */
int NativeOps::lengthForShapeBufferPointer(Nd4jPointer buffer) {
    auto shapeBuffer = reinterpret_cast<Nd4jLong *>(buffer);
    return shape::shapeInfoLength(shape::rank(shapeBuffer));
}


/**
  * The pointer to get the address for
  *
  * @param address the address to get the pointer
  * @return the pointer for the given address
  */

Nd4jPointer NativeOps::pointerForAddress(Nd4jLong address) {
	return reinterpret_cast<Nd4jPointer >(address);
}

void NativeOps::tear(Nd4jPointer *extras,
					 void *x, Nd4jLong *xShapeInfo,
					 void *dX, Nd4jLong *dXShapeInfo,
					 Nd4jPointer *targets,
					 Nd4jLong *zShapeInfo,
					 Nd4jLong *tadShapeInfo,
					 Nd4jLong *tadOffsets) {

    hipStream_t *stream = reinterpret_cast<hipStream_t *>(extras[1]);
    dim3 launchDims(512, 512, 512);
    auto xType = nd4j::ArrayOptions::dataType(xShapeInfo);
    BUILD_SINGLE_SELECTOR(xType, tearKernelGeneric, (launchDims, stream, dX, dXShapeInfo, targets, zShapeInfo, tadShapeInfo, tadOffsets), LIBND4J_TYPES);

    nd4j::DebugHelper::checkErrorCode(stream, "tearFloat(...) failed");
}


void prescanArrayRecursive(Nd4jPointer *extras, int *dZ, int *dX, int numElements, int level) {

    auto stream = reinterpret_cast<hipStream_t *>(extras[1]);
    auto g_scanBlockSums = reinterpret_cast<int **>(extras[2]);

    int blockSize = 512; // max size of the thread blocks
    int numBlocks = nd4j::math::nd4j_max<int>(1, static_cast<int>(ceil(static_cast<float>(numElements) / (2.f * blockSize))));
    int numThreads;

    if (numBlocks > 1)
        numThreads = blockSize;
    else if (nd4j::isPowerOfTwo(numElements))
        numThreads = numElements / 2;
    else
        numThreads = nd4j::floorPow2(numElements);

    int numEltsPerBlock = numThreads * 2;

    // if this is a non-power-of-2 array, the last block will be non-full
    // compute the smallest power of 2 able to compute its scan.
    int numEltsLastBlock =
            numElements - (numBlocks-1) * numEltsPerBlock;
    int numThreadsLastBlock = nd4j::math::nd4j_max<int>(1, numEltsLastBlock / 2);
    int np2LastBlock = 0;
    int sharedMemLastBlock = 0;

    if (numEltsLastBlock != numEltsPerBlock) {
        np2LastBlock = 1;

        if(!isPowerOfTwo(numEltsLastBlock))
            numThreadsLastBlock = floorPow2(numEltsLastBlock);

        unsigned int extraSpace = (2 * numThreadsLastBlock) / NUM_BANKS;
        sharedMemLastBlock = sizeof(int) * (2 * numThreadsLastBlock + extraSpace);
    }

    // padding space is used to avoid shared memory bank conflicts
    int extraSpace = numEltsPerBlock / NUM_BANKS;
    int sharedMemSize = sizeof(int) * (numEltsPerBlock + extraSpace);

    // setup execution parameters
    // if NP2, we process the last block separately
    dim3 grid(max(1, numBlocks - np2LastBlock), 1, 1);
    dim3 threads(numThreads, 1, 1);
    dim3 gridOnes(1, 1, 1);
    dim3 threadsOnes(numThreadsLastBlock, 1, 1);

    if (sharedMemSize < 2048)
        sharedMemSize = 2048;

    if (sharedMemLastBlock < 2048)
        sharedMemLastBlock = 2048;

    // execute the scan
    if (numBlocks > 1) {
        nd4j::prescanLauncher<true, false>(grid, threads, sharedMemSize, stream, dZ, dX, g_scanBlockSums[level], numThreads * 2, 0, 0);
        if (np2LastBlock) {
            nd4j::prescanLauncher<true, true>(gridOnes, threadsOnes, sharedMemLastBlock, stream, dZ, dX, g_scanBlockSums[level], numEltsLastBlock, numBlocks - 1, numElements - numEltsLastBlock);
        }

        // After scanning all the sub-blocks, we are mostly done.  But now we
        // need to take all of the last values of the sub-blocks and scan those.
        // This will give us a new value that must be sdded to each block to
        // get the final results.
        // recursive (CPU) call
        prescanArrayRecursive(extras, g_scanBlockSums[level], g_scanBlockSums[level], numBlocks, level+1);

        nd4j::uniformAdd<<<grid, threads, 1024, *stream>>>(dZ, g_scanBlockSums[level], numElements - numEltsLastBlock, 0, 0);

        if (np2LastBlock) {
            nd4j::uniformAdd<<<1, numThreadsLastBlock, 1024, *stream>>>(dZ, g_scanBlockSums[level], numEltsLastBlock, numBlocks - 1, numElements - numEltsLastBlock);
        }
    } else if (isPowerOfTwo(numElements)) {
        nd4j::prescanLauncher<false, false>(grid, threads, sharedMemSize, stream, dZ, dX, 0, numThreads * 2, 0, 0);
    } else {
        nd4j::prescanLauncher<false, true>(grid, threads, sharedMemSize, stream, dZ, dX, 0, numElements, 0, 0);
    }

    nd4j::DebugHelper::checkErrorCode(stream, "prescanArray(...) failed");
}


void NativeOps::encodeThresholdP1(Nd4jPointer *extras, void *dx, Nd4jLong *hXShapeInfo, Nd4jLong N, int *dz, float threshold) {

    hipStream_t *stream = reinterpret_cast<hipStream_t *>(extras[1]);

    int blockSize = 1024;
    int numBlocks = N / blockSize + (N % blockSize ? 1 : 0);

    dim3 launchDims(numBlocks, blockSize, 1024);
    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    BUILD_SINGLE_SELECTOR(xType, encoderKernelP1Generic, (launchDims, stream, dx, N, dz, threshold), LIBND4J_TYPES);

    nd4j::DebugHelper::checkErrorCode(stream, "encodeThresholdP1Float(...) failed");
}



void NativeOps::encodeThresholdP2Int(Nd4jPointer *extraPointers, int *dx, Nd4jLong N, int *dz) {

    hipStream_t *stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);
    //encoderKernelP2Float<<<numBlocks, blockSize , 1024 * sizeof(float), *stream>>>(dx, N, dz);
    prescanArrayRecursive(extraPointers, dz, dx + 1, (int) N, 0);
    nd4j::DebugHelper::checkErrorCode(stream, "encodeThresholdP2Int(...) failed");
}

void NativeOps::encodeThresholdP3(Nd4jPointer *extraPointers, void *dx, Nd4jLong *hXShapeInfo, int *offsets, Nd4jLong N, int *dz){

    hipStream_t *stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);

    int blockSize = 1024;
    int numBlocks = N / blockSize + (N % blockSize ? 1 : 0);

    dim3 launchDims(numBlocks, blockSize, 4096);
    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    BUILD_SINGLE_SELECTOR(xType, encoderKernelP3Generic, (launchDims, stream, dx, offsets, N, dz), LIBND4J_TYPES);

    nd4j::DebugHelper::checkErrorCode(stream, "encodeThresholdP3Float(...) failed");
}

void NativeOps::decodeThreshold(Nd4jPointer *extraPointers, void *dx, Nd4jLong N, void *dz, Nd4jLong *zShapeInfo){

    hipStream_t *stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);

    // we probably want to have smaller blocks here, memory writes are misaligned anyway
    int blockSize = 128;
    int numBlocks = N / blockSize + (N % blockSize ? 1 : 0);

    dim3 launchDims(numBlocks, blockSize, 1024);
    auto zType = nd4j::ArrayOptions::dataType(zShapeInfo);
    BUILD_SINGLE_SELECTOR(zType, decoderKernelGeneric, (launchDims, stream, dx, N, dz), LIBND4J_TYPES);

    nd4j::DebugHelper::checkErrorCode(stream, "decodeThresholdFloat(...) failed");
}

////////////////////////////////////////////////////////////////////////
void NativeOps::execReduce3All(Nd4jPointer *extraPointers,
									int opNum,
									void *hX, Nd4jLong *hXShapeInfo,
                            		void *dX, Nd4jLong *dXShapeInfo,
                            		void *extraParamsVals,
									void *hY, Nd4jLong *hYShapeInfo,
                            		void *dY, Nd4jLong *dYShapeInfo,
                            		void *hZ, Nd4jLong *hZShapeInfo,
                            		void *dZ, Nd4jLong *dZShapeInfo,
							   		void *hDimension, Nd4jLong *hDimensionShape,
							   		void *dDimension, Nd4jLong *dDimensionShape,
									Nd4jLong *xTadShapeInfo, Nd4jLong *xOffsets,
									Nd4jLong *yTadShapeInfo, Nd4jLong *yOffsets) {
	auto dimension = reinterpret_cast<int *>(dDimension);
	int dimensionLength = static_cast<int>(shape::length(hDimensionShape));

    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
    NativeOpExecutioner::execReduce3All(&lc, opNum, hX, hXShapeInfo, dX, dXShapeInfo, extraParamsVals, hY, hYShapeInfo, dY, dYShapeInfo, hZ, hZShapeInfo, dZ, dZShapeInfo, dimension, dimensionLength, xTadShapeInfo, xOffsets, yTadShapeInfo, yOffsets);
}


void NativeOps::sort(Nd4jPointer *extraPointers,
					 void *x, Nd4jLong *xShapeInfo,
					 void *dX, Nd4jLong *dXShapeInfo,
					 bool descending) {

    hipStream_t *stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);

    auto xLength = shape::length(xShapeInfo);
    auto xEWS = shape::elementWiseStride(xShapeInfo);
    auto xType = nd4j::ArrayOptions::dataType(xShapeInfo);


    // check if xLength is a power of 2, and use bitonic sort, if that's the case
    if ((xLength != 0) && ((xLength & (xLength - 1)) == 0) && (xLength <= 1024 * 1024 * 10)) {
        int numThreads = nd4j::math::nd4j_min<int>(512, xLength);
        int numBlocks = xLength / numThreads;
        if (xLength % numThreads > 0 || numBlocks == 0)
            numBlocks++;

        dim3 launchDims(numBlocks, numThreads, 32768);

        for (int k = 2; k <= xLength; k = 2*k) {
            for (int j = k >> 1; j > 0; j = j >> 1) {
				BUILD_SINGLE_SELECTOR(xType, bitonicSortStepGeneric, (launchDims, stream, dX, dXShapeInfo, j, k, xLength, descending), LIBND4J_TYPES);
			}
        }
    } else {
    	int numThreads = nd4j::math::nd4j_min<int>(512, xLength);
    	int numBlocks = xLength / numThreads;
    	if (xLength % numThreads > 0 || numBlocks == 0)
    		numBlocks++;

    	numBlocks = nd4j::math::nd4j_min<int>(512, numBlocks);
    	dim3 launchDims(numBlocks, numThreads, 32768);

    	int max = 2, dg = 0;
    	while (max < xLength) {
    		max <<= 1;
    		dg++;
    	}
    	max <<= 1;

    	for (int window = 2; window < max; window<<=1) {
    		int n = window;
    		int rev = 0;
    		do{
    			int half = n >> 1;
    			BUILD_SINGLE_SELECTOR(xType, bitonicArbitraryStepGeneric, (launchDims, stream, dX, dXShapeInfo, n, xLength, rev, descending), LIBND4J_TYPES);
    			n>>=1;
    			rev = 1;
    		} while(n > 1);
    	}
    }

    nd4j::DebugHelper::checkErrorCode(stream, "sort(...) failed");
}


void NativeOps::sortTad(Nd4jPointer *extraPointers,
						void *x, Nd4jLong *xShapeInfo,
						void *dX, Nd4jLong *dXShapeInfo,
						int *dimension,
						int dimensionLength,
						Nd4jLong *tadShapeInfo,
						Nd4jLong *tadOffsets,
						bool descending) {
    // to be implemented
    auto stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);
    auto context = extraPointers[0] == 0 ? LaunchContext::defaultContext(): reinterpret_cast<LaunchContext*>(extraPointers[0]);
    auto tadPack = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(xShapeInfo, dimension, dimensionLength);
    dim3 launchDims(tadPack.numberOfTads(), 1024, 33768);
	auto xType = nd4j::ArrayOptions::dataType(xShapeInfo);
    BUILD_SINGLE_SELECTOR(xType, oesTadGeneric, (launchDims, stream, dX, dXShapeInfo, nullptr, dimensionLength, tadShapeInfo, tadOffsets, descending), LIBND4J_TYPES);

    nd4j::DebugHelper::checkErrorCode(stream, "sortTadFloat(...) failed");
}

void NativeOps::sortCooIndices(Nd4jPointer *extraPointers, Nd4jLong *indices, void *values, Nd4jLong length, int rank) {
	throw std::runtime_error("sortCooIndices:: Not implemented yet");
}


Nd4jLong NativeOps::encodeBitmap(Nd4jPointer *extraPointers,
								void *dx, Nd4jLong *hXShapeInfo,
								Nd4jLong N,
								int *dz,
								float threshold) {

    hipStream_t *stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);
    int *resultPointer = reinterpret_cast<int *>(extraPointers[2]);
    int *reductionPointer = reinterpret_cast<int *>(extraPointers[3]);

    dim3 launchDims(512, 512, 32768);
    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    BUILD_SINGLE_SELECTOR(xType, cudaEncodeBitmapGeneric, (launchDims, stream, dx, N, dz, resultPointer, reductionPointer, threshold), LIBND4J_TYPES);

    nd4j::DebugHelper::checkErrorCode(stream, "encodeBitmapFloat(...) failed");

    Nd4jLong dZ = (Nd4jLong) resultPointer[0];
    resultPointer[0] = 0;

    return dZ;
}


void NativeOps::decodeBitmap(Nd4jPointer *extraPointers,
							void *dx,
							Nd4jLong N,
							void *dz, Nd4jLong *zShapeInfo) {

    hipStream_t *stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);
    dim3 launchDims(512, 512, 16384);
    auto xType = nd4j::ArrayOptions::dataType(zShapeInfo);
    BUILD_SINGLE_SELECTOR(xType, cudaDecodeBitmapGeneric, (launchDims, stream, dx, N, dz), LIBND4J_TYPES);

    nd4j::DebugHelper::checkErrorCode(stream, "decodeBitmapFloat(...) failed");
}

Nd4jLong* NativeOps::mmapFile(Nd4jPointer *extraPointers, const char *fileName, Nd4jLong length) {
	return nullptr;
}

void NativeOps::munmapFile(Nd4jPointer *extraPointers, Nd4jLong* ptrMap, Nd4jLong length) {

}


nd4j::graph::ResultWrapper* NativeOps::executeFlatGraph(Nd4jPointer *extraPointers, Nd4jPointer flatBufferPointer) {
    return nd4j::graph::GraphExecutioner::executeFlatBuffer(flatBufferPointer);
}


const char* NativeOps::getAllCustomOps() {
	return nd4j::ops::OpRegistrator::getInstance()->getAllCustomOperations();
}


nd4j::ShapeList* _calculateOutputShapes(Nd4jPointer* extraPointers, nd4j::ops::DeclarableOp* op, Nd4jPointer* inputBuffers, Nd4jPointer* inputShapes, int numInputShapes, double* tArgs, int numTArgs, Nd4jLong *iArgs, int numIArgs, bool *bArgs, int numBArgs) {
    nd4j::graph::VariableSpace varSpace;
    Context block(2, &varSpace);
    nd4j::ShapeList inShapes;

    for (int e = 0; e < numIArgs; e++)
        block.getIArguments()->push_back(iArgs[e]);

    for (int e = 0; e < numTArgs; e++)
        block.getTArguments()->push_back(tArgs[e]);

	for (int e = 0; e < numBArgs; e++)
		block.getBArguments()->push_back(bArgs[e]);

	for (int e = 0; e < numInputShapes; e++) {
		auto shape_ = reinterpret_cast<Nd4jLong *>(inputShapes[e]);

		// we shouldn't copy buffer if that's empty array
		void *buffer_ = nd4j::ArrayOptions::arrayType(shape_) == ArrayType::EMPTY ? nullptr : inputBuffers[e];

		auto array = new nd4j::NDArray(buffer_, shape_);

		// block should contain references to proper variable
		varSpace.putVariable(1, e, array);
		block.pickInput(1, e);

		inShapes.push_back(shape_);
	}

    auto shapeList = op->calculateOutputShape(&inShapes, block);

    if (varSpace.launchContext()->getWorkspace() != nullptr)
        shapeList->detach();

    return shapeList;
}

nd4j::ShapeList* NativeOps::calculateOutputShapes(Nd4jPointer* extraPointers, Nd4jLong hash, Nd4jPointer* inputBuffers, Nd4jPointer* inputShapes, int numInputShapes, double* tArgs, int numTArgs, Nd4jLong *iArgs, int numIArgs, bool *bArgs, int numBArgs) {
    auto op = nd4j::ops::OpRegistrator::getInstance()->getOperation(hash);

    return _calculateOutputShapes(extraPointers, op, inputBuffers, inputShapes, numInputShapes, tArgs, numTArgs, iArgs, numIArgs, bArgs, numBArgs);
}

nd4j::ShapeList* _calculateOutputShapes(Nd4jPointer* extraPointers, nd4j::ops::DeclarableOp* op, Nd4jPointer* inputShapes, int numInputShapes, double* tArgs, int numTArgs, Nd4jLong *iArgs, int numIArgs) {
    Context block(1);
	nd4j::ShapeList inShapes;

	for (int e = 0; e < numIArgs; e++)
		block.getIArguments()->push_back(iArgs[e]);

	for (int e = 0; e < numTArgs; e++)
		block.getTArguments()->push_back(tArgs[e]);

	for (int e = 0; e < numInputShapes; e++)
		inShapes.push_back(reinterpret_cast<Nd4jLong *>(inputShapes[e]));

	auto shapeList = op->calculateOutputShape(&inShapes, block);

	return shapeList;
}

nd4j::ShapeList* NativeOps::calculateOutputShapes(Nd4jPointer* extraPointers, Nd4jLong hash, Nd4jPointer* inputShapes, int numInputShapes, double* tArgs, int numTArgs, Nd4jLong *iArgs, int numIArgs) {
	auto op = nd4j::ops::OpRegistrator::getInstance()->getOperation(hash);

	return _calculateOutputShapes(extraPointers, op, inputShapes, numInputShapes, tArgs, numTArgs, iArgs, numIArgs);
}


static FORCEINLINE Nd4jStatus realExec(nd4j::ops::DeclarableOp* op, Nd4jPointer* extraPointers, Nd4jLong hash, Nd4jPointer* inputBuffers, Nd4jPointer* inputShapes, int numInputs, Nd4jPointer* outputBuffers, Nd4jPointer* outputShapes, int numOutputs, double* tArgs, int numTArgs, Nd4jLong *iArgs, int numIArgs, bool* bArgs, int numBArgs, bool isInplace) {
	if (op == nullptr)
		nd4j_printf("Can't find requested operation: [%lld]\n", hash);

	// we're using the same fake nodeId everywhere here

	std::vector<nd4j::NDArray*> inputs(numInputs);
	std::vector<nd4j::NDArray*> outputs(numOutputs);
	std::vector<double> ttArgs(numTArgs);
	std::vector<bool> bbArgs(numBArgs);
	std::vector<Nd4jLong> iiArgs(numIArgs);

	// filling block now with inputs
	for (int e = 0; e < numInputs; e++) {
		auto shape = reinterpret_cast<Nd4jLong *>(inputShapes[e]);
		void *buffer = nd4j::ArrayOptions::arrayType(shape) == ArrayType::EMPTY ? nullptr : inputBuffers[e];
        void *bufferD = nd4j::ArrayOptions::arrayType(shape) == ArrayType::EMPTY ? nullptr : inputBuffers[e + numInputs];

		inputs[e] = new nd4j::NDArray(buffer, bufferD, shape);
	}

	// if not inplace - transferring output arrays

	if (!isInplace)
		for (int e = 0; e < numOutputs; e++) {
			// we want to keep original output shape intact
			auto shape = shape::copyShape(reinterpret_cast<Nd4jLong *>(outputShapes[e]));
			void *buffer = nd4j::ArrayOptions::arrayType(shape) == ArrayType::EMPTY ? nullptr : outputBuffers[e];
            void *bufferD = nd4j::ArrayOptions::arrayType(shape) == ArrayType::EMPTY ? nullptr : outputBuffers[e + numOutputs];

			// FIXME: revisit this.
			bool canNullify = true;
			for (int i = 0; i < numInputs; i++) {
				void *ibuffer = nd4j::ArrayOptions::arrayType(shape) == ArrayType::EMPTY ? nullptr : inputBuffers[i];
				if (ibuffer == buffer) {
					canNullify = false;
					break;
				}
			}

			if (canNullify)
				memset((uint8_t *) buffer, '\0', shape::length(shape) * DataTypeUtils::sizeOfElement(ArrayOptions::dataType(shape)));

			auto array = new nd4j::NDArray(buffer, bufferD, shape);
			outputs[e] = array;
		}

	for (int e = 0; e < numIArgs; e++)
		iiArgs[e] = iArgs[e];

	for (int e = 0; e < numTArgs; e++)
		ttArgs[e] = tArgs[e];

    for (int e = 0; e < numBArgs; e++)
        bbArgs[e] = bArgs[e];


	// hypothetically at this point we have everything filled
	auto dZ = op->execute(inputs, outputs, ttArgs, iiArgs, bbArgs, isInplace);
	//auto dZ = op->execute(inputs, ttArgs, iiArgs, isInplace);


	if (!isInplace)
		for (int e = 0; e < numOutputs; e++) {
			//shape::printShapeInfoLinear("JVM output shape", (int *) outputShapes[e]);
			//shape::printShapeInfoLinear("C++ output shape", (int *) outputs[e]->shapeInfo());
			//outputs[e]->printIndexedBuffer("C++ raw output");
			//outputs[e]->printBuffer("C++ indexed output");

			if (outputs[e]->ordering() != shape::order(reinterpret_cast<Nd4jLong *>(outputShapes[e])))
				outputs[e]->streamline(shape::order(reinterpret_cast<Nd4jLong *>(outputShapes[e])));
		}

	for (auto v: inputs)
		delete v;

	for (auto v: outputs)
		delete v;

	return Status::OK();
}


int NativeOps::execCustomOp(Nd4jPointer* extraPointers, Nd4jLong hash, Nd4jPointer* inputBuffers, Nd4jPointer* inputShapes, int numInputs, Nd4jPointer* outputBuffers, Nd4jPointer* outputShapes, int numOutputs, double* tArgs, int numTArgs, Nd4jLong *iArgs, int numIArgs, bool* bArgs, int numBArgs, bool isInplace) {
	auto op = nd4j::ops::OpRegistrator::getInstance()->getOperation(hash);

	return realExec(op, extraPointers, hash, inputBuffers, inputShapes, numInputs, outputBuffers, outputShapes, numOutputs, tArgs, numTArgs, iArgs, numIArgs, bArgs, numBArgs, isInplace);
}

int NativeOps::execCustomOp(Nd4jPointer* extraPointers, Nd4jLong hash, Nd4jPointer opContext) {
    auto op = nd4j::ops::OpRegistrator::getInstance()->getOperation(hash);
    auto context = reinterpret_cast<Context*>(opContext);

    auto result = op->execute(context);

    // FIXME: remove once CUDA backend is 100% ready
    for (auto v:context->fastpath_in()) {
        v->makeBothActual();
    }

    for (auto v:context->fastpath_out()) {
        v->makeBothActual();
    }

    return result;
}

int NativeOps::registerGraph(Nd4jPointer *extraPointers, Nd4jLong graphId, Nd4jPointer flatBufferPointer) {

	auto graph = nd4j::graph::GraphExecutioner::importFromFlatPointer(flatBufferPointer);

	nd4j::graph::GraphHolder::getInstance()->registerGraph(graphId, graph);

	return ND4J_STATUS_OK;
}


static VariablesSet* executeStoredGraphT(Nd4jPointer *extraPointers, Nd4jLong graphId, Nd4jPointer *inputBuffers, Nd4jPointer *inputShapes, int* inputIndices, int numInputs) {
	auto graph = nd4j::graph::GraphHolder::getInstance()->pullGraph(graphId);
	auto varSpace = graph->getVariableSpace()->clone();

	std::vector<nd4j::NDArray*> handles;

	for (int e = 0; e < numInputs; e++) {
		auto idx = inputIndices[e];

		// we'll delete this array later, together with cloned VariableSpace
		auto array = new nd4j::NDArray(inputBuffers[e], reinterpret_cast<Nd4jLong *>(inputShapes[e]));
		handles.emplace_back(array);

		if (varSpace->hasVariable(idx)) {
			auto var = varSpace->getVariable(idx);
			if (var->hasNDArray())
				delete var->getNDArray();

			var->setNDArray(array);
		} else
			varSpace->putVariable(idx, array);
	}

	auto dZ = nd4j::graph::GraphExecutioner::execute(graph, varSpace);
	auto varSet = new nd4j::graph::VariablesSet(dZ);

	if (dZ == ND4J_STATUS_OK) {
		// pull back results, and provide them
		auto outputs = graph->fetchOutputs();
		for (int e = 0; e < outputs->size(); e++) {
			// we're only getting variable ID/Index from original grap. values will be taken from cloned workspace
			std::pair<int, int> varId(outputs->at(e)->id(), outputs->at(e)->index());

			auto var = varSpace->getVariable(varId);

			varSet->push_back(var->clone());
		}

		delete outputs;
	}

	delete varSpace;

	return varSet;
}

VariablesSet* NativeOps::executeStoredGraph(Nd4jPointer *extraPointers, Nd4jLong graphId, Nd4jPointer *inputBuffers, Nd4jPointer *inputShapes, int* inputIndices, int numInputs) {
	return executeStoredGraphT(extraPointers, graphId, inputBuffers, inputShapes, inputIndices, numInputs);
}

int NativeOps::unregisterGraph(Nd4jPointer *extraPointers, Nd4jLong graphId) {

	nd4j::graph::GraphHolder::getInstance()->dropGraphAny(graphId);

	return ND4J_STATUS_OK;
}

void NativeOps::deletePointerArray(Nd4jPointer pointer) {
    Nd4jPointer *ptr = reinterpret_cast<Nd4jPointer *>(pointer);
    delete[] ptr;
}

void NativeOps::deleteIntArray(Nd4jPointer pointer) {
	auto ptr = reinterpret_cast<int *>(pointer);
	delete[] ptr;
}

void NativeOps::deleteLongArray(Nd4jPointer pointer) {
	auto ptr = reinterpret_cast<Nd4jLong *>(pointer);
	delete[] ptr;
}

template <typename T>
static void deleteVariablesSetT(Nd4jPointer pointer) {
	nd4j::graph::VariablesSet* ptr = reinterpret_cast<nd4j::graph::VariablesSet*>(pointer);
	delete ptr;
}

void NativeOps::deleteVariablesSet(Nd4jPointer pointer) {
	deleteVariablesSetT<double>(pointer);
}

void NativeOps::deleteShapeList(Nd4jPointer shapeList) {
    nd4j::ShapeList* list = reinterpret_cast<nd4j::ShapeList*>(shapeList);

    //list->destroy();
    delete list;
}

const char* NativeOps::getAllOperations() {
    return nd4j::OpTracker::getInstance()->exportOperations();
}

Nd4jPointer NativeOps::getGraphState(Nd4jLong id) {
    return (Nd4jPointer) new nd4j::graph::GraphState(id);
}


void NativeOps::deleteGraphState(Nd4jPointer state) {
    auto stateP = reinterpret_cast<nd4j::graph::GraphState*>(state);
    delete stateP;
}


Nd4jStatus execCustomOpWithScope(Nd4jPointer *extraPointers, nd4j::graph::GraphState *state, Nd4jLong opHash, Nd4jLong *scopes, int numScopes, Nd4jPointer *inputBuffers, Nd4jPointer *inputShapes, int numInputs, Nd4jPointer *outputBuffers, Nd4jPointer *outputShapes, int numOutputs) {
    /**
     * That's basically exec, with VariableSpace provided in GraphState:
     * depending on operation (i.e. while of if), different logic executors could be used
     */

    auto graph = state->graph();
    auto varSpace = state->variableSpace();

    // Node is dynamically created, and has nothing beyond it: only inputs and outputs
    // this node has id of 0, and inputs are
    Node node(OpType_LOGIC, opHash, 0);

    // mapping inputs
    for (int e = 0; e < numInputs; e++) {
        auto buffer = inputBuffers[e];
        auto shapeInfo = reinterpret_cast<Nd4jLong *>(inputShapes[e]);

        auto array = new nd4j::NDArray(buffer, shapeInfo, varSpace->launchContext());

        // now we just put array to VarSpace
        varSpace->putVariable(0, e, array);
        node.pickInput(0, e);
    }

    // mapping scopes
    for (int e = 0; e < numScopes; e++) {
        // we should check scope existence in GraphState/Graph
        int scopeId = (int) scopes[e];
        if (!state->hasScope(scopeId)) {
            // nd4j_printf("execCustomOpWithScope: referenced scope [%i] doesn't exist\n", scopeId);
            return Status::THROW();
        }
        node.pickInput(scopeId, 0);
    }

    auto dZ = LogicExecutor::processNode(graph, &node);
    if (dZ != Status::OK())
        return dZ;

    // mapping outputs

    for (int e = 0; e < numOutputs; e++) {
        auto buffer = outputBuffers[e];
        auto shapeInfo = reinterpret_cast<Nd4jLong *>(outputShapes[e]);

        NDArray array(buffer, shapeInfo, varSpace->launchContext());

        // now we just put array to VarSpace to the same ID
        //varSpace->putVariable(0, e, array);

        auto t = varSpace->getVariable(0, e)->getNDArray();
        array.assign(t);
    }

    // removing input variables
    for (int e = 0; e < numInputs; e++) {
        varSpace->dropVariable(0, e);
    }

    // after some bla-bla-bla we should have Graph and Node for current op
    return Status::OK();
}


Nd4jStatus NativeOps::execCustomOpWithScope(Nd4jPointer *extraPointers, Nd4jPointer state, Nd4jLong opHash, Nd4jLong *scopes, int numScopes, Nd4jPointer *inputBuffers, Nd4jPointer *inputShapes, int numInputs, Nd4jPointer *outputBuffers, Nd4jPointer *outputShapes, int numOutputs) {

    return execCustomOpWithScope(extraPointers, reinterpret_cast<nd4j::graph::GraphState*>(state), opHash, scopes, numScopes, inputBuffers, inputShapes, numInputs, outputBuffers, outputShapes, numOutputs);
}

void NativeOps::deleteResultWrapper(Nd4jPointer ptr) {
	// just 0 room for compiler s@!t
	auto p = reinterpret_cast<nd4j::graph::ResultWrapper *>(ptr);
	delete p;
}

int NativeOps::estimateThreshold(Nd4jPointer *extraPointers, Nd4jPointer dX, Nd4jLong *dXShapeInfo, int N, float threshold) {
	throw std::runtime_error("estimateThreshold: Not implemented yet");
}

/*
 * TypeDef:
 *     void convertTypes(Nd4jPointer *extras, int srcType, Nd4jPointer dX, long N, int dstType, Nd4jPointer dZ);
 */
void NativeOps::convertTypes(Nd4jPointer *extras, int srcType, Nd4jPointer dX, Nd4jLong N, int dstType, Nd4jPointer dZ) {
 	auto dx = reinterpret_cast<void *>(dX);
	auto dz = reinterpret_cast<void *>(dZ);

    if (srcType == ND4J_FLOAT8) {
        if (dstType == ND4J_FLOAT8) {
            // convertKernel<double, nd4j::float8>(extras, dx, N, dz);
        } else if (dstType == ND4J_INT8) {
            //nd4j::TypeCast::convertGenericCuda<nd4j::float8, nd4j::int8>(extras, dx, N, dz);
        } else if (dstType == ND4J_UINT8) {
            //nd4j::TypeCast::convertGenericCuda<nd4j::float8, nd4j::uint8>(extras, dx, N, dz);
        } else if (dstType == ND4J_FLOAT16) {
            //nd4j::TypeCast::convertGenericCuda<nd4j::float8, float16>(extras, dx, N, dz);
        } else if (dstType == ND4J_INT16) {
            //nd4j::TypeCast::convertGenericCuda<nd4j::float8, nd4j::int16>(extras, dx, N, dz);
        } else if (dstType == ND4J_UINT16) {
            //nd4j::TypeCast::convertGenericCuda<nd4j::float8, nd4j::uint16>(extras, dx, N, dz);
        } else if (dstType == ND4J_FLOAT24) {

        } else if (dstType == ND4J_FLOAT32) {
            //nd4j::TypeCast::convertGenericCuda<nd4j::float8, float>(extras, dx, N, dz);
        } else if (dstType == ND4J_DOUBLE) {
            //nd4j::TypeCast::convertGenericCuda<nd4j::float8, double>(extras, dx, N, dz);
        } else {
            nd4j_printf("Unsupported types conversion: [%i] -> [%i]\n", srcType, dstType);
        }
    } else if (srcType == ND4J_INT8) {
        if (dstType == ND4J_FLOAT8) {
            //nd4j::TypeCast::convertGenericCuda<nd4j::int8, nd4j::float8>(extras, dx, N, dz);
        } else if (dstType == ND4J_INT8) {
            //convertKernel<nd4j::int8, nd4j::int8>(extras, dx, N, dz);
        } else if (dstType == ND4J_UINT8) {
            nd4j::TypeCast::convertGenericCuda<int8_t, uint8_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_FLOAT16) {
            nd4j::TypeCast::convertGenericCuda<int8_t, float16>(extras, dx, N, dz);
        } else if (dstType == ND4J_INT16) {
            nd4j::TypeCast::convertGenericCuda<int8_t, int16_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_UINT16) {
            nd4j::TypeCast::convertGenericCuda<int8_t, uint16_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_FLOAT24) {
            // TODO: eventually we might want to add it
        } else if (dstType == ND4J_FLOAT32) {
            nd4j::TypeCast::convertGenericCuda<int8_t, float>(extras, dx, N, dz);
        } else if (dstType == ND4J_DOUBLE) {
            nd4j::TypeCast::convertGenericCuda<int8_t, double>(extras, dx, N, dz);
        } else {
            nd4j_printf("Unsupported types conversion: [%i] -> [%i]\n", srcType, dstType);
        }
    } else if (srcType == ND4J_UINT8) {
        if (dstType == ND4J_FLOAT8) {
            //nd4j::TypeCast::convertGenericCuda<uint8_t, nd4j::float8>(extras, dx, N, dz);
        } else if (dstType == ND4J_INT8) {
            nd4j::TypeCast::convertGenericCuda<uint8_t, int8_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_UINT8) {
            nd4j::TypeCast::convertGenericCuda<uint8_t, uint8_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_FLOAT16) {
            nd4j::TypeCast::convertGenericCuda<uint8_t, float16>(extras, dx, N, dz);
        } else if (dstType == ND4J_INT16) {
            nd4j::TypeCast::convertGenericCuda<uint8_t, int16_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_UINT16) {
            nd4j::TypeCast::convertGenericCuda<uint8_t, uint16_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_FLOAT24) {
            // TODO: still might want to add
        } else if (dstType == ND4J_FLOAT32) {
            nd4j::TypeCast::convertGenericCuda<uint8_t, float>(extras, dx, N, dz);
        } else if (dstType == ND4J_DOUBLE) {
            nd4j::TypeCast::convertGenericCuda<uint8_t, double>(extras, dx, N, dz);
        } else {
            nd4j_printf("Unsupported types conversion: [%i] -> [%i]\n", srcType, dstType);
        }
    } else if (srcType == ND4J_FLOAT16) {
        if (dstType == ND4J_FLOAT8) {
            //nd4j::TypeCast::convertGenericCuda<float16, nd4j::float8>(extras, dx, N, dz);
        } else if (dstType == ND4J_INT8) {
            nd4j::TypeCast::convertGenericCuda<float16, int8_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_UINT8) {
            nd4j::TypeCast::convertGenericCuda<float16, uint8_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_FLOAT16) {
            nd4j::TypeCast::convertGenericCuda<float16, float16>(extras, dx, N, dz);
        } else if (dstType == ND4J_INT16) {
            nd4j::TypeCast::convertGenericCuda<float16, int16_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_UINT16) {
            nd4j::TypeCast::convertGenericCuda<float16, uint16_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_FLOAT24) {
            // TODO: .... ^^^
        } else if (dstType == ND4J_FLOAT32) {
            nd4j::TypeCast::convertGenericCuda<float16, float>(extras, dx, N, dz);
        } else if (dstType == ND4J_DOUBLE) {
            nd4j::TypeCast::convertGenericCuda<float16, double>(extras, dx, N, dz);
        } else if (dstType == ND4J_THRESHOLD) {
            //nd4j::convertToThreshold<float16>(nullptr, dx, N, dz);
        } else {
            nd4j_printf("Unsupported types conversion: [%i] -> [%i]\n", srcType, dstType);
        }
    } else if (srcType == ND4J_INT16) {
        if (dstType == ND4J_FLOAT8) {
            //nd4j::TypeCast::convertGenericCuda<int16_t, nd4j::float8>(extras, dx, N, dz);
        } else if (dstType == ND4J_INT8) {
            nd4j::TypeCast::convertGenericCuda<int16_t, int8_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_UINT8) {
            nd4j::TypeCast::convertGenericCuda<int16_t, uint8_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_FLOAT16) {
            nd4j::TypeCast::convertGenericCuda<int16_t, float16>(extras, dx, N, dz);
        } else if (dstType == ND4J_INT16) {
            nd4j::TypeCast::convertGenericCuda<int16_t, int16_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_UINT16) {
            nd4j::TypeCast::convertGenericCuda<int16_t, uint16_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_FLOAT24) {
            // TODO...
        } else if (dstType == ND4J_FLOAT32) {
            nd4j::TypeCast::convertGenericCuda<int16_t, float>(extras, dx, N, dz);
        } else if (dstType == ND4J_DOUBLE) {
            nd4j::TypeCast::convertGenericCuda<int16_t, double>(extras, dx, N, dz);
        } else {
            printf("Unsupported types conversion: [%i] -> [%i]\n", srcType, dstType);
        }
    } else if (srcType == ND4J_FLOAT24) {

    } else if (srcType == ND4J_FLOAT32) {
        if (dstType == ND4J_FLOAT8) {
            //nd4j::TypeCast::convertGenericCuda<float, nd4j::float8>(extras, dx, N, dz);
        } else if (dstType == ND4J_INT8) {
            nd4j::TypeCast::convertGenericCuda<float, int8_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_UINT8) {
            nd4j::TypeCast::convertGenericCuda<float, uint8_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_FLOAT16) {
            nd4j::TypeCast::convertGenericCuda<float, float16>(extras, dx, N, dz);
        } else if (dstType == ND4J_INT16) {
            nd4j::TypeCast::convertGenericCuda<float, int16_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_UINT16) {
            nd4j::TypeCast::convertGenericCuda<float, uint16_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_FLOAT24) {

        } else if (dstType == ND4J_DOUBLE) {
            nd4j::TypeCast::convertGenericCuda<float, double>(extras, dx, N, dz);
        } else if (dstType == ND4J_THRESHOLD) {
            //nd4j::convertToThreshold<float>(nullptr, dx, N, dz);
        } else {
            nd4j_printf("Unsupported types conversion: [%i] -> [%i]\n", srcType, dstType);
        }
    } else if (srcType == ND4J_DOUBLE) {
        if (dstType == ND4J_FLOAT8) {
            //nd4j::TypeCast::convertGenericCuda<double, nd4j::float8>(extras, dx, N, dz);
        } else if (dstType == ND4J_INT8) {
            nd4j::TypeCast::convertGenericCuda<double, int8_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_UINT8) {
            nd4j::TypeCast::convertGenericCuda<double, uint8_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_FLOAT16) {
            nd4j::TypeCast::convertGenericCuda<double, float16>(extras, dx, N, dz);
        } else if (dstType == ND4J_INT16) {
            nd4j::TypeCast::convertGenericCuda<double, int16_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_UINT16) {
            nd4j::TypeCast::convertGenericCuda<double, uint16_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_FLOAT24) {

        } else if (dstType == ND4J_FLOAT32) {
            nd4j::TypeCast::convertGenericCuda<double, float>(extras, dx, N, dz);
        } else if (dstType == ND4J_DOUBLE) {
            //
        } else if (dstType == ND4J_THRESHOLD) {
            //nd4j::convertToThreshold<double>(nullptr, dx, N, dz);
        } else {
            nd4j_printf("Unsupported types conversion: [%i] -> [%i]\n", srcType, dstType);
        }
    } else if (srcType == ND4J_THRESHOLD) {
        if (dstType == ND4J_FLOAT16) {
            //nd4j::convertFromThreshold<float16>(nullptr, dx, N, dz);
        } else if (dstType == ND4J_FLOAT32) {
            //nd4j::convertFromThreshold<float>(nullptr, dx, N, dz);
        } else if (dstType == ND4J_DOUBLE) {
            //nd4j::convertFromThreshold<double>(nullptr, dx, N, dz);
        } else {
            nd4j_printf("Unsupported types conversion: [%i] -> [%i]\n", srcType, dstType);
        }
    } else {
        nd4j_printf("Unsupported types conversion: [%i] -> [%i]\n", srcType, dstType);
    }
}

Nd4jPointer NativeOps::createUtf8String(Nd4jPointer *extraPointers, const char *string, int length) {
    auto u = new nd4j::utf8string(string, length);
    return reinterpret_cast<Nd4jPointer>(u);
}

void NativeOps::deleteUtf8String(Nd4jPointer *extraPointers, Nd4jPointer ptr) {
    delete(reinterpret_cast<nd4j::utf8string*>(ptr));
}

///////////////////////////////////////////////////////////////////
template<typename T>
__global__ static void scatterUpdateCuda(const int opCode, const int numOfSubArrs,
										      void* vx, const Nd4jLong *xShapeInfo, const Nd4jLong *xOffsets,
										      void* vy, const Nd4jLong *yShapeInfo, const Nd4jLong *yOffsets,
										      const int* indexes) {

    __shared__ T *x, *y;
    __shared__ Nd4jLong arrLenX, arrLenY;

    for (int e = 0; e < numOfSubArrs; e++ ) {

        const auto xIndex = indexes[e];
        const bool isOwner = xIndex < gridDim.x ? blockIdx.x == xIndex : blockIdx.x == xIndex % gridDim.x;

        if (!isOwner)
            continue;

        if (threadIdx.x == 0) {
            x = reinterpret_cast<T*>(vx) + xOffsets[xIndex];
            y = reinterpret_cast<T*>(vy) + yOffsets[e];
            arrLenX = shape::length(xShapeInfo);
            arrLenY = shape::length(yShapeInfo);
        }

        __syncthreads();

        if (arrLenX != arrLenY)
            return;

        for (Nd4jLong i = threadIdx.x; i < arrLenX; i += blockDim.x) {

            const auto xOffset = shape::getIndexOffset(i, xShapeInfo, arrLenX);
            const auto yOffset = shape::getIndexOffset(i, yShapeInfo, arrLenY);

            switch (opCode) {
                case 0:
                    x[xOffset] += y[yOffset];
                    break;
                case 1:
                    x[xOffset] -= y[yOffset];
                    break;
                case 2:
                    x[xOffset] *= y[yOffset];
                    break;
                case 3:
                    x[xOffset] /= y[yOffset];
                    break;
                case 4:
                    x[xOffset] = y[yOffset] - x[xOffset];
                    break;
                case 5:
                    x[xOffset] = y[yOffset] / x[xOffset];
                    break;
                case 6:
                    x[xOffset] = y[yOffset];
                    break;
                default:
                    continue;
            }
        }
        __syncthreads();
    }
}

template<typename T>
__host__ static void scatterUpdateCudaLauncher(const hipStream_t* stream, const int opCode, const int numOfSubArrs, void* vx, const Nd4jLong *xShapeInfo, const Nd4jLong *xOffsets, void* vy, const Nd4jLong *yShapeInfo, const Nd4jLong *yOffsets, const int* indexes) {

    scatterUpdateCuda<T><<<512, 256, MAX_NUM_THREADS, *stream>>>(opCode, numOfSubArrs, vx, xShapeInfo, xOffsets, vy, yShapeInfo, yOffsets, indexes);
}


//////////////////////////////////////////////////////////////////////////
void NativeOps::scatterUpdate(Nd4jPointer *extraPointers, int opCode, int numOfSubArrs,
                      			void* hX, Nd4jLong* hXShapeInfo, Nd4jLong* hXOffsets,
                      			void* dX, Nd4jLong* dXShapeInfo, Nd4jLong* dXOffsets,
                      			void* hY, Nd4jLong* hYShapeInfo, Nd4jLong* hYOffsets,
                      			void* dY, Nd4jLong* dYShapeInfo, Nd4jLong* dYOffsets,
                      			int* hIindexes, int* dIndexes) {

	auto stream = reinterpret_cast<hipStream_t *>(extraPointers[1]);

	nd4j::DataType type = ArrayOptions::dataType(hXShapeInfo);

    BUILD_SINGLE_SELECTOR(type, scatterUpdateCudaLauncher, (stream, opCode, numOfSubArrs, dX, dXShapeInfo, dXOffsets, dY, dYShapeInfo, dYOffsets, dIndexes), LIBND4J_TYPES);
    nd4j::DebugHelper::checkErrorCode(stream, "scatterUpdate(...) failed");
}

void NativeOps::inspectArray(Nd4jPointer *extraPointers, Nd4jPointer buffer, Nd4jLong *shapeInfo, Nd4jPointer specialBuffer, Nd4jLong *specialShapeInfo, Nd4jPointer debugInfo) {
    LaunchContext lc(extraPointers[1], extraPointers[4], extraPointers[5], extraPointers[3]);
    auto p = reinterpret_cast<nd4j::DebugInfo*>(debugInfo);
    NDArray array(buffer, specialBuffer, shapeInfo, &lc);
    nd4j::DebugHelper::retrieveDebugStatistics(p, &array);
}

void __global__ tryPointerKernel(void* p, int len) {
    auto buf = reinterpret_cast<int8_t*>(p);
    auto tid = threadIdx.x + blockIdx.x * blockDim.x;
    __shared__ int b;
    if (tid < len)
        atomicAdd(&b, buf[tid]);

    __syncthreads();

    if (threadIdx.x ==0 && blockIdx.x == 0)
        printf("Pointer check complete: %i\n", b);
}

void NativeOps::tryPointer(Nd4jPointer extra, Nd4jPointer p, int len) {

    hipStream_t stream;
    hipStreamCreate(&stream);

    tryPointerKernel<<<256, 512, len+64, stream>>>(p, len);
    auto e = hipStreamSynchronize(stream);

    if (e != 0)
        throw std::runtime_error("tryPointer failed");

    hipStreamDestroy(stream);
}

int NativeOps::dataTypeFromNpyHeader(void *header) {
    return (int) cnpy::dataTypeFromHeader(reinterpret_cast<char *>(header));
}
nd4j::ConstantDataBuffer* NativeOps::shapeBuffer(int rank, Nd4jLong *shape, Nd4jLong *strides, nd4j::DataType dtype, char order, Nd4jLong ews, bool empty) {
    auto buffer = new ConstantDataBuffer();
    *buffer = nd4j::ConstantShapeHelper::getInstance()->bufferForShapeInfo(ShapeDescriptor(dtype, order, shape, strides, rank, ews, empty));
    return buffer;
}

nd4j::ConstantDataBuffer* NativeOps::constantBuffer(nd4j::DataType dtype, Nd4jLong *data, int length) {
    return nd4j::ConstantHelper::getInstance()->constantBuffer(ConstantDescriptor(data, length), dtype);
}

nd4j::ConstantDataBuffer* NativeOps::constantBuffer(nd4j::DataType dtype, double *data, int length) {
    return nd4j::ConstantHelper::getInstance()->constantBuffer(ConstantDescriptor(data, length), dtype);
}

nd4j::ConstantDataBuffer* NativeOps::constantBuffer(nd4j::DataType dtype, nd4j::ConstantDescriptor *descriptor) {
    return nd4j::ConstantHelper::getInstance()->constantBuffer(*descriptor, dtype);
}

Nd4jPointer NativeOps::shapeBufferForNumpy(Nd4jPointer npyArray) {
    cnpy::NpyArray arr = cnpy::loadNpyFromPointer(reinterpret_cast<char *>(npyArray));
    unsigned int shapeSize = arr.shape.size();
    std::vector<Nd4jLong> shape(shapeSize);
    bool _empty = false;
    for(unsigned int i = 0; i < shapeSize; i++) {
        shape[i] = arr.shape[i];

        if (arr.shape[i] == 0)
            _empty = true;
    }

    auto dtype = cnpy::dataTypeFromHeader(reinterpret_cast<char *>(npyArray));

    Nd4jLong *shapeBuffer;
    if (_empty) {
        if (shapeSize > 0)
            shapeBuffer = nd4j::ShapeBuilders::emptyShapeInfo(dtype, arr.fortranOrder ? 'f' : 'c', shape);
        else
            shapeBuffer = nd4j::ShapeBuilders::emptyShapeInfo(dtype);
    } else {
        shapeBuffer = nd4j::ShapeBuilders::createShapeInfo(dtype, arr.fortranOrder ? 'f' : 'c', shape);
    }
    return reinterpret_cast<Nd4jPointer>(shapeBuffer);
}
